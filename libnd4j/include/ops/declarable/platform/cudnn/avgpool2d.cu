/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com)
//

#include <ops/declarable/helpers/convolutions.h>

#include "cudnnUtils.h"

namespace sd {
namespace ops {
namespace platforms {

//////////////////////////////////////////////////////////////////////////
PLATFORM_IMPL(avgpool2d, ENGINE_CUDA) {
  auto input = INPUT_VARIABLE(0);
  auto output = OUTPUT_VARIABLE(0);

  // 0,1 - kernel Height/Width; 2,3 - stride Height/Width; 4,5 - pad Height/Width; 6,7 - dilation Height/Width; 8 - same
  // mode;
  const LongType kH = INT_ARG(0);
  const LongType kW = INT_ARG(1);
  const LongType sH = INT_ARG(2);
  const LongType sW = INT_ARG(3);
  LongType pH = INT_ARG(4);
  LongType pW = INT_ARG(5);
  const LongType dH = INT_ARG(6);
  const LongType dW = INT_ARG(7);
  const auto paddingMode = static_cast<bool>(INT_ARG(8));
  const auto extraParam0 = INT_ARG(9);
  const int isNCHW = block.getIArguments()->size() > 10 ? !INT_ARG(10) : 1;  // INT_ARG(10): 0-NCHW, 1-NHWC

  REQUIRE_TRUE(input->rankOf() == 4, 0, "AVGPOOL2D CUDNN op: input should have rank of 4, but got %i instead",
               input->rankOf());
  REQUIRE_TRUE(dH != 0 && dW != 0, 0, "AVGPOOL2D CUDNN op: dilation must not be zero, but got instead {%i, %i}", dH,
               dW);

  LongType oH = 0;
  LongType oW = 0;

  const LongType iH = static_cast<LongType>(isNCHW ? input->sizeAt(2) : input->sizeAt(1));
  const LongType iW = static_cast<LongType>(isNCHW ? input->sizeAt(3) : input->sizeAt(2));

  ConvolutionUtils::calcOutSizePool2D(oH, oW, kH, kW, sH, sW, pH, pW, dH, dW, iH, iW, paddingMode);

  if (paddingMode) ConvolutionUtils::calcPadding2D(pH, pW, oH, oW, iH, iW, kH, kW, sH, sW, dH, dW);

  const hipdnnPoolingMode_t mode =
      (extraParam0 == 0) ? HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING : HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING;

  pooling2dCUDNN(block.launchContext(), input, output, kH, kW, sH, sW, pH, pW, dH, dW, isNCHW, mode);

  return Status::OK;
}

//////////////////////////////////////////////////////////////////////////
PLATFORM_CHECK(avgpool2d, ENGINE_CUDA) {
  auto input = INPUT_VARIABLE(0);
  auto output = OUTPUT_VARIABLE(0);

  Requirements req("CUDNN AVGPOOL2d OP");
  req.expectEq(makeInfoVariable(input->dataType(), TYPE_MSG_INPUT),
               makeInfoVariable(output->dataType(), TYPE_MSG_OUTPUT)) &&
      req.expectIn(makeInfoVariable(input->dataType(), TYPE_MSG_INPUT),
                   {INT32, HALF, FLOAT32, DOUBLE});
  req.logTheSuccess();
  return req;
}

//////////////////////////////////////////////////////////////////////////
PLATFORM_IMPL(avgpool2d_bp, ENGINE_CUDA) {
  auto input = INPUT_VARIABLE(0);   // [bS, iH, iW, iC] (NHWC) or [bS, iC, iH, iW] (NCHW)
  auto gradO = INPUT_VARIABLE(1);   // [bS, oH, oW, oC] (NHWC) or [bS, oC, oH, oW] (NCHW), epsilon_next
  auto gradI = OUTPUT_VARIABLE(0);  // [bS, iH, iW, iC] (NHWC) or [bS, iC, iH, iW] (NCHW), epsilon

  const LongType kH = INT_ARG(0);           // filter(kernel) height
  const LongType kW = INT_ARG(1);           // filter(kernel) width
  const LongType sH = INT_ARG(2);           // strides height
  const LongType sW = INT_ARG(3);           // strides width
  LongType pH = INT_ARG(4);                 // paddings height
  LongType pW = INT_ARG(5);                 // paddings width
  const LongType dH = INT_ARG(6);           // dilations height
  const LongType dW = INT_ARG(7);           // dilations width
  const auto paddingMode = INT_ARG(8);  // 0-VALID, 1-SAME
  const auto extraParam0 = INT_ARG(9);
  const auto isNCHW = block.getIArguments()->size() > 10 ? !INT_ARG(10) : 1;  // INT_ARG(10): 0-NCHW, 1-NHWC

  REQUIRE_TRUE(input->rankOf() == 4, 0, "AVGPOOL2D_BP CUDNN op: input should have rank of 4, but got %i instead",
               input->rankOf());
  REQUIRE_TRUE(dH != 0 && dW != 0, 0, "AVGPOOL2D_BP CUDNN op: dilation must not be zero, but got instead {%i, %i}", dH,
               dW);

  LongType bS, iC, iH, iW, oC, oH,
      oW;  // batch size, input channels, input height/width, output channels, output height/width;
  LongType indIOioC, indIiH, indWoC, indWiC, indWkH, indOoH;  // corresponding indexes
  ConvolutionUtils::getSizesAndIndexesConv2d(isNCHW, 0, *input, *gradO, bS, iC, iH, iW, oC, oH, oW, indIOioC, indIiH,
                                             indWiC, indWoC, indWkH, indOoH);

  std::vector<LongType> expectedGradOShape =
      ShapeUtils::composeShapeUsingDimsAndIdx({bS, iC, oH, oW, 0, indIOioC, indIiH, indIiH + 1});
  std::vector<LongType> expectedGradIShape =
      ShapeUtils::composeShapeUsingDimsAndIdx({bS, iC, iH, iW, 0, indIOioC, indIiH, indIiH + 1});
  REQUIRE_TRUE(gradO->isSameShape(expectedGradOShape), 0,
               "AVGPOOL2D_BP CUDNN op: wrong shape of output's gradients array (next epsilon), expected is %s, but got "
               "%s instead !",
               ShapeUtils::shapeAsString(expectedGradOShape).c_str(), ShapeUtils::shapeAsString(gradO).c_str());
  REQUIRE_TRUE(
      gradI->isSameShape(expectedGradIShape), 0,
      "AVGPOOL2D_BP CUDNN op: wrong shape of input's gradients array (epsilon), expected is %s, but got %s instead !",
      ShapeUtils::shapeAsString(expectedGradIShape).c_str(), ShapeUtils::shapeAsString(gradI).c_str());

  if (paddingMode)  // SAME
    ConvolutionUtils::calcPadding2D(pH, pW, oH, oW, iH, iW, kH, kW, sH, sW, dH, dW);

  const hipdnnPoolingMode_t mode =
      (extraParam0 == 0) ? HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING : HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING;

  pooling2dBpCUDNN(block.launchContext(), input, gradO, gradI, kH, kW, sH, sW, pH, pW, dH, dW, isNCHW, mode);

  return Status::OK;
}

PLATFORM_CHECK(avgpool2d_bp, ENGINE_CUDA) {
  auto input = INPUT_VARIABLE(0);   // [bS, iH, iW, iC] (NHWC) or [bS, iC, iH, iW] (NCHW)
  auto gradO = INPUT_VARIABLE(1);   // [bS, oH, oW, oC] (NHWC) or [bS, oC, oH, oW] (NCHW), epsilon_next
  auto gradI = OUTPUT_VARIABLE(0);  // [bS, iH, iW, iC] (NHWC) or [bS, iC, iH, iW] (NCHW), epsilon

  Requirements req("CUDNN AVGPOOL2d_BP OP");
  req.expectEq(makeInfoVariable(input->dataType(), TYPE_MSG_INPUT0),
               makeInfoVariable(gradO->dataType(), TYPE_MSG_INPUT1)) &&
      req.expectEq(makeInfoVariable(input->dataType(), TYPE_MSG_INPUT),
                   makeInfoVariable(gradI->dataType(), TYPE_MSG_OUTPUT)) &&
      req.expectIn(makeInfoVariable(input->dataType(), TYPE_MSG_INPUT),
                   {INT32, HALF, FLOAT32, DOUBLE}) &&
      req.expect(
          makeShapeInfoVariable(input, SHAPE_MSG_INPUT0), makeShapeInfoVariable(gradI, SHAPE_MSG_OUTPUT),
          [](const decltype(input)& l, const decltype(gradI)& r) {
            return shape::haveSameShapeAndStrides(l->shapeInfo(), r->shapeInfo());
          },
          EXPECTED_EQ_MSG);
  req.logTheSuccess();
  return req;
}

}  // namespace platforms
}  // namespace ops
}  // namespace sd
