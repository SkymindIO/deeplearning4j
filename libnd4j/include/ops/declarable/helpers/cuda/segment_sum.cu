#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author GS <sgazeos@gmail.com>
//
#include <array/NDArrayFactory.h>
#include <exceptions/cuda_exception.h>
#include <execution/cuda/LaunchDims.h>
#include <helpers/ConstantTadHelper.h>
#include <helpers/PointersManager.h>
#include <helpers/ShapeUtils.h>
#include <helpers/TAD.h>
#include <ops/declarable/helpers/segment.h>
#include <ops/declarable/helpers/segment_common.h>

#include "helpers/DebugHelper.h"


namespace sd {
namespace ops {
namespace helpers {
// -------------------------------------------------------------------------------------------------------------- //
// Segment ops linear kernels
// -------------------------------------------------------------------------------------------------------------- //
template <typename T, typename I>
static SD_KERNEL void segmentSumLinearKernel(const void* input, const LongType* inputShape, LongType* starts,
                                             LongType* lengths, LongType numOfClasses, void* output,
                                             const LongType* outputShape) {
  __shared__ T* val;
  __shared__ LongType xLen, zLen, segment, zIndex;
  __shared__ const T* x;
  __shared__ T* z;
  __shared__ int threadsPerSegment, start, finish;

  if (threadIdx.x == 0) {
    threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
    segment = blockIdx.x / threadsPerSegment;
    x = reinterpret_cast<const T*>(input);
    z = reinterpret_cast<T*>(output);

    xLen = shape::length(inputShape);
    zLen = shape::length(outputShape);

    if (segment < numOfClasses) {
      zIndex = shape::getIndexOffset(segment, outputShape);
      if(zIndex >= zLen)
        return;
      start = starts[segment];
      finish = start + lengths[segment];
      z[zIndex] = x[shape::getIndexOffset(start, inputShape)];
    }
  }
  __syncthreads();

  for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
    auto xIndex = shape::getIndexOffset(e, inputShape);
    if (xIndex >= xLen) return;
    math::atomics::sd_atomicAdd(&z[zIndex], x[xIndex]);
  }
}
// -------------------------------------------------------------------------------------------------------------- //

template <typename T, typename I>
static SD_KERNEL void unsortedSegmentSumLinearKernel(const void* input, const LongType* inputShape,
                                                     const void* indices, const LongType* indicesShape, LongType* starts, LongType* lengths,
                                                     LongType numOfClasses, void* output,
                                                     const LongType* outputShape) {
  __shared__ T* val;
  __shared__ LongType xLen, zLen, segment, zIndex;
  __shared__ const T* x;
  __shared__ T* z;
  __shared__ const I* y;

  if (threadIdx.x == 0) {
    segment = blockIdx.x;
    x = reinterpret_cast<const T*>(input);
    z = reinterpret_cast<T*>(output);
    y = reinterpret_cast<const I*>(indices);
    xLen = shape::length(inputShape);
    zLen = shape::length(outputShape);

    zIndex = shape::getIndexOffset(segment, outputShape);
    if (lengths[segment] > 0)
      z[zIndex] = x[shape::getIndexOffset(starts[segment], inputShape)];
    else
      z[zIndex] = 0;
  }
  __syncthreads();

  if (lengths[segment] > 0)
    for (auto e = threadIdx.x; e < xLen; e += blockDim.x) {
      auto xIndex = shape::getIndexOffset(e, inputShape);
      auto yIndex = shape::getIndexOffset(e, indicesShape);
      if (y[yIndex] == segment && e != starts[segment]) {
        math::atomics::sd_atomicAdd(&z[zIndex], x[xIndex]);
      }
    }
}
// -------------------------------------------------------------------------------------------------------------- //
// SegmentSum kernel
template <typename T, typename I>
static SD_KERNEL void segmentSumTadKernel(void* inputBuf, const LongType* inputShape,
                                          const LongType* inputTads, const LongType* inputTadOffsets,
                                          const I* indices, LongType* starts,
                                          LongType* lengths, LongType numOfClasses, void* outputBuf, const LongType* outputShape,
                                          const LongType* outputTads, const LongType* outputTadOffsets, LongType numIndices) {


   __shared__ LongType len, total;

   if (threadIdx.x == 0) {
     total = shape::sizeAt(inputShape, 0);
     len = shape::length(inputTads);
   }
   __syncthreads();

   for (auto idx = blockIdx.x; idx < total; idx += gridDim.x) {
     auto x = reinterpret_cast<T*>(inputBuf) + inputTadOffsets[idx];
     auto segment = indices[idx];
     auto z = reinterpret_cast<T*>(outputBuf) + outputTadOffsets[segment];
     auto start = starts[segment];
     auto finish = start + lengths[segment];
     if (lengths[segment] == 0) continue;
     for (auto e = threadIdx.x; e < len; e += blockDim.x) {
       auto xIndex = shape::getIndexOffset(e, inputTads);
       auto zIndex = shape::getIndexOffset(e, outputTads);
      math::atomics::sd_atomicAdd(&z[zIndex], x[xIndex]);
     }
   }
}
// -------------------------------------------------------------------------------------------------------------- //

template <typename T, typename I>
static void segmentSumFunctor_(LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
  auto stream = context->getCudaStream();
  LongType numClasses = indices->e<LongType>(indices->lengthOf() - 1) + 1;
  NDArray classesRangesLens = NDArrayFactory::create<LongType>('c', {numClasses}, context);
  NDArray classesRangesBegs = NDArrayFactory::create<LongType>('c', {numClasses}, context);

  classesRangesBegs.assign(indices->lengthOf());
  classesRangesLens.assign(0);

  fillUpSegments(indices, numClasses, classesRangesBegs, classesRangesLens);
  LongType* begins = reinterpret_cast<LongType*>(classesRangesBegs.specialBuffer());
  LongType* lengths = reinterpret_cast<LongType*>(classesRangesLens.specialBuffer());

  if (input->isVector() || input->isScalar()) {
    segmentSumLinearKernel<T, I><<<numClasses, input->lengthOf(), numClasses * 32 + 32, *stream>>>(
        input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numClasses, output->specialBuffer(),
        output->specialShapeInfo());
    sd::DebugHelper::checkErrorCode(stream, "segmentSumLinearKernel failed");

  } else {
    LongType zero = 0;
    std::vector<LongType> *dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), 1,&zero);
    auto packX = ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
    auto packZ = ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
    auto inputTads = packX->specialShapeInfo();
    auto inputTadOffsets = packX->specialOffsets();
    auto outputTads = packZ->specialShapeInfo();
    auto outputTadOffsets = packZ->specialOffsets();
    dim3 segmentTadDims = segmentTad(input->sizeAt(0));
    segmentSumTadKernel<T, I><<<segmentTadDims.y,segmentTadDims.x,segmentTadDims.z, *stream>>>(
        input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets,
        reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numClasses, output->specialBuffer(),
        output->specialShapeInfo(), outputTads, outputTadOffsets, indices->lengthOf());
    sd::DebugHelper::checkErrorCode(stream, "segmentSumTadKernel failed");

    delete dimensions;
  }
}
// -------------------------------------------------------------------------------------------------------------- //
void segmentSumFunctor(LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
  NDArray::prepareSpecialUse({output}, {input, indices});
  output->nullify();
  BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), segmentSumFunctor_, (context, input, indices, output),
                        SD_NUMERIC_TYPES, SD_INDEXING_TYPES);
  NDArray::registerSpecialUse({output}, {input, indices});
}

// -------------------------------------------------------------------------------------------------------------- //
template <typename T, typename I>
static void unsortedSegmentSumFunctor_(LaunchContext* context, NDArray* input, NDArray* indices, LongType numOfClasses, NDArray* output) {
  auto stream = context->getCudaStream();
  NDArray classesRangesBegs = NDArrayFactory::create<LongType>('c', {numOfClasses}, context);
  NDArray classesRangesLens = NDArrayFactory::create<LongType>('c', {numOfClasses}, context);
  classesRangesBegs.assign(indices->lengthOf());
  classesRangesLens.assign(0);
  dim3 dims = getSegmentSumDims(numOfClasses,indices->lengthOf());
  fillUpSegments(indices, numOfClasses, classesRangesBegs, classesRangesLens);
  LongType* begins = reinterpret_cast<LongType*>(classesRangesBegs.specialBuffer());
  LongType* lengths = reinterpret_cast<LongType*>(classesRangesLens.specialBuffer());

  if (input->isVector() || input->isScalar()) {
    unsortedSegmentSumLinearKernel<T, I><<<dims.x, dims.y, dims.z, *stream>>>(
        input->specialBuffer(), input->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(),
        begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo());
        sd::DebugHelper::checkErrorCode(stream, "unsortedSegmentSumLinearKernel failed");

  } else {

    output->assign(0);
    LongType zero = 0;
    std::vector<LongType> *dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(),1,&zero);
    auto packX = ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
    auto packZ = ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
    auto inputTads = packX->specialShapeInfo();
    auto inputTadOffsets = packX->specialOffsets();
    auto outputTads = packZ->specialShapeInfo();
    auto outputTadOffsets = packZ->specialOffsets();
    dim3 dims = segmentTad(input->sizeAt(0));
    segmentSumTadKernel<T, I><<<dims.x, dims.y, dims.z, *stream>>>(
        input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets,
        reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numOfClasses, output->specialBuffer(),
        output->specialShapeInfo(), outputTads, outputTadOffsets, indices->lengthOf());
    sd::DebugHelper::checkErrorCode(stream, "segmentSumTadKernel failed");

    delete dimensions;
    dimensions = nullptr;
  }
}
// -------------------------------------------------------------------------------------------------------------- //
void unsortedSegmentSumFunctor(LaunchContext* context, NDArray* input, NDArray* indices, LongType numOfClasses,
                               NDArray* output) {
  NDArray::prepareSpecialUse({output}, {input, indices});
  output->nullify();
  BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), unsortedSegmentSumFunctor_,
                        (context, input, indices, numOfClasses, output), SD_NUMERIC_TYPES, SD_INDEXING_TYPES);
  NDArray::registerSpecialUse({output}, {input, indices});
}

// -------------------------------------------------------------------------------------------------------------- //
// Backpropagate ops
// -------------------------------------------------------------------------------------------------------------- //
// Sorted sum backpropagate
template <typename T, typename I>
static SD_KERNEL void segmentSumBPLinearKernel(const void* inputBuf, const LongType* inputShape, const void* eps,
                                               const LongType* epsShape, const void* indicesBuf,
                                               const LongType* indicesShape, void* outputBuf,
                                               const LongType* outputShape) {
  auto x = reinterpret_cast<const T*>(inputBuf);
  auto y = reinterpret_cast<const I*>(indicesBuf);
  auto z = reinterpret_cast<T*>(outputBuf);
  auto gradOut = reinterpret_cast<const T*>(eps);
  __shared__ LongType xLen, gradLen;

  if (threadIdx.x == 0) {
    xLen = shape::length(inputShape);
    gradLen = shape::length(epsShape);
  }
  __syncthreads();

  auto start = blockIdx.x * blockDim.x + threadIdx.x;
  auto step = gridDim.x * blockDim.x;

  for (auto e = start; e < xLen; e += step) {
    auto zOffset = shape::getIndexOffset(e, outputShape);
    auto xOffset = shape::getIndexOffset(e, inputShape);
    auto yOffset = shape::getIndexOffset(e, indicesShape);
    auto classIndex = y[yOffset];
    auto gradOffsetO = shape::getIndexOffset(classIndex, epsShape);

    z[zOffset] = gradOut[gradOffsetO];
  }
}
// -------------------------------------------------------------------------------------------------------------- //
template <typename T, typename I>
static SD_KERNEL void segmentSumBPTadKernel(const void* inputBuf, const LongType* inputShape, const void* eps,
                                            const LongType* epsShape, const void* indicesBuf,
                                            const LongType* indicesShape, void* outputBuf,
                                            const LongType* outputShape, const LongType* inputTad,
                                            const LongType* inputOffsets, const LongType* gradOutTad,
                                            const LongType* gradOutOffsets, const LongType* outTad,
                                            const LongType* outOffsets) {
  __shared__ const T* x;
  __shared__ const T* gradOut;
  __shared__ const I* y;
  __shared__ T* z;
  __shared__ LongType xLen, yLen, gradLen, currentLen;

  if (threadIdx.x == 0) {
    xLen = shape::length(inputShape);
    x = reinterpret_cast<const T*>(inputBuf);
    y = reinterpret_cast<const I*>(indicesBuf);
    z = reinterpret_cast<T*>(outputBuf);
    yLen = shape::length(indicesShape);
    gradOut = reinterpret_cast<const T*>(eps);
    gradLen = shape::length(epsShape);
    currentLen = shape::length(outTad);
  }
  __syncthreads();

  for (auto i = blockIdx.x; i < yLen; i += gridDim.x) {
    auto yIndex = shape::getIndexOffset(i, indicesShape);
    auto segment = y[yIndex];
    auto currentOut = z + outOffsets[i];
    auto outGrad = gradOut + gradOutOffsets[segment];

    for (auto e = threadIdx.x; e < currentLen; e += blockDim.x) {
      currentOut[e] = outGrad[e];
    }
  }
}
// -------------------------------------------------------------------------------------------------------------- //
template <typename T, typename I>
Status segmentSumFunctorBP_(LaunchContext* context, NDArray* input, NDArray* indices, NDArray* gradOut,
                                NDArray* output) {
  auto stream = context->getCudaStream();
  NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
  if (input->isVector()  || input->isScalar()) {
    LongType loop_size = input->lengthOf();
    auto numOfClasses = gradOut->lengthOf();
    segmentSumBPLinearKernel<T, I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(
        input->specialBuffer(), input->specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
        indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo());
    sd::DebugHelper::checkErrorCode(stream, "segmentSumBPLinearKernel failed");

  } else {
    LongType zero = 0;
    std::vector<LongType> *dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), 1,&zero);
    auto packX = ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
    auto packZ = ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
    auto packGradOut = ConstantTadHelper::getInstance().tadForDimensions(gradOut->shapeInfo(), dimensions);
    auto inputTads = packX->specialShapeInfo();
    auto inputTadOffsets = packX->specialOffsets();
    auto outputTads = packZ->specialShapeInfo();
    auto outputTadOffsets = packZ->specialOffsets();
    auto gradOutTads = packGradOut->specialShapeInfo();
    auto gradOutTadOffsets = packGradOut->specialOffsets();

    segmentSumBPTadKernel<T, I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(
        input->specialBuffer(), input->specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
        indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
        inputTads, inputTadOffsets, gradOutTads, gradOutTadOffsets, outputTads, outputTadOffsets);
    sd::DebugHelper::checkErrorCode(stream, "segmentSumBPTadKernel failed");

    delete dimensions;
  }
  NDArray::registerSpecialUse({output}, {input, indices, gradOut});
  return Status::OK;
}
// -------------------------------------------------------------------------------------------------------------- //

Status segmentSumFunctorBP(LaunchContext* context, NDArray* input, NDArray* indices, NDArray* gradOut,
                               NDArray* output) {
  NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
  BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return segmentSumFunctorBP_,
                        (context, input, indices, gradOut, output), SD_FLOAT_TYPES, SD_INDEXING_TYPES);
  NDArray::registerSpecialUse({output}, {input, indices, gradOut});
}

template <typename T, typename I>
static Status unsortedSegmentSumFunctorBP_(LaunchContext* context, NDArray* input, NDArray* indices,
                                               NDArray* gradOut,
                                           LongType numOfClasses, NDArray* output) {
  auto stream = context->getCudaStream();
  NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
  if (input->isVector()  || input->isScalar()) {
    LongType loop_size = input->lengthOf();
    auto numOfClasses = gradOut->lengthOf();
    segmentSumBPLinearKernel<T, I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(
        input->specialBuffer(), input->specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
        indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo());
    sd::DebugHelper::checkErrorCode(stream, "segmentSumBPLinearKernel failed");

  } else {
    LongType zero = 0;
    std::vector<LongType> *dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), 1,&zero);
    auto packX = ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
    auto packZ = ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
    auto packGradOut = ConstantTadHelper::getInstance().tadForDimensions(gradOut->shapeInfo(), dimensions);
    auto inputTads = packX->specialShapeInfo();
    auto inputTadOffsets = packX->specialOffsets();
    auto outputTads = packZ->specialShapeInfo();
    auto outputTadOffsets = packZ->specialOffsets();
    auto gradOutTads = packGradOut->specialShapeInfo();
    auto gradOutTadOffsets = packGradOut->specialOffsets();

    segmentSumBPTadKernel<T, I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(
        input->specialBuffer(), input->specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
        indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
        inputTads, inputTadOffsets, gradOutTads, gradOutTadOffsets, outputTads, outputTadOffsets);
    sd::DebugHelper::checkErrorCode(stream, "segmentSumBPTadKernel failed");

    delete dimensions;
  }
  NDArray::registerSpecialUse({output}, {input, indices, gradOut});
  return Status::OK;
}
// -------------------------------------------------------------------------------------------------------------- //
Status unsortedSegmentSumFunctorBP(LaunchContext* context, NDArray* input, NDArray* indices, NDArray* gradOut,
                                   LongType numOfClasses, NDArray* output) {
  NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
  BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return unsortedSegmentSumFunctorBP_,
                        (context, input, indices, gradOut, numOfClasses, output), SD_FLOAT_TYPES, SD_INDEXING_TYPES);
  NDArray::registerSpecialUse({output}, {input, indices, gradOut});
}

}  // namespace helpers
}  // namespace ops
}  // namespace sd
