/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//

#include <ops/declarable/helpers/top_k.h>
#include <MmulHelper.h>
#include <NDArrayFactory.h>
#include <Status.h>

namespace nd4j {
namespace ops {
namespace helpers {

    template <typename T> 
    static void _swapRows(NDArray* matrix, int theFirst, int theSecond) {

    }
    BUILD_SINGLE_TEMPLATE(template void _swapRows, (NDArray* matrix, int theFirst, int theSecond), FLOAT_TYPES);

    void swapRows(NDArray* matrix, int theFirst, int theSecond) {
        BUILD_SINGLE_SELECTOR(matrix->dataType(), _swapRows, (matrix, theFirst, theSecond), FLOAT_TYPES);
    }

    template <typename T>
    static void _invertLowerMatrix(NDArray* inputMatrix, NDArray* invertedMatrix) {

    }

    BUILD_SINGLE_TEMPLATE(template void _invertLowerMatrix, (NDArray* inputMatrix, NDArray* invertedMatrix);, FLOAT_TYPES);

    void invertLowerMatrix(NDArray* inputMatrix, NDArray* invertedMatrix) {
        BUILD_SINGLE_SELECTOR(inputMatrix->dataType(), _invertLowerMatrix, (inputMatrix, invertedMatrix), FLOAT_TYPES);
    }

    template <typename T>
    static void _invertUpperMatrix(NDArray* inputMatrix, NDArray* invertedMatrix) {

    }

    BUILD_SINGLE_TEMPLATE(template void _invertUpperMatrix, (NDArray* inputMatrix, NDArray* invertedMatrix);, FLOAT_TYPES);

    void invertUpperMatrix(NDArray* inputMatrix, NDArray* invertedMatrix) {
        BUILD_SINGLE_SELECTOR(inputMatrix->dataType(), _invertUpperMatrix, (inputMatrix, invertedMatrix), FLOAT_TYPES);
    }


    template <typename T>
    static NDArray _lup(NDArray* input, NDArray* compound, NDArray* permutation) {
        NDArray determinant = NDArrayFactory::create<T>(1.f);

        return determinant;
    }
    BUILD_SINGLE_TEMPLATE(template NDArray _lup, (NDArray* input, NDArray* output, NDArray* permutation), FLOAT_TYPES);

    template <typename T>
    static int _determinant(NDArray* input, NDArray* output) {
        return Status::OK();
    }

    BUILD_SINGLE_TEMPLATE(template int _determinant, (NDArray* input, NDArray* output), FLOAT_TYPES);

    int determinant(NDArray* input, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), return _determinant, (input, output), FLOAT_TYPES);
    }

    template <typename T>
    int log_abs_determinant_(NDArray* input, NDArray* output) {
        return ND4J_STATUS_OK;
    }

    BUILD_SINGLE_TEMPLATE(template int log_abs_determinant_, (NDArray* input, NDArray* output), FLOAT_TYPES);

    int log_abs_determinant(NDArray* input, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), return log_abs_determinant_, (input, output), FLOAT_TYPES);
    }

    template <typename T>
    static int _inverse(NDArray* input, NDArray* output) {
        return Status::OK();
    }

    int inverse(NDArray* input, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), return _inverse, (input, output), FLOAT_TYPES);
    }

    template <typename T>
    int cholesky_(NDArray* input, NDArray* output, bool inplace) {
        return Status::OK();
    }

    int cholesky(NDArray* input, NDArray* output, bool inplace) {
        BUILD_SINGLE_SELECTOR(input->dataType(), return cholesky_, (input, output, inplace), FLOAT_TYPES);
    }    
    BUILD_SINGLE_TEMPLATE(template int cholesky_, (NDArray* input, NDArray* output, bool inplace), FLOAT_TYPES);
    BUILD_SINGLE_TEMPLATE(template int _inverse, (NDArray* input, NDArray* output), FLOAT_TYPES);

}
}
}
