#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//

#include <ops/declarable/helpers/top_k.h>
#include <MmulHelper.h>
#include <NDArrayFactory.h>
#include <Status.h>
#include <ConstantTadHelper.h>
#include <ShapeUtils.h>

#include <hipsolver.h>
#include <cuda_exception.h>

namespace nd4j {
namespace ops {
namespace helpers {

    template <typename T>
    static __device__ void swapRows_(T* matrix, Nd4jLong* shape, int theFirst, int theSecond, Nd4jLong N) {
        if (theFirst != theSecond) {
            auto start = threadIdx.x + blockIdx.x * blockDim.x;
            auto step = blockDim.x * gridDim.x;
            for (auto i = start; i < N; i += step) {
                Nd4jLong iCoord1[] = {theFirst, i};
                Nd4jLong iCoord2[] = {theSecond, i};
                auto iIndex1 = shape::getOffset(0, shape::shapeOf(shape), shape::stride(shape), iCoord1, 2);
                auto iIndex2 = shape::getOffset(0, shape::shapeOf(shape), shape::stride(shape), iCoord2, 2);
                //atomicExch(&matrix[iIndex1], matrix[iIndex2]);
                T e0 = matrix[iIndex1];
                T e1 = matrix[iIndex2];
                matrix[iIndex1] = e0;
                matrix[iIndex2] = e1;
            }
        }
    }
//    BUILD_SINGLE_TEMPLATE(template void swapRows_, (NDArray* matrix, int theFirst, int theSecond), FLOAT_TYPES);
//
//    void swapRows(NDArray* matrix, int theFirst, int theSecond) {
//        BUILD_SINGLE_SELECTOR(matrix->dataType(), swapRows_, (matrix, theFirst, theSecond), FLOAT_TYPES);
//    }
    template <typename T>
    static __global__ void invertKernelLow(void* invertedBuf, Nd4jLong* invertedShape, void* inputBuf, Nd4jLong* inputShape, Nd4jLong n) {
        __shared__ T* inverted;
        __shared__ T* input;

        if (threadIdx.x == 0) {
            inverted = reinterpret_cast<T*>(invertedBuf);
            input = reinterpret_cast<T*>(inputBuf);
        }
        __syncthreads();

        auto start = threadIdx.x + blockIdx.x * blockDim.x;
        auto step = blockDim.x * gridDim.x;

        for (int i = start + 1; i < n; i += step) {
            Nd4jLong pos[] = {i, i - 1};
            auto xIndex = shape::getOffset(0, shape::shapeOf(inputShape), shape::stride(inputShape), pos, 2);
            auto zIndex = shape::getOffset(0, shape::shapeOf(invertedShape), shape::stride(invertedShape), pos, 2);
            inverted[zIndex] = -input[xIndex];
        }
    }

    template <typename T>
    static __global__ void upvertKernel(void* invertedBuf, Nd4jLong* invertedShape, void* inputBuf, Nd4jLong* inputShape, Nd4jLong n) {
        __shared__ T* inverted;
        __shared__ T* input;

        if (threadIdx.x == 0) {
            inverted = reinterpret_cast<T*>(invertedBuf);
            input = reinterpret_cast<T*>(inputBuf);
        }
        __syncthreads();

        auto start = threadIdx.x + blockIdx.x * blockDim.x;
        auto step = blockDim.x * gridDim.x;

        for (int i = start + 1; i < n; i += step) {
            Nd4jLong pos[] = {i, i};
            auto xIndex = shape::getOffset(0, shape::shapeOf(inputShape), shape::stride(inputShape), pos, 2);
            auto zIndex = shape::getOffset(0, shape::shapeOf(invertedShape), shape::stride(invertedShape), pos, 2);
            inverted[zIndex] /= input[xIndex];
        }
    }

    template <typename T>
    static __global__ void upvertKernelUp(void* invertedBuf, Nd4jLong* invertedShape, void* inputBuf, Nd4jLong* inputShape, Nd4jLong n) {
        __shared__ T* inverted;
        __shared__ T* input;

        if (threadIdx.x == 0) {
            inverted = reinterpret_cast<T*>(invertedBuf);
            input = reinterpret_cast<T*>(inputBuf);
        }
        __syncthreads();

        auto start = threadIdx.x + blockIdx.x * blockDim.x;
        auto step = blockDim.x * gridDim.x;

        for (int i = start + 1; i < n - 1; i += step) {
            Nd4jLong pos[] = {i, i + 1};
            Nd4jLong posY[] = {i, i};
            Nd4jLong posX[] = {i + 1, i + 1};
            auto xIndex = shape::getOffset(0, shape::shapeOf(inputShape), shape::stride(inputShape), pos, 2);
            auto yIndex = shape::getOffset(0, shape::shapeOf(inputShape), shape::stride(inputShape), pos, 2);
//            auto yIndex = shape::getOffset(0, shape::shapeOf(inputShape), shape::stride(inputShape), pos, 2);
            auto iIndex = shape::getOffset(0, shape::shapeOf(invertedShape), shape::stride(invertedShape), posX, 2);
            auto zIndex = shape::getOffset(0, shape::shapeOf(invertedShape), shape::stride(invertedShape), pos, 2);
            inverted[zIndex] -= input[xIndex] * inverted[iIndex] / input[yIndex];
            //inputMatrix->t<T>(i, i + 1) * invertedMatrix->t<T>(i + 1, i + 1) / inputMatrix->t<T>(i, i)
        }
    }

    template <typename T>
    static __global__ void invertLowKernel(void* invertedBuf, Nd4jLong* invertedShape, void* inputBuf, Nd4jLong* inputShape, Nd4jLong n) {
        __shared__ T* inverted;
        __shared__ T* input;

        if (threadIdx.x == 0) {
            inverted = reinterpret_cast<T*>(invertedBuf);
            input = reinterpret_cast<T*>(inputBuf);
        }
        __syncthreads();

//        auto start = threadIdx.x + blockIdx.x * blockDim.x;
//        auto step = blockDim.x * gridDim.x;

        for (int i = blockIdx.x + 2; i < n; i += gridDim.x) {
            for (int j = i - 2; j > -1; --j)
                for (int k = threadIdx.x; k < i; k+= blockDim.x) {
                    Nd4jLong posZ[] = {i, j};
                    Nd4jLong posX[] = {k, j};
                    Nd4jLong posY[] = {i, k};

                    auto xIndex = shape::getOffset(0, shape::shapeOf(inputShape), shape::stride(inputShape), posX, 2);
                    auto yIndex = shape::getOffset(0, shape::shapeOf(invertedShape), shape::stride(invertedShape), posY, 2);
                    auto zIndex = shape::getOffset(0, shape::shapeOf(invertedShape), shape::stride(invertedShape), posZ, 2);
                    inverted[zIndex] -= inverted[yIndex] * input[xIndex];
                }
        }
    }

    template <typename T>
    static __global__ void invertUpKernel(void* invertedBuf, Nd4jLong* invertedShape, void* inputBuf, Nd4jLong* inputShape, Nd4jLong n) {
        __shared__ T* inverted;
        __shared__ T* input;

        if (threadIdx.x == 0) {
            inverted = reinterpret_cast<T*>(invertedBuf);
            input = reinterpret_cast<T*>(inputBuf);
        }
        __syncthreads();

//        auto start = threadIdx.x + blockIdx.x * blockDim.x;
//        auto step = blockDim.x * gridDim.x;

        for (int i = n - blockIdx.x - 2; i >= 0; i -= gridDim.x) {
            for (int j = i + 2; j < n; j++)
                for (int k = i + threadIdx.x; k < n; k+= blockDim.x) {
                    Nd4jLong posZ[] = {i, j};
                    Nd4jLong posY[] = {k, j};
                    Nd4jLong posX[] = {i, k};
                    Nd4jLong posD[] = {i, i};

                    auto xIndex = shape::getOffset(0, shape::shapeOf(inputShape), shape::stride(inputShape), posX, 2);
                    auto yIndex = shape::getOffset(0, shape::shapeOf(invertedShape), shape::stride(invertedShape), posY, 2);
                    auto dIndex = shape::getOffset(0, shape::shapeOf(inputShape), shape::stride(inputShape), posD, 2);
                    auto zIndex = shape::getOffset(0, shape::shapeOf(invertedShape), shape::stride(invertedShape), posZ, 2);
                    inverted[zIndex] -= inverted[yIndex] * input[xIndex] / input[dIndex];
                }
        }
    }

    template <typename T>
    static void invertLowerMatrix_(NDArray* inputMatrix, NDArray* invertedMatrix) {
        int n = inputMatrix->rows();
        invertedMatrix->setIdentity();

        if (inputMatrix->isIdentityMatrix()) return;
        LaunchContext* context = inputMatrix->getContext();
        auto stream = context->getCudaStream();

        invertKernelLow<T><<<1, n, 128, *stream>>>(invertedMatrix->specialBuffer(), invertedMatrix->specialShapeInfo(), inputMatrix->specialBuffer(), inputMatrix->specialShapeInfo(), n);
        invertLowKernel<T><<<n, n, 128, *stream>>>(invertedMatrix->specialBuffer(), invertedMatrix->specialShapeInfo(), inputMatrix->specialBuffer(), inputMatrix->specialShapeInfo(), n);
    }

    BUILD_SINGLE_TEMPLATE(template void invertLowerMatrix_, (NDArray* inputMatrix, NDArray* invertedMatrix);, FLOAT_TYPES);

    void invertLowerMatrix(NDArray* inputMatrix, NDArray* invertedMatrix) {
        BUILD_SINGLE_SELECTOR(inputMatrix->dataType(), invertLowerMatrix_, (inputMatrix, invertedMatrix), FLOAT_TYPES);
    }

    template <typename T>
    static void invertUpperMatrix_(NDArray* inputMatrix, NDArray* invertedMatrix) {
        int n = inputMatrix->rows();
        invertedMatrix->setIdentity();
        auto stream = inputMatrix->getContext()->getCudaStream();
        if (inputMatrix->isIdentityMatrix()) { // the inverse for I is I
            return;
        }

        upvertKernel<T><<<1, n, 128, *stream>>>(invertedMatrix->specialBuffer(), invertedMatrix->specialShapeInfo(), inputMatrix->specialBuffer(), inputMatrix->specialShapeInfo(), n);
        upvertKernelUp<T><<<1, n, 128, *stream>>>(invertedMatrix->specialBuffer(), invertedMatrix->specialShapeInfo(), inputMatrix->specialBuffer(), inputMatrix->specialShapeInfo(), n);
        invertUpKernel<T><<<n, n, 256, *stream>>>(invertedMatrix->specialBuffer(), invertedMatrix->specialShapeInfo(), inputMatrix->specialBuffer(), inputMatrix->specialShapeInfo(), n);
    }

    BUILD_SINGLE_TEMPLATE(template void invertUpperMatrix_, (NDArray* inputMatrix, NDArray* invertedMatrix);, FLOAT_TYPES);

    void invertUpperMatrix(NDArray* inputMatrix, NDArray* invertedMatrix) {
        BUILD_SINGLE_SELECTOR(inputMatrix->dataType(), invertUpperMatrix_, (inputMatrix, invertedMatrix), FLOAT_TYPES);
    }

    template <typename T>
    static __global__ void lupKernel(T* compound, Nd4jLong* compoundShape, T* permutation, Nd4jLong* permutationShape, Nd4jLong rowNum) {
        int swapCount = 0;
        for(int i = blockIdx.x; i < rowNum; i += gridDim.x ) {
            auto pivotValue = T(0.0);
            auto pivot = -1;

            for(int rowCounter = i; rowCounter < rowNum; rowCounter++ ) {
                Nd4jLong rowCoord[] = {rowCounter, i};
                auto rowPos = shape::getOffset(0, shape::shapeOf(compoundShape), shape::stride(compoundShape), rowCoord, 2);
                if(nd4j::math::nd4j_abs(compound[rowPos]) > pivotValue ) {
                    pivotValue = nd4j::math::nd4j_abs(compound[rowPos]);
                    pivot = rowCounter;
                }
            }

            if( pivotValue != T(0.0) ) {
                swapRows_<T>(compound, compoundShape, pivot, i, rowNum);
                swapRows_<T>(permutation, permutationShape, pivot, i, rowNum);
                if (pivot != i)
                    swapCount++;

                for( int j = i + 1; j < rowNum; j++ ) {
                    Nd4jLong posJIbuf[] = {j, i};
                    Nd4jLong posIIbuf[] = {i, i};
                    auto posJI = shape::getOffset(0, shape::shapeOf(compoundShape), shape::stride(compoundShape), posJIbuf, 2);
                    auto posII = shape::getOffset(0, shape::shapeOf(compoundShape), shape::stride(compoundShape), posIIbuf, 2);

                    compound[posJI] /= compound[posII];
                    for( int k = i + 1; k < rowNum; k++ ) {
                        Nd4jLong posJKbuf[] = {j, k};
                        Nd4jLong posIKbuf[] = {i, k};
                        auto posJK = shape::getOffset(0, shape::shapeOf(compoundShape), shape::stride(compoundShape), posJKbuf, 2);
                        auto posIK = shape::getOffset(0, shape::shapeOf(compoundShape), shape::stride(compoundShape), posIKbuf, 2);
                        T arg = compound[posJI] * compound[posIK];
                        compound[posJK] -= arg;
                    }
                }
            }
        }
    }

    template <typename T, typename F>
    static __global__ void determinantKernel(T* compound, T* result, Nd4jLong len) {
        __shared__ F tempRes;
        if (blockIdx.x == 0) {
            tempRes = (F)result[0];
        }
        __syncthreads();

        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;
        for (auto i = start; i < len; i += step) {
            auto pos = i * len + i; //shape::getOffset(0, shape::shapeOf(shape), shape::stride(shape), di, 2);
            math::atomics::nd4j_atomicMul<F>(&tempRes, (F)compound[pos]);
        }
        __syncthreads();

        if (blockIdx.x == 0) {
            result[0] = (T)tempRes;
        }
    }

        template <typename T, typename F>
        static __global__ void determinantLogKernel(T* compound, T* result, Nd4jLong len) {
            __shared__ F tempRes;
            if (blockIdx.x == 0) {
                tempRes = (F)result[0];
            }
            __syncthreads();

            auto start = blockIdx.x * blockDim.x + threadIdx.x;
            auto step = blockDim.x * gridDim.x;
            for (auto i = start; i < len; i += step) {
                auto pos = i * len + i; //shape::getOffset(0, shape::shapeOf(shape), shape::stride(shape), di, 2);
                math::atomics::nd4j_atomicMul<F>(&tempRes, (F)compound[pos]);
            }
            __syncthreads();

            if (blockIdx.x == 0) {
                result[0] = (T)math::nd4j_log<F,F>(math::nd4j_abs(tempRes));
            }
        }

    template <typename T, typename F>
    static __global__ void fillMatrix(void* output, Nd4jLong* outShape, void* input, Nd4jLong* inputShape, Nd4jLong pos, Nd4jLong rowLen) {
        __shared__ F* matrix;
        __shared__ T* inputBuf;
        __shared__ Nd4jLong inputLen;
        __shared__ Nd4jLong n2;

        if (threadIdx.x == 0) {
            matrix = reinterpret_cast<F*>(output);
            inputBuf = reinterpret_cast<T*>(input);
            inputLen = shape::length(inputShape);
            n2 = rowLen * rowLen;
        }
        __syncthreads();
        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;

        for (int k = pos + start, j = start; j < n2; k += step, j += step) {
            auto xIndex = shape::getIndexOffset(k, inputShape, inputLen);
            matrix[j] = (F)inputBuf[xIndex];
        }
    }
    template <typename F>
    static __global__ void fillUpPermutation(void* output, Nd4jLong* shape, int* source, int rowNum) {
        __shared__ F* permutation;

        if (threadIdx.x == 0) {
            permutation = reinterpret_cast<F*>(output);
        }
        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;
        for (auto i = start; i < rowNum; i += step) {
            int val = source[i] - 1;
            Nd4jLong posF[] = {i, val};
            auto pos = shape::getOffset(0, shape::shapeOf(shape), shape::stride(shape), posF, 2);
            permutation[pos] = F(1.f);
        }
    }

    template <typename T>
    static void lup_(LaunchContext* context, NDArray* input, NDArray* compound, NDArray* permutation) {
        auto stream = context->getCudaStream();
        auto n = input->rows();
        hipsolverHandle_t cusolverH = nullptr;
        hipsolverStatus_t status = hipsolverDnCreate(&cusolverH);
        if (HIPSOLVER_STATUS_SUCCESS != status) {
            throw cuda_exception::build("Cannot create cuSolver handle", status);
        }
        status = hipsolverSetStream(cusolverH, *stream);
        if (HIPSOLVER_STATUS_SUCCESS != status) {
            throw cuda_exception::build("Cannot set up stream for cuda solver", status);
        }
        int lwork = 0;
        int *d_info = nullptr;

        auto err = hipMalloc((void **) &d_info, sizeof(int));
        if (err) {
            throw cuda_exception::build("helpers::lup_: Cannot allocate memory for solver info buffer", err);
        }

        DataType dtype = input->dataType();
        switch(dtype) {

            case DataType::DOUBLE: {
                double *d_work = nullptr;
                err = hipMalloc((void **) &d_work, sizeof(float) * lwork);
                if (err) {
                    throw cuda_exception::build("helpers::lup_: Cannot allocate memory for solver data buffer", err);
                }
                double *matrix = reinterpret_cast<double*>(input->specialBuffer());
                status = hipsolverDnDgetrf_bufferSize(
                        cusolverH,
                        n,
                        n,
                        matrix,
                        n,
                        &lwork);
                if (HIPSOLVER_STATUS_SUCCESS != status) {
                    throw cuda_exception::build("helpers::lup_: Cannot create cuSolver handle", status);
                }
                if (permutation == nullptr)
                    status = hipsolverDnDgetrf(
                            cusolverH,
                            n,
                            n,
                            matrix,
                            n,
                            d_work,
                            nullptr,
                            d_info);
                else {
                    NDArray permutVector('c', {n}, nd4j::DataType::INT32, context);
                    int *permutationBuf = reinterpret_cast<int *>(permutVector.specialBuffer());
                    status = hipsolverDnDgetrf(
                            cusolverH,
                            n,
                            n,
                            matrix,
                            n,
                            d_work,
                            permutationBuf,
                            d_info);
                    fillUpPermutation<double><<<n, n, 128, *stream>>>(permutation->specialBuffer(), permutation->specialShapeInfo(), permutationBuf, n);
                    permutation->tickWriteDevice();
                }
                err = hipFree(d_work);
                if (err) {
                    throw cuda_exception::build("helpers::lup_: Cannot deallocate memory for solver data buffer", err);
                }
            }
                break;
            case DataType::FLOAT32: {
                float *matrix = reinterpret_cast<float*>(input->specialBuffer());
                float *d_work = nullptr;
                err = hipMalloc((void **) &d_work, sizeof(float) * lwork);
                if (err) {
                    throw cuda_exception::build("helpers::lup_: Cannot allocate memory for solver data buffer", err);
                }

                status = hipsolverDnSgetrf_bufferSize(
                        cusolverH,
                        n,
                        n,
                        matrix,
                        n,
                        &lwork);
                if (HIPSOLVER_STATUS_SUCCESS != status) {
                    throw cuda_exception::build("helpers::lup_: Cannot create cuSolver handle", status);
                }

                if (permutation == nullptr)
                    status = hipsolverDnSgetrf(
                            cusolverH,
                            n,
                            n,
                            matrix,
                            n,
                            d_work,
                            nullptr,
                            d_info);
                else {
                    NDArray permutVector('c', {n}, nd4j::DataType::INT32, context);
                    int *permutationBuf = reinterpret_cast<int *>(permutVector.specialBuffer());
                    status = hipsolverDnSgetrf(
                            cusolverH,
                            n,
                            n,
                            matrix,
                            n,
                            d_work,
                            permutationBuf,
                            d_info);
                    fillUpPermutation<float><<<n, n, 128, *stream>>>(permutation->specialBuffer(), permutation->specialShapeInfo(), permutationBuf, n);
                    permutation->tickWriteDevice();
                }
                err = hipFree(d_work);
                if (err) {
                    throw cuda_exception::build("helpers::lup_: Cannot deallocate memory for solver data buffer", err);
                }

            }
        }
        if (HIPSOLVER_STATUS_SUCCESS != status) {
            throw cuda_exception::build("helpers::lup_: Cannot make LU decomposition", status);
        }
        err = hipFree(d_info);
        if (err) {
            throw cuda_exception::build("helpers::lup_: Cannot deallocate memory for solver info buffer", err);
        }
        hipsolverDnDestroy(cusolverH);
//        NDArray::registerSpecialUse({input}, {input});
        input->tickWriteDevice();
    }
    BUILD_SINGLE_TEMPLATE(template void lup_, (LaunchContext* context, NDArray* input, NDArray* output, NDArray* permutation), FLOAT_TYPES);

    template <typename T>
    static int determinant_(nd4j::LaunchContext* context, NDArray* input, NDArray* output) {
        Nd4jLong n = input->sizeAt(-1);
        Nd4jLong n2 = n * n;
        std::vector<int> dims();
        auto packX = ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), {input->rankOf() - 2, input->rankOf() - 1});
        //auto packZ = ConstantTadHelper::getInstance()->tadForDimensions(output->shapeInfo(), {output->rankOf() - 1});
        DataType dtype = input->dataType();
        if (dtype != DataType::DOUBLE)
            dtype = DataType::FLOAT32;

        auto matrix = NDArrayFactory::create(input->ordering(), {n, n}, dtype, input->getContext()); //, block.getWorkspace());
        auto det = NDArrayFactory::create<T>(1);
        auto stream = context->getCudaStream();
        NDArray::prepareSpecialUse({output}, {input});
        dim3 launchDims(256, 256, 1024);
        output->assign(1.f);
        for (int e = 0; e < output->lengthOf(); e++) {
            Nd4jLong pos = e * n2;
            if (matrix.dataType() == input->dataType())
                fillMatrix<T, T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(matrix.specialBuffer(), matrix.specialShapeInfo(), input->specialBuffer(), input->specialShapeInfo(), pos, n);
            else
                fillMatrix<T, float><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(matrix.specialBuffer(), matrix.specialShapeInfo(), input->specialBuffer(), input->specialShapeInfo(), pos, n);

            if (matrix.dataType() == input->dataType())
                lup_<T>(context, &matrix, nullptr, nullptr);
            else
                lup_<float>(context, &matrix, nullptr, nullptr);
            auto offset = shape::getIndexOffset(e, output->shapeInfo(), output->lengthOf());
            auto inputBuf = reinterpret_cast<T*>(matrix.specialBuffer());
            auto outputBuf = reinterpret_cast<T*>(output->specialBuffer()) + offset;
            if (matrix.dataType() == input->dataType())
                determinantKernel<T, T><<<launchDims.x, launchDims.y, launchDims.z, *stream >>> (inputBuf, outputBuf, n);
            else
                determinantKernel<T, float><<<launchDims.x, launchDims.y, launchDims.z, *stream >>> (inputBuf, outputBuf, n);
        }
        NDArray::registerSpecialUse({output}, {input});

        return Status::OK();
    }

    BUILD_SINGLE_TEMPLATE(template int determinant_, (nd4j::LaunchContext* context, NDArray* input, NDArray* output), FLOAT_TYPES);

    int determinant(nd4j::LaunchContext * context, NDArray* input, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), return determinant_, (context, input, output), FLOAT_TYPES);
    }

    template <typename T>
    int logAbsDeterminant_(LaunchContext* context, NDArray* input, NDArray* output) {

        Nd4jLong n = input->sizeAt(-1);
        Nd4jLong n2 = n * n;
        std::vector<int> dims();
        auto packX = ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), {input->rankOf() - 2, input->rankOf() - 1});
        //auto packZ = ConstantTadHelper::getInstance()->tadForDimensions(output->shapeInfo(), {output->rankOf() - 1});
        DataType dtype = input->dataType();
        if (dtype != DataType::DOUBLE)
            dtype = DataType::FLOAT32;

        auto matrix = NDArrayFactory::create(input->ordering(), {n, n}, dtype, input->getContext()); //, block.getWorkspace());
        auto det = NDArrayFactory::create<T>(1);
        auto stream = context->getCudaStream();
        NDArray::prepareSpecialUse({output}, {input});
        dim3 launchDims(256, 256, 1024);
        output->assign(1.f);
        for (int e = 0; e < output->lengthOf(); e++) {
            Nd4jLong pos = e * n2;
            if (matrix.dataType() == input->dataType())
                fillMatrix<T, T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(matrix.specialBuffer(), matrix.specialShapeInfo(), input->specialBuffer(), input->specialShapeInfo(), pos, n);
            else
                fillMatrix<T, float><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(matrix.specialBuffer(), matrix.specialShapeInfo(), input->specialBuffer(), input->specialShapeInfo(), pos, n);

            if (matrix.dataType() == input->dataType())
                lup_<T>(context, &matrix, nullptr, nullptr);
            else
                lup_<float>(context, &matrix, nullptr, nullptr);
            auto offset = shape::getIndexOffset(e, output->shapeInfo(), output->lengthOf());
            auto inputBuf = reinterpret_cast<T*>(matrix.specialBuffer());
            auto outputBuf = reinterpret_cast<T*>(output->specialBuffer()) + offset;
            if (matrix.dataType() == input->dataType())
                determinantLogKernel<T, T><<<launchDims.x, launchDims.y, launchDims.z, *stream >>> (inputBuf, outputBuf, n);
            else
                determinantLogKernel<T, float><<<launchDims.x, launchDims.y, launchDims.z, *stream >>> (inputBuf, outputBuf, n);
        }
        NDArray::registerSpecialUse({output}, {input});

        return Status::OK();

        return ND4J_STATUS_OK;
    }

    BUILD_SINGLE_TEMPLATE(template int logAbsDeterminant_, (LaunchContext* context, NDArray* input, NDArray* output), FLOAT_TYPES);

    int logAbsDeterminant(nd4j::LaunchContext * context, NDArray* input, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), return logAbsDeterminant_, (context, input, output), FLOAT_TYPES);
    }

    template <typename T>
    static __global__ void fillLowerUpperKernel(void* lowerBuf, Nd4jLong* lowerShape, void* upperBuf, Nd4jLong* upperShape, void* matrixBuf, Nd4jLong* matrixShape, Nd4jLong n) {

        __shared__ Nd4jLong* xShapeOf;
        __shared__ Nd4jLong* yShapeOf;
        __shared__ Nd4jLong* zShapeOf;
        __shared__ Nd4jLong* xStrideOf;
        __shared__ Nd4jLong* yStrideOf;
        __shared__ Nd4jLong* zStrideOf;
        __shared__ T* lowerMatrix;
        __shared__ T* upperMatrix;
        __shared__ T* matrix;

        if (threadIdx.x == 0) {
            xShapeOf = shape::shapeOf(lowerShape);
            yShapeOf = shape::shapeOf(upperShape);
            zShapeOf = shape::shapeOf(matrixShape);
            xStrideOf = shape::stride(lowerShape);
            yStrideOf = shape::stride(upperShape);
            zStrideOf = shape::stride(matrixShape);
            lowerMatrix = reinterpret_cast<T*>(lowerBuf);
            upperMatrix = reinterpret_cast<T*>(upperBuf);
            matrix = reinterpret_cast<T*>(matrixBuf);
        }
        __syncthreads();

        for (int k = blockIdx.x; k < n; k += gridDim.x) {  // and then put all values under main diagonal on to it
            for (int j = threadIdx.x; j < n; j += blockDim.x) {
                Nd4jLong posX[] = {j, k};

                auto xPos = shape::getOffset(0, xShapeOf, xStrideOf, posX, 2);
                auto yPos = shape::getOffset(0, yShapeOf, yStrideOf, posX, 2);
                auto pos =  shape::getOffset(0, zShapeOf, zStrideOf, posX, 2);
                if (k <= j)
                    lowerMatrix[xPos] = matrix[pos];//(k, j);
                else
                    upperMatrix[yPos] = matrix[pos]; //k, j);
            }
        }
    }

    template <typename T>
    static int inverse_(nd4j::LaunchContext* context, NDArray* input, NDArray* output) {
        auto n = input->sizeAt(-1);
        auto n2 = n * n;
        auto dtype = input->dataType();
        if (dtype != DataType::DOUBLE)
            dtype = DataType::FLOAT32;
        NDArray matrix = NDArrayFactory::create('c', {n, n}, dtype, input->getContext());
        NDArray upper = NDArrayFactory::create('c', {n, n}, dtype, input->getContext());
        NDArray lower = NDArrayFactory::create('c', {n, n}, dtype, input->getContext());
        NDArray compound = NDArrayFactory::create('c', {n, n}, dtype, input->getContext());
        NDArray permutation = NDArrayFactory::create('c', {n, n}, dtype, input->getContext());
        auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), {input->rankOf() - 2, input->rankOf() - 1});
        auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), {output->rankOf() - 2, output->rankOf() - 1});
        auto stream = context->getCudaStream();

        for (auto i = 0LL; i < packX.numberOfTads(); i++) {
            fillMatrix<T, float><<<1, n2, 128, *stream>>>(matrix.specialBuffer(), matrix.specialShapeInfo(), input->specialBuffer(), input->specialShapeInfo(), i * n2, n);
            permutation.assign(0.f);
            lup_<float>(context, &matrix, &compound, &permutation);
            matrix.tickWriteDevice();
            permutation.tickWriteDevice();
            permutation.printIndexedBuffer("PERMUTE");
            lower.setIdentity(); // set up U to identity matrix
            upper.setIdentity();
            fillLowerUpperKernel<float><<<1, n2, 128>>>(lower.specialBuffer(), lower.specialShapeInfo(), upper.specialBuffer(), upper.specialShapeInfo(), matrix.specialBuffer(), matrix.specialShapeInfo(), n);
            lower.tickWriteDevice();
            upper.tickWriteDevice();
            invertUpperMatrix(&upper, &matrix);
            invertLowerMatrix(&lower, &upper);
            lower.tickWriteDevice();
            upper.tickWriteDevice();
            lower.printIndexedBuffer("LOWER");
            upper.printIndexedBuffer("UPPER");

            nd4j::MmulHelper::mmul(&matrix, &upper, &compound, 1.0, 0.0);
            nd4j::MmulHelper::mmul(&compound, &permutation, &matrix, 1.0, 0.0);
//            for (int k = e * n2, row = 0; k < (e + 1) * n2; k++) {
//                output->t<T>(k) = matrix.template t<T>(row++);
//            }
        }


        return Status::OK();
    }

    int inverse(nd4j::LaunchContext * context, NDArray* input, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), return inverse_, (context, input, output), FLOAT_TYPES);
    }

    bool checkCholeskyInput(nd4j::LaunchContext * context, NDArray const* input) {
        return true;
    }

    template <typename F>
    __global__ void fillBatchKernel(F** dArrayBatch, F* buf, Nd4jLong* offsets, Nd4jLong batchSize) {
        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;

        for (auto i = start; i < batchSize; i += step) {
            dArrayBatch[i] = buf + offsets[i];
        }
    }

    template <typename F>
    __global__ void adjustResultsKernel(F* dArray, Nd4jLong* shape, Nd4jLong* offsets, Nd4jLong batchSize, Nd4jLong n) {
        //auto i = blockIdx.x * blockDim.x + threadIdx.x;
        __shared__ Nd4jLong* shapeOf;
        __shared__ Nd4jLong* strideOf;
        if (blockIdx.x == 0 && threadIdx.x == 0) {
            shapeOf = shape::shapeOf(shape);
            strideOf = shape::stride(shape);
        }
        __syncthreads();

        for (auto i = blockIdx.x; i < batchSize; i+= gridDim.x) {
            auto current = dArray + offsets[i];
            for (auto r = threadIdx.x; r < n; r += blockDim.x) {
                for (auto c = r + 1; c < n; c++) {
                    Nd4jLong posRC[] = {r, c};
                    auto pos = r * n + c; //shape::getOffset(0, shapeOf, strideOf, posRC, 2);
                    current[pos] = 0.;
                }
            }
        }
    }

    template <typename F>
    int cholesky__(LaunchContext* context, NDArray* input, NDArray* output, bool inplace) {
        if (!inplace)
            output->assign(input);
        std::unique_ptr<NDArray> tempOutput(output->dup());
        hipsolverHandle_t handle = nullptr;
        auto n = input->sizeAt(-1);
        auto n2 = n * n;
        NDArray::prepareSpecialUse({output}, {input});
        auto status = hipsolverDnCreate(&handle);
        if (HIPSOLVER_STATUS_SUCCESS != status) {
            throw cuda_exception::build("helpers::cholesky_: Cannot create solver handle", status);
        }
        F** dArrayBatch = nullptr;
        auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(tempOutput->getShapeInfo(), {tempOutput->rankOf() - 2, tempOutput->rankOf() - 1});
        const Nd4jLong batchSize = packX.numberOfTads();
        int* dInfoArray = nullptr;
        auto err = hipMalloc((void**)&dArrayBatch, sizeof(F*) * batchSize);
        if (err) {
            throw cuda_exception::build("helpers::cholesky_: Cannot allocate memory for solver batch data buffer", err);
        }
        err = hipMalloc ((void**)&dInfoArray, sizeof(int) * batchSize);
        if (err) {
            throw cuda_exception::build("helpers::cholesky_: Cannot allocate memory for solver errors buffer", err);
        }
        auto stream = context->getCudaStream();
        fillBatchKernel<F><<<1, batchSize, 128, *stream>>>(dArrayBatch, reinterpret_cast<F*>(tempOutput->specialBuffer()), packX.specialOffsets(), batchSize);

        status = hipsolverSetStream(handle, *stream);
        if (HIPSOLVER_STATUS_SUCCESS != status) {
            throw cuda_exception::build("helpers::cholesky_: Cannot set stream to solver handle", status);
        }
        const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER;
        if (input->dataType() == DataType::DOUBLE)
        status = hipsolverDnDpotrfBatched(
                handle,
                uplo,
                n,
                (double**)dArrayBatch,
                n,
                dInfoArray,
                batchSize);
        else
        status = hipsolverDnSpotrfBatched(
                    handle,
                    uplo,
                    n,
                    (float**)dArrayBatch,
                    n,
                    dInfoArray,
                    batchSize);

        if (HIPSOLVER_STATUS_SUCCESS != status) {
            throw cuda_exception::build("helpers::cholesky_: Cholesky factorization failed for batch", status);
        }
        adjustResultsKernel<F><<<batchSize, n2, 128, *stream>>>(reinterpret_cast<F*>(tempOutput->specialBuffer()), packX.specialShapeInfo(), packX.specialOffsets(), batchSize, n);

        err = hipFree(dArrayBatch);
        if (err) {
            throw cuda_exception::build("helpers::cholesky_: Cannot deallocate memory for solver batch data buffer", err);
        }
        err = hipFree(dInfoArray);
        if (err) {
            throw cuda_exception::build("helpers::cholesky_: Cannot allocate memory for solver errors buffer", err);
        }

        if(!inplace)
            output->assign(tempOutput.get());

        NDArray::registerSpecialUse({output}, {input});
        return Status::OK();
    }

//    template <typename T>
    int cholesky_(LaunchContext* context, NDArray* input, NDArray* output, bool inplace) {
        if (input->dataType() == DataType::DOUBLE)
            cholesky__<double>(context, input, output, inplace);
        else if (input->dataType() == DataType::FLOAT32)
            cholesky__<float>(context, input, output, inplace);
        else {
            std::unique_ptr<NDArray> tempOutput(NDArrayFactory::create_('c', input->getShapeAsVector(), DataType::FLOAT32, input->getContext()));
            tempOutput->assign(input);
            cholesky__<float>(context, tempOutput.get(), tempOutput.get(), true);
            output->assign(tempOutput.get());
        }
        return Status::OK();
    }

    int cholesky(nd4j::LaunchContext* context, NDArray* input, NDArray* output, bool inplace) {
//        BUILD_SINGLE_SELECTOR(input->dataType(), return cholesky_, (context, input, output, inplace), FLOAT_TYPES);
        return cholesky_(context, input, output, inplace);
    }
//    BUILD_SINGLE_TEMPLATE(template int cholesky_, (LaunchContext* context, NDArray* input, NDArray* output, bool inplace), FLOAT_TYPES);
    BUILD_SINGLE_TEMPLATE(template int inverse_, (nd4j::LaunchContext* context, NDArray* input, NDArray* output), FLOAT_TYPES);

    __global__ void logDetKernel(void* inputBuf, Nd4jLong* inputShape, Nd4jLong batchNum, Nd4jLong* tadShape, Nd4jLong* tadOffsets, void* outputBuf, Nd4jLong* outputShape) {
        __shared__ double* output;
        __shared__ double* input;
        __shared__ int n2;
        if (threadIdx.x == 0) {
            output = reinterpret_cast<double*>(outputBuf);
            input = reinterpret_cast<double*>(inputBuf);
            n2 = shape::sizeAt(inputShape, -1) * shape::sizeAt(inputShape, -1);
        }
        __syncthreads();

        for (Nd4jLong i = blockIdx.x; i < batchNum; i += gridDim.x) {
            double* current = input + tadOffsets[i];
            Nd4jLong* shapeOf = shape::shapeOf(tadShape);
            Nd4jLong* strideOf = shape::stride(tadShape);
            auto zIndex = shape::getIndexOffset(i, outputShape, batchNum);
            for (Nd4jLong e = threadIdx.x; e < n2; e += blockDim.x) {
                Nd4jLong diag[] = {e, e};
                auto xIndex = shape::getOffset(0, shapeOf, strideOf, diag, 2);
                math::atomics::nd4j_atomicAdd(&output[zIndex], math::nd4j_log<double,double>(current[xIndex] * current[xIndex]));
            }
        }
    }

    int logdetFunctor(nd4j::LaunchContext* context, NDArray* input, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input});
        auto tempOutput = input->dup('c');
        auto n2 = input->sizeAt(-1) * input->sizeAt(-2);
        auto stream = context->getCudaStream();
        cholesky(context, tempOutput, tempOutput, true);
        auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(tempOutput->getShapeInfo(), {tempOutput->rankOf() - 2, tempOutput->rankOf() - 1});
        //for (Nd4jLong e = 0; e < output->lengthOf(); e++) {
        auto outputBuf = reinterpret_cast<double*>(output->specialBuffer()); // + e * n2;
        logDetKernel<<<packX.numberOfTads(), n2, 128, *stream>>>(tempOutput->specialBuffer(), tempOutput->specialShapeInfo(), packX.numberOfTads(), packX.specialShapeInfo(), packX.specialOffsets(), outputBuf, output->specialShapeInfo());
        //}
        NDArray::registerSpecialUse({output}, {input});
        delete tempOutput;
        return Status::OK();
    }
}
}
}
