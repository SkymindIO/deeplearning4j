#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//

#include <ops/declarable/helpers/top_k.h>
#include <MmulHelper.h>
#include <NDArrayFactory.h>
#include <Status.h>
#include <ConstantTadHelper.h>

namespace nd4j {
namespace ops {
namespace helpers {

    template <typename T> 
    static __device__ void _swapRows(T* matrix, Nd4jLong* shape, int theFirst, int theSecond, Nd4jLong N) {
        if (theFirst != theSecond) {
            auto start = threadIdx.x + blockIdx.x * blockDim.x;
            auto step = blockDim.x * gridDim.x;
            for (auto i = start; i < N; i += step) {
                Nd4jLong iCoord1[] = {theFirst, i};
                Nd4jLong iCoord2[] = {theSecond, i};
                auto iIndex1 = shape::getOffset(0, shape::shapeOf(shape), shape::stride(shape), iCoord1, 2);
                auto iIndex2 = shape::getOffset(0, shape::shapeOf(shape), shape::stride(shape), iCoord2, 2);
                //atomicExch(&matrix[iIndex1], matrix[iIndex2]);
                T e0 = matrix[iIndex1];
                T e1 = matrix[iIndex2];
                matrix[iIndex1] = e0;
                matrix[iIndex2] = e1;
            }
        }
    }
//    BUILD_SINGLE_TEMPLATE(template void _swapRows, (NDArray* matrix, int theFirst, int theSecond), FLOAT_TYPES);
//
//    void swapRows(NDArray* matrix, int theFirst, int theSecond) {
//        BUILD_SINGLE_SELECTOR(matrix->dataType(), _swapRows, (matrix, theFirst, theSecond), FLOAT_TYPES);
//    }

    template <typename T>
    static void _invertLowerMatrix(NDArray* inputMatrix, NDArray* invertedMatrix) {

    }

    BUILD_SINGLE_TEMPLATE(template void _invertLowerMatrix, (NDArray* inputMatrix, NDArray* invertedMatrix);, FLOAT_TYPES);

    void invertLowerMatrix(NDArray* inputMatrix, NDArray* invertedMatrix) {
        BUILD_SINGLE_SELECTOR(inputMatrix->dataType(), _invertLowerMatrix, (inputMatrix, invertedMatrix), FLOAT_TYPES);
    }

    template <typename T>
    static void _invertUpperMatrix(NDArray* inputMatrix, NDArray* invertedMatrix) {

    }

    BUILD_SINGLE_TEMPLATE(template void _invertUpperMatrix, (NDArray* inputMatrix, NDArray* invertedMatrix);, FLOAT_TYPES);

    void invertUpperMatrix(NDArray* inputMatrix, NDArray* invertedMatrix) {
        BUILD_SINGLE_SELECTOR(inputMatrix->dataType(), _invertUpperMatrix, (inputMatrix, invertedMatrix), FLOAT_TYPES);
    }

    template <typename T>
    static __global__ void lupKernel(T* compound, Nd4jLong* compoundShape, T* permutation, Nd4jLong* permutationShape, Nd4jLong rowNum) {
        int swapCount = 0;
        for(int i = blockIdx.x; i < rowNum; i += gridDim.x ) {
            auto pivotValue = T(0.0);
            auto pivot = -1;

            for(int rowCounter = i; rowCounter < rowNum; rowCounter++ ) {
                Nd4jLong rowCoord[] = {rowCounter, i};
                auto rowPos = shape::getOffset(0, shape::shapeOf(compoundShape), shape::stride(compoundShape), rowCoord, 2);
                if(nd4j::math::nd4j_abs(compound[rowPos]) > pivotValue ) {
                    pivotValue = nd4j::math::nd4j_abs(compound[rowPos]);
                    pivot = rowCounter;
                }
            }

            if( pivotValue != T(0.0) ) {
                _swapRows<T>(compound, compoundShape, pivot, i, rowNum);
                _swapRows<T>(permutation, permutationShape, pivot, i, rowNum);
                if (pivot != i)
                    swapCount++;

                for( int j = i + 1; j < rowNum; j++ ) {
                    Nd4jLong posJIbuf[] = {j, i};
                    Nd4jLong posIIbuf[] = {i, i};
                    auto posJI = shape::getOffset(0, shape::shapeOf(compoundShape), shape::stride(compoundShape), posJIbuf, 2);
                    auto posII = shape::getOffset(0, shape::shapeOf(compoundShape), shape::stride(compoundShape), posIIbuf, 2);

                    compound[posJI] /= compound[posII];
                    for( int k = i + 1; k < rowNum; k++ ) {
                        Nd4jLong posJKbuf[] = {j, k};
                        Nd4jLong posIKbuf[] = {i, k};
                        auto posJK = shape::getOffset(0, shape::shapeOf(compoundShape), shape::stride(compoundShape), posJKbuf, 2);
                        auto posIK = shape::getOffset(0, shape::shapeOf(compoundShape), shape::stride(compoundShape), posIKbuf, 2);
                        T arg = compound[posJI] * compound[posIK];
                        compound[posJK] -= arg;
                    }
                }
            }
        }
    }
    template <typename T>
    static __global__ void determinantKernel(T* compound, Nd4jLong* shape, T* result) {
        __shared__ Nd4jLong len;

        if (threadIdx.x == 0) {
            len = shape::length(shape);
        }
        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;
        for (auto i = start; i < len; i += step) {
            Nd4jLong di[] = {i, i};
            auto pos = shape::getOffset(0, shape::shapeOf(shape), shape::stride(shape), di, 2);
            math::atomics::nd4j_atomicMul(result, compound[pos]);
        }
    }
    template <typename T>
    static __global__ void determinantFullKernel(T* input, Nd4jLong* inputShape, T* output, Nd4jLong* outputShape, Nd4jLong* tadShape, Nd4jLong* tadOffsets) {

    }

    template <typename T>
    static NDArray _lup(LaunchContext* context, NDArray* input, NDArray* compound, NDArray* permutation) {
        NDArray determinant = NDArrayFactory::create<T>(1.f);
        auto rowNum = input->rows();
        auto columnNum = input->columns();

        NDArray compoundMatrix = *input; // copy
        NDArray permutationMatrix(input, false, input->getContext()); // has same shape as input and contiguous strides
        permutationMatrix.setIdentity();

        T pivotValue; // = T(0.0);
        int pivot; // = -1;
        int swapCount = 0;
        T* compoundBuf = reinterpret_cast<T*>(compoundMatrix.specialBuffer());
        T* permutationBuf = reinterpret_cast<T*>(permutationMatrix.specialBuffer());
        auto stream = context->getCudaStream();
        lupKernel<T><<<256, 256, 1024, *stream>>>(compoundBuf, compoundMatrix.specialShapeInfo(), permutationBuf, permutationMatrix.specialShapeInfo(), rowNum);
        determinantKernel<T><<<256, 256, 1024, *stream>>>(compoundBuf, compoundMatrix.specialShapeInfo(), reinterpret_cast<T*>(determinant.specialBuffer()));
//        for (int e = 0; e < rowNum; e++) {
//            // nd4j_printf("Compound matrix diag %i %f.\n", e, (*compoundMatrix)(e, e));
//            determinant *= compoundMatrix.e<T>(e, e);
//        }
        if (swapCount % 2) determinant = -determinant;
        if (compound != nullptr)
            compound->assign(compoundMatrix);
        if (permutation != nullptr)
            permutation->assign(permutationMatrix);
        return determinant;
    }
    BUILD_SINGLE_TEMPLATE(template NDArray _lup, (LaunchContext* context, NDArray* input, NDArray* output, NDArray* permutation), FLOAT_TYPES);

    template <typename T>
    static int _determinant(nd4j::LaunchContext* context, NDArray* input, NDArray* output) {
        Nd4jLong n = input->sizeAt(-1);
        Nd4jLong n2 = n * n;
        std::vector<int> dims();
        auto packX = ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), {input->rankOf() - 2, input->rankOf() - 1});
        //auto packZ = ConstantTadHelper::getInstance()->tadForDimensions(output->shapeInfo(), {output->rankOf() - 1});

        //auto matrix = NDArrayFactory::create(input->ordering(), {n, n}, input->dataType(), input->getContext()); //, block.getWorkspace());
        auto stream = context->getCudaStream();
        auto inputBuf = reinterpret_cast<T*>(input->specialBuffer());
        auto outputBuf = reinterpret_cast<T*>(output->specialBuffer());
        dim3 launchDims(256, 256, 1024);
        determinantFullKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(inputBuf, input->specialShapeInfo(), outputBuf, output->specialShapeInfo(), packX.specialShapeInfo(), packX.specialOffsets());
//        for (int e = 0; e < output->lengthOf(); e++) {
//            for (int k = e * n2, row = 0; k < (e + 1) * n2; ++k, ++row)
//                matrix.p(row, input->e<T>(k));
////            output->p(e, lup_<T>(&matrix, (NDArray*)nullptr, (NDArray*)nullptr));
//        }

        return Status::OK();
    }

    BUILD_SINGLE_TEMPLATE(template int _determinant, (nd4j::LaunchContext* context, NDArray* input, NDArray* output), FLOAT_TYPES);

    int determinant(nd4j::LaunchContext * context, NDArray* input, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), return _determinant, (context, input, output), FLOAT_TYPES);
    }

    template <typename T>
    int log_abs_determinant_(NDArray* input, NDArray* output) {
        return ND4J_STATUS_OK;
    }

    BUILD_SINGLE_TEMPLATE(template int log_abs_determinant_, (NDArray* input, NDArray* output), FLOAT_TYPES);

    int log_abs_determinant(nd4j::LaunchContext * context, NDArray* input, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), return log_abs_determinant_, (input, output), FLOAT_TYPES);
    }

    template <typename T>
    static int _inverse(NDArray* input, NDArray* output) {
        return Status::OK();
    }

    int inverse(nd4j::LaunchContext * context, NDArray* input, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), return _inverse, (input, output), FLOAT_TYPES);
    }

    bool checkCholeskyInput(nd4j::LaunchContext * context, NDArray const* input) {
        return false;
    }

    template <typename T>
    int cholesky_(NDArray* input, NDArray* output, bool inplace) {
        return Status::OK();
    }

    int cholesky(nd4j::LaunchContext * context, NDArray* input, NDArray* output, bool inplace) {
        BUILD_SINGLE_SELECTOR(input->dataType(), return cholesky_, (input, output, inplace), FLOAT_TYPES);
    }    
    BUILD_SINGLE_TEMPLATE(template int cholesky_, (NDArray* input, NDArray* output, bool inplace), FLOAT_TYPES);
    BUILD_SINGLE_TEMPLATE(template int _inverse, (NDArray* input, NDArray* output), FLOAT_TYPES);


    int logdetFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* output) {
        return 119;
    }
}
}
}
