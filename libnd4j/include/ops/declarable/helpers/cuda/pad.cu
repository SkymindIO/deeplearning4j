#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 20.04.2018
//

#include <array/NDArrayFactory.h>
#include <array/ResultSet.h>
#include <exceptions/cuda_exception.h>
#include <helpers/ConstantTadHelper.h>
#include <helpers/PointersManager.h>
#include <helpers/ShapeUtils.h>
#include <helpers/TAD.h>
#include <ops/declarable/helpers/transforms.h>

#include <numeric>

#include "execution/cuda/LaunchDims.h"


namespace sd {
namespace ops {
namespace helpers {
///////////////////////////////////////////////////////////////////
// x - input, y - paddings, z - output
template <typename X, typename Y>
SD_KERNEL static void padCuda(const int mode, const void* vx, const LongType* xShapeInfo, const void* vy,
                              const LongType* yShapeInfo, void* vz, const LongType* zShapeInfo,
                              const void* vPadVal) {
  const X padVal = *reinterpret_cast<const X*>(vPadVal);

  const auto x = reinterpret_cast<const X*>(vx);
  const auto y = reinterpret_cast<const Y*>(vy);
  auto z = reinterpret_cast<X*>(vz);

  __shared__ int rank, rankMinusOne;
  __shared__ LongType zLen, totalThreads, *coords, *xShape, *zShape, shift1, shift2, yStride0;

  if (threadIdx.x == 0) {
    extern __shared__ unsigned char shmem[];
    coords = reinterpret_cast<LongType*>(shmem);
    zLen = shape::length(zShapeInfo);
    xShape = shape::shapeOf(const_cast<LongType*>(xShapeInfo));
    zShape = shape::shapeOf(const_cast<LongType*>(zShapeInfo));
    yStride0 = shape::stride(const_cast<LongType*>(yShapeInfo))[0];
    rank = shape::rank(xShapeInfo);
    zLen = shape::length(zShapeInfo);
    rankMinusOne = rank - 1;
    totalThreads = gridDim.x * blockDim.x;
    shift1 = mode == 1 ? 0 : 1;  // REFLECT : SYMMETRIC
    shift2 = mode == 1 ? 2 : 1;  // REFLECT : SYMMETRIC
  }

  __syncthreads();

  auto xzCoord = coords + threadIdx.x * rank;  // we use xzCoord storage both for x and z arrays

  const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (mode == 0) {  // CONSTANT case

    for (LongType i = tid; i < zLen; i += totalThreads) {
      shape::index2coords(i, zShapeInfo, xzCoord);
      const auto zOffset = shape::getOffset(zShapeInfo, xzCoord);

      bool within = true;
      for (int j = rankMinusOne; j >= 0; --j) {
        if (xShape[j] == zShape[j]) continue;
        const auto left = y[shape::getIndexOffset(yStride0 * j, yShapeInfo)];
        if (xzCoord[j] < left || xzCoord[j] >= left + xShape[j]) {
          within = false;
          break;
        } else {
          xzCoord[j] = xzCoord[j] - left;
        }
      }

      if (within)
        z[zOffset] = x[shape::getOffset(xShapeInfo, xzCoord)];
      else
        z[zOffset] = padVal;
    }
  } else {  // REFLECT and SYMMETRIC cases

    for (LongType i = tid; i < zLen; i += totalThreads) {
      shape::index2coords(i, zShapeInfo, xzCoord);
      const auto zOffset = shape::getOffset(zShapeInfo, xzCoord);

      for (int j = rankMinusOne; j >= 0; --j) {
        if (xShape[j] == zShape[j]) continue;
        xzCoord[j] =
            xzCoord[j] - y[shape::getIndexOffset(
                yStride0 * j, yShapeInfo)];  // are ready to fill middle (within input dimension range)
        if (xzCoord[j] < 0)
          xzCoord[j] = -xzCoord[j] - shift1;  // means fill from left
        else if (xzCoord[j] >= xShape[j])
          xzCoord[j] = 2 * xShape[j] - xzCoord[j] - shift2;  // means fill from right
      }

      const auto xOffset = shape::getOffset(xShapeInfo, xzCoord);
      z[zOffset] = x[xOffset];
    }
  }
}

///////////////////////////////////////////////////////////////////
template <typename X, typename Y>
static void padCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem,
                            const hipStream_t* stream, const int mode, const void* vx, const LongType* xShapeInfo,
                            const void* vy, const LongType* yShapeInfo, void* vz, const LongType* zShapeInfo,
                            const void* padVal) {
  padCuda<X, Y><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(mode, vx, xShapeInfo, vy, yShapeInfo, vz,
                                                                        zShapeInfo, padVal);
  sd::DebugHelper::checkErrorCode(const_cast<hipStream_t *>(stream), "padCuda failed");

}

///////////////////////////////////////////////////////////////////
void pad(LaunchContext* context, const int mode, const NDArray& input, const NDArray& paddings, NDArray& output,
         const NDArray& padValue) {
  PointersManager manager(context, "pad");

  NDArray::prepareSpecialUse({&output}, {&input, &paddings, &padValue});

  dim3 padLaunch = padDims(output.lengthOf(),output.rankOf());
  const auto xType = input.dataType();
  const auto yType = paddings.dataType();

  BUILD_DOUBLE_SELECTOR(
      xType, yType, padCudaLauncher,
      (padLaunch.y, padLaunch.x, padLaunch.z, context->getCudaStream(), mode, input.specialBuffer(),
          input.specialShapeInfo(), paddings.specialBuffer(), paddings.specialShapeInfo(), output.specialBuffer(),
          output.specialShapeInfo(), padValue.specialBuffer()),
      SD_COMMON_TYPES, SD_INDEXING_TYPES);

  NDArray::registerSpecialUse({&output}, {&input, &paddings, &padValue});
  manager.synchronize();
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
template <typename T>
static SD_KERNEL void mirrorPadLinearKernel(void const* vx, const LongType* xShape, void* vz,
                                            const LongType* zShape,
                                            LongType leftSide, LongType leftSideCorrected, LongType xLen, LongType len,
                                            LongType zLen) {
  __shared__ T const* x;
  __shared__ T* z;
  if (threadIdx.x == 0) {
    x = reinterpret_cast<T const*>(vx);
    z = reinterpret_cast<T*>(vz);
  }
  __syncthreads();
  auto start = blockIdx.x * blockDim.x + threadIdx.x;
  auto step = blockDim.x * gridDim.x;

  for (int i = start; i < zLen; i += step) {
    auto zIndex = shape::getIndexOffset(i, zShape);
    auto xIndex = shape::getIndexOffset(len - i, xShape);
    if (i < leftSide)  // left side
      xIndex = shape::getIndexOffset(leftSideCorrected - i, xShape);

    else if (i >= leftSide && i < leftSide + xLen)  // middle
      xIndex = shape::getIndexOffset(i - leftSide, xShape);

    if(zIndex >= 0 && xIndex >= 0 && zIndex < zLen && xIndex < xLen)
      z[zIndex] = x[xIndex];
  }
}

template <typename F, typename I>
static SD_KERNEL void mirrorPadKernel(void const* vx, const LongType* xShape, void* vz, const LongType* zShape,
                                      LongType outLen, void const* paddings, const LongType* paddingShape,
                                      int reflBorder) {
  __shared__ F const* x;
  __shared__ I const* pads;
  __shared__ F* z;
  __shared__ LongType zRank, rank;
  __shared__ LongType* xIdx;
  if (threadIdx.x == 0) {
    extern __shared__ unsigned char shmem[];
    xIdx = reinterpret_cast<LongType*>(shmem);
    rank = shape::rank(xShape);

    x = reinterpret_cast<F const*>(vx);  //
    pads = reinterpret_cast<I const*>(paddings);
    z = reinterpret_cast<F*>(vz);
  }
  __syncthreads();
  auto start = threadIdx.x + blockIdx.x * blockDim.x;
  auto step = blockDim.x * gridDim.x;

  for (LongType i = start; i < outLen; i += step) {
    auto xzCoord = xIdx + threadIdx.x * rank;
    shape::index2coords(i, zShape, xzCoord);
    auto outOffset = shape::getOffset(zShape, xzCoord);
    for (LongType j = 0; j < rank; j++) {
      const LongType inLen = shape::sizeAt(xShape, j);
      LongType coords[2] = {j, 0};
      auto padOffset = shape::getOffset(paddingShape, coords);  // padding already has rank 2
      const auto leftSide = pads[padOffset];
      const auto leftSideCorrected = leftSide - reflBorder;
      const LongType len = 2 * (inLen - 1) + leftSide + reflBorder;

      if (xzCoord[j] < leftSide)  // left side
        xzCoord[j] = leftSideCorrected - xzCoord[j];

      else if (xzCoord[j] >= leftSide && xzCoord[j] < leftSide + inLen)  // middle
        xzCoord[j] = xzCoord[j] - leftSide;

      else if (len > xzCoord[j])  // right side
        xzCoord[j] = len - xzCoord[j];
      else
        xzCoord[j] = xzCoord[j] - len;
    }

    auto inOffset = shape::getOffset(xShape, xzCoord);
    z[outOffset] = x[inOffset];
  }
}

template <typename F, typename I>
static void mirrorPad_(LaunchContext* context, const NDArray& input, const NDArray& paddings, NDArray& output,
                       const int mode) {
  // mode:  0 - REFLECT, else - SYMMETRIC
  const int reflBorder = (bool)mode ? 1 : 0;
  const LongType rank = input.rankOf();
  const LongType outLen = output.lengthOf();
  auto stream = context->getCudaStream();
  NDArray::prepareSpecialUse({&output}, {&input, &paddings});

  if (rank <= 1) {
    const LongType inLen = input.isScalar() ? 1 : input.lengthOf();
    const auto leftSide = paddings.e<LongType>(0);
    const auto leftSideCorrected = leftSide - reflBorder;
    const LongType len = 2 * (inLen - 1) + leftSide + reflBorder;

    dim3 mirrorPadLinearDims2 = mirrorPadLinearDims(len);
    mirrorPadLinearKernel<F><<<mirrorPadLinearDims2.y, mirrorPadLinearDims2.x, mirrorPadLinearDims2.z, *stream>>>(
        input.specialBuffer(), input.specialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), leftSide,
        leftSideCorrected, inLen, len, outLen);
    DebugHelper::checkErrorCode(stream, "helpers::mirrorPadLinearKernel(...) failed");
  } else {
    dim3 mirrorPadDims = mirrorPadTad(output.lengthOf(),input.rankOf());
    mirrorPadKernel<F, I><<<mirrorPadDims.y, mirrorPadDims.x, mirrorPadDims.z, *stream>>>(
        input.specialBuffer(), input.specialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), outLen,
        paddings.specialBuffer(), paddings.specialShapeInfo(), reflBorder);
    DebugHelper::checkErrorCode(stream, "helpers::mirrorPadKernel(...) failed");
  }
  NDArray::registerSpecialUse({&output}, {&input, &paddings});
}

void mirrorPad(LaunchContext* context, const NDArray& input, const NDArray& paddings, NDArray& output,
               const int mode) {
  BUILD_DOUBLE_SELECTOR(input.dataType(), paddings.dataType(), mirrorPad_, (context, input, paddings, output, mode),
                        SD_COMMON_TYPES, SD_INDEXING_TYPES);
}

}  // namespace helpers
}  // namespace ops
}  // namespace sd
