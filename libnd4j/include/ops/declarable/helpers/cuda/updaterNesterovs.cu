#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Oleh Semeniv (oleg.semeniv@gmail.com)
//
#include <helpers/PointersManager.h>
#include <math/platformmath.h>
#include <math/templatemath.h>
#include <ops/declarable/helpers/updatersHelpers.h>
#include <system/op_boilerplate.h>

#include "execution/cuda/LaunchDims.h"
#include "helpers/DebugHelper.h"


namespace sd {
namespace ops {
namespace helpers {

///////////////////////////////////////////////////////////////////
template <typename T>
SD_KERNEL void nesterovsUpdaterCuda(const void* vx, const LongType* xShapeInfo, const void* vin,
                                    const LongType* inShapeInfo, void* vz, const LongType* zShapeInfo,
                                    void* vst, const LongType* stShapeInfo, const T lr, const T momentum) {
  const auto grad = reinterpret_cast<const T*>(vx);
  const auto init = reinterpret_cast<const T*>(vin);
  auto up = reinterpret_cast<T*>(vz);
  auto st = reinterpret_cast<T*>(vst);

  __shared__ LongType xLen;
  __shared__ T momentumT;
  __shared__ bool bEWS, bOrdering, bXZsame, bXInSame, bXStSame;

  if (threadIdx.x == 0) {
    xLen = shape::length(xShapeInfo);
    momentumT = (-momentum - 1);

    bEWS = 1 == shape::elementWiseStride(xShapeInfo) && 1 == shape::elementWiseStride(zShapeInfo) &&
           1 == shape::elementWiseStride(stShapeInfo) && 1 == shape::elementWiseStride(inShapeInfo);
    bOrdering = shape::order(xShapeInfo) == shape::order(zShapeInfo) &&
                shape::order(xShapeInfo) == shape::order(inShapeInfo) &&
                shape::order(xShapeInfo) == shape::order(stShapeInfo);

    bXZsame = shape::haveSameShapeAndStrides(xShapeInfo, zShapeInfo);
    bXInSame = shape::haveSameShapeAndStrides(xShapeInfo, inShapeInfo);
    bXStSame = shape::haveSameShapeAndStrides(xShapeInfo, stShapeInfo);
  }
  __syncthreads();

  LongType coords[SD_MAX_RANK];

  for (LongType i = blockIdx.x * blockDim.x + threadIdx.x; i < xLen; i += gridDim.x * blockDim.x) {
    LongType xOffset = i, zOffset = i, initOffset = i, stOffset = i;

    if (!bEWS || !bOrdering) {
      shape::index2coords(i, xShapeInfo, coords);
      xOffset = shape::getOffset(xShapeInfo, coords);
      zOffset = bXZsame ? xOffset : shape::getOffset(zShapeInfo, coords);
      initOffset = bXInSame ? xOffset : shape::getOffset(inShapeInfo, coords);
      stOffset = bXStSame ? xOffset : shape::getOffset(stShapeInfo, coords);
    }

    T prevState = momentum * init[initOffset];
    st[stOffset] = prevState - lr * grad[xOffset];
    up[zOffset] = prevState + momentumT * st[stOffset];
  }
}

///////////////////////////////////////////////////////////////////
template <typename T>
void nesterovsUpdaterCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMemory,
                                  const hipStream_t* stream, const void* vx, const LongType* xShapeInfo,
                                  const void* vin, const LongType* inShapeInfo, void* vz,
                                  const LongType* zShapeInfo, void* vst, const LongType* stShapeInfo,
                                  const double dLr, const double dMomentum) {
  const T lr = static_cast<T>(dLr);
  const T momentum = static_cast<T>(dMomentum);
  nesterovsUpdaterCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMemory, *stream>>>(vx, xShapeInfo, vin, inShapeInfo, vz,
                                                                            zShapeInfo, vst, stShapeInfo, lr, momentum);
  sd::DebugHelper::checkErrorCode(const_cast<hipStream_t *>(stream), "nesterovsUpdaterCuda failed");

}

///////////////////////////////////////////////////////////////////
void updaterNesterovs(LaunchContext* context, const NDArray& gradient, const NDArray& initState, NDArray& update,
                      NDArray& stateV, const double dLr, const double dMomentum) {
  PointersManager manager(context, "nesterovsUpdater");

  dim3 launchDims = updaterDims(gradient.lengthOf());
  NDArray::prepareSpecialUse({&update, &stateV}, {&gradient, &initState});
  BUILD_SINGLE_SELECTOR(
      gradient.dataType(), nesterovsUpdaterCudaLauncher,
      (launchDims.y, launchDims.x,launchDims.z, context->getCudaStream(), gradient.specialBuffer(), gradient.specialShapeInfo(),
       initState.specialBuffer(), initState.specialShapeInfo(), update.specialBuffer(), update.specialShapeInfo(),
       stateV.specialBuffer(), stateV.specialShapeInfo(), dLr, dMomentum),
      SD_FLOAT_TYPES);
  NDArray::registerSpecialUse({&update, &stateV}, {&gradient, &initState});

  manager.synchronize();
}

}  // namespace helpers
}  // namespace ops
}  // namespace sd
