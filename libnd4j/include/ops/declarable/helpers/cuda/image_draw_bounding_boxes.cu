#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author sgazeos@gmail.com
//
#include <array/NDArray.h>
#include <system/op_boilerplate.h>

#include "execution/cuda/LaunchDims.h"


namespace sd {
namespace ops {
namespace helpers {

typedef NDArray ColorTable_t;
static NDArray DefaultColorTable(int depth, LaunchContext* context) {
  // std::vector<std::vector<float>> colorTable;
  const LongType kDefaultTableLength = 10;
  const LongType kDefaultChannelLength = 4;
  std::vector<sd::LongType> shape = {kDefaultTableLength, kDefaultChannelLength};
  NDArray colorTable('c', shape,
                     {
                         1,   1,   0,   1,  // yellow
                         0,   0,   1,   1,  // 1: blue
                         1,   0,   0,   1,  // 2: red
                         0,   1,   0,   1,  // 3: lime
                         0.5, 0,   0.5, 1,  // 4: purple
                         0.5, 0.5, 0,   1,  // 5: olive
                         0.5, 0,   0,   1,  // 6: maroon
                         0,   0,   0.5, 1,  // 7: navy blue
                         0,   1,   1,   1,  // 8: aqua
                         1,   0,   1,   1   // 9: fuchsia
                     },
                     FLOAT32, context);

  if (depth == 1) {
    colorTable.assign(1.f);  // all to white when black and white colors
  }
  return colorTable;
}

template <typename T>
static SD_KERNEL void drawBoundingBoxesKernel(T const* images, const LongType* imagesShape, float const* boxes,
                                              const LongType* boxesShape, float const* colorTable,
                                              const LongType* colorTableShape, T* output,
                                              const LongType* outputShape,
                                              LongType batchSize, LongType width, LongType height, LongType channels,
                                              LongType boxSize, LongType colorTableLen) {
  for (auto batch = blockIdx.x; batch < (int)batchSize; batch += gridDim.x) {  // loop by batch
    for (auto boxIndex = 0; boxIndex < boxSize; ++boxIndex) {
      // box with shape
      // auto internalBox = &boxes[b * colorSetSize * 4 + c * 4];//(*boxes)(b, {0})(c, {0});//internalBoxes->at(c);
      auto colorIndex = boxIndex % colorTableLen;  // colorSet->at(c);
      LongType indices0[] = {batch, boxIndex, 0};
      LongType indices1[] = {batch, boxIndex, 1};
      LongType indices2[] = {batch, boxIndex, 2};
      LongType indices3[] = {batch, boxIndex, 3};
      auto rowStart = LongType((height - 1) * boxes[shape::getOffset(boxesShape, indices0, 0)]);
      auto rowStartBound = math::sd_max(LongType(0), rowStart);
      auto rowEnd = LongType((height - 1) * boxes[shape::getOffset(boxesShape, indices2, 0)]);
      auto rowEndBound = math::sd_min(LongType(height - 1), rowEnd);
      auto colStart = LongType((width - 1) * boxes[shape::getOffset(boxesShape, indices1, 0)]);
      auto colStartBound = math::sd_max(LongType(0), colStart);
      auto colEnd = LongType((width - 1) * boxes[shape::getOffset(boxesShape, indices3, 0)]);
      auto colEndBound = math::sd_min(LongType(width - 1), colEnd);
      if (rowStart > rowEnd || colStart > colEnd) {
        continue;
      }
      if (rowStart >= height || rowEnd < 0 || colStart >= width || colEnd < 0) {
        continue;
      }

      // Draw upper line
      if (rowStart >= 0) {
        for (auto j = colStartBound + threadIdx.x; j <= colEndBound; j += blockDim.x)
          for (auto c = 0; c < channels; c++) {
            LongType zPos[] = {batch, rowStart, j, c};
            LongType cPos[] = {colorIndex, c};
            auto cIndex = shape::getOffset(colorTableShape, cPos, 0);
            auto zIndex = shape::getOffset(outputShape, zPos, 0);
            output[zIndex] = (T)colorTable[cIndex];
          }
      }
      // Draw bottom line.
      if (rowEnd < height) {
        for (auto j = colStartBound + threadIdx.x; j <= colEndBound; j += blockDim.x)
          for (auto c = 0; c < channels; c++) {
            LongType zPos[] = {batch, rowEnd, j, c};
            LongType cPos[] = {colorIndex, c};
            auto cIndex = shape::getOffset(colorTableShape, cPos, 0);
            auto zIndex = shape::getOffset(outputShape, zPos, 0);
            output[zIndex] = (T)colorTable[cIndex];
          }
      }

      // Draw left line.
      if (colStart >= 0) {
        for (auto i = rowStartBound + threadIdx.x; i <= rowEndBound; i += blockDim.x)
          for (auto c = 0; c < channels; c++) {
            LongType zPos[] = {batch, i, colStart, c};
            LongType cPos[] = {colorIndex, c};
            auto cIndex = shape::getOffset(colorTableShape, cPos, 0);
            auto zIndex = shape::getOffset(outputShape, zPos, 0);
            output[zIndex] = (T)colorTable[cIndex];
          }
      }
      // Draw right line.
      if (colEnd < width) {
        for (auto i = rowStartBound + threadIdx.x; i <= rowEndBound; i += blockDim.x)
          for (auto c = 0; c < channels; c++) {
            LongType zPos[] = {batch, i, colEnd, c};
            LongType cPos[] = {colorIndex, c};
            auto cIndex = shape::getOffset(colorTableShape, cPos, 0);
            auto zIndex = shape::getOffset(outputShape, zPos, 0);
            output[zIndex] = (T)colorTable[cIndex];
          }
      }
    }
  }
}

template <typename T>
void drawBoundingBoxesH(LaunchContext* context, NDArray * images, NDArray * boxes, NDArray * colors,
                        NDArray* output) {
  auto batchSize = images->sizeAt(0);
  auto height = images->sizeAt(1);
  auto width = images->sizeAt(2);
  auto channels = images->sizeAt(3);
  auto stream = context->getCudaStream();
  auto boxSize = boxes->sizeAt(1);
  NDArray colorsTable = DefaultColorTable(channels, context);
  if ((colors != nullptr && colors->lengthOf() > 0)) {
    colorsTable = *colors;
  }

  auto imagesBuf = images->getDataBuffer()->specialAsT<T>();
  auto boxesBuf = boxes->getDataBuffer()->specialAsT<float>();             // boxes should be float32
  auto colorsTableBuf = colorsTable.getDataBuffer()->specialAsT<float>();  // color table is float32
  auto outputBuf = output->dataBuffer()->specialAsT<T>();
  dim3 launchDims = getLaunchDims("draw_bounding_boxes");
  drawBoundingBoxesKernel<<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(
      imagesBuf, images->specialShapeInfo(), boxesBuf, boxes->specialShapeInfo(), colorsTableBuf,
      colorsTable.specialShapeInfo(), outputBuf, output->specialShapeInfo(), batchSize, width, height, channels,
      boxSize, colorsTable.lengthOf());
}

void drawBoundingBoxesFunctor(LaunchContext* context, NDArray* images, NDArray* boxes, NDArray* colors,
                              NDArray* output) {
  // images - batch of 3D images with BW (last dim = 1), RGB (last dim = 3) or RGBA (last dim = 4) channel set
  // boxes - batch of 2D bounds with last dim (y_start, x_start, y_end, x_end) to compute i and j as
  // floor((height - 1 ) * y_start) => rowStart, floor((height - 1) * y_end) => rowEnd
  // floor((width - 1 ) * x_start) => colStart, floor((width - 1) * x_end) => colEnd
  // height = images->sizeAt(1), width = images->sizeAt(2)
  // colors - colors for each box given
  // set up color for each box as frame
  NDArray::prepareSpecialUse({output}, {images, boxes, colors});
  output->assign(images);
  BUILD_SINGLE_SELECTOR(output->dataType(), drawBoundingBoxesH, (context, images, boxes, colors, output),
                        SD_FLOAT_TYPES);
  NDArray::registerSpecialUse({output}, {images, boxes, colors});
}

}  // namespace helpers
}  // namespace ops
}  // namespace sd
