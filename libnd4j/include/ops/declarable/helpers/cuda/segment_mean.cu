#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author GS <sgazeos@gmail.com>
//
#include <array/NDArrayFactory.h>
#include <exceptions/cuda_exception.h>
#include <helpers/ConstantTadHelper.h>
#include <helpers/PointersManager.h>
#include <helpers/ShapeUtils.h>
#include <helpers/TAD.h>
#include <ops/declarable/helpers/segment.h>
#include <ops/declarable/helpers/segment_common.h>

#include <execution/cuda/LaunchDims.h>

namespace sd {
namespace ops {
namespace helpers {
// -------------------------------------------------------------------------------------------------------------- //
// Segment ops linear kernels
// -------------------------------------------------------------------------------------------------------------- //
template <typename T, typename I>
static SD_KERNEL void segmentMeanLinearKernel(void* input, sd::LongType const* inputShape,
                                              sd::LongType* indices, sd::LongType* lengths,
                                              sd::LongType numOfClasses, void* output,
                                              sd::LongType const* outputShape) {
  __shared__ T* val;
  __shared__ sd::LongType xLen, zLen, zIndex;
  __shared__ T* x;
  __shared__ T* z;
  __shared__ sd::LongType threadsPerSegment, start, finish;

  auto segment = blockIdx.x;
  if (threadIdx.x == 0) {
    x = reinterpret_cast<T*>(input);
    z = reinterpret_cast<T*>(output);
    extern __shared__ unsigned char shmem[];
    val = reinterpret_cast<T*>(shmem);
    xLen = shape::length(inputShape);
    zLen = shape::length(outputShape);

    if (segment < numOfClasses) {
      zIndex = shape::getIndexOffset(segment, outputShape);
      start = indices[segment];
      finish = start + lengths[segment];
      auto xOffset = shape::getIndexOffset(start, inputShape);
      zIndex = shape::getIndexOffset(segment, outputShape);
      if (lengths[segment] > 0)
        z[zIndex] = T(x[shape::getIndexOffset(indices[segment], inputShape)] / T(lengths[segment]));
      else
        z[zIndex] = 0;
    }
      val[segment] = z[zIndex];

  }
  __syncthreads();

  for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
    auto xIndex = shape::getIndexOffset(e, inputShape);
    sd::math::atomics::sd_atomicAdd(&z[zIndex], T(x[xIndex] / static_cast<T>(lengths[segment])));
  }
}
// -------------------------------------------------------------------------------------------------------------- //
template <typename T, typename I>
static SD_KERNEL void unsortedSegmentMeanLinearKernel(void* input, sd::LongType const* inputShape, void* indices,
                                                      sd::LongType const* indicesShape, sd::LongType* starts, sd::LongType* lengths,
                                                      sd::LongType numOfClasses, void* output,
                                                      sd::LongType const* outputShape) {
  __shared__ sd::LongType xLen, zLen, zIndex;
  __shared__ T* x;
  __shared__ T* z;
  __shared__ I* y;
  auto segment = blockIdx.x;
  if (threadIdx.x == 0) {
    x = reinterpret_cast<T*>(input);
    z = reinterpret_cast<T*>(output);
    y = reinterpret_cast<I*>(indices);
    xLen = shape::length(inputShape);
    zLen = shape::length(outputShape);

    zIndex = shape::getIndexOffset(segment, outputShape);
    if (lengths[segment] > 0)
      z[zIndex] = T(x[shape::getIndexOffset(starts[segment], inputShape)] / T(lengths[segment]));
    else
      z[zIndex] = 0;
  }
  __syncthreads();
  if (lengths[segment] > 0)
    for (auto e = threadIdx.x; e < xLen; e += blockDim.x) {
      auto xIndex = shape::getIndexOffset(e, inputShape);
      auto yIndex = shape::getIndexOffset(e, indicesShape);
      if (y[yIndex] == segment && e != starts[segment]) {
        sd::math::atomics::sd_atomicAdd(&z[zIndex], T(x[xIndex] / T(lengths[segment])));
      }
    }
}
// -------------------------------------------------------------------------------------------------------------- //
// SegmentMean kernel
template <typename T, typename I>
static SD_KERNEL void segmentMeanTadKernel(void* inputBuf, sd::LongType const* inputShape,
                                           sd::LongType const* inputTads, sd::LongType const* inputTadOffsets,
                                           I* indices, sd::LongType* starts, sd::LongType* lengths,
                                           sd::LongType numOfClasses, void* outputBuf, sd::LongType const* outputShape,
                                           sd::LongType const* outputTads, sd::LongType const* outputTadOffsets,
                                           sd::LongType indicesLen) {
  __shared__ T* val;
  __shared__ sd::LongType len, zIndex, total;
  __shared__ T* z;
  __shared__ int threadsPerSegment, start, finish;
  if(blockIdx.x >= indicesLen)
    return;



  auto segment = indices[blockIdx.x];  // / threadsPerSegment;

  if (threadIdx.x == 0) {
    z = reinterpret_cast<T*>(outputBuf) + outputTadOffsets[segment];
    len = shape::length(inputTads);
    start = starts[segment];
    finish = start + lengths[segment];
    total = shape::sizeAt(inputShape, 0);
  }
  __syncthreads();

  auto idx = blockIdx.x;
  if (blockIdx.x <= total) {
    auto x = reinterpret_cast<T*>(inputBuf) + inputTadOffsets[idx];
    if (blockIdx.x == start) {
      for (auto e = threadIdx.x; e < len; e += blockDim.x) {
        auto xIndex = shape::getIndexOffset(e, inputTads);
        auto zIndex = shape::getIndexOffset(e, outputTads);
        sd::math::atomics::sd_atomicAdd(&z[zIndex], T(x[xIndex] / lengths[segment]));
      }
    } else {
      for (auto e = threadIdx.x; e < len; e += blockDim.x) {
        auto xIndex = shape::getIndexOffset(e, inputTads);
        auto zIndex = shape::getIndexOffset(e, outputTads);
        if (lengths[segment]) sd::math::atomics::sd_atomicAdd(&z[zIndex], T(x[xIndex] / lengths[segment]));
      }
    }
  }
}
// -------------------------------------------------------------------------------------------------------------- //
// segment mean
template <typename T, typename I>
static void segmentMeanFunctor_(LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
  auto stream = context->getCudaStream();
  sd::LongType numClasses = indices->e<sd::LongType>(indices->lengthOf() - 1) + 1;
  NDArray classesRangesLens = NDArrayFactory::create<sd::LongType>('c', {numClasses}, context);
  NDArray classesRangesBegs = NDArrayFactory::create<sd::LongType>('c', {numClasses}, context);

  classesRangesBegs.assign(indices->lengthOf());
  classesRangesLens.assign(0);
  NDArray::prepareSpecialUse({output}, {input, indices});
  sd::LongType* begins = reinterpret_cast<sd::LongType*>(classesRangesBegs.specialBuffer());
  sd::LongType* lengths = reinterpret_cast<sd::LongType*>(classesRangesLens.specialBuffer());
  fillUpSegments(indices, numClasses, classesRangesBegs, classesRangesLens);

  if (input->isVector()) {
    dim3 launchDims = segmentDims(numClasses,input->lengthOf());
    segmentMeanLinearKernel<T, I><<<launchDims.y, launchDims.x, launchDims.z, *stream>>>(
        input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numClasses, output->specialBuffer(),
        output->specialShapeInfo());
  } else {
    sd::LongType zero = 0;
    std::vector<sd::LongType> *dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), 1,&zero);
    auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
    auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
    auto inputTads = packX->specialShapeInfo();
    auto inputTadOffsets = packX->specialOffsets();
    auto outputTads = packZ->specialShapeInfo();
    auto outputTadOffsets = packZ->specialOffsets();
    dim3 launchDims = segmentTad(input->sizeAt(0));
    segmentMeanTadKernel<T, I><<<launchDims.y, launchDims.x, launchDims.z, *stream>>>(
        input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets,
        reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numClasses, output->specialBuffer(),
        output->specialShapeInfo(), outputTads, outputTadOffsets,indices->lengthOf());
    delete dimensions;
  }
  NDArray::registerSpecialUse({output}, {input, indices});
}
// -------------------------------------------------------------------------------------------------------------- //
void segmentMeanFunctor(sd::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
  NDArray::prepareSpecialUse({output}, {input, indices});
  BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), segmentMeanFunctor_, (context, input, indices, output),
                        SD_NUMERIC_TYPES, SD_INDEXING_TYPES);
  NDArray::registerSpecialUse({output}, {input, indices});
}

// -------------------------------------------------------------------------------------------------------------- //
template <typename T, typename I>
static void unsortedSegmentMeanFunctor_(sd::LaunchContext* context, NDArray* input, NDArray* indices,
                                        sd::LongType numOfClasses, NDArray* output) {
  auto stream = context->getCudaStream();

  NDArray classesRangesBegs = NDArrayFactory::create<sd::LongType>('c', {numOfClasses}, context);
  NDArray classesRangesLens = NDArrayFactory::create<sd::LongType>('c', {numOfClasses}, context);

  classesRangesBegs.assign(indices->lengthOf());
  classesRangesLens.assign(0);
  dim3 dims = getFillUpSegmentsDims(numOfClasses, indices->lengthOf());
  fillUpSegments(indices, numOfClasses, classesRangesBegs, classesRangesLens);
  sd::LongType* begins = reinterpret_cast<sd::LongType*>(classesRangesBegs.specialBuffer());
  sd::LongType* lengths = reinterpret_cast<sd::LongType*>(classesRangesLens.specialBuffer());

  if (input->isVector()) {
    unsortedSegmentMeanLinearKernel<T, I><<<dims.x, dims.y, dims.z, *stream>>>(
        input->specialBuffer(), input->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(),
        begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo());
  } else {
    output->assign(0);
    sd::LongType zero = 0;
    std::vector<sd::LongType> *dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), 1,&zero);
    auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
    auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
    sd::LongType const* inputTads = packX->specialShapeInfo();
    sd::LongType const* inputTadOffsets = packX->specialOffsets();
    sd::LongType const* outputTads = packZ->specialShapeInfo();
    sd::LongType const* outputTadOffsets = packZ->specialOffsets();
    dims.x = input->sizeAt(0);
    segmentMeanTadKernel<T, I><<<dims.x, dims.y, dims.z, *stream>>>(
        input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets,
        reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numOfClasses, output->specialBuffer(),
        output->specialShapeInfo(), outputTads, outputTadOffsets, indices->lengthOf());
    delete dimensions;
  }
}
// -------------------------------------------------------------------------------------------------------------- //
void unsortedSegmentMeanFunctor(sd::LaunchContext* context, NDArray* input, NDArray* indices, sd::LongType numOfClasses,
                                NDArray* output) {
  NDArray::prepareSpecialUse({output}, {input, indices});
  BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), unsortedSegmentMeanFunctor_,
                        (context, input, indices, numOfClasses, output), SD_NUMERIC_TYPES, SD_INDEXING_TYPES);
  NDArray::registerSpecialUse({output}, {input, indices});
}

// -------------------------------------------------------------------------------------------------------------- //
template <typename T, typename I>
static SD_KERNEL void segmentMeanBPLinearKernel(void* inputBuf, sd::LongType const* inputShape, void* eps,
                                                sd::LongType const* epsShape, void* indicesBuf,
                                                sd::LongType const* indicesShape, sd::LongType* lengths, void* outputBuf,
                                                sd::LongType const* outputShape) {
  __shared__ T* x;
  __shared__ T* gradIn;
  __shared__ T* gradOut;
  __shared__ I* y;
  __shared__ T* z;
  __shared__ sd::LongType xLen, gradLen;

  if (threadIdx.x == 0) {
    xLen = shape::length(inputShape);
    x = reinterpret_cast<T*>(inputBuf);
    y = reinterpret_cast<I*>(indicesBuf);
    z = reinterpret_cast<T*>(outputBuf);
    gradOut = reinterpret_cast<T*>(eps);
    gradLen = shape::length(epsShape);
  }
  __syncthreads();

  auto start = blockIdx.x * blockDim.x + threadIdx.x;
  auto step = gridDim.x * blockDim.x;

  for (auto e = start; e < xLen; e += step) {
    auto zOffset = shape::getIndexOffset(e, outputShape);
    auto xOffset = shape::getIndexOffset(e, inputShape);
    auto yOffset = shape::getIndexOffset(e, indicesShape);
    auto classIndex = y[yOffset];
    auto gradOffsetO = shape::getIndexOffset(classIndex, epsShape);

    z[zOffset] = T(gradOut[gradOffsetO] / float(lengths[classIndex]));
  }
}
// -------------------------------------------------------------------------------------------------------------- //
template <typename T, typename I>
static SD_KERNEL void segmentMeanBPTadKernel(void* inputBuf, sd::LongType const* inputShape, void* eps,
                                             sd::LongType const* epsShape, void* indicesBuf,
                                             sd::LongType const* indicesShape, sd::LongType* lengths, void* outputBuf,
                                             sd::LongType const* outputShape, sd::LongType const* inputTad,
                                             sd::LongType const* inputOffsets, sd::LongType const* gradOutTad,
                                             sd::LongType const* gradOutOffsets, sd::LongType const* outTad,
                                             sd::LongType const* outOffsets) {
  __shared__ T* x;
  __shared__ T* gradOut;
  __shared__ I* y;
  __shared__ T* z;
  __shared__ sd::LongType xLen, yLen, gradLen, currentLen;

  if (threadIdx.x == 0) {
    xLen = shape::length(inputShape);
    x = reinterpret_cast<T*>(inputBuf);
    y = reinterpret_cast<I*>(indicesBuf);
    z = reinterpret_cast<T*>(outputBuf);
    yLen = shape::length(indicesShape);
    gradOut = reinterpret_cast<T*>(eps);
    gradLen = shape::length(epsShape);
    currentLen = shape::length(outTad);
  }
  __syncthreads();

  for (auto i = blockIdx.x; i < yLen; i += gridDim.x) {
    auto segment = y[i];
    T* currentOut = z + outOffsets[i];
    T* outGrad = gradOut + gradOutOffsets[segment];

    for (auto e = threadIdx.x; e < currentLen; e += blockDim.x) {
      auto zIndex = shape::getIndexOffset(e, outTad);
      auto gradIndex = shape::getIndexOffset(e, gradOutTad);
      if (lengths[segment] > 0) currentOut[zIndex] = T(outGrad[gradIndex] / float(lengths[segment]));
    }
  }
}
// -------------------------------------------------------------------------------------------------------------- //
// backrop for mean
template <typename T, typename I>
sd::Status segmentMeanFunctorBP_(sd::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* gradOut,
                                 NDArray* output) {
  auto stream = context->getCudaStream();
  NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
  auto numClasses = indices->e<sd::LongType>(indices->lengthOf() - 1) + 1;
  NDArray classesRangesLens = NDArrayFactory::create<sd::LongType>('c', {numClasses}, context);
  NDArray classesRangesBegs = NDArrayFactory::create<sd::LongType>('c', {numClasses}, context);

  classesRangesBegs.assign(indices->lengthOf());
  classesRangesLens.assign(0);
  fillUpSegments(indices, numClasses, classesRangesBegs, classesRangesLens);
  sd::LongType* begins = reinterpret_cast<sd::LongType*>(classesRangesBegs.specialBuffer());
  sd::LongType* lengths = reinterpret_cast<sd::LongType*>(classesRangesLens.specialBuffer());

  if (input->isVector()) {
    sd::LongType loop_size = input->lengthOf();
    auto numOfClasses = gradOut->lengthOf();  // indices->e<sd::LongType>(loop_size - 1);
    dim3 segmentBpDims2 = segmentBpDims(gradOut->lengthOf(),input->lengthOf());
    segmentMeanBPLinearKernel<T, I><<<segmentBpDims2.y, segmentBpDims2.x, segmentBpDims2.z, *stream>>>(
        input->specialBuffer(), input->specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
        indices->specialBuffer(), indices->specialShapeInfo(), lengths, output->specialBuffer(),
        output->specialShapeInfo());
  } else {
    sd::LongType zero = 0;
    std::vector<sd::LongType> *dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), 1,&zero);
    auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
    auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
    auto packGradOut = sd::ConstantTadHelper::getInstance().tadForDimensions(gradOut->shapeInfo(), dimensions);
    sd::LongType const* inputTads = packX->specialShapeInfo();
    sd::LongType const* inputTadOffsets = packX->specialOffsets();
    sd::LongType const* outputTads = packZ->specialShapeInfo();
    sd::LongType const* outputTadOffsets = packZ->specialOffsets();
    sd::LongType const* gradOutTads = packGradOut->specialShapeInfo();
    sd::LongType const* gradOutTadOffsets = packGradOut->specialOffsets();
    dim3 segmentBpTad2 = segmentBpTad(indices->lengthOf(),input->lengthOf());

    segmentMeanBPTadKernel<T, I><<<segmentBpTad2.y, segmentBpTad2.x, segmentBpTad2.z, *stream>>>(
        input->specialBuffer(), input->specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
        indices->specialBuffer(), indices->specialShapeInfo(), lengths, output->specialBuffer(),
        output->specialShapeInfo(), inputTads, inputTadOffsets, gradOutTads, gradOutTadOffsets, outputTads,
        outputTadOffsets);
    delete dimensions;
  }
  NDArray::registerSpecialUse({output}, {input, indices, gradOut});
  return sd::Status::OK;
}
// -------------------------------------------------------------------------------------------------------------- //
// segmen mean bp main
sd::Status segmentMeanFunctorBP(sd::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* gradOut,
                                NDArray* output) {
  NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
  BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return segmentMeanFunctorBP_,
                        (context, input, indices, gradOut, output), SD_FLOAT_TYPES, SD_INDEXING_TYPES);
  NDArray::registerSpecialUse({output}, {input, indices, gradOut});
}
// -------------------------------------------------------------------------------------------------------------- //

template <typename T, typename I>
static sd::Status unsortedSegmentMeanFunctorBP_(sd::LaunchContext* context, NDArray* input, NDArray* indices,
                                                NDArray* gradOut, sd::LongType numOfClasses, NDArray* output) {
  auto stream = context->getCudaStream();
  NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
  auto numClasses = indices->e<sd::LongType>(indices->lengthOf() - 1) + 1;
  NDArray classesRangesLens = NDArrayFactory::create<sd::LongType>('c', {numClasses}, context);
  NDArray classesRangesBegs = NDArrayFactory::create<sd::LongType>('c', {numClasses}, context);

  classesRangesBegs.assign(indices->lengthOf());
  classesRangesLens.assign(0);
  fillUpSegments(indices, numClasses, classesRangesBegs, classesRangesLens);
  sd::LongType* begins = reinterpret_cast<sd::LongType*>(classesRangesBegs.specialBuffer());
  sd::LongType* lengths = reinterpret_cast<sd::LongType*>(classesRangesLens.specialBuffer());

  if (input->isVector()) {
    sd::LongType loop_size = input->lengthOf();
    auto numOfClasses = gradOut->lengthOf();
    dim3 segmentBpDims2 = segmentBpDims(gradOut->lengthOf(),input->lengthOf());
    segmentMeanBPLinearKernel<T, I><<<segmentBpDims2.y,segmentBpDims2.x,segmentBpDims2.z, *stream>>>(
        input->specialBuffer(), input->specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
        indices->specialBuffer(), indices->specialShapeInfo(), lengths, output->specialBuffer(),
        output->specialShapeInfo());
  } else {
    sd::LongType zero = 0;
    std::vector<sd::LongType> *dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(),1, &zero);
    auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
    auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);

    auto packGradOut = sd::ConstantTadHelper::getInstance().tadForDimensions(gradOut->shapeInfo(), dimensions);
    sd::LongType const* inputTads = packX->specialShapeInfo();
    sd::LongType const* inputTadOffsets = packX->specialOffsets();
    sd::LongType const* outputTads = packZ->specialShapeInfo();
    sd::LongType const* outputTadOffsets = packZ->specialOffsets();
    sd::LongType const* gradOutTads = packGradOut->specialShapeInfo();
    sd::LongType const* gradOutTadOffsets = packGradOut->specialOffsets();
    dim3 segmentBpTad2 = segmentBpTad(indices->lengthOf(),input->lengthOf());

    segmentMeanBPTadKernel<T, I><<<segmentBpTad2.y,segmentBpTad2.x, segmentBpTad2.z, *stream>>>(
        input->specialBuffer(), input->specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
        indices->specialBuffer(), indices->specialShapeInfo(), lengths, output->specialBuffer(),
        output->specialShapeInfo(), inputTads, inputTadOffsets, gradOutTads, gradOutTadOffsets, outputTads,
        outputTadOffsets);
    delete dimensions;
  }
  NDArray::registerSpecialUse({output}, {input, indices, gradOut});
  return sd::Status::OK;
}
// -------------------------------------------------------------------------------------------------------------- //
sd::Status unsortedSegmentMeanFunctorBP(sd::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* gradOut,
                                        sd::LongType numOfClasses, NDArray* output) {
  NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
  BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return unsortedSegmentMeanFunctorBP_,
                        (context, input, indices, gradOut, numOfClasses, output), SD_FLOAT_TYPES, SD_INDEXING_TYPES);
  NDArray::registerSpecialUse({output}, {input, indices, gradOut});
}

}  // namespace helpers
}  // namespace ops
}  // namespace sd
