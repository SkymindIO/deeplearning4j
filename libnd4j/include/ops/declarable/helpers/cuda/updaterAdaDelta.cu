#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Oleh Semeniv (oleg.semeniv@gmail.com)
//
#include <helpers/PointersManager.h>
#include <math/platformmath.h>
#include <math/templatemath.h>
#include <ops/declarable/helpers/updatersHelpers.h>
#include <system/op_boilerplate.h>

#include "execution/cuda/LaunchDims.h"
#include "helpers/DebugHelper.h"


namespace sd {
namespace ops {
namespace helpers {

///////////////////////////////////////////////////////////////////
template <typename T>
SD_KERNEL void adaDeltaUpdaterCuda(const void* vx, const LongType* xShapeInfo, const void* vinMsg,
                                   const LongType* inMsgShapeInfo, const void* vinMsdx,
                                   const LongType* inMsdxShapeInfo, void* vz, const LongType* zShapeInfo,
                                   void* vstMsg, const LongType* stMsgShapeInfo, void* vstMsdx,
                                   const LongType* stMsdxShapeInfo, const T rho, const T epsilon) {
  const auto grad = reinterpret_cast<const T*>(vx);
  const auto initMsg = reinterpret_cast<const T*>(vinMsg);
  const auto initMsdx = reinterpret_cast<const T*>(vinMsdx);

  auto up = reinterpret_cast<T*>(vz);
  auto stMsg = reinterpret_cast<T*>(vstMsg);
  auto stMsdx = reinterpret_cast<T*>(vstMsdx);

  __shared__ LongType xLen;
  __shared__ T rhoT;
  __shared__ bool bOrdering, bXZsame, bXInMsgSame, bXStMsgSame, bXInMsdxSame, bXStMsdxSame;

  // Cache shape information in shared memory
  __shared__ LongType xRank, zRank, inMsgRank, stMsgRank, inMsdxRank, stMsdxRank;
  __shared__ LongType *xShape, *zShape, *inMsgShape, *stMsgShape, *inMsdxShape, *stMsdxShape;
  __shared__ LongType *xStride, *zStride, *inMsgStride, *stMsgStride, *inMsdxStride, *stMsdxStride;

  if (threadIdx.x == 0) {
    xLen = shape::length(xShapeInfo);
    rhoT = (1 - rho);

    // Cache ranks
    xRank = shape::rank(xShapeInfo);
    zRank = shape::rank(zShapeInfo);
    inMsgRank = shape::rank(inMsgShapeInfo);
    stMsgRank = shape::rank(stMsgShapeInfo);
    inMsdxRank = shape::rank(inMsdxShapeInfo);
    stMsdxRank = shape::rank(stMsdxShapeInfo);

    // Cache shapes
    xShape = shape::shapeOf(xShapeInfo);
    zShape = shape::shapeOf(zShapeInfo);
    inMsgShape = shape::shapeOf(inMsgShapeInfo);
    stMsgShape = shape::shapeOf(stMsgShapeInfo);
    inMsdxShape = shape::shapeOf(inMsdxShapeInfo);
    stMsdxShape = shape::shapeOf(stMsdxShapeInfo);

    // Cache strides
    xStride = shape::stride(xShapeInfo);
    zStride = shape::stride(zShapeInfo);
    inMsgStride = shape::stride(inMsgShapeInfo);
    stMsgStride = shape::stride(stMsgShapeInfo);
    inMsdxStride = shape::stride(inMsdxShapeInfo);
    stMsdxStride = shape::stride(stMsdxShapeInfo);

    bOrdering = shape::order(xShapeInfo) == shape::order(zShapeInfo) &&
                shape::order(zShapeInfo) == shape::order(stMsgShapeInfo) &&
                shape::order(stMsgShapeInfo) == shape::order(inMsgShapeInfo) &&
                shape::order(inMsgShapeInfo) == shape::order(stMsdxShapeInfo) &&
                shape::order(stMsdxShapeInfo) == shape::order(inMsdxShapeInfo);

    bXZsame = shape::haveSameShapeAndStrides(xShapeInfo, zShapeInfo);
    bXInMsgSame = shape::haveSameShapeAndStrides(xShapeInfo, inMsgShapeInfo);
    bXStMsgSame = shape::haveSameShapeAndStrides(xShapeInfo, stMsgShapeInfo);
    bXInMsdxSame = shape::haveSameShapeAndStrides(xShapeInfo, inMsdxShapeInfo);
    bXStMsdxSame = shape::haveSameShapeAndStrides(xShapeInfo, stMsdxShapeInfo);
  }
  __syncthreads();

  LongType coords[SD_MAX_RANK];

  for (LongType i = blockIdx.x * blockDim.x + threadIdx.x; i < xLen; i += gridDim.x * blockDim.x) {
    LongType xOffset, zOffset, initMsgOffset, initMsdxOffset, stMsgOffset, stMsdxOffset;

    INDEX2COORDS(i, xRank, xShape, coords);
    COORDS2INDEX(xRank, xStride, coords, xOffset);

    if (bXZsame) {
      zOffset = xOffset;
    } else {
      COORDS2INDEX(zRank, zStride, coords, zOffset);
    }

    if (bXInMsgSame) {
      initMsgOffset = xOffset;
    } else {
      COORDS2INDEX(inMsgRank, inMsgStride, coords, initMsgOffset);
    }

    if (bXStMsgSame) {
      stMsgOffset = xOffset;
    } else {
      COORDS2INDEX(stMsgRank, stMsgStride, coords, stMsgOffset);
    }

    if (bXInMsdxSame) {
      initMsdxOffset = xOffset;
    } else {
      COORDS2INDEX(inMsdxRank, inMsdxStride, coords, initMsdxOffset);
    }

    if (bXStMsdxSame) {
      stMsdxOffset = xOffset;
    } else {
      COORDS2INDEX(stMsdxRank, stMsdxStride, coords, stMsdxOffset);
    }

    stMsg[stMsgOffset] = rho * initMsg[initMsgOffset] + grad[xOffset] * grad[xOffset] * rhoT;

    up[zOffset] = grad[xOffset] * (math::sd_sqrt<T, T>(initMsdx[initMsdxOffset] + epsilon) /
                                   math::sd_sqrt<T, T>(stMsg[stMsgOffset] + epsilon));

    stMsdx[stMsdxOffset] = rho * initMsdx[initMsdxOffset] + up[zOffset] * up[zOffset] * rhoT;
  }
}
///////////////////////////////////////////////////////////////////
template <typename T>
void adaDeltaUpdaterCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMemory,
                                 const hipStream_t* stream, const void* vx, const LongType* xShapeInfo,
                                 const void* vinMsg, const LongType* inMsgShapeInfo, const void* vinMsdx,
                                 const LongType* inMsdxShapeInfo, void* vz, const LongType* zShapeInfo,
                                 void* vstMsg, const LongType* stMsgShapeInfo, void* vstMsdx,
                                 const LongType* stMsdxShapeInfo, const double dRho, const double dEpsilon) {
  const T rho = static_cast<T>(dRho);
  T epsilon = static_cast<T>(dEpsilon);
  //fp16 to prevent underflow
  if(epsilon == 0.0) {
    epsilon = static_cast<T>(1e-7);
  }
  adaDeltaUpdaterCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMemory, *stream>>>(
      vx, xShapeInfo, vinMsg, inMsgShapeInfo, vinMsdx, inMsdxShapeInfo, vz, zShapeInfo, vstMsg, stMsgShapeInfo, vstMsdx,
      stMsdxShapeInfo, rho, epsilon);
  sd::DebugHelper::checkErrorCode(const_cast<hipStream_t *>(stream), "adaDeltaUpdaterCuda failed");

}

///////////////////////////////////////////////////////////////////
void updaterAdaDelta(LaunchContext* context, NDArray& gradient, NDArray& initStateMsg,
                     NDArray& initStateMsdx, NDArray& update, NDArray& stateMsg, NDArray& stateMsdx,
                     const double dRho, const double dEpsilon) {
  PointersManager manager(context, "adaDeltaUpdater");
  dim3 updater2Dims = updaterDims(gradient.lengthOf());

  NDArray::prepareSpecialUse({&update, &stateMsg, &stateMsdx}, {&gradient, &initStateMsg, &initStateMsdx});
  BUILD_SINGLE_SELECTOR(
      gradient.dataType(), adaDeltaUpdaterCudaLauncher,
      (updater2Dims.y, updater2Dims.x,updater2Dims.z, context->getCudaStream(), gradient.specialBuffer(), gradient.specialShapeInfo(),
          initStateMsg.specialBuffer(), initStateMsg.specialShapeInfo(), initStateMsdx.specialBuffer(),
          initStateMsdx.specialShapeInfo(), update.specialBuffer(), update.specialShapeInfo(), stateMsg.specialBuffer(),
          stateMsg.specialShapeInfo(), stateMsdx.specialBuffer(), stateMsdx.specialShapeInfo(), dRho, dEpsilon),
      SD_FLOAT_TYPES);
  NDArray::registerSpecialUse({&update, &stateMsg, &stateMsdx}, {&gradient, &initStateMsg, &initStateMsdx});

  manager.synchronize();
}

}  // namespace helpers
}  // namespace ops
}  // namespace sd
