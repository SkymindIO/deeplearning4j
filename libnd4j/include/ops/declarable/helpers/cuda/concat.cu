#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 20.04.2018
//


#include<ops/declarable/helpers/transforms.h>
#include <array/ResultSet.h>
#include <helpers/ShapeUtils.h>
#include <numeric>
#include <NDArrayFactory.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <PointersManager.h>
#include <ConstantTadHelper.h>

namespace nd4j {
    namespace ops {
        namespace helpers {
            ///////////////////////////////////////////////////////////////////
            template<typename T>
            __global__ static void concatCuda(const int numOfArrs, void* pVx,  void* pxShapeInfo, void* pVz, void* pzShapeInfo) {

                __shared__ int arrIdx, blocksPerArr;
                __shared__ T *x, *z;
                __shared__ Nd4jLong *zShapeInfo, *xShapeInfo, arrLen, arrLenPerBlock, start, end;

                if (threadIdx.x == 0) {

                    blocksPerArr = (gridDim.x + numOfArrs - 1) / numOfArrs;     // ceil
                    arrIdx = blockIdx.x / blocksPerArr;

                    x = reinterpret_cast<T*>(reinterpret_cast<void**>(pVx)[arrIdx]);
                    z = reinterpret_cast<T*>(reinterpret_cast<void**>(pVz)[arrIdx]);
                    xShapeInfo = reinterpret_cast<Nd4jLong**>(pxShapeInfo)[arrIdx];
                    zShapeInfo = reinterpret_cast<Nd4jLong**>(pzShapeInfo)[arrIdx];
                    arrLen = shape::length(xShapeInfo);

                    arrLenPerBlock = (arrLen + blocksPerArr - 1) / blocksPerArr;  // ceil

                    start = (blockIdx.x % blocksPerArr) * arrLenPerBlock;
                    end   = (start + arrLenPerBlock) > arrLen ? arrLen : (start + arrLenPerBlock);
                }

                __syncthreads();

                for (Nd4jLong i = start + threadIdx.x; i < end; i += blockDim.x)
                    z[shape::getIndexOffset(i, zShapeInfo, arrLen)] = x[shape::getIndexOffset(i, xShapeInfo, arrLen)];
            }

///////////////////////////////////////////////////////////////////
            template<typename T>
            __host__ static void concatCudaLauncher(const int numOfArrs, const hipStream_t *stream,  void* pVx, void* pxShapeInfo, void* pVz, void* pzShapeInfo) {

                concatCuda<T><<<512, 256, 1024, *stream>>>(numOfArrs, pVx, pxShapeInfo, pVz, pzShapeInfo);
            }
            BUILD_SINGLE_TEMPLATE(template void concatCudaLauncher,  (const int numOfArrs, const hipStream_t *stream, void* pVx, void* pxShapeInfo, void* pVz, void* pzShapeInfo), LIBND4J_TYPES);

            //////////////////////////////////////////////////////////////////////////
            void concat(nd4j::LaunchContext * context, const std::vector<NDArray*>& inArrs, NDArray& output, const int axis) {

                const int numOfArrs = inArrs.size();
                for(int i = 0; i < numOfArrs; ++i)
                    if(!inArrs[i]->isActualOnDeviceSide()) inArrs[i]->syncToDevice();

                const int rank  = inArrs[0]->rankOf();
                const int rank2 = 2*rank;
                std::vector<std::vector<Nd4jLong>> indices(numOfArrs, std::vector<Nd4jLong>(rank2,0));

                // take into account indices for first array
                indices[0][2 * axis + 1] = inArrs[0]->sizeAt(axis);

                // loop through the rest of input arrays
                for(int i = 1; i < numOfArrs; ++i) {
                    indices[i][2 * axis]     = indices[i-1][2 * axis + 1];                                // index start from
                    indices[i][2 * axis + 1] = indices[i-1][2 * axis + 1] + inArrs[i]->sizeAt(axis);      // index end with (excluding)
                }

                std::vector<NDArray*> outSubArrs(numOfArrs);
                for(int i = 0; i < numOfArrs; ++i)
                    outSubArrs[i] = new NDArray(output(indices[i], true));

                // prepare arrays of pointers on buffers and shapes
                std::vector<void*>     hOutBuffers(numOfArrs), hInBuffers(numOfArrs);
                std::vector<Nd4jLong*> hOutShapeInfo(numOfArrs), hInShapeInfo(numOfArrs);
                for(int i = 0; i < numOfArrs; ++i) {
                    hOutBuffers[i]   = outSubArrs[i]->getSpecialBuffer();
                    hInBuffers[i]    =     inArrs[i]->getSpecialBuffer();
                    hOutShapeInfo[i] = outSubArrs[i]->getSpecialShapeInfo();
                    hInShapeInfo[i]  =     inArrs[i]->getSpecialShapeInfo();
                }

                // allocate and copy all buffers and shapes arrays to global memory
                PointersManager manager(context, "helpers::concat");
                void* dOutBuffers	= manager.replicatePointer(hOutBuffers.data(),   hOutBuffers.size() * sizeof(void*));
                void* dInBuffers	= manager.replicatePointer(hInBuffers.data(),    hInBuffers.size() * sizeof(void*));
                void* dInShapeInfo  = manager.replicatePointer(hInShapeInfo.data(),  hInShapeInfo.size() * sizeof(Nd4jLong*));
                void* dOutShapeInfo = manager.replicatePointer(hOutShapeInfo.data(), hOutShapeInfo.size() * sizeof(Nd4jLong*));

                BUILD_SINGLE_SELECTOR(inArrs[0]->dataType(), concatCudaLauncher, (numOfArrs, context->getCudaStream(), dInBuffers, dInShapeInfo, dOutBuffers, dOutShapeInfo), LIBND4J_TYPES);

                manager.synchronize();

                for(int i = 0; i < numOfArrs; ++i)
                    delete outSubArrs[i];

                for(int i = 0; i < numOfArrs; ++i)
                    inArrs[i]->tickReadHost();

                output.tickWriteDevice();
            }
        }
    }
}