/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author sgazeos@gmail.com
//

#include <ops/declarable/helpers/image_suppression.h>
//#include <blas/NDArray.h>

namespace nd4j {
namespace ops {
namespace helpers {

    template <typename T>
    static void nonMaxSuppressionV2_(NDArray* boxes, NDArray* scales, int maxSize, double threshold, NDArray* output) {

    }

    void nonMaxSuppressionV2(NDArray* boxes, NDArray* scales, int maxSize, double threshold, NDArray* output) {
        BUILD_SINGLE_SELECTOR(output->dataType(), nonMaxSuppressionV2_, (boxes, scales, maxSize, threshold, output), NUMERIC_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template void nonMaxSuppressionV2_, (NDArray* boxes, NDArray* scales, int maxSize, double threshold, NDArray* output), NUMERIC_TYPES);

}
}
}