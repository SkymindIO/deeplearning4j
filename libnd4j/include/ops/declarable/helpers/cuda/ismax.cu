/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma, created on 21.09.2018
// @author raver119@gmail.com
//


#include <helpers/TAD.h>
#include<ops/declarable/helpers/ismax.h>

namespace nd4j 	  {
namespace ops 	  {
namespace helpers {

template <typename T>
static void ismax_(const NDArray* input, NDArray* output, const std::vector<int>& dimensions) {

}


void ismax(const NDArray *input, NDArray *output, const std::vector<int>& dimensions) {
    BUILD_SINGLE_SELECTOR(input->dataType(), ismax_, (input, output, dimensions), LIBND4J_TYPES);
}

BUILD_SINGLE_TEMPLATE(template void ismax_, (const NDArray *input, NDArray *output, const std::vector<int>& dimensions), LIBND4J_TYPES);

}
}
}

