#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author GS <sgazeos@gmail.com>
//
#include <array/NDArrayFactory.h>
#include <exceptions/cuda_exception.h>
#include <helpers/ConstantTadHelper.h>
#include <helpers/PointersManager.h>
#include <helpers/ShapeUtils.h>
#include <helpers/TAD.h>
#include <ops/declarable/helpers/segment.h>
#include <ops/declarable/helpers/segment_common.h>
#include <execution/cuda/LaunchDims.h>
namespace sd {
namespace ops {
namespace helpers {
// -------------------------------------------------------------------------------------------------------------- //
template <typename T, typename I>
static SD_KERNEL void unsortedSegmentSqrtNLinearKernel(T* input, sd::LongType const* inputShape, I* indices,
                                                       sd::LongType const* indicesShape, sd::LongType* starts, sd::LongType* lengths,
                                                       sd::LongType numOfClasses, T* output,
                                                       sd::LongType const* outputShape) {
  __shared__ sd::LongType xLen, zLen;

  if (threadIdx.x == 0) {
    xLen = shape::length(inputShape);
    zLen = shape::length(outputShape);
  }
  __syncthreads();

  auto start = threadIdx.x + blockIdx.x * blockDim.x;
  auto step = blockDim.x * gridDim.x;

  for (auto idx = start; idx < xLen; idx += step) {
    auto yIndex = shape::getIndexOffset(idx, indicesShape);
    auto segment = indices[yIndex];
    auto zIndex = shape::getIndexOffset(segment, outputShape);
    if (lengths[segment] == 0) continue;
    auto xIndex = shape::getIndexOffset(idx, inputShape);
    if(xIndex >= xLen)
      continue;
    sd::math::atomics::sd_atomicAdd(&output[zIndex], input[xIndex] / sd::math::sd_sqrt<sd::LongType, T>(lengths[segment]));
  }
}
// -------------------------------------------------------------------------------------------------------------- //
// SegmentSqrtN kernel
template <typename T, typename I>
static SD_KERNEL void segmentSqrtNTadKernel(T* inputBuf, sd::LongType const* inputShape, sd::LongType const* inputTads,
                                            sd::LongType const* inputTadOffsets, I* indices, sd::LongType* starts,
                                            sd::LongType* lengths, sd::LongType numOfClasses, void* outputBuf,
                                            sd::LongType const* outputShape, sd::LongType const* outputTads,
                                            sd::LongType const* outputTadOffsets, sd::LongType numIndices) {

  if(blockIdx.x >= numIndices)
    return;
  __shared__ sd::LongType len, total;


  if (threadIdx.x == 0) {
    total = shape::sizeAt(inputShape, 0);
    len = shape::length(inputTads);
  }
  __syncthreads();

  for (auto idx = blockIdx.x; idx < total; idx += gridDim.x) {
    auto segment = indices[idx];
    auto x = inputBuf + inputTadOffsets[idx];
    auto z = reinterpret_cast<T*>(outputBuf) + outputTadOffsets[segment];
    auto start = starts[segment];
    auto finish = start + lengths[segment];

    for (auto e = threadIdx.x; e < len; e += blockDim.x) {
      auto xIndex = shape::getIndexOffset(e, inputTads);
      auto zIndex = shape::getIndexOffset(e, outputTads);
      sd::math::atomics::sd_atomicAdd(&z[zIndex], x[xIndex] / sd::math::sd_sqrt<sd::LongType, T>(lengths[segment]));
    }
  }
}
// -------------------------------------------------------------------------------------------------------------- //
template <typename T, typename I>
static void unsortedSegmentSqrtNFunctor_(sd::LaunchContext* context, NDArray* input, NDArray* indices,
                                         sd::LongType numOfClasses, NDArray* output) {
  auto stream = context->getCudaStream();
  NDArray classesRangesBegs = NDArrayFactory::create<sd::LongType>('c', {numOfClasses}, context);
  NDArray classesRangesLens = NDArrayFactory::create<sd::LongType>('c', {numOfClasses}, context);
  classesRangesBegs.assign(indices->lengthOf());
  classesRangesLens.assign(0);
  dim3 dims= getLaunchDims("segmentSqrtN");
  fillUpSegments(indices, numOfClasses, classesRangesBegs, classesRangesLens);
  sd::LongType* begins = reinterpret_cast<sd::LongType*>(classesRangesBegs.specialBuffer());
  sd::LongType* lengths = reinterpret_cast<sd::LongType*>(classesRangesLens.specialBuffer());
  output->nullify();
  if (input->isVector()) {
    unsortedSegmentSqrtNLinearKernel<T, I><<<dims.x, dims.y, dims.z, *stream>>>(
        input->dataBuffer()->specialAsT<T>(), input->specialShapeInfo(), indices->dataBuffer()->specialAsT<I>(),
        indices->specialShapeInfo(), begins, lengths, numOfClasses, output->dataBuffer()->specialAsT<T>(),
        output->specialShapeInfo());
  } else {
    output->nullify();
    sd::LongType zero = 0;
    std::vector<sd::LongType> *dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), 1,&zero);
    auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
    auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
    auto inputTads = packX->specialShapeInfo();
    auto inputTadOffsets = packX->specialOffsets();
    auto outputTads = packZ->specialShapeInfo();
    auto outputTadOffsets = packZ->specialOffsets();
    dims.x = input->sizeAt(0);
    segmentSqrtNTadKernel<T, I><<<dims.x, dims.y, dims.z, *stream>>>(
        input->dataBuffer()->specialAsT<T>(), input->specialShapeInfo(), inputTads, inputTadOffsets,
        indices->dataBuffer()->specialAsT<I>(), begins, lengths, numOfClasses, output->specialBuffer(),
        output->specialShapeInfo(), outputTads, outputTadOffsets, indices->lengthOf());
    delete dimensions;
  }
}
// -------------------------------------------------------------------------------------------------------------- //
void unsortedSegmentSqrtNFunctor(sd::LaunchContext* context, NDArray* input, NDArray* indices,
                                 sd::LongType numOfClasses, NDArray* output) {
  NDArray::prepareSpecialUse({output}, {input, indices});
  BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), unsortedSegmentSqrtNFunctor_,
                        (context, input, indices, numOfClasses, output), SD_FLOAT_TYPES, SD_INDEXING_TYPES);
  NDArray::registerSpecialUse({output}, {input, indices});
}
// -------------------------------------------------------------------------------------------------------------- //
template <typename T, typename I>
static SD_KERNEL void segmentSqrtNBPLinearKernel(void* inputBuf, sd::LongType const* inputShape, void* eps,
                                                 sd::LongType const* epsShape, void* indicesBuf,
                                                 sd::LongType const* indicesShape, sd::LongType* lengths, void* outputBuf,
                                                 sd::LongType const* outputShape) {
  __shared__ T* x;
  __shared__ T* gradIn;
  __shared__ T* gradOut;
  __shared__ I* y;
  __shared__ T* z;
  __shared__ sd::LongType xLen, gradLen;

  if (threadIdx.x == 0) {
    xLen = shape::length(inputShape);
    x = reinterpret_cast<T*>(inputBuf);
    y = reinterpret_cast<I*>(indicesBuf);
    z = reinterpret_cast<T*>(outputBuf);
    gradOut = reinterpret_cast<T*>(eps);
    gradLen = shape::length(epsShape);
  }
  __syncthreads();

  auto start = blockIdx.x * blockDim.x + threadIdx.x;
  auto step = gridDim.x * blockDim.x;

  for (auto e = start; e < xLen; e += step) {
    auto zOffset = shape::getIndexOffset(e, outputShape);
    auto xOffset = shape::getIndexOffset(e, inputShape);
    auto yOffset = shape::getIndexOffset(e, indicesShape);
    auto classIndex = y[yOffset];
    auto gradOffsetO = shape::getIndexOffset(classIndex, epsShape);

    z[zOffset] = T(gradOut[gradOffsetO] / math::sd_sqrt<sd::LongType, float>(lengths[classIndex]));
  }
}
// -------------------------------------------------------------------------------------------------------------- //

template <typename T, typename I>
static SD_KERNEL void segmentSqrtNBPTadKernel(void* inputBuf, sd::LongType const* inputShape, void* eps,
                                              sd::LongType const* epsShape, void* indicesBuf,
                                              sd::LongType const* indicesShape, sd::LongType* lengths, void* outputBuf,
                                              sd::LongType const* outputShape, sd::LongType const* inputTad,
                                              sd::LongType const* inputOffsets, sd::LongType const* gradOutTad,
                                              sd::LongType const* gradOutOffsets, sd::LongType const* outTad,
                                              sd::LongType const* outOffsets) {
  __shared__ T* x;
  __shared__ T* gradOut;
  __shared__ I* y;
  __shared__ T* z;
  __shared__ sd::LongType xLen, yLen, gradLen, currentLen;

  if (threadIdx.x == 0) {
    xLen = shape::length(inputShape);
    x = reinterpret_cast<T*>(inputBuf);
    y = reinterpret_cast<I*>(indicesBuf);
    z = reinterpret_cast<T*>(outputBuf);
    yLen = shape::length(indicesShape);
    gradOut = reinterpret_cast<T*>(eps);
    gradLen = shape::length(epsShape);
    currentLen = shape::length(outTad);
  }
  __syncthreads();

  for (auto i = blockIdx.x; i < yLen; i += gridDim.x) {
    auto segment = y[i];  // yIndex];
    T* currentOut = z + outOffsets[i];
    T* outGrad = gradOut + gradOutOffsets[segment];

    for (auto e = threadIdx.x; e < currentLen; e += blockDim.x) {
      auto zIndex = shape::getIndexOffset(e, outTad);
      auto gradIndex = shape::getIndexOffset(e, gradOutTad);
      if (lengths[segment] > 0)
        currentOut[zIndex] = T(outGrad[gradIndex] / math::sd_sqrt<sd::LongType, float>(lengths[segment]));
    }
  }
}
// -------------------------------------------------------------------------------------------------------------- //

template <typename T, typename I>
static sd::Status unsortedSegmentSqrtNFunctorBP_(sd::LaunchContext* context, NDArray* input, NDArray* indices,
                                                 NDArray* gradOut, sd::LongType numOfClasses, NDArray* output) {
  auto stream = context->getCudaStream();
  NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
  auto numClasses = indices->e<sd::LongType>(indices->lengthOf() - 1) + 1;
  NDArray classesRangesLens = NDArrayFactory::create<sd::LongType>('c', {numClasses}, context);
  NDArray classesRangesBegs = NDArrayFactory::create<sd::LongType>('c', {numClasses}, context);

  classesRangesBegs.assign(indices->lengthOf());
  classesRangesLens.assign(0);
  fillUpSegments(indices, numClasses, classesRangesBegs, classesRangesLens);
  sd::LongType* begins = reinterpret_cast<sd::LongType*>(classesRangesBegs.specialBuffer());
  sd::LongType* lengths = reinterpret_cast<sd::LongType*>(classesRangesLens.specialBuffer());

  if (input->isVector()) {
    sd::LongType loop_size = input->lengthOf();
    auto numOfClasses = gradOut->lengthOf();
    segmentSqrtNBPLinearKernel<T, I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(
        input->specialBuffer(), input->specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
        indices->specialBuffer(), indices->specialShapeInfo(), lengths, output->specialBuffer(),
        output->specialShapeInfo());
  } else {
    sd::LongType zero = 0;
    std::vector<sd::LongType> *dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), 1,&zero);
    auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
    auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
    auto packGradOut = sd::ConstantTadHelper::getInstance().tadForDimensions(gradOut->shapeInfo(), dimensions);
    auto inputTads = packX->specialShapeInfo();
    auto inputTadOffsets = packX->specialOffsets();
    auto outputTads = packZ->specialShapeInfo();
    auto outputTadOffsets = packZ->specialOffsets();
    auto gradOutTads = packGradOut->specialShapeInfo();
    auto gradOutTadOffsets = packGradOut->specialOffsets();
    dim3 segmentBpTad2 = segmentBpTad(indices->lengthOf(),input->lengthOf());

    segmentSqrtNBPTadKernel<T, I><<<segmentBpTad2.y, segmentBpTad2.x, segmentBpTad2.z, *stream>>>(
        input->specialBuffer(), input->specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
        indices->specialBuffer(), indices->specialShapeInfo(), lengths, output->specialBuffer(),
        output->specialShapeInfo(), inputTads, inputTadOffsets, gradOutTads, gradOutTadOffsets, outputTads,
        outputTadOffsets);
    delete dimensions;
  }
  NDArray::registerSpecialUse({output}, {input, indices, gradOut});

  return sd::Status::OK;
}
// -------------------------------------------------------------------------------------------------------------- //
sd::Status unsortedSegmentSqrtNFunctorBP(sd::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* gradOut,
                                         sd::LongType numOfClasses, NDArray* output) {
  NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
  BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return unsortedSegmentSqrtNFunctorBP_,
                        (context, input, indices, gradOut, numOfClasses, output), SD_FLOAT_TYPES, SD_INDEXING_TYPES);
  NDArray::registerSpecialUse({output}, {input, indices, gradOut});
}
}  // namespace helpers
}  // namespace ops
}  // namespace sd
