#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author GS <sgazeos@gmail.com>
//

#include <ops/declarable/helpers/bds.h>
#include <Status.h>


namespace nd4j {
namespace ops {
namespace helpers {


    template <typename T>
    static __global__ void bdsLoopKernel(void const* inputX, Nd4jLong const* inputXshape, void const* inputY, Nd4jLong const* inputYshape, void* output, Nd4jLong* outputShape) {
        __shared__ T const* x;
        __shared__ T const* y;
        __shared__ T* z;
        __shared__ bool speedWay;
        //__shared__ int indexX, indexY;
        __shared__ Nd4jLong xLen, yLen, outputLen;
        if (threadIdx.x == 0) {
            x = reinterpret_cast<T const*>(inputX);
            y = reinterpret_cast<T const*>(inputY);
            z = reinterpret_cast<T*>(output);
            xLen = shape::length(inputXshape);
            yLen = shape::length(inputYshape);
            outputLen = shape::length(outputShape);
            speedWay = true;
            speedWay = speedWay && (shape::elementWiseStride(inputXshape) == 1);
            speedWay = speedWay && (shape::elementWiseStride(inputYshape) == 1);
            speedWay = speedWay && (shape::elementWiseStride(outputShape) == 1);

        }
        __syncthreads();

        auto tid = threadIdx.x + blockIdx.x * blockDim.x;
        auto step = blockDim.x * gridDim.x;
        for (int e = tid; e < outputLen; e += step) {
            T val;
            if (speedWay) {
                if (e < nd4j::math::nd4j_min(yLen, xLen)) {
                    val = nd4j::math::nd4j_max(x[e], y[e]);
                } else if (e < xLen) {
                    val = nd4j::math::nd4j_max(x[e], y[yLen - 1]);
                } else {
                    val = nd4j::math::nd4j_max(x[xLen - 1], y[e]);
                }
                z[e] = val;
            }
            else {
                auto xIndex = e < xLen?shape::getIndexOffset(e, inputXshape, xLen):shape::getIndexOffset(xLen, inputXshape, xLen);
                auto yIndex = e < yLen?shape::getIndexOffset(e, inputYshape, yLen):shape::getIndexOffset(yLen - 1, inputYshape, yLen);
                auto zIndex = shape::getIndexOffset(e, outputShape, outputLen);
                z[zIndex] = nd4j::math::nd4j_max(x[xIndex], y[yIndex]);
            }
        }
    }

    template <typename T>
    static void bdsLoopH(hipStream_t* stream, void const* inputX, Nd4jLong const* inputXshape, void const* inputY, Nd4jLong const* inputYshape, void* output, Nd4jLong* outputShape) {
        bdsLoopKernel<T><<<1, 256, 512, *stream>>>(inputX, inputXshape, inputY, inputYshape, output, outputShape);

    }

    Nd4jStatus bdsFunctor(nd4j::LaunchContext * context, NDArray* x_shape, NDArray* y_shape, NDArray* output) {
        //int e = 0, x = 0, y = 0;
        NDArray::prepareSpecialUse({output}, {x_shape, y_shape});
        if (x_shape->lengthOf() == 1 || y_shape->lengthOf() == 1) {// except case
            x_shape->syncToHost(); y_shape->syncToHost();
            if (x_shape->lengthOf() == y_shape->lengthOf()) {
                auto greater = (x_shape->e<Nd4jLong>(0) < y_shape->e<Nd4jLong>(0) ? y_shape : x_shape);
                output->assign(greater);
            }
            else {
                auto lesser = (x_shape->lengthOf() == 1 ? x_shape : y_shape);
                auto greater = (x_shape->lengthOf() == 1 ? y_shape : x_shape);
                output->assign(greater);
                auto lastG = greater->lengthOf() - 1;
                auto lastL = lesser->lengthOf() - 1;
                if (greater->e<Nd4jLong>(lastG) < lesser->e<Nd4jLong>(lastL))
                    output->p(lastG, lesser->e(lastL));
                output->syncToDevice();
            }
        }
        else {
            //bdsLoopH(context->getCudaStream(), x->getSpecialBuffer(), x->getSpecialShapeInfo(), y->getSpecialBuffer(), y->getSpecialShape(), output->specialBuffer(), output->specialShapeInfo())
            BUILD_SINGLE_SELECTOR(output->dataType(), bdsLoopH, (context->getCudaStream(), x_shape->getSpecialBuffer(), x_shape->getSpecialShapeInfo(), y_shape->getSpecialBuffer(), y_shape->getSpecialShapeInfo(), output->specialBuffer(), output->specialShapeInfo()), NUMERIC_TYPES);
        }
        NDArray::registerSpecialUse({output}, {x_shape, y_shape});
        return Status::OK();
        return Status::OK();
    }

}
}
}