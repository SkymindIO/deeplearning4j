/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author GS (sgazeos@gmail.com), created on 10/1/2018
//


#include<ops/declarable/helpers/cross.h>
#include <helpers/ShapeUtils.h>
#include <ops/declarable/CustomOperations.h>

namespace nd4j 	  {
namespace ops 	  {
namespace helpers {


//////////////////////////////////////////////////////////////////////////
template <typename T>
static void weightedCrossEntropyWithLogitsFunctor_(NDArray const* targets, NDArray const* input, NDArray const* weights, NDArray* output) {

}

void weightedCrossEntropyWithLogitsFunctor(NDArray const* targets, NDArray const* input, NDArray const* weights, NDArray* output) {
    BUILD_SINGLE_SELECTOR(targets->dataType(), weightedCrossEntropyWithLogitsFunctor_, (targets, input, weights, output), FLOAT_TYPES);
}
BUILD_SINGLE_TEMPLATE(template void weightedCrossEntropyWithLogitsFunctor_, (NDArray const* targets, NDArray const* input, NDArray const* weights, NDArray* output), FLOAT_TYPES);

}
}
}