#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Oleh Semeniv (oleg.semeniv@gmail.com)
// @author Abdelrauf (rauf@konduit.ai)

// https://arxiv.org/pdf/2010.07468.pdf
#include <helpers/PointersManager.h>
#include <math/platformmath.h>
#include <math/templatemath.h>
#include <ops/declarable/helpers/updatersHelpers.h>
#include <system/op_boilerplate.h>

#include "execution/cuda/LaunchDims.h"
#include "helpers/DebugHelper.h"


namespace sd {
namespace ops {
namespace helpers {

///////////////////////////////////////////////////////////////////
template <typename T>
SD_KERNEL void adaBeliefUpdaterCuda(const void* vx, const LongType* xShapeInfo, const void* vinv,
                                    const LongType* invShapeInfo, const void* vinm,
                                    const LongType* inmShapeInfo, void* vz, const LongType* zShapeInfo,
                                    void* vstV, const LongType* stvShapeInfo, void* vstM,
                                    const LongType* stmShapeInfo, const T lr, const T beta1, const T beta2,
                                    const T epsilon, const T iteration) {
  const auto grad = reinterpret_cast<const T*>(vx);
  const auto initU = reinterpret_cast<const T*>(vinv);
  const auto initM = reinterpret_cast<const T*>(vinm);

  auto up = reinterpret_cast<T*>(vz);
  auto stU = reinterpret_cast<T*>(vstV);
  auto stM = reinterpret_cast<T*>(vstM);

  __shared__ LongType xLen;
  __shared__ T epsilonT;
  __shared__ bool bOrdering, bXZsame, bXInUSame, bXStUSame, bXInMSame, bXStMSame;

  if (threadIdx.x == 0) {
    xLen = shape::length(xShapeInfo);

    T beta1T = math::sd_pow<T, T, T>(beta1, (iteration + 1));
    T beta2T = math::sd_pow<T, T, T>(beta2, (iteration + 1));

    epsilonT = lr * math::sd_sqrt<T, T>(1. - beta2T) / (1.0 - beta1T);
    if (math::sd_isnan(epsilonT) || 0 == epsilonT || math::sd_isinf(epsilonT)) epsilonT = epsilon;

    bOrdering = shape::order(xShapeInfo) == shape::order(zShapeInfo) &&
                shape::order(zShapeInfo) == shape::order(stmShapeInfo) &&
                shape::order(stmShapeInfo) == shape::order(inmShapeInfo) &&
                shape::order(inmShapeInfo) == shape::order(stvShapeInfo) &&
                shape::order(stvShapeInfo) == shape::order(invShapeInfo);

    bXZsame = shape::haveSameShapeAndStrides(xShapeInfo, zShapeInfo);
    bXInUSame = shape::haveSameShapeAndStrides(xShapeInfo, invShapeInfo);
    bXStUSame = shape::haveSameShapeAndStrides(xShapeInfo, stvShapeInfo);
    bXInMSame = shape::haveSameShapeAndStrides(xShapeInfo, inmShapeInfo);
    bXStMSame = shape::haveSameShapeAndStrides(xShapeInfo, stmShapeInfo);
  }
  __syncthreads();

  LongType coords[SD_MAX_RANK];

  for (LongType i = blockIdx.x * blockDim.x + threadIdx.x; i < xLen; i += gridDim.x * blockDim.x) {
    LongType xOffset, zOffset, initMOffset, initUOffset, stMOffset, stUOffset;

    INDEX2COORDS(i, xShapeInfo, coords);
    COORDS2INDEX(shape::rank(xShapeInfo), shape::shapeOf(xShapeInfo), coords, xOffset);
    zOffset = bXZsame ? xOffset : COORDS2INDEX(shape::rank(zShapeInfo), shape::shapeOf(zShapeInfo), coords, zOffset);
    initUOffset = bXInUSame ? xOffset : COORDS2INDEX(shape::rank(invShapeInfo), shape::shapeOf(invShapeInfo), coords, initUOffset);
    stUOffset = bXStUSame ? xOffset : COORDS2INDEX(shape::rank(stvShapeInfo), shape::shapeOf(stvShapeInfo), coords, stUOffset);
    initMOffset = bXInMSame ? xOffset : COORDS2INDEX(shape::rank(inmShapeInfo), shape::shapeOf(inmShapeInfo), coords, initMOffset);
    stMOffset = bXStMSame ? xOffset : COORDS2INDEX(shape::rank(stmShapeInfo), shape::shapeOf(stmShapeInfo), coords, stMOffset);

    stM[stMOffset] = beta1 * initM[initMOffset] + grad[xOffset] * (1 - beta1);
    stU[stUOffset] = beta2 * initU[initUOffset] +
                     (grad[xOffset] - stM[stMOffset]) * (grad[xOffset] - stM[stMOffset]) * (1 - beta2) + epsilon;

    up[zOffset] = (stM[stMOffset] * epsilonT) / (math::sd_sqrt<T, T>(stU[stUOffset]) + epsilon);
  }
}
///////////////////////////////////////////////////////////////////
template <typename T>
void adaBeliefUpdaterCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMemory,
                                  const hipStream_t* stream, const void* vx, const LongType* xShapeInfo,
                                  const void* vinv, const LongType* invShapeInfo, const void* vinm,
                                  const LongType* inmShapeInfo, void* vz, const LongType* zShapeInfo,
                                  void* vstV, const LongType* stvShapeInfo, void* vstM,
                                  const LongType* stmShapeInfo, const double dLr, const double dBeta1,
                                  const double dBeta2, const double dEpsilon, const int nIteration) {
  const T lr = static_cast<T>(dLr);
  const T beta1 = static_cast<T>(dBeta1);
  const T beta2 = static_cast<T>(dBeta2);
  T epsilon = static_cast<T>(dEpsilon);
  //fp16 to prevent underflow
  if(epsilon == 0.0) {
    epsilon = static_cast<T>(1e-7);
  }
  const T iteration = static_cast<T>(nIteration);
  adaBeliefUpdaterCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMemory, *stream>>>(
      vx, xShapeInfo, vinv, invShapeInfo, vinm, inmShapeInfo, vz, zShapeInfo, vstV, stvShapeInfo, vstM, stmShapeInfo,
      lr, beta1, beta2, epsilon, iteration);
  sd::DebugHelper::checkErrorCode(const_cast<hipStream_t *>(stream), "adaBeliefUpdaterCuda failed");

}

///////////////////////////////////////////////////////////////////
void updaterAdaBelief(LaunchContext* context, NDArray& gradient, NDArray& initStateU,
                      NDArray& initStateM, NDArray& update, NDArray& stateU, NDArray& stateM, const double dLr,
                      const double dBeta1, const double dBeta2, const double dEpsilon, const int nIteration) {
  PointersManager manager(context, "adamUpdater");

  dim3 updaterDims2 = updaterDims(gradient.lengthOf());
  NDArray::prepareSpecialUse({&update, &stateU, &stateM}, {&gradient, &initStateU, &initStateM});

  BUILD_SINGLE_SELECTOR(gradient.dataType(), adaBeliefUpdaterCudaLauncher,
                        (updaterDims2.y, updaterDims2.x, updaterDims2.z,context->getCudaStream(), gradient.specialBuffer(),
                            gradient.specialShapeInfo(), initStateU.specialBuffer(), initStateU.specialShapeInfo(),
                            initStateM.specialBuffer(), initStateM.specialShapeInfo(), update.specialBuffer(),
                            update.specialShapeInfo(), stateU.specialBuffer(), stateU.specialShapeInfo(),
                            stateM.specialBuffer(), stateM.specialShapeInfo(), dLr, dBeta1, dBeta2, dEpsilon, nIteration),
                        SD_FLOAT_TYPES);

  NDArray::registerSpecialUse({&update, &stateU, &stateM}, {&gradient, &initStateU, &initStateM});

  manager.synchronize();
}

}  // namespace helpers
}  // namespace ops
}  // namespace sd
