/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//

#include <ops/declarable/helpers/max_pooling.h>
#include <ops/declarable/generic/helpers/convolutions.h>


namespace nd4j {
namespace ops {
namespace helpers {

    template <typename T>
    static void maxPoolingFunctor_(nd4j::graph::Context& block, NDArray* input, NDArray* values, std::vector<int> const& params, NDArray* indices) {

    }

    void maxPoolingFunctor(nd4j::graph::Context& block, NDArray* input, NDArray* values, std::vector<int> const& params, NDArray* indices) {
        BUILD_SINGLE_SELECTOR(input->dataType(), maxPoolingFunctor_, (block, input, values, params, indices), FLOAT_TYPES);
    }


    BUILD_SINGLE_TEMPLATE(template void maxPoolingFunctor_, (nd4j::graph::Context& block, NDArray* input, NDArray* values, std::vector<int> const& params, NDArray* indices), FLOAT_TYPES);

}
}
}