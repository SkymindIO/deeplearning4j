/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 19.04.2018
// @author raver119@gmail.com
//

#include <ops/declarable/helpers/activations.h>
#include <ShapeUtils.h>
#include <numeric>

namespace nd4j    {
namespace ops     {
namespace helpers {

    template <typename T>
    void _softMaxForVector(void *input, Nd4jLong *inShapeInfo, void *output, Nd4jLong *outShapeInfo) {

    }

    template <typename T>
    void _logSoftMaxForVector(void *input, Nd4jLong *inShapeInfo, void *output, Nd4jLong *outShapeInfo) {

    }

    ///////////////////////////////////////////////////////////////////
    void softMaxForVector(const NDArray& input, NDArray& output) {

        if(!input.isVector() || !output.isVector())
            throw std::runtime_error("ops::helpers::softMaxForVector function: input and output arrays must be vectors !");

        auto xType = input.dataType();
        BUILD_SINGLE_SELECTOR(xType, _softMaxForVector, (input.getBuffer(), input.getShapeInfo(), output.buffer(), output.shapeInfo()), FLOAT_TYPES);
    }


    ///////////////////////////////////////////////////////////////////
    void logSoftMaxForVector(const NDArray& input, NDArray& output) {

        if(!input.isVector() || !output.isVector())
            throw std::runtime_error("ops::helpers::logSoftMaxForVector function input and output arrays must be vectors !");

        auto xType = input.dataType();
        BUILD_SINGLE_SELECTOR(xType, _logSoftMaxForVector, (input.getBuffer(), input.getShapeInfo(), output.buffer(), output.shapeInfo()), FLOAT_TYPES);
    }

    //////////////////////////////////////////////////////////////////////////
    void softmax(const NDArray& input, NDArray& output, const int dimension) {

        const int rank = input.rankOf();

        if(input.isVector()) {
        
            if(rank == 1 || input.sizeAt(dimension) != 1)
                softMaxForVector(input, output);
            else
                output = 1.;
        }
        else {
            auto maxAlongDim = const_cast<NDArray&>(input).reduceAlongDims(reduce::Max, {dimension}, true);
            auto exponents = (input - maxAlongDim).transform(transform::Exp);
            auto sumAlongDim = exponents.reduceAlongDims(reduce::Sum, {dimension}, true);

            // FIXME: assign?
            output.assign(exponents / sumAlongDim);
        }
    }

    //////////////////////////////////////////////////////////////////////////
    void prelu(const NDArray& input, const NDArray& alpha, NDArray& output) {

    }

    //////////////////////////////////////////////////////////////////////////
    void preluBP(const NDArray& input, const NDArray& alpha, const NDArray& dLdO, NDArray& dLdI, NDArray& dLdA) {

    }

    BUILD_SINGLE_TEMPLATE(template void _softMaxForVector, (void *input, Nd4jLong *inShapeInfo, void *output, Nd4jLong *outShapeInfo), FLOAT_TYPES);
    BUILD_SINGLE_TEMPLATE(template void _logSoftMaxForVector, (void *input, Nd4jLong *inShapeInfo, void *output, Nd4jLong *outShapeInfo), FLOAT_TYPES);

    bool checkAlphaShapeLen(std::vector<Nd4jLong> const& expectedShape, Nd4jLong shapeLen) {
        Nd4jLong expectedAlphaLen = std::accumulate(expectedShape.cbegin(), expectedShape.cend(), 1, std::multiplies<Nd4jLong>());
        return expectedAlphaLen == shapeLen;
    }

    template <typename T>
    static void thresholdRelu_(NDArray const& input, double threshold, NDArray& output) {
        auto routine = LAMBDA_T(_x, threshold) {
            return _x > (T)threshold? _x: (T)0.f;
        };
        const_cast<NDArray&>(input).applyLambda<T>(routine, &output);
    }

    void thresholdRelu(NDArray const& input, double threshold, NDArray& output) {
        BUILD_SINGLE_SELECTOR(input.dataType(), thresholdRelu_, (input, threshold, output), FLOAT_TYPES);
    }

    template <typename T>
    static void thresholdReluDerivative_(NDArray* input, double theta, NDArray* dLdO, NDArray* output) {

    }

    void thresholdReluDerivative(NDArray* input, double threshold, NDArray* dLdO, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), thresholdReluDerivative_, (input, threshold, dLdO, output), FLOAT_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void thresholdReluDerivative_, (NDArray* input, double threshold, NDArray* dLdO, NDArray* output), FLOAT_TYPES);

}
}
}

