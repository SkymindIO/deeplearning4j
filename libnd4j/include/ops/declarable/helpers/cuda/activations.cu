#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 19.04.2018
// @author raver119@gmail.com
//
#include <helpers/ConstantTadHelper.h>
#include <helpers/PointersManager.h>
#include <helpers/ShapeUtils.h>
#include <ops/declarable/helpers/activations.h>
#include <system/op_boilerplate.h>

#include <numeric>

#include "execution/cuda/LaunchDims.h"


namespace sd {
namespace ops {
namespace helpers {

///////////////////////////////////////////////////////////////////
template <typename X, typename Y>
void SD_KERNEL preluCuda(const void *vx, const LongType *xShapeInfo, const void *vy, const LongType *yShapeInfo,
                         void *vz) {
  const auto x = reinterpret_cast<const X *>(vx);
  const auto y = reinterpret_cast<const Y *>(vy);
  auto z = reinterpret_cast<X *>(vz);

  __shared__ LongType xzLen;
  __shared__ int xzRank, yRank;

  if (threadIdx.x == 0) {
    xzLen = shape::length(xShapeInfo);

    xzRank = shape::rank(xShapeInfo);
    yRank = shape::rank(yShapeInfo);
  }
  __syncthreads();

  const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
  LongType coords[SD_MAX_RANK];

  for (int i = tid; i < xzLen; i += blockDim.x * gridDim.x) {
    shape::index2coords(i, xShapeInfo, coords);

    const auto xzOffset = shape::getOffset(xShapeInfo, coords);
    const auto xVal = x[xzOffset];

    if (xVal < 0) {
      for (LongType j = 0; j < yRank; ++j)
        if (yShapeInfo[j + 1] == 1) coords[j + 1] = 0;

      z[xzOffset] = xVal * y[shape::getOffset(yShapeInfo, coords + 1)];
    } else
      z[xzOffset] = xVal;
  }
}

///////////////////////////////////////////////////////////////////
template <typename X, typename Y>
void preluCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem,
                       const hipStream_t *stream, const void *vx, const LongType *xShapeInfo, const void *vy,
                       const LongType *yShapeInfo, void *vz) {
  preluCuda<X, Y><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vy, yShapeInfo, vz);
  sd::DebugHelper::checkGlobalErrorCode("prelu  failed");

}

///////////////////////////////////////////////////////////////////
void prelu(LaunchContext *context, const NDArray &input, const NDArray &alpha, NDArray &output) {
  PointersManager manager(context, "prelu");

  dim3 launchDims = getLaunchDims("prelu");

  const auto xType = input.dataType();
  const auto yType = alpha.dataType();

  NDArray::prepareSpecialUse({&output}, {&input, &alpha});
  BUILD_SINGLE_SELECTOR_TWICE(
      xType, preluCudaLauncher,
      (launchDims.x, launchDims.y, launchDims.z, context->getCudaStream(), input.specialBuffer(),
          input.specialShapeInfo(), alpha.specialBuffer(), alpha.specialShapeInfo(), output.specialBuffer()),
      SD_FLOAT_TYPES);
  NDArray::registerSpecialUse({&output}, {&input, &alpha});

  manager.synchronize();
}

///////////////////////////////////////////////////////////////////
template <typename X, typename Y>
void SD_KERNEL preluBPCuda(const void *vIn, const LongType *inShapeInfo, const void *vAlpha,
                           const LongType *alphaShapeInfo, const void *vdLdO, const LongType *dLdOShapeInfo,
                           void *vdLdI, const LongType *dLdIShapeInfo, void *vdLdA,
                           const LongType *dLdAShapeInfo) {
  const auto in = reinterpret_cast<const X *>(vIn);
  const auto alpha = reinterpret_cast<const Y *>(vAlpha);
  const auto dLdO = reinterpret_cast<const Y *>(vdLdO);
  auto dLdI = reinterpret_cast<Y *>(vdLdI);
  auto dLdA = reinterpret_cast<Y *>(vdLdA);

  __shared__ LongType inLen, totalThreads;
  __shared__ int inRank, alphaRank;

  if (threadIdx.x == 0) {
    inLen = shape::length(inShapeInfo);
    totalThreads = gridDim.x * blockDim.x;

    inRank = shape::rank(inShapeInfo);
    alphaRank = shape::rank(alphaShapeInfo);
  }
  __syncthreads();

  const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
  LongType coords[SD_MAX_RANK];

  for (int i = tid; i < inLen; i += totalThreads) {
    shape::index2coords(i, inShapeInfo, coords);

    const auto inOffset = shape::getOffset(inShapeInfo, coords);
    const auto dLdOOffset = shape::getOffset(dLdOShapeInfo, coords);
    const auto dLdIOffset = shape::getOffset(dLdIShapeInfo, coords);

    const auto xVal = in[inOffset];
    const auto grO = dLdO[dLdOOffset];

    if (xVal < 0) {
      for (LongType j = 0; j < alphaRank; ++j)
        if (alphaShapeInfo[j + 1] == 1) coords[j + 1] = 0;

      const auto alphaOffset = shape::getOffset(alphaShapeInfo, coords + 1);
      const auto dLdAOffset = shape::getOffset(dLdAShapeInfo, coords + 1);

      dLdI[dLdIOffset] = grO * alpha[alphaOffset];

      math::atomics::sd_atomicAdd<Y>(&dLdA[dLdAOffset], static_cast<Y>(grO * xVal));
    } else
      dLdI[dLdIOffset] = grO;
  }
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Y>
void SD_HOST preluBPCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem,
                                 const hipStream_t *stream, const void *vIn, const LongType *inShapeInfo,
                                 const void *vAlpha, const LongType *alphaShapeInfo, const void *vdLdO,
                                 const LongType *dLdOShapeInfo, void *vdLdI, const LongType *dLdIShapeInfo,
                                 void *vdLdA, const LongType *dLdAShapeInfo) {
  preluBPCuda<X, Y><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(
      vIn, inShapeInfo, vAlpha, alphaShapeInfo, vdLdO, dLdOShapeInfo, vdLdI, dLdIShapeInfo, vdLdA, dLdAShapeInfo);
  sd::DebugHelper::checkGlobalErrorCode("prelu bp failed");

}

//////////////////////////////////////////////////////////////////////////
void preluBP(LaunchContext *context, const NDArray &input, const NDArray &alpha, const NDArray &dLdO, NDArray &dLdI,
             NDArray &dLdA) {
  dLdA.nullify();

  PointersManager manager(context, "preluBP");

  dim3 launchDims = getLaunchDims("prelu");

  const auto xType = input.dataType();
  const auto zType = alpha.dataType();

  NDArray::prepareSpecialUse({&dLdI, &dLdA}, {&input, &alpha, &dLdO});
  BUILD_SINGLE_SELECTOR_TWICE(
      xType, preluBPCudaLauncher,
      (launchDims.x, launchDims.y, launchDims.z, context->getCudaStream(), input.specialBuffer(),
          input.specialShapeInfo(), alpha.specialBuffer(), alpha.specialShapeInfo(), dLdO.specialBuffer(),
          dLdO.specialShapeInfo(), dLdI.specialBuffer(), dLdI.specialShapeInfo(), dLdA.specialBuffer(),
          dLdA.specialShapeInfo()),
      SD_FLOAT_TYPES);
  NDArray::registerSpecialUse({&dLdI, &dLdA}, {&input, &alpha, &dLdO});

  manager.synchronize();
}

///////////////////////////////////////////////////////////////////
template <typename T>
SD_DEVICE void softMaxForVectorCuda(const void *vx, const LongType *xShapeInfo, void *vz,
                                    const LongType *zShapeInfo) {
  auto inBuff = reinterpret_cast<const T *>(vx);
  auto outBuff = reinterpret_cast<T *>(vz);

  __shared__ T shmemMax;
  __shared__ T shmemSum;
  __shared__ LongType tadLen;
  if (threadIdx.x == 0) {
    tadLen = shape::length(xShapeInfo);
    shmemMax = -DataTypeUtils::max<T>();
    shmemSum = 0.f;
  }
  __syncthreads();

  T max = -DataTypeUtils::max<T>();
  T sum = 0.f;

  // Calculate max
  for (LongType j = 0; j < tadLen; ++j) {
    LongType offset = shape::getIndexOffset(j, xShapeInfo);
    max = math::sd_max<T>(max, inBuff[offset]);
  }


  // Calculate exp(x - max) and sum
  for (LongType j = 0; j < tadLen; ++j) {
    LongType offset = shape::getIndexOffset(j, xShapeInfo);
    T temp = math::sd_exp<T, T>(inBuff[offset] - max);
    outBuff[offset] = temp;
    sum += temp;
  }

  // Final division step
  for (LongType j = 0; j < tadLen; ++j) {
    LongType offset = shape::getIndexOffset(j, zShapeInfo);
    outBuff[offset] /= sum;
  }
}

template <typename T>
void SD_KERNEL softMaxForVectorCudaGlobal(const void *vx, const LongType *xShapeInfo, void *vz,
                                          const LongType *zShapeInfo, LongType numOfSubArrs) {
  softMaxForVectorCuda<T>(vx, xShapeInfo, vz, zShapeInfo);
}

///////////////////////////////////////////////////////////////////
template <typename T>
void softMaxForVectorCudaLauncher(const hipStream_t *stream, const void *vx, const LongType *xShapeInfo, void *vz,
                                  const LongType *zShapeInfo, LongType numTads) {

  softMaxForVectorCudaGlobal<T><<<1, SD_CUDA_BLOCK_SIZE, 1024, *stream>>>(vx, xShapeInfo, vz, zShapeInfo, numTads);
  sd::DebugHelper::checkGlobalErrorCode("softmax  failed");

}

///////////////////////////////////////////////////////////////////

template <typename T>
SD_KERNEL void softmaxEws1Kernel(const T *input, const LongType *inputOffsets, T *output,
                                 const LongType *outputOffsets,
                                 LongType numOfSubArrs, LongType tadLen) {
  int i = blockIdx.x;  // Each block handles one TAD

  if (i >= numOfSubArrs) return;  // Out-of-bounds check for TADs

  auto inBuff = input + inputOffsets[i];
  auto outBuff = output + outputOffsets[i];

  __shared__ T shmemMax;
  __shared__ T shmemSum;

  if (threadIdx.x == 0) {
    shmemMax = -DataTypeUtils::max<T>();
    shmemSum = 0.f;
  }
  __syncthreads();


  // Calculate max
  for (LongType j = threadIdx.x; j < tadLen; j+= gridDim.x) {
    math::atomics::sd_atomicMax(&shmemMax, inBuff[j]);
  }
  __syncthreads();

  // Calculate exp(x - max) and sum
  for (LongType j = threadIdx.x; j < tadLen; j += gridDim.x) {
    T temp = math::sd_exp<T, T>(inBuff[j] - shmemMax);
    outBuff[j] = temp;
    math::atomics::sd_atomicAdd(&shmemSum, temp);
  }
  __syncthreads();

  // Final division step
  for (LongType j = threadIdx.x; j < tadLen; j += blockDim.x) {
    outBuff[j] /= shmemSum;
  }


}
template <typename T>
SD_KERNEL static void softMaxCuda(const void *vx, const LongType *xTadShapeInfo, const LongType *xOffsets,
                                  void *vz, const LongType *zTadShapeInfo, const LongType *zOffsets, LongType numTads) {
  int i = blockIdx.x;
  if(i >= numTads) return;

  const auto x = reinterpret_cast<const T *>(vx);
  auto z = reinterpret_cast<T *>(vz);

  const auto *xTad = x + xOffsets[blockIdx.x];
  auto *zTad = z + zOffsets[blockIdx.x];
  softMaxForVectorCuda<T>(xTad, xTadShapeInfo, zTad, zTadShapeInfo);
}

///////////////////////////////////////////////////////////////////

template <typename T>
static void softMaxEws1CudaLauncher(const int blocksPerGrid,
                                    const int threadsPerBlock,
                                    const int sharedMem,
                                    const hipStream_t *stream,
                                    const void *vx, const LongType *xOffsets, void *vz,
                                    const LongType *zOffsets, LongType numTads, LongType tadLength) {



  auto reCastInputs = reinterpret_cast<const T *>(vx);
  auto reCastOutputs = reinterpret_cast<T *>(vz);
  softmaxEws1Kernel<T>
  <<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(reCastInputs,
                                                           xOffsets,
                                                           reCastOutputs,
                                                           zOffsets,
                                                           numTads,
                                                           tadLength);
  sd::DebugHelper::checkGlobalErrorCode("softmaxews  failed");

}

template <typename T>
static void softMaxCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem,
                                const hipStream_t *stream, const void *vx, const LongType *xTadShapeInfo,
                                const LongType *xOffsets, void *vz, const LongType *zTadShapeInfo,
                                const LongType *zOffsets, LongType numTads) {


  softMaxCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xTadShapeInfo, xOffsets, vz, zTadShapeInfo,
                                                                         zOffsets ,numTads);
  sd::DebugHelper::checkGlobalErrorCode("softmax  failed");

}

//////////////////////////////////////////////////////////////////////////
void softmax(LaunchContext *context, const NDArray &input, NDArray &output, const int dimension) {
  const int rank = input.rankOf();

  PointersManager manager(context, "helpers::softmax");

  if (input.isVector()) {
    if (rank == 1 || input.sizeAt(dimension) != 1) {
      NDArray::prepareSpecialUse({&output}, {&input});
      BUILD_SINGLE_SELECTOR(input.dataType(), softMaxForVectorCudaLauncher,
                            (context->getCudaStream(), input.specialBuffer(), input.specialShapeInfo(),
                                output.specialBuffer(), output.specialShapeInfo(),1),
                            SD_FLOAT_TYPES);
      NDArray::registerSpecialUse({&output}, {&input});
    } else
      output = 1.;
  } else if(shape::ews(input.shapeInfo()) == 1) {
    auto packX = ConstantTadHelper::getInstance().tadForDimensions(input.shapeInfo(), {dimension});
    auto packZ = ConstantTadHelper::getInstance().tadForDimensions(output.shapeInfo(), {dimension});
    dim3 softmaxDims = getSoftmaxDims(packZ->numberOfTads());
    manager.synchronize();
    NDArray::prepareSpecialUse({&output}, {&input});
    //TODO: look in to why TAD shape info for cuda is 100 but it's 10 on cpu
    auto tadLength = shape::length(packX->primaryShapeInfo());
    BUILD_SINGLE_SELECTOR(input.dataType(), softMaxEws1CudaLauncher,
                          (softmaxDims.x, softmaxDims.y,
                              softmaxDims.z,
                              context->getCudaStream(),
                              input.specialBuffer(),
                              packX->specialOffsets(),
                              output.specialBuffer(),
                              packZ->specialOffsets(),
                              packX->numberOfTads(),
                              tadLength),
                          SD_FLOAT_TYPES);
    NDArray::registerSpecialUse({&output}, {&input});
  }

  else {
    auto packX = ConstantTadHelper::getInstance().tadForDimensions(input.shapeInfo(), {dimension});
    auto packZ = ConstantTadHelper::getInstance().tadForDimensions(output.shapeInfo(), {dimension});

    dim3 softmaxDims = getSoftmaxDims(packZ->numberOfTads());


    NDArray::prepareSpecialUse({&output}, {&input});
    BUILD_SINGLE_SELECTOR(input.dataType(), softMaxCudaLauncher,
                          (softmaxDims.x, softmaxDims.y,
                              softmaxDims.z,
                              context->getCudaStream(),
                              input.specialBuffer(),
                              packX->specialShapeInfo(),
                              packX->specialOffsets(), output.specialBuffer(),
                              packZ->specialShapeInfo(),
                              packZ->specialOffsets(),packX->numberOfTads()),
                          SD_FLOAT_TYPES);
    NDArray::registerSpecialUse({&output}, {&input});

  }

  manager.synchronize();

  output.tickWriteDevice();
}

///////////////////////////////////////////////////////////////////
template <typename T>
void SD_KERNEL logSoftMaxForVectorCuda(const void *vx, const LongType *xzShapeInfo, void *vz) {
  // logic of this kernel is based on assumption gridDim = 1

  const auto x = reinterpret_cast<const T *>(vx);
  auto z = reinterpret_cast<T *>(vz);

  __shared__ LongType len;
  __shared__ int numOfIters;
  __shared__ T shmem[SD_CUDA_BLOCK_SIZE];

  if (threadIdx.x == 0) {
    len = shape::length(xzShapeInfo);
    numOfIters = (len + blockDim.x - 1) / blockDim.x;  // ceil (len / blockDim.x)
  }
  __syncthreads();

  T temp =
      -DataTypeUtils::max<T>();  // set start value to compare with at first iteration, FIXME: what if T is unsigned ??

  // ************ evaluate max element in input array x ************ //
  for (int i = 0; i < numOfIters; ++i) {
    const LongType elemIdx = i * blockDim.x + threadIdx.x;
    if (elemIdx < len) {
      const LongType offset = shape::getIndexOffset(elemIdx, xzShapeInfo);
      shmem[threadIdx.x] =
          (threadIdx.x != 0)
          ? x[offset]
          : math::sd_max<T>(
              x[offset],
              temp);  // take into account max element evaluated on previous iteration and stored in temp
    } else
      shmem[threadIdx.x] = -DataTypeUtils::max<T>();  // FIXME: what if T is unsigned ??

    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s /= 2) {
      if (threadIdx.x < s) shmem[threadIdx.x] = math::sd_max<T>(shmem[threadIdx.x], shmem[threadIdx.x + s]);
      __syncthreads();
    }

    temp = shmem[0];  // save max value calculated at current iteration
  }

  const T max = temp;
  temp = 0;

  // ************ evaluate value of exp(x[offset] - max) per each element, store it to shared memory shmem ************
  // // at the same time evaluate sum of exponents, sum will be stored in shmem[0]
  for (int i = 0; i < numOfIters; ++i) {
    const LongType elemIdx = i * blockDim.x + threadIdx.x;
    if (elemIdx < len) {
      const LongType offset = shape::getIndexOffset(elemIdx, xzShapeInfo);
      z[offset] = math::sd_exp<T, T>(x[offset] - max);
      shmem[threadIdx.x] =
          (threadIdx.x != 0)
          ? z[offset]
          : (z[offset] + temp);  // take into account sum element evaluated on previous iteration and stored in temp
    } else
      shmem[threadIdx.x] = 0;

    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s /= 2) {
      if (threadIdx.x < s) shmem[threadIdx.x] += shmem[threadIdx.x + s];
      __syncthreads();
    }

    temp = shmem[0];  // save sum calculated at current iteration
  }

  // ************ evaluate log(z[offset] / sum)  ************ //
  for (int i = 0; i < numOfIters; ++i) {
    const LongType elemIdx = i * blockDim.x + threadIdx.x;
    const LongType offset = shape::getIndexOffset(elemIdx, xzShapeInfo);
    z[offset] = math::sd_log<T, T>(z[offset] / shmem[0]);
  }
}

///////////////////////////////////////////////////////////////////
template <typename T>
void logSoftMaxForVectorCudaLauncher(const hipStream_t *stream, const void *vx, const LongType *xzShapeInfo,
                                     void *vz) {
  dim3 launchDims = getLaunchDims("softmax");
  logSoftMaxForVectorCuda<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(vx, xzShapeInfo, vz);
  sd::DebugHelper::checkGlobalErrorCode("logsoftmax  failed");

}

//////////////////////////////////////////////////////////////////////////
void logSoftmax(LaunchContext *context, const NDArray &input, NDArray &output, const int dimension) {
  if (!input.isActualOnDeviceSide()) input.syncToDevice();
  const int rank = input.rankOf();

  if (input.isVector()) {
    if (rank == 1 || input.sizeAt(dimension) != 1) {
      BUILD_SINGLE_SELECTOR(
          input.dataType(), logSoftMaxForVectorCudaLauncher,
          (context->getCudaStream(), input.specialBuffer(), input.specialShapeInfo(), output.specialBuffer()),
          SD_FLOAT_TYPES);
      input.tickReadDevice();
    } else
      output = 0.;
  } else {
    std::vector<LongType> dim = {static_cast<LongType>(dimension)};
    auto maxAlongDim = const_cast<NDArray &>(input).reduceAlongDimension(reduce::Max, &dim, true);
    (input - maxAlongDim).applyTransform(transform::Exp, output);  // output contains exponents temporarily
    auto sumAlongDim = output.reduceAlongDimension(reduce::Sum, &dim, true);
    output /= sumAlongDim;
    output.applyTransform(transform::Log, output);
    input.tickReadDevice();
  }

  PointersManager manager(context, "helpers::logSoftmax");
  manager.synchronize();

  output.tickWriteDevice();
}

///////////////////////////////////////////////////////////////////
template <typename T>
void SD_KERNEL softMaxDerivForVectorCuda(const void *vx, const LongType *xzShapeInfo, void *vz) {
  // logic of this kernel is based on assumption gridDim = 1

  const auto x = reinterpret_cast<const T *>(vx);
  auto z = reinterpret_cast<T *>(vz);

  __shared__ LongType len;
  __shared__ int numOfIters;
  __shared__ T shmem[SD_CUDA_BLOCK_SIZE];

  if (threadIdx.x == 0) {
    len = shape::length(xzShapeInfo);
    numOfIters = (len + blockDim.x - 1) / blockDim.x;  // ceil (len / blockDim.x)
  }
  __syncthreads();

  T temp =
      -DataTypeUtils::max<T>();  // set start value to compare with at first iteration, FIXME: what if T is unsigned ??

  // ************ evaluate max element in input array x ************ //
  for (int i = 0; i < numOfIters; ++i) {
    const LongType elemIdx = i * blockDim.x + threadIdx.x;
    if (elemIdx < len) {
      const LongType offset = shape::getIndexOffset(elemIdx, xzShapeInfo);
      shmem[threadIdx.x] =
          (threadIdx.x != 0)
          ? x[offset]
          : math::sd_max<T>(
              x[offset],
              temp);  // take into account max element evaluated on previous iteration and stored in temp
    } else
      shmem[threadIdx.x] = -DataTypeUtils::max<T>();  // FIXME: what if T is unsigned ??

    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s /= 2) {
      if (threadIdx.x < s) shmem[threadIdx.x] = math::sd_max<T>(shmem[threadIdx.x], shmem[threadIdx.x + s]);
      __syncthreads();
    }

    temp = shmem[0];  // save max value calculated at current iteration
  }

  const T max = temp;
  temp = 0;

  // ************ evaluate value of exp(x[offset] - max) per each element, store it to shared memory shmem ************
  // // at the same evaluate sum of exponents, sum will be stored in shmem[0]
  for (int i = 0; i < numOfIters; ++i) {
    const LongType elemIdx = i * blockDim.x + threadIdx.x;
    if (elemIdx < len) {
      const LongType offset = shape::getIndexOffset(elemIdx, xzShapeInfo);
      z[offset] = math::sd_exp<T, T>(x[offset] - max);
      shmem[threadIdx.x] =
          (threadIdx.x != 0)
          ? z[offset]
          : (z[offset] + temp);  // take into account sum element evaluated on previous iteration and stored in temp
    } else
      shmem[threadIdx.x] = 0;

    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s /= 2) {
      if (threadIdx.x < s) shmem[threadIdx.x] += shmem[threadIdx.x + s];
      __syncthreads();
    }

    temp = shmem[0];  // save sum calculated at current iteration
  }

  // ************ evaluate (z[offset] / sum) and derivative z[offset] = z[offset] * (1 - z[offset]) ************ //
  for (int i = 0; i < numOfIters; ++i) {
    const LongType elemIdx = i * blockDim.x + threadIdx.x;
    if (elemIdx >= len) continue;
    const LongType offset = shape::getIndexOffset(elemIdx, xzShapeInfo);
    z[offset] /= shmem[0];
    z[offset] *= (1.f - z[offset]);  // derivative
  }
}

///////////////////////////////////////////////////////////////////
template <typename T>
void softMaxDerivForVectorCudaLauncher(const hipStream_t *stream, const void *vx, const LongType *xzShapeInfo,
                                       void *vz) {
  dim3 launchDims = getLaunchDims("softmax");

  softMaxDerivForVectorCuda<T><<<launchDims.x,launchDims.y, launchDims.z, *stream>>>(vx, xzShapeInfo, vz);
  sd::DebugHelper::checkGlobalErrorCode("softmax derivative  failed");

}

///////////////////////////////////////////////////////////////////
void softmaxDerivative(LaunchContext *context, const NDArray &input, NDArray &output, const int dimension) {
  if (!input.isActualOnDeviceSide()) input.syncToDevice();
  const int rank = input.rankOf();
  LongType temp;

  if (shape::isCommonVector(input.shapeInfo(), temp)) {
    BUILD_SINGLE_SELECTOR(
        input.dataType(), softMaxDerivForVectorCudaLauncher,
        (context->getCudaStream(), input.specialBuffer(), input.specialShapeInfo(), output.specialBuffer()),
        SD_FLOAT_TYPES);
    input.tickReadDevice();
  } else {
    std::vector<LongType> dim = {static_cast<LongType>(dimension)};
    auto maxAlongDim = const_cast<NDArray &>(input).reduceAlongDimension(reduce::Max, &dim, true);
    (input - maxAlongDim).applyTransform(transform::Exp, output);  // output contains exponents temporarily
    auto sumAlongDim = output.reduceAlongDimension(reduce::Sum, &dim, true);
    output /= sumAlongDim;
    output *= (1.f - output);  // derivative
    input.tickReadDevice();
  }

  PointersManager manager(context, "helpers::softmaxDerivative");
  manager.synchronize();

  output.tickWriteDevice();
}

template <typename T>
void thresholdRelu_(NDArray const &input, double threshold, NDArray &output) {
  auto routine = LAMBDA_T(_x, threshold) { return _x > (T)threshold ? _x : (T)0.f; };
  const_cast<NDArray &>(input).applyLambda(routine, output);
}

void thresholdRelu(LaunchContext *context, NDArray const &input, double threshold, NDArray &output) {
  BUILD_SINGLE_SELECTOR(input.dataType(), thresholdRelu_, (input, threshold, output), SD_FLOAT_TYPES);
}

template <typename T>
void thresholdReluDerivative_(NDArray *input, double theta, NDArray *dLdO, NDArray *output) {
  auto derivative = LAMBDA_TT(_x, grO, theta) {
    if (_x > theta)
      return grO;
    else
      return static_cast<T>(0);
  };

  input->applyPairwiseLambda(*dLdO, derivative, *output);
}

void thresholdReluDerivative(LaunchContext *context, NDArray *input, double threshold, NDArray *dLdO,
                             NDArray *output) {
  BUILD_SINGLE_SELECTOR(input->dataType(), thresholdReluDerivative_, (input, threshold, dLdO, output), SD_FLOAT_TYPES);
}

}  // namespace helpers
}  // namespace ops
}  // namespace sd
