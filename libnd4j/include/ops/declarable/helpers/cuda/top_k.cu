#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com)
//
#include <helpers/ConstantTadHelper.h>
#include <helpers/PointersManager.h>
#include <ops/declarable/helpers/top_k.h>

#include "execution/cuda/LaunchDims.h"
#include "helpers/DebugHelper.h"


namespace sd {
namespace ops {
namespace helpers {

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Y>
SD_KERNEL static void inTopKCuda(const void* vx, const LongType* xShapeInfo, const void* vy,
                                 const LongType* yShapeInfo, void* vz, const LongType* zShapeInfo,
                                 const LongType* xTadShapeInfo, const LongType* xTadOffsets,
                                 const LongType k) {
  const auto y = reinterpret_cast<const Y*>(vy);
  auto z = reinterpret_cast<bool*>(vz);

  __shared__ LongType sharedMem[SD_CUDA_BLOCK_SIZE];
  __shared__ X elemToCompare;
  __shared__ const X* xTad;
  __shared__ LongType idx, xTadLen;

  if (threadIdx.x == 0) {
    xTadLen = shape::length(xTadShapeInfo);

    xTad = reinterpret_cast<const X*>(vx) + xTadOffsets[blockIdx.x];
    LongType yCoords[SD_MAX_RANK];
    LongType yOffset;
    INDEX2COORDS(blockIdx.x, shape::rank(yShapeInfo), yShapeInfo, yCoords);
    COORDS2INDEX(shape::rank(yShapeInfo), shape::shapeOf(yShapeInfo), yCoords, yOffset);
    idx = y[yOffset];
    LongType xCoords[SD_MAX_RANK];
    LongType xOffset;
    INDEX2COORDS(idx, shape::rank(xTadShapeInfo), xTadShapeInfo, xCoords);
    COORDS2INDEX(shape::rank(xTadShapeInfo), shape::shapeOf(xTadShapeInfo), xCoords, xOffset);
    elemToCompare = xTad[xOffset];
  }

  __syncthreads();

  sharedMem[threadIdx.x] = 0;
  for (LongType i = threadIdx.x; i < xTadLen; i += blockDim.x) {
    LongType xCoords[SD_MAX_RANK];
    LongType xOffset;
    INDEX2COORDS(i, shape::rank(xTadShapeInfo), xTadShapeInfo, xCoords);
    COORDS2INDEX(shape::rank(xTadShapeInfo), shape::shapeOf(xTadShapeInfo), xCoords, xOffset);
    if (elemToCompare < xTad[xOffset]) ++sharedMem[threadIdx.x];
  }

  __syncthreads();

  // aggregate sum
  for (LongType activeThreads = blockDim.x / 2; activeThreads > 0; activeThreads /= 2) {
    if (threadIdx.x < activeThreads) sharedMem[threadIdx.x] += sharedMem[threadIdx.x + activeThreads];
    __syncthreads();
  }

  if (threadIdx.x == 0) {
    LongType zCoords[SD_MAX_RANK];
    LongType zOffset;
    INDEX2COORDS(blockIdx.x, shape::rank(zShapeInfo), zShapeInfo, zCoords);
    COORDS2INDEX(shape::rank(zShapeInfo), shape::shapeOf(zShapeInfo), zCoords, zOffset);
    z[zOffset] = *sharedMem < k;
  }
}
///////////////////////////////////////////////////////////////////
template <typename X, typename Y>
static void inTopKCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem,
                               const hipStream_t* stream, const void* vx, const LongType* xShapeInfo,
                               const void* vy, const LongType* yShapeInfo, void* vz, const LongType* zShapeInfo,
                               const LongType* xTadShapeInfo, const LongType* xTadOffsets,
                               const LongType k) {
  inTopKCuda<X, Y><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vy, yShapeInfo, vz,
                                                                           zShapeInfo, xTadShapeInfo, xTadOffsets, k);
  sd::DebugHelper::checkErrorCode(const_cast<hipStream_t *>(stream), "inTopKCudaLauncher failed");

}

///////////////////////////////////////////////////////////////////
Status inTopKFunctor(LaunchContext* context, NDArray* predictions, NDArray* targets,
                         NDArray* output, const LongType k) {
  PointersManager manager(context, "in_top_k");

  const auto packX = ConstantTadHelper::getInstance().tadForDimensions(predictions->shapeInfo(), {1});

  dim3 topkDims2 = topkDims(packX->numberOfTads());
  const auto xType = predictions->dataType();
  const auto yType = targets->dataType();

  NDArray::prepareSpecialUse({output}, {predictions, targets});
  BUILD_DOUBLE_SELECTOR(
      xType, yType, inTopKCudaLauncher,
      (topkDims2.y,topkDims2.x, topkDims2.z, context->getCudaStream(), predictions->specialBuffer(),
          predictions->specialShapeInfo(), targets->specialBuffer(), targets->specialShapeInfo(), output->specialBuffer(),
          output->specialShapeInfo(), packX->specialShapeInfo(), packX->specialOffsets(), k),
      SD_FLOAT_TYPES, SD_INDEXING_TYPES);
  NDArray::registerSpecialUse({output}, {predictions, targets});

  manager.synchronize();

  return Status::OK;
}

template <typename X, typename Y>
static SD_KERNEL void topValuesMover(void const* vx, LongType const* xTadShapeInfo, LongType const* xTadOffsets,
                                     void const* vi, LongType const* iTadShapeInfo, LongType const* iTadOffsets,
                                     void* vz, LongType const* zTadShapeInfo, LongType const* zTadOffsets,
                                     LongType tadLength, int numTads, int k) {
  for (int t = blockIdx.x; t < numTads; t += gridDim.x) {
    auto x = reinterpret_cast<X const*>(vx) + xTadOffsets[t];
    auto i = reinterpret_cast<Y const*>(vi) + iTadOffsets[t];
    auto z = reinterpret_cast<X*>(vz) + zTadOffsets[t];

    LongType iCoords[SD_MAX_RANK];
    LongType zCoords[SD_MAX_RANK];
    LongType xCoords[SD_MAX_RANK];
    LongType iOffset;
    LongType zOffset;
    LongType xOffset;

    for (int e = threadIdx.x; e < k; e += blockDim.x) {
      INDEX2COORDS(e, shape::rank(iTadShapeInfo), iTadShapeInfo, iCoords);
      COORDS2INDEX(shape::rank(iTadShapeInfo), shape::shapeOf(iTadShapeInfo), iCoords, iOffset);
      auto idx = i[iOffset];

      INDEX2COORDS(e, shape::rank(zTadShapeInfo), zTadShapeInfo, zCoords);
      COORDS2INDEX(shape::rank(zTadShapeInfo), shape::shapeOf(zTadShapeInfo), zCoords, zOffset);

      INDEX2COORDS(idx, shape::rank(xTadShapeInfo), xTadShapeInfo, xCoords);
      COORDS2INDEX(shape::rank(xTadShapeInfo), shape::shapeOf(xTadShapeInfo), xCoords, xOffset);

      z[zOffset] = x[xOffset];
    }
  }
}

template <typename X, typename Y>
static SD_KERNEL void indicesAlongDimension(void const* vx, LongType const* xTadShapeInfo, LongType const* xTadOffsets, void* vi, LongType const* iTadShapeInfo, LongType const* iTadOffsets,
                                            void* vz, LongType const* zTadShapeInfo, LongType const* zTadOffsets,
                                            LongType tadLength, int numTads, int k,
                                            int scanWidth, bool needSort) {
  extern __shared__ char _shmem[];

  X* tempValues = reinterpret_cast<X*>(_shmem) + threadIdx.x * scanWidth;
  Y* tempIndices =
      reinterpret_cast<Y*>(reinterpret_cast<X*>(_shmem) + blockDim.x * scanWidth) + threadIdx.x * scanWidth;

  __shared__ X localMaximum;
  if (threadIdx.x == 0) localMaximum = -DataTypeUtils::max<X>();
  __syncthreads();

  for (int t = blockIdx.x; t < numTads; t += gridDim.x) {
    auto x = reinterpret_cast<X const*>(vx) + xTadOffsets[t];
    auto i = reinterpret_cast<Y*>(vi) + iTadOffsets[t];
    auto z = reinterpret_cast<X*>(vz) + zTadOffsets[t];

    // we'll do multiple reads here
    for (int p = 0; p < k; p += scanWidth) {
      // resetting temporary storage
      for (int p = 0; p < scanWidth; p++) {
        tempValues[p] = -DataTypeUtils::max<X>();
        tempIndices[p] = DataTypeUtils::max<Y>();
      }

      // local max values/indices
      for (int e = threadIdx.x; e < tadLength; e++) {
        LongType xCoords[SD_MAX_RANK];
        LongType xOffset;
        INDEX2COORDS(e, shape::rank(xTadShapeInfo), xTadShapeInfo, xCoords);
        COORDS2INDEX(shape::rank(xTadShapeInfo), shape::shapeOf(xTadShapeInfo), xCoords, xOffset);
        auto value = x[xOffset];

        // we'll compare this value to current stored ones
        for (int f = 0; f < scanWidth; f++) {
          if (value > tempValues[f] && (p == 0 || value < localMaximum)) {
            tempValues[f] = value;
            tempIndices[f] = e;
          }
        }
      }
      __syncthreads();

      // at this point we have local part ready for merge and define global maximum for this iteration, and local
      // maximum for next iteration
      for (LongType activeThreads = blockDim.x / 2; activeThreads > 0; activeThreads /= 2) {
        if (threadIdx.x < activeThreads) {
          if (tempValues[0] < tempValues[0 + activeThreads * scanWidth]) {
            tempValues[0] = tempValues[0 + activeThreads * scanWidth];
            tempIndices[0] = tempIndices[0 + activeThreads * scanWidth];
          }
        }
        __syncthreads();
      }
      __syncthreads();

      // at this point we know local minimum for next iteration
      if (threadIdx.x == 0) {
        localMaximum = tempValues[scanWidth - 1];
        LongType zCoords[SD_MAX_RANK];
        LongType zOffset;
        INDEX2COORDS(p, shape::rank(zTadShapeInfo), zTadShapeInfo, zCoords);
        COORDS2INDEX(shape::rank(zTadShapeInfo), shape::shapeOf(zTadShapeInfo), zCoords, zOffset);
        z[zOffset] = tempValues[scanWidth - 1];
        LongType iCoords[SD_MAX_RANK];
        LongType iOffset;
        INDEX2COORDS(p, shape::rank(iTadShapeInfo), iTadShapeInfo, iCoords);
        COORDS2INDEX(shape::rank(iTadShapeInfo), shape::shapeOf(iTadShapeInfo), iCoords, iOffset);
        i[iOffset] = tempIndices[scanWidth - 1];
      }
      __syncthreads();
    }

    __syncthreads();
    if (!needSort) {
      // if we don't need sort, we need to return values based on their indices (ascending)
      for (int m = 0; m < k; m++) {
        if (m % 2 == 0) {
          for (int tid = threadIdx.x; tid < k; tid += blockDim.x) {
            auto top = 2 * tid + 1;
            if (top < k) {
              LongType t0Coords[SD_MAX_RANK];
              LongType t0Offset;
              INDEX2COORDS(top - 1, shape::rank(iTadShapeInfo), iTadShapeInfo, t0Coords);
              COORDS2INDEX(shape::rank(iTadShapeInfo), shape::shapeOf(iTadShapeInfo), t0Coords, t0Offset);
              LongType t1Coords[SD_MAX_RANK];
              LongType t1Offset;
              INDEX2COORDS(top, shape::rank(iTadShapeInfo), iTadShapeInfo, t1Coords);
              COORDS2INDEX(shape::rank(iTadShapeInfo), shape::shapeOf(iTadShapeInfo), t1Coords, t1Offset);

              if (i[t0Offset] > i[t1Offset]) {
                // swap indices first
                Y di0 = i[t0Offset];
                i[t0Offset] = i[t1Offset];
                i[t1Offset] = di0;

                // swap values next
                LongType zT0Coords[SD_MAX_RANK];
                LongType zT0Offset;
                INDEX2COORDS(top - 1, shape::rank(zTadShapeInfo), zTadShapeInfo, zT0Coords);
                COORDS2INDEX(shape::rank(zTadShapeInfo), shape::shapeOf(zTadShapeInfo), zT0Coords, zT0Offset);
                LongType zT1Coords[SD_MAX_RANK];
                LongType zT1Offset;
                INDEX2COORDS(top, shape::rank(zTadShapeInfo), zTadShapeInfo, zT1Coords);
                COORDS2INDEX(shape::rank(zTadShapeInfo), shape::shapeOf(zTadShapeInfo), zT1Coords, zT1Offset);

                X dz0 = z[zT0Offset];
                z[zT0Offset] = z[zT1Offset];
                z[zT1Offset] = dz0;
              }
            }
          }
        } else {
          for (int tid = threadIdx.x; tid < k; tid += blockDim.x) {
            auto top = 2 * tid + 2;
            if (top < k) {
              LongType t0Coords[SD_MAX_RANK];
              LongType t0Offset;
              INDEX2COORDS(top - 1, shape::rank(iTadShapeInfo), iTadShapeInfo, t0Coords);
              COORDS2INDEX(shape::rank(iTadShapeInfo), shape::shapeOf(iTadShapeInfo), t0Coords, t0Offset);
              LongType t1Coords[SD_MAX_RANK];
              LongType t1Offset;
              INDEX2COORDS(top, shape::rank(iTadShapeInfo), iTadShapeInfo, t1Coords);
              COORDS2INDEX(shape::rank(iTadShapeInfo), shape::shapeOf(iTadShapeInfo), t1Coords, t1Offset);

              if (i[t0Offset] > i[t1Offset]) {
                // swap indices first
                Y di0 = i[t0Offset];
                i[t0Offset] = i[t1Offset];
                i[t1Offset] = di0;

                // swap values next
                LongType zT0Coords[SD_MAX_RANK];
                LongType zT0Offset;
                INDEX2COORDS(top - 1, shape::rank(zTadShapeInfo), zTadShapeInfo, zT0Coords);
                COORDS2INDEX(shape::rank(zTadShapeInfo), shape::shapeOf(zTadShapeInfo), zT0Coords, zT0Offset);
                LongType zT1Coords[SD_MAX_RANK];
                LongType zT1Offset;
                INDEX2COORDS(top, shape::rank(zTadShapeInfo), zTadShapeInfo, zT1Coords);
                COORDS2INDEX(shape::rank(zTadShapeInfo), shape::shapeOf(zTadShapeInfo), zT1Coords, zT1Offset);

                X dz0 = z[zT0Offset];
                z[zT0Offset] = z[zT1Offset];
                z[zT1Offset] = dz0;
              }
            }
          }
        }
        __syncthreads();
      }
    }
  }
}

template <typename X, typename Y>
static Status topKFunctor_(LaunchContext* context, NDArray* input, NDArray* values, NDArray* indices,
                           const LongType k, bool needSort) {
  auto packX = ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), {input->rankOf() - 1});
  auto packI = ConstantTadHelper::getInstance().tadForDimensions(indices->shapeInfo(), {input->rankOf() - 1});
  auto packZ = ConstantTadHelper::getInstance().tadForDimensions(values->shapeInfo(), {input->rankOf() - 1});

  auto tadLength = shape::length(packX->primaryShapeInfo());

  // we get top K values first
  if (k == 1) {
    std::vector<LongType> dims = {input->rankOf() - 1};
    input->applyIndexReduce(indexreduce::IndexMax, *indices, &dims);

    dim3 launchDims = getLaunchDims("top_k_mover");
    // copy values on specified indices
    topValuesMover<X, Y><<<launchDims.y, launchDims.x, launchDims.z, *context->getCudaStream()>>>(
        input->specialBuffer(), packX->platformShapeInfo(), packX->platformOffsets(), indices->specialBuffer(),
        packI->platformShapeInfo(), packI->platformOffsets(), values->specialBuffer(), packZ->platformShapeInfo(),
        packZ->platformOffsets(), tadLength, packX->numberOfTads(), k);
    sd::DebugHelper::checkErrorCode(context->getCudaStream(), "topValuesMover failed");

  } else {
    int scanWidth = 1;
    dim3 topKIndices2 = topKIndices(scanWidth, sizeof(X), sizeof(Y));
    indicesAlongDimension<X, Y><<<topKIndices2.y, topKIndices2.x, topKIndices2.z, *context->getCudaStream()>>>(
        input->specialBuffer(), packX->platformShapeInfo(), packX->platformOffsets(), indices->specialBuffer(),
        packI->platformShapeInfo(), packI->platformOffsets(), values->specialBuffer(), packZ->platformShapeInfo(),
        packZ->platformOffsets(), tadLength, packX->numberOfTads(), k, scanWidth, needSort);
    sd::DebugHelper::checkErrorCode(context->getCudaStream(), "indicesAlongDimension failed");

  }

  return Status::OK;
}

Status topKFunctor(LaunchContext* context, NDArray* input, NDArray* values, NDArray* indices,
                       const LongType k, bool needSort) {
  input->syncToDevice();

  BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), topKFunctor_,
                        (context, input, values, indices, k, needSort), SD_COMMON_TYPES, SD_INDEXING_TYPES);

  values->tickWriteDevice();
  indices->tickWriteDevice();

  return Status::OK;
}

}  // namespace helpers
}  // namespace ops
}  // namespace sd
