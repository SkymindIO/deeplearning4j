/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 16.04.2018
//

// function nnCell implements an Elman RNN cell: output = activation(Wx*x + bx  +  Wh*ht  + bh)

#include<ops/declarable/helpers/rnn.h>
#include <helpers/BlasHelper.h>


namespace nd4j    {
namespace ops     {
namespace helpers {


    //////////////////////////////////////////////////////////////////////////
    static FORCEINLINE NDArray activation(const NDArray& arr) {
        return (const_cast<NDArray&>(arr)).transform(transform::Tanh);
    }


    //////////////////////////////////////////////////////////////////////////
    void rnnCell(const NDArray* xt, const NDArray* Wx, const NDArray* Wh, const NDArray* b, const NDArray* ht_1, NDArray* ht) {

    }


    //////////////////////////////////////////////////////////////////////////
    void rnnTimeLoop(const NDArray* x, const NDArray* Wx, const NDArray* Wh, const NDArray* b, const NDArray* h0, const NDArray* maxTimeStep, NDArray* h, NDArray* hFinal) {

    }

}
}
}

