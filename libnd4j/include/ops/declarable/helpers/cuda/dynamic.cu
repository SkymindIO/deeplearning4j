/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// Created by george on 05.04.18.
//
#include <ops/declarable/helpers/dynamic.h>

namespace nd4j {
    namespace ops {
        namespace helpers {

            template <typename T>
            static void _dynamicPartitionFunctor(NDArray const* input, NDArray const* indices, std::vector<NDArray*>& outputList) {

            }

            template <typename T>
            static int _dynamicStitchFunctor(std::vector<NDArray*> const& inputs, std::vector<NDArray*> const& indices, NDArray* output){
                return Status::OK();
            }

            template <typename T>
            static void _dynamicPartitionFunctorBP(NDArray const* input, NDArray const* indices, std::vector<NDArray*> const& inputGradientList, std::vector<NDArray*>& outputList) {

            }

            void dynamicPartitionFunctor(NDArray const* input, NDArray const* indices, std::vector<NDArray*>& outputList) {
                auto xType = input->dataType();

                BUILD_SINGLE_SELECTOR(xType, _dynamicPartitionFunctor, (input, indices, outputList), LIBND4J_TYPES);
            }

            template <typename T>
            static int _dynamicStitchFunctorBP(std::vector<NDArray*> const& inputs, std::vector<NDArray*> const& indices, NDArray const* gradInput, std::vector<NDArray*>& outputList){
                throw std::runtime_error("Not umplemented yet");
            }

            int dynamicStitchFunctor(std::vector<NDArray*> const& inputs, std::vector<NDArray*> const& indices, NDArray* output){
                auto xType = inputs.at(0)->dataType();

                BUILD_SINGLE_SELECTOR(xType, return _dynamicStitchFunctor, (inputs, indices, output), LIBND4J_TYPES);
            }

            int dynamicStitchFunctorBP(std::vector<NDArray*> const& inputs, std::vector<NDArray*> const& indices, NDArray const* gradInput, std::vector<NDArray*>& outputList) {
                auto xType = inputs.at(0)->dataType();

                BUILD_SINGLE_SELECTOR(xType, return _dynamicStitchFunctorBP, (inputs, indices, gradInput, outputList), LIBND4J_TYPES);
            }

            void dynamicPartitionFunctorBP(NDArray const* input, NDArray const* indices, std::vector<NDArray*> const& inputGradientList, std::vector<NDArray*>& outputList) {
                auto xType = input->dataType();

                BUILD_SINGLE_SELECTOR(xType, _dynamicPartitionFunctorBP, (input, indices, inputGradientList, outputList), LIBND4J_TYPES);
            }

            BUILD_SINGLE_TEMPLATE(template void _dynamicPartitionFunctorBP, (NDArray const* input, NDArray const* indices, std::vector<NDArray*> const& inputGradientList, std::vector<NDArray*>& outputList);, LIBND4J_TYPES);
            BUILD_SINGLE_TEMPLATE(template int _dynamicStitchFunctorBP, (std::vector<NDArray*> const& inputs, std::vector<NDArray*> const& indices, NDArray const* gradInput, std::vector<NDArray*>& outputList);, LIBND4J_TYPES);

            BUILD_SINGLE_TEMPLATE(template void _dynamicPartitionFunctor, (NDArray const* input, NDArray const* indices, std::vector<NDArray*>& outputList);, LIBND4J_TYPES);
            BUILD_SINGLE_TEMPLATE(template int _dynamicStitchFunctor, (std::vector<NDArray*> const& inputs, std::vector<NDArray*> const& indices, NDArray* output);, LIBND4J_TYPES);


        }
    }
}

