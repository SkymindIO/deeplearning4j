#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com, created on 30.11.17.
// @author Yurii Shyrma (iuriish@yahoo.com)
//
#include <helpers/PointersManager.h>
#include <ops/declarable/helpers/col2im.h>

namespace sd {
namespace ops {
namespace helpers {

//////////////////////////////////////////////////////////////////////////
// columns [bS, iC, kH, kW, oH, oW] to be de-convoluted to image [bS, iC, iH, iW]
template <typename T>
static SD_KERNEL void col2imCuda(const void* columns, const sd::LongType* colShapeInfo, void* image,
                                 const sd::LongType* imShapeInfo, const LongType sH, const LongType sW, const LongType pH,
                                 const LongType pW, const LongType dH, const LongType dW) {
  const T* col = reinterpret_cast<const T*>(columns);
  T* im = reinterpret_cast<T*>(image);

  __shared__ sd::LongType kH, kW, oH, oW, *sharedMem;
  __shared__ sd::LongType imLen;

  if (threadIdx.x == 0) {
    extern __shared__ unsigned char shmem[];
    sharedMem = reinterpret_cast<sd::LongType*>(shmem);

    kH = dH * (colShapeInfo[3] - 1) + 1;
    kW = dW * (colShapeInfo[4] - 1) + 1;

    oH = colShapeInfo[5];
    oW = colShapeInfo[6];

    imLen = shape::length(imShapeInfo);
  }
  __syncthreads();

  auto coords = sharedMem + threadIdx.x * 6;

  const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

  for (sd::LongType i = tid; i < imLen; i += gridDim.x * blockDim.x) {
    shape::index2coords(i, imShapeInfo, coords);

    const auto imOffset = shape::getOffset(imShapeInfo, coords);

    const auto bSiCoffset = coords[0] * colShapeInfo[7] + coords[1] * colShapeInfo[8];

    const sd::LongType imH = coords[2] + pH;
    const sd::LongType imW = coords[3] + pW;

    const sd::LongType colHstart = (imH < kH) ? 0 : (imH - kH) / sH + 1;
    const sd::LongType colWstart = (imW < kW) ? 0 : (imW - kW) / sW + 1;

    const sd::LongType colHend = sd::math::sd_min<sd::LongType>(imH / sH + 1, oH);
    const sd::LongType colWend = sd::math::sd_min<sd::LongType>(imW / sW + 1, oW);

    T val = 0;

    for (coords[4] = colHstart; coords[4] < colHend; ++coords[4]) {
      coords[2] = imH - coords[4] * sH;
      if (coords[2] % dH != 0) continue;

      for (coords[5] = colWstart; coords[5] < colWend; ++coords[5]) {
        coords[3] = imW - coords[5] * sW;
        if (coords[3] % dW != 0) continue;

        val += col[bSiCoffset + (coords[2] / dH) * colShapeInfo[9] + (coords[3] / dW) * colShapeInfo[10] +
                   coords[4] * colShapeInfo[11] + coords[5] * colShapeInfo[12]];
      }
    }
    im[imOffset] = val;
  }
}

////////////////////////////////////////////////////////////////////////
// columns [bS, iC, kH, kW, oH, oW] to be de-convoluted to image [bS, iC, iH, iW]
template <typename T>
SD_KERNEL static void col2imCuda2(const void* columns, void* image, const sd::LongType* colShapeInfo,
                                  const sd::LongType* imShapeInfo, const int sH, const int sW, const int pH,
                                  const int pW, const int dH, const int dW) {
  const auto col = reinterpret_cast<const T*>(columns);
  auto im = reinterpret_cast<T*>(image);

  auto colShape = shape::shapeOf(const_cast<sd::LongType*>(colShapeInfo));
  auto colStride = shape::stride(const_cast<sd::LongType*>(colShapeInfo));

  int colStride0 = colStride[0];
  int colStride1 = colStride[1];
  int colStride2 = colStride[2];
  int colStride3 = colStride[3];
  int colStride4 = colStride[4];
  int colStride5 = colStride[5];

  int kH = colShape[2];
  int kW = colShape[3];

  auto imShape = shape::shapeOf(const_cast<sd::LongType*>(imShapeInfo));
  auto imOrder = shape::order(const_cast<sd::LongType*>(imShapeInfo));
  auto imStride = shape::stride(const_cast<sd::LongType*>(imShapeInfo));

  LongType bS = imShape[0];
  LongType iC = imShape[1];
  LongType iH = imShape[2];
  LongType iW = imShape[3];

  LongType oH = colShape[4];  //(iH + 2 * pH - kH) / sW + 1;
  LongType oW = colShape[5];  //(iW + 2 * pW - kW) / sH + 1;

  int n = bS * iC * iH * iW;

  // Effective kernel size, accounting for dilation
  LongType kHeff = kH + (kH - 1) * (dH - 1);
  LongType kWeff = kW + (kW - 1) * (dW - 1);

  for (int i = (blockDim.x * blockIdx.x) + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
    T val = 0;

    LongType w_im = i % iW + pW;
    LongType h_im = (i / iW) % iH + pH;
    LongType c_im = i / (iW * iH);
    LongType b = c_im / iC;
    LongType c = c_im % iC;

    // compute the start and end of the output
    // These are the indexes for dimensions ??? in the 6d col matrix
    LongType w_col_start = (w_im < kWeff) ? 0 : (w_im - kWeff) / sW + 1;
    LongType w_col_end = sd::math::sd_min<LongType>(w_im / sW + 1, oW);

    LongType h_col_start = (h_im < kHeff) ? 0 : (h_im - kHeff) / sH + 1;
    LongType h_col_end = sd::math::sd_min<LongType>(h_im / sH + 1, oH);

    // Iterate over col entries in the 6d array... these are added up
    for (int colH = h_col_start; colH < h_col_end; colH += 1) {
      for (int colW = w_col_start; colW < w_col_end; colW += 1) {
        LongType kRow = (h_im - colH * sH);
        LongType kCol = (w_im - colW * sW);

        if (kRow % dH == 0 && kCol % dW == 0) {
          kRow /= dH;
          kCol /= dW;

          int data_col_index = b * colStride0 + c * colStride1 + kRow * colStride2 + kCol * colStride3 +
                               colH * colStride4 + colW * colStride5;
          val += col[data_col_index];
        }
      }
    }

    LongType i_f = 0;
    LongType i_c = i;
    for (int dim = 3; dim >= 0; dim--) {
      i_f += (i_c % imShape[dim]) * imStride[dim];
      i_c = i_c / imShape[dim];
    }

    im[i_f] = val;
  }
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
static void col2imCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem,
                               const hipStream_t* stream, const void* columns, const sd::LongType* colShapeInfo,
                               void* image, const sd::LongType* imShapeInfo, const LongType sH, const LongType sW, const LongType pH,
                               const LongType pW, const LongType dH, const LongType dW) {
  // col2imCuda2<T><<<512, 512, 1024, *stream>>>(columns, image, colShapeInfo, imShapeInfo, sH, sW, pH, pW, dH, dW);
  col2imCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(columns, colShapeInfo, image, imShapeInfo, sH,
                                                                        sW, pH, pW, dH, dW);
}

//////////////////////////////////////////////////////////////////////////
void col2im(sd::LaunchContext& context, const NDArray& col, NDArray& im, const LongType sH, const LongType sW, const LongType pH,
            const LongType pW, const LongType iH, const LongType iW, const LongType dH, const LongType dW) {
  PointersManager manager(&context, "col2im");

  const int threadsPerBlock = SD_MAX_NUM_THREADS / 2;
  const int blocksPerGrid = (im.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
  const int sharedMem = col.rankOf() * sizeof(sd::LongType) * threadsPerBlock + 256;

  NDArray::prepareSpecialUse({&im}, {&col});
  BUILD_SINGLE_SELECTOR(im.dataType(), col2imCudaLauncher,
                        (blocksPerGrid, threadsPerBlock, sharedMem, context.getCudaStream(), col.specialBuffer(),
                         col.specialShapeInfo(), im.specialBuffer(), im.specialShapeInfo(), sH, sW, pH, pW, dH, dW),
                        SD_FLOAT_TYPES);
  NDArray::registerSpecialUse({&im}, {&col});

  manager.synchronize();
}

}  // namespace helpers
}  // namespace ops
}  // namespace sd
