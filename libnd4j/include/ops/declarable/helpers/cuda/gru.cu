#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 15.02.2018
//

// implementation of gated Recurrent Unit cell
// (cf. http://arxiv.org/abs/1406.1078).
// Kyunghyun Cho, Bart van Merrienboer, Caglar Gulcehre, Dzmitry Bahdanau, Fethi Bougares, Holger Schwenk, Yoshua Bengio
// "Learning Phrase Representations using RNN Encoder-Decoder for Statistical Machine Translation"


#include<ops/declarable/helpers/gru.h>
#include <ops/declarable/CustomOperations.h>
#include<ops/declarable/helpers/transforms.h>
#include <MmulHelper.h>

namespace nd4j 	  {
namespace ops 	  {
namespace helpers {


//////////////////////////////////////////////////////////////////////////
void gruCell(nd4j::LaunchContext * context, const NDArray* x, const NDArray* hLast, const NDArray* Wru, const NDArray* Wc,
             const NDArray* bru, const NDArray* bc,
             NDArray* r, NDArray* u, NDArray* c, NDArray* h) {

    //Inputs:
    // x        input [bS x inSize]
    // hLast    previous cell output [bS x numUnits],  that is at previous time step t-1
    // Wru      RU weights - [bS, 2*numUnits] - reset and update gates
    // Wc       C weights - [bS, numUnits] - cell gate
    // bru      r and u biases, [2*numUnits] - reset and update gates
    // bc       c biases, [numUnits] - cell gate

    //Outputs:
    // r        Reset gate output [bS, numUnits]
    // u        Update gate output [bS, numUnits]
    // c        Cell gate output [bS, numUnits]
    // h        current cell output [bS, numUnits]

    const int nIn = x->sizeAt(1);
    const int nU = hLast->sizeAt(1);                // number of units

    //Concat inputs: [x, yt-1]: concat([bs,nIn],[bs,nOut]) -> [bs, (nIn+nOut)]
    nd4j::ops::concat concatOp;
    std::vector<NDArray*> inputs;
    std::vector<double> targs;
    std::vector<Nd4jLong> iargs({1});   //Axis = 1
    std::vector<bool> bargs;
    inputs.emplace_back(const_cast<NDArray*>(x));
    inputs.emplace_back(const_cast<NDArray*>(hLast));

    auto result = concatOp.execute(inputs, targs, iargs, bargs);
    auto concatOut = result->at(0);

    //mmul/z for reset and update gates: (x * weight_ux + hLast * weight_xr + b_u)
    auto m = mmul(*concatOut, *Wru);    //mmul: [bs, (nIn+numUnits)]* [(inSize+numUnits), 2*numUnits] = [bs, 4*numUnits]
    m += (*bru);

    sigmoidInplace(m);  //sigmoid(rz) and sigmoid(uz)
    auto mr = m({0,0, 0, nU});
    auto mu = m({0,0, nU, 2*nU});

    r->assign(&mr);
    u->assign(&mu);

    //Concatenated inputs: [x, yt-1 .* r]
    auto yr = (*concatOut)({0,0, nIn, nIn+nU});
    yr *= (*r);

    //c = tanh(x * weight_cx + (hLast .* r) * weight_cr + b_c)
    MmulHelper::mmul(concatOut, const_cast<NDArray*>(Wc), c, 1.0, 0.0);       //c = 1.0 * concatOut * Wc + 0.0 * c
    *c += *bc;
    tanhInplace(*c);

    //Output: h = (1-u).*c + u .* hPrev
    //auto hResult = (*u) * (*hLast) + (1.0f - *u) * (*c); const_cast<NDArray*>(h)->assign(&hResult);
    u->applyPairwiseTransform(pairwise::Multiply, hLast, h, nullptr);        //h = u * hLast
    auto temp = (1.0f - *u);
    temp *= (*c);
    (*h) += temp;

    delete result;
}

//////////////////////////////////////////////////////////////////////////
void gruTimeLoop(nd4j::LaunchContext * context, const NDArray* x, const NDArray* h0, const NDArray* Wx, const NDArray* Wh, const NDArray* b, NDArray* h) {

}

//////////////////////////////////////////////////////////////////////////
void gruCellBP(nd4j::LaunchContext * context, const NDArray* x, const NDArray* h0, const NDArray* Wx, const NDArray* Wh, const NDArray* b, const NDArray* dLdh, const NDArray* dLdWx0,
               const NDArray* dLdWh0, const NDArray* dLdb0, NDArray* dLdx, NDArray* dLdh0, NDArray* dLdWx, NDArray* dLdWh, NDArray* dLdb) {

    // x                        input [bS, iS]
    // h0                       previous cell output [bS, nU],  that is at previous time step t-1
    // Wx                       input-to-hidden  weights, [iS, 3*nU]
    // Wh                       hidden-to-hidden weights, [nU, 3*nU]
    // b                        biases, [3*nU]
    // dLdh                     gradient wrt output, [bS,nU], that is epsilon_next
    // dLdWx0                   gradient wrt Wx at previous time step, [iS, 3*nU]
    // dLdWh0                   gradient wrt Wh at previous time step, [nU, 3*nU]
    // dLdb0                    gradient wrt b at previous time step,  [3*nU]

    // dLdx                   gradient wrt x,  [bS, iS], that is epsilon
    // dLdh0                  gradient wrt h0, [bS, nU]
    // dLdWx                  gradient wrt Wx, [iS, 3*nU]
    // dLdWh                  gradient wrt Wh, [nU, 3*nU]
    // dLdb                   gradient wrt b at previous time step,  [3*nU]

    // h is current cell output [bS, nU], that is at current time step t

    const int nU = h0->sizeAt(1);

    // ***** feed forward step ***** //
    // gates = sigmoid(x*Wx + h0*Wh + b)
    auto gates = sigmoid(mmul(*x, (*Wx)({0,0, 0,2*nU})) + mmul(*h0, (*Wh)({0,0, 0,2*nU})) + (*b)({0,2*nU}));       // [bS, 2*nU] + [bS, 2*nU] + [1, 2*nU] = [bS, 2*nU]
    // reset gate
    auto r = gates({0,0, 0, nU});               // [bS, nU]
    // update gate
    auto u = gates({0,0, nU, 2*nU});            // [bS, nU]
    // ◦ means element-wise product or so called Hadamard product
    // n = tanh(x*Wx + (r◦h0)*Wh + b)
    auto n = tanh(mmul(*x, (*Wx)({0,0, 2*nU,3*nU})) + mmul((*h0)*r, (*Wh)({0,0, 2*nU,3*nU})) + (*b)({2*nU,3*nU}));     // [bS, nU]

    // ***** back prop step ***** //
    auto Wxr  = (*Wx)({0,0, 0,   nU});
    auto Wxu  = (*Wx)({0,0, nU,  2*nU});
    auto Wxn  = (*Wx)({0,0, 2*nU,3*nU});
    auto Whr  = (*Wh)({0,0, 0,   nU});
    auto Whu  = (*Wh)({0,0, nU,  2*nU});
    auto Whn  = (*Wh)({0,0, 2*nU,3*nU});
    auto WxrT = Wxr.transpose();
    auto WxuT = Wxu.transpose();
    auto WxnT = Wxn.transpose();
    auto WhrT = Whr.transpose();
    auto WhuT = Whu.transpose();
    auto WhnT = Whn.transpose();
    auto xT   = x->transpose();
    auto h0T  = h0->transpose();

    auto dLdWxr = (*dLdWx)({0,0, 0,     nU});
    auto dLdWxu = (*dLdWx)({0,0, nU,  2*nU});
    auto dLdWxn = (*dLdWx)({0,0, 2*nU,3*nU});

    auto dLdWhr = (*dLdWh)({0,0, 0,     nU});
    auto dLdWhu = (*dLdWh)({0,0, nU,  2*nU});
    auto dLdWhn = (*dLdWh)({0,0, 2*nU,3*nU});

    auto dLdbr = (*dLdb)({0,     nU});
    auto dLdbu = (*dLdb)({nU,  2*nU});
    auto dLdbn = (*dLdb)({2*nU,3*nU});

    auto dhdu   = *h0  - n;              // [bS, nU]
    auto dhdn   = 1.f - u;               // [bS, nU]
    auto dSigdu = u * (1.f - u);         // [bS, nU]
    auto dSigdr = r * (1.f - r);         // [bS, nU]
    auto dActdn = 1.f - n * n;           // [bS, nU]
    auto dndr   = mmul(dActdn * (*h0), WhnT);
    auto drdh0  = mmul(dSigdr, WhrT);

    auto dLdn = (*dLdh) * dhdn;
    auto dLdu = (*dLdh) * dhdu;
    auto dLdr = dLdn * dndr;

    dLdx->assign( mmul(dLdu * dSigdu, WxuT) + mmul(dLdr * dSigdr, WxrT) + mmul(dLdn * dActdn, WxnT) );      // [bS,iS]
    dLdh0->assign( mmul(dLdu * dSigdu, WhuT) + mmul(dLdn * dActdn * (r + drdh0), WhnT) + (*dLdh)*u );       // [bS,nU]

    dLdWxr.assign( mmul(xT, dSigdr * dLdr) );                                                               //  [iS,nU]
    dLdWhr.assign( mmul(h0T, dSigdr * dLdr) );                                                              //  [nU,nU]

    dLdWxu.assign( mmul(xT, dSigdu * dLdu) );                                                               //  [iS,nU]
    dLdWhu.assign( mmul(h0T, dSigdu * dLdu) );                                                              //  [nU,nU]

    dLdWxn.assign( mmul(xT, dActdn * dLdn) );                                                               //  [iS,nU]
    dLdWhn.assign( mmul((r*(*h0)).transpose(), dActdn * dLdn) );                                               //  [nU,nU]

    dLdbr.assign( (dSigdr * dLdr).reduceAlongDims(reduce::Sum, {0}));                          // [nU]
    dLdbu.assign( (dSigdu * dLdu).reduceAlongDims(reduce::Sum, {0}));                          // [nU]
    dLdbn.assign( (dActdn * dLdn).reduceAlongDims(reduce::Sum, {0}));                          // [nU]

    if(dLdWx0 != nullptr)
        *dLdWx += *dLdWx0;

    if(dLdWh0 != nullptr)
        *dLdWh += *dLdWh0;

    if(dLdb0 != nullptr)
        *dLdb += *dLdb0;

}


}
}
}

