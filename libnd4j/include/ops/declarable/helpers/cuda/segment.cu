#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author GS <sgazeos@gmail.com>
//

#include <ops/declarable/helpers/segment.h>
#include <NDArrayFactory.h>
#include <helpers/ShapeUtils.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <PointersManager.h>
#include <ConstantTadHelper.h>

namespace nd4j {
namespace ops {
namespace helpers {

    template <typename T, typename I>
    static __global__ void segmentMaxLinearKernel(void* input, Nd4jLong* inputShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
         __shared__ T* val;
         __shared__ Nd4jLong xLen, zLen, segment, zIndex;
         __shared__ T* x;
         __shared__ T* z;
         __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            extern __shared__ unsigned char shmem[];
            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

            //[zIndex] =
            if (segment < numOfClasses) {
                zIndex = shape::getIndexOffset(segment, outputShape, zLen);
                start = starts[segment];
                finish = start + lengths[segment];
                //val[segment] = ;
                z[zIndex] = x[shape::getIndexOffset(start, inputShape, xLen)];
                val[segment] = z[zIndex];
            }

        }
        __syncthreads();
//         auto tid = threadIdx.x + blockIdx.x * blockDim.x;
//         auto step = blockDim.x * gridDim.x;

         for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
             auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
             //val[segment] = nd4j::math::nd4j_max<T>(x[xIndex], val[segment]);
//             if (val[segment] < x[xIndex])
//                 val[segment] = x[xIndex];
             nd4j::math::atomics::nd4j_atomicMax(&z[zIndex], x[xIndex]);
         }
//        __syncthreads();
//        for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
//            auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
//            //val[segment] = nd4j::math::nd4j_max<T>(x[xIndex], val[segment]);
//            if (val[segment] < x[xIndex])
//                val[segment] = x[xIndex];
//        }
//        __syncthreads();
//
//        if (threadIdx.x == 0) {
//            z[zIndex] = val[segment];
//        }

    }

    template <typename T, typename I>
    static __global__ void segmentMinLinearKernel(void* input, Nd4jLong* inputShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            extern __shared__ unsigned char shmem[];
            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

            //[zIndex] =
            if (segment < numOfClasses) {
                zIndex = shape::getIndexOffset(segment, outputShape, zLen);
                start = starts[segment];
                finish = start + lengths[segment];
                //val[segment] = ;
                z[zIndex] = x[shape::getIndexOffset(start, inputShape, xLen)];
                val[segment] = z[zIndex];
            }

        }
        __syncthreads();
//         auto tid = threadIdx.x + blockIdx.x * blockDim.x;
//         auto step = blockDim.x * gridDim.x;

        for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
            auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
            //val[segment] = nd4j::math::nd4j_max<T>(x[xIndex], val[segment]);
           nd4j::math::atomics::nd4j_atomicMin(&z[zIndex], x[xIndex]);
//            if (val[segment] > x[xIndex])
//                val[segment] = x[xIndex];
//            printf("%d(%lld): %lf > %lf\n", e, segment, x[xIndex], val[segment]);
        }
//        __syncthreads();
//        for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
//            auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
//            //val[segment] = nd4j::math::nd4j_max<T>(x[xIndex], val[segment]);
//            if (val[segment] > x[xIndex])
//                val[segment] = x[xIndex];
//        }
//        __syncthreads();
//
//        if (threadIdx.x == 0) {
//            z[zIndex] = val[segment];
//        }

    }
    template <typename T, typename I>
    static __global__ void segmentSumLinearKernel(void* input, Nd4jLong* inputShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);

            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);


            if (segment < numOfClasses) {
                zIndex = shape::getIndexOffset(segment, outputShape, zLen);
                start = starts[segment];
                finish = start + lengths[segment];
                //val[segment] = ;
                z[zIndex] = x[shape::getIndexOffset(start, inputShape, xLen)];
//                val[segment] = z[zIndex];
            }

        }
        __syncthreads();

        for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
            auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
            nd4j::math::atomics::nd4j_atomicAdd(&z[zIndex], x[xIndex]);
        }
    }
    template <typename T, typename I>
    static __global__ void segmentMeanLinearKernel(void* input, Nd4jLong* inputShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            extern __shared__ unsigned char shmem[];
            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

            //[zIndex] =
            if (segment < numOfClasses) {
                zIndex = shape::getIndexOffset(segment, outputShape, zLen);
                start = starts[segment];
                finish = start + lengths[segment];
                //val[segment] = ;
                z[zIndex] = x[shape::getIndexOffset(start, inputShape, xLen)];
                val[segment] = z[zIndex];
            }

        }
        __syncthreads();
//         auto tid = threadIdx.x + blockIdx.x * blockDim.x;
//         auto step = blockDim.x * gridDim.x;

        for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
            auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
            //val[segment] = nd4j::math::nd4j_max<T>(x[xIndex], val[segment]);
            nd4j::math::atomics::nd4j_atomicAdd(&val[segment], x[xIndex]);
        }
        __syncthreads();

        if (threadIdx.x == 0) {
            z[zIndex] = val[segment] / lengths[segment];
        }
    }

    template <typename T, typename I>
    static __global__ void segmentProdLinearKernel(void* input, Nd4jLong* inputShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            extern __shared__ unsigned char shmem[];
            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

            if (segment < numOfClasses) {
                zIndex = shape::getIndexOffset(segment, outputShape, zLen);
                start = starts[segment];
                finish = start + lengths[segment];
                //val[segment] = ;
                z[zIndex] = x[shape::getIndexOffset(start, inputShape, xLen)];
                val[segment] = z[zIndex];
            }

        }
        __syncthreads();
//         auto tid = threadIdx.x + blockIdx.x * blockDim.x;
//         auto step = blockDim.x * gridDim.x;

        for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
            auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
            nd4j::math::atomics::nd4j_atomicMul(&val[segment], x[xIndex]);
        }
        __syncthreads();

        if (threadIdx.x == 0) {
            z[zIndex] = val[segment];
        }

    }

    template <typename I>
    static __global__ void fillUpSegmentsKernel(void* indices, Nd4jLong* indexShape, int numClasses, int* classesRangesStart, int* classesRangesLenghts) {
        __shared__ I* idxBuf;
        __shared__ Nd4jLong idxLen;
        __shared__ int* result;
        if (threadIdx.x == 0) {
            idxBuf = reinterpret_cast<I*>(indices);
            idxLen = shape::length(indexShape);
            //extern __shared__ unsigned char shmem[];
            //result = reinterpret_cast<int*>(shmem);
            //result[0] = 0; //idxBuf[0];
        }
        __syncthreads();

        auto tid = threadIdx.x + blockDim.x * blockIdx.x;
        auto step = blockDim.x * gridDim.x;

        for (auto j = tid; j < idxLen; j += step) {
            auto pos = idxBuf[j];
//             if (classesRangesStart[pos] == idxLen)
//                 classesRangesStart[pos] = j;
//            result[pos] = nd4j::math::nd4j_min<int>(classesRangesStart[pos], j);
            //atomicMin(&classesRangesStart[pos], j);
            nd4j::math::atomics::nd4j_atomicMin(&classesRangesStart[pos], (int)j);
//             = nd4j::math::nd4j_min<int>(classesRangesStart[pos], result[pos]);
            nd4j::math::atomics::nd4j_atomicAdd(&classesRangesLenghts[pos], 1);
        }
    }
    // segment max
    template <typename T, typename I>
    static __global__ void segmentMaxTadKernel(void* inputBuf, Nd4jLong* inputShape, Nd4jLong* inputTads, Nd4jLong* inputTadOffsets, int* starts, int* lengths, Nd4jLong numOfClasses, void* outputBuf, Nd4jLong* outputShape, Nd4jLong* outputTads, Nd4jLong* outputTadOffsets) {
        __shared__ T* val;
        __shared__ Nd4jLong len, segment, zIndex, total;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x / threadsPerSegment;
            //x = reinterpret_cast<T*>(input) + inputTadOffsets[segment];
            z = reinterpret_cast<T*>(outputBuf) + outputTadOffsets[segment];
            len = shape::length(inputTads);
            // = shape::length(outputShape);

            if (segment < numOfClasses) {
//                zIndex = shape::getIndexOffset(segment, outputShape, zLen);
                start = starts[segment];
                finish = start + lengths[segment];
                //val[segment] = ;
//                if (lengths[segment] > 0) {
//                    z[zIndex] = x[shape::getIndexOffset(start, inputShape, xLen)];
//                }
                //val[segment] = z[zIndex];
//                auto x = reinterpret_cast<T*>(inputBuf) + inputTadOffsets[segment];

            }
            //printf("Segment is %d\n", segment);
            total = shape::sizeAt(inputShape, 0);
            printf("Total rows %lld. %lld per each.\n", total, len);
            auto x = reinterpret_cast<T*>(inputBuf) + inputTadOffsets[starts[segment]];
            for (auto e = 0; e < len; e++) {
                auto xIndex = shape::getIndexOffset(e, inputTads, len);
                auto zIndex = shape::getIndexOffset(e, outputTads, len);
                z[xIndex] = x[xIndex];
            }

        }
        __syncthreads();

//        for (auto idx = start + blockIdx.x; idx < finish; idx += gridDim.x ){
//            printf("Segment: %d; Idx: %d (%d)\n", segment, idx, starts[segment]);
//            auto x = reinterpret_cast<T*>(inputBuf) + inputTadOffsets[idx];
//            //auto currentSegment = indices[idx];
//            if (idx == starts[segment]) {
//                x = reinterpret_cast<T*>(inputBuf) + inputTadOffsets[start];
//                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
//                    auto xIndex = shape::getIndexOffset(e, inputTads, len);
//                    auto zIndex = shape::getIndexOffset(e, outputTads, len);
//
//                    z[zIndex] = x[xIndex];
//                }
//            }
//            else
        for (auto idx = start + blockIdx.x + 1; idx < finish; idx += gridDim.x) {
            auto x = reinterpret_cast<T*>(inputBuf) + inputTadOffsets[idx];
            //printf("Segment: %d; Idx: %d (%d)\n", segment, idx, starts[segment]);
            for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                auto xIndex = shape::getIndexOffset(e, inputTads, len);
                auto zIndex = shape::getIndexOffset(e, outputTads, len);
                nd4j::math::atomics::nd4j_atomicMax(&z[zIndex], x[xIndex]);
            }
        }
}
    template <typename T, typename I>
    static void segmentMaxFunctor_(LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
        //int numClasses = output->sizeAt(0);
        // if input is a vector: (as if in doc sample)
        //Nd4jLong idx = indices->e<Nd4jLong>(0);
        auto stream = context->getCudaStream();
        Nd4jLong numClasses = indices->e<Nd4jLong>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses});

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);

        dim3 dims(256, 512, 256);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numClasses, begins, lengths);

        if (input->isVector()) {
            segmentMaxLinearKernel<T,I><<<numClasses, input->lengthOf(), numClasses * 32 + 32, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            segmentMaxTadKernel<T,I><<<input->sizeAt(0), 512, 2048, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }
    }

    // segmen min 
    template <typename T, typename I>
    static void segmentMinFunctor_(LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
        auto stream = context->getCudaStream();
        Nd4jLong numClasses = indices->e<Nd4jLong>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses});

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);

        dim3 dims(numClasses, indices->lengthOf(), numClasses * 32 + 32);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numClasses, begins, lengths);

        if (input->isVector()) {
            segmentMinLinearKernel<T,I><<<numClasses, input->lengthOf(), numClasses * 32 + 32, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {

        }
    }

    // segmen mean
    template <typename T, typename I>
    static void segmentMeanFunctor_(LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
        auto stream = context->getCudaStream();
        Nd4jLong numClasses = indices->e<Nd4jLong>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses});

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);

        dim3 dims(numClasses, indices->lengthOf(), numClasses * 32 + 32);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numClasses, begins, lengths);

        if (input->isVector()) {
            segmentMeanLinearKernel<T,I><<<numClasses, input->lengthOf(), numClasses * 32 + 32, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {

        }

    }

    template <typename T, typename I>
    static void segmentSumFunctor_(nd4j::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
        auto stream = context->getCudaStream();
        Nd4jLong numClasses = indices->e<Nd4jLong>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses});

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);

        dim3 dims(numClasses, indices->lengthOf(), numClasses * 32 + 32);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numClasses, begins, lengths);

        if (input->isVector()) {
            segmentSumLinearKernel<T,I><<<numClasses, input->lengthOf(), numClasses * 32 + 32, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {

        }

    }

    template <typename T, typename I>
    static void segmentProdFunctor_(nd4j::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
        auto stream = context->getCudaStream();
        Nd4jLong numClasses = indices->e<Nd4jLong>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses});

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);

        dim3 dims(numClasses, indices->lengthOf(), numClasses * 32 + 32);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numClasses, begins, lengths);

        if (input->isVector()) {
            segmentProdLinearKernel<T,I><<<numClasses, input->lengthOf(), numClasses * 32 + 32, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {

        }

    }

    template <typename T, typename I>
    static bool segmentIndicesValidate_(NDArray* indices, NDArray& aexpected, NDArray& aoutput) {
        return true;
    }

    void segmentMaxFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), segmentMaxFunctor_, (context, input, indices, output), NUMERIC_TYPES, INTEGER_TYPES);
    }

    void segmentMinFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), segmentMinFunctor_, (context, input, indices, output), NUMERIC_TYPES, INTEGER_TYPES);
    }

    void segmentMeanFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), segmentMeanFunctor_, (context, input, indices, output), NUMERIC_TYPES, INTEGER_TYPES);
    }

    void segmentSumFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), segmentSumFunctor_, (context, input, indices, output), NUMERIC_TYPES, INTEGER_TYPES);
    }

    void segmentProdFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), segmentProdFunctor_, (context, input, indices, output), FLOAT_TYPES, INTEGER_TYPES);
    }

    bool segmentIndicesValidate(nd4j::LaunchContext * context, NDArray* indices, NDArray& expected, NDArray& output) {
        BUILD_DOUBLE_SELECTOR(output.dataType(), indices->dataType(), return segmentIndicesValidate_, (indices, expected, output), NUMERIC_TYPES, INTEGER_TYPES);
    }

    BUILD_DOUBLE_TEMPLATE(template bool segmentIndicesValidate_, (NDArray*, NDArray&, NDArray&), NUMERIC_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template void segmentProdFunctor_, (nd4j::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output), FLOAT_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template void segmentSumFunctor_, (nd4j::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output), NUMERIC_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template void segmentMeanFunctor_, (nd4j::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output), NUMERIC_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template void segmentMinFunctor_, (nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* output), NUMERIC_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template void segmentMaxFunctor_, (LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output), NUMERIC_TYPES, INTEGER_TYPES);
    // -------------------------------------------------------------------------------------------------------------- //
    // Unsorted segment ops
    // -------------------------------------------------------------------------------------------------------------- //

    bool unsortedSegmentIndicesValidate(nd4j::LaunchContext * context, NDArray* indices, Nd4jLong expected, Nd4jLong& output) {
        return true;
    }

    template <typename T>
    static void unsortedSegmentMaxFunctor_(NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {

    }

    void unsortedSegmentMaxFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), unsortedSegmentMaxFunctor_, (input, indices, numOfClasses, output), NUMERIC_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template void unsortedSegmentMaxFunctor_, (NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES);

    template <typename T>
    static void unsortedSegmentMinFunctor_(NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {

    }

    void unsortedSegmentMinFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), unsortedSegmentMinFunctor_, (input, indices, numOfClasses, output),
                              NUMERIC_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void unsortedSegmentMinFunctor_, (NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES);

    void unsortedSegmentMeanFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {

    }

    void unsortedSegmentSumFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {

    }

    void unsortedSegmentProdFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
      //  BUILD_SINGLE_SELECTOR(input->dataType(), unsortedSegmentProdFunctor_, (input, indices, numOfClasses, output), NUMERIC_TYPES);
    }
    //BUILD_SINGLE_TEMPLATE(template void unsortedSegmentProdFunctor_, (NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES);

    void unsortedSegmentSqrtNFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {

    }

    // -------------------------------------------------------------------------------------------------------------- //
    // Backpropagate ops helpers
    // -------------------------------------------------------------------------------------------------------------- //
    // Sorted backpropagate ops
    //

    // segment max
    template <typename T>
    int segmentMaxFunctorBP_(NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        return Status::OK();
    }

    int segmentMaxFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        BUILD_SINGLE_SELECTOR(output->dataType(), return segmentMaxFunctorBP_, (input, indices, gradOut, output), NUMERIC_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template int segmentMaxFunctorBP_, (NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output), NUMERIC_TYPES);

    // segmen min
    int segmentMinFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        return Status::OK();
    }

    // segmen mean
    int segmentMeanFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        return Status::OK();
    }

    int segmentSumFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        return Status::OK();
    }

    // -------------------------------------------------------------------------------------------------------------- //
    // Unsorted backpropagate segment ops
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T>
    static int unsortedSegmentMaxFunctorBP_(NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        return Status::OK();
    }

    int unsortedSegmentMaxFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_SINGLE_SELECTOR(output->dataType(), return unsortedSegmentMaxFunctorBP_, (input, indices, gradOut, numOfClasses, output), NUMERIC_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template int unsortedSegmentMaxFunctorBP_, (NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES);

    template <typename T>
    static int unsortedSegmentMinFunctorBP_(NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        return Status::OK();
    }

    int unsortedSegmentMinFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_SINGLE_SELECTOR(output->dataType(), return unsortedSegmentMinFunctorBP_, (input, indices, gradOut, numOfClasses, output), NUMERIC_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template int unsortedSegmentMinFunctorBP_, (NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES);

    int unsortedSegmentMeanFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        return Status::OK();
    }

    int unsortedSegmentSumFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        return Status::OK();
    }

    int unsortedSegmentProdFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        return Status::OK();
    }

//    template <typename T>
    int unsortedSegmentSqrtNFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        return Status::OK();
    }

//    int unsortedSegmentSqrtNFunctorBP(NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
//        BUILD_SINGLE_SELECTOR(output->dataType(), return unsortedSegmentSqrtNFunctorBP_, (input, indices, gradOut, numOfClasses, output), FLOAT_TYPES);
//    }
//    BUILD_SINGLE_TEMPLATE(template int unsortedSegmentSqrtNFunctorBP_, (NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output), FLOAT_TYPES);
}
}
}