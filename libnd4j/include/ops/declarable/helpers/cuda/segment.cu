#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author GS <sgazeos@gmail.com>
//

#include <ops/declarable/helpers/segment.h>

namespace nd4j {
namespace ops {
namespace helpers {

    // segment max
    template <typename T>
    static void segmentMaxFunctor_(NDArray* input, NDArray* indices, NDArray* output) {

    }

    // segmen min 
    template <typename T>
    static void segmentMinFunctor_(NDArray* input, NDArray* indices, NDArray* output) {

    }

    // segmen mean
    template <typename T>
    static void segmentMeanFunctor_(NDArray* input, NDArray* indices, NDArray* output) {

    }

    template <typename T>
    static void segmentSumFunctor_(NDArray* input, NDArray* indices, NDArray* output) {

    }

    template <typename T>
    static void segmentProdFunctor_(NDArray* input, NDArray* indices, NDArray* output) {

    }

    template <typename T>
    static bool segmentIndicesValidate_(NDArray* indices, NDArray& aexpected, NDArray& aoutput) {
        return true;
    }

    void segmentMaxFunctor(NDArray* input, NDArray* indices, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), segmentMaxFunctor_, (input, indices, output), LIBND4J_TYPES);
    }

    void segmentMinFunctor(NDArray* input, NDArray* indices, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), segmentMinFunctor_, (input, indices, output), LIBND4J_TYPES);
    }

    void segmentMeanFunctor(NDArray* input, NDArray* indices, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), segmentMeanFunctor_, (input, indices, output), LIBND4J_TYPES);
    }

    void segmentSumFunctor(NDArray* input, NDArray* indices, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), segmentSumFunctor_, (input, indices, output), LIBND4J_TYPES);
    }

    void segmentProdFunctor(NDArray* input, NDArray* indices, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), segmentProdFunctor_, (input, indices, output), LIBND4J_TYPES);
    }

    bool segmentIndicesValidate(NDArray* indices, NDArray& expected, NDArray& output) {
        BUILD_SINGLE_SELECTOR(output.dataType(), return segmentIndicesValidate_, (indices, expected, output), LIBND4J_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template bool segmentIndicesValidate_, (NDArray*, NDArray&, NDArray&), LIBND4J_TYPES);
    BUILD_SINGLE_TEMPLATE(template void segmentProdFunctor_, (NDArray* input, NDArray* indices, NDArray* output), LIBND4J_TYPES);
    BUILD_SINGLE_TEMPLATE(template void segmentSumFunctor_, (NDArray* input, NDArray* indices, NDArray* output), LIBND4J_TYPES);
    BUILD_SINGLE_TEMPLATE(template void segmentMeanFunctor_, (NDArray* input, NDArray* indices, NDArray* output), LIBND4J_TYPES);
    BUILD_SINGLE_TEMPLATE(template void segmentMinFunctor_, (NDArray* input, NDArray* indices, NDArray* output), LIBND4J_TYPES);
    BUILD_SINGLE_TEMPLATE(template void segmentMaxFunctor_, (NDArray* input, NDArray* indices, NDArray* output), LIBND4J_TYPES);
    // -------------------------------------------------------------------------------------------------------------- //
    // Unsorted segment ops
    // -------------------------------------------------------------------------------------------------------------- //

    bool unsortedSegmentIndicesValidate(NDArray* indices, Nd4jLong expected, Nd4jLong& output) {
        return true;
    }

    template <typename T>
    static void unsortedSegmentMaxFunctor_(NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {

    }

    void unsortedSegmentMaxFunctor(NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), unsortedSegmentMaxFunctor_, (input, indices, numOfClasses, output), NUMERIC_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template void unsortedSegmentMaxFunctor_, (NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES);

    template <typename T>
    static void unsortedSegmentMinFunctor_(NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {

    }

    void unsortedSegmentMinFunctor(NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), unsortedSegmentMinFunctor_, (input, indices, numOfClasses, output),
                              NUMERIC_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void unsortedSegmentMinFunctor_, (NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES);

    void unsortedSegmentMeanFunctor(NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {

    }

    void unsortedSegmentSumFunctor(NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {

    }

    void unsortedSegmentProdFunctor(NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
      //  BUILD_SINGLE_SELECTOR(input->dataType(), unsortedSegmentProdFunctor_, (input, indices, numOfClasses, output), NUMERIC_TYPES);
    }
    //BUILD_SINGLE_TEMPLATE(template void unsortedSegmentProdFunctor_, (NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES);

    void unsortedSegmentSqrtNFunctor(NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {

    }

    // -------------------------------------------------------------------------------------------------------------- //
    // Backpropagate ops helpers
    // -------------------------------------------------------------------------------------------------------------- //
    // Sorted backpropagate ops
    //

    // segment max
    template <typename T>
    int segmentMaxFunctorBP_(NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        return Status::OK();
    }

    int segmentMaxFunctorBP(NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        BUILD_SINGLE_SELECTOR(output->dataType(), return segmentMaxFunctorBP_, (input, indices, gradOut, output), NUMERIC_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template int segmentMaxFunctorBP_, (NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output), NUMERIC_TYPES);

    // segmen min
    int segmentMinFunctorBP(NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        return Status::OK();
    }

    // segmen mean
    int segmentMeanFunctorBP(NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        return Status::OK();
    }

    int segmentSumFunctorBP(NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        return Status::OK();
    }

    // -------------------------------------------------------------------------------------------------------------- //
    // Unsorted backpropagate segment ops
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T>
    static int unsortedSegmentMaxFunctorBP_(NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        return Status::OK();
    }

    int unsortedSegmentMaxFunctorBP(NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_SINGLE_SELECTOR(output->dataType(), return unsortedSegmentMaxFunctorBP_, (input, indices, gradOut, numOfClasses, output), NUMERIC_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template int unsortedSegmentMaxFunctorBP_, (NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES);

    template <typename T>
    static int unsortedSegmentMinFunctorBP_(NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        return Status::OK();
    }

    int unsortedSegmentMinFunctorBP(NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_SINGLE_SELECTOR(output->dataType(), return unsortedSegmentMinFunctorBP_, (input, indices, gradOut, numOfClasses, output), NUMERIC_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template int unsortedSegmentMinFunctorBP_, (NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES);

    int unsortedSegmentMeanFunctorBP(NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        return Status::OK();
    }

    int unsortedSegmentSumFunctorBP(NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        return Status::OK();
    }

    int unsortedSegmentProdFunctorBP(NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        return Status::OK();
    }

//    template <typename T>
    int unsortedSegmentSqrtNFunctorBP(NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        return Status::OK();
    }

//    int unsortedSegmentSqrtNFunctorBP(NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
//        BUILD_SINGLE_SELECTOR(output->dataType(), return unsortedSegmentSqrtNFunctorBP_, (input, indices, gradOut, numOfClasses, output), FLOAT_TYPES);
//    }
//    BUILD_SINGLE_TEMPLATE(template int unsortedSegmentSqrtNFunctorBP_, (NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output), FLOAT_TYPES);
}
}
}