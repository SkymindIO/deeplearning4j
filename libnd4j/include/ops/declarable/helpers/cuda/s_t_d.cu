/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//
//

#include <ops/declarable/helpers/s_t_d.h>

namespace nd4j {
namespace ops {
namespace helpers {

    template <typename T>
    static void _spaceTodepth_(NDArray *input, NDArray *output, int block_size, bool isNHWC) {

    }

    void _spaceTodepth(NDArray *input, NDArray *output, int block_size, bool isNHWC) {
        BUILD_SINGLE_SELECTOR(input->dataType(), _spaceTodepth_, (input, output, block_size, isNHWC), LIBND4J_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void _spaceTodepth_, (NDArray *input, NDArray *output, int block_size, bool isNHWC), LIBND4J_TYPES);

}
}
}