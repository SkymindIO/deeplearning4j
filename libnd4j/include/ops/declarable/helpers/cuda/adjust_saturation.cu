#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma (iuriish@yahoo.com)
//
#include <helpers/ConstantTadHelper.h>
#include <helpers/PointersManager.h>
#include <ops/declarable/helpers/adjust_hue.h>
#include <ops/declarable/helpers/adjust_saturation.h>

namespace sd {
namespace ops {
namespace helpers {

///////////////////////////////////////////////////////////////////
template <typename T>
static void SD_KERNEL adjustSaturationCuda(const void* vx, const sd::LongType* xShapeInfo,
                                           const sd::LongType* xTadOffsets, void* vz, const sd::LongType* zShapeInfo,
                                           const sd::LongType* zTadOffsets, const sd::LongType numOfTads,
                                           const T factor, const int dimC) {
  const T* x = reinterpret_cast<const T*>(vx);
  T* z = reinterpret_cast<T*>(vz);

  __shared__ int rank;
  __shared__ sd::LongType xDimCstride, zDimCstride;

  if (threadIdx.x == 0) {
    rank = shape::rank(xShapeInfo);
    xDimCstride = shape::stride(xShapeInfo)[dimC];
    zDimCstride = shape::stride(zShapeInfo)[dimC];
  }
  __syncthreads();

  const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

  for (sd::LongType i = tid; i < numOfTads; i += gridDim.x * blockDim.x) {
    const T* xTad = x + xTadOffsets[i];
    T* zTad = z + zTadOffsets[i];

    T h, s, v;

    rgbToHsv<T>(xTad[0], xTad[xDimCstride], xTad[2 * xDimCstride], h, s, v);

    s *= factor;
    if (s > 1.f)
      s = 1.f;
    else if (s < 0.f)
      s = 0.f;

    hsvToRgb<T>(h, s, v, zTad[0], zTad[zDimCstride], zTad[2 * zDimCstride]);
  }
}

///////////////////////////////////////////////////////////////////
template <typename T>
static SD_HOST void adjustSaturationCudaLauncher(const int blocksPerGrid, const int threadsPerBlock,
                                                 const hipStream_t* stream, const void* vx,
                                                 const sd::LongType* xShapeInfo, const sd::LongType* xTadOffsets,
                                                 void* vz, const sd::LongType* zShapeInfo,
                                                 const sd::LongType* zTadOffsets, const sd::LongType numOfTads,
                                                 const NDArray* factorScalarArr, const int dimC) {
  adjustSaturationCuda<T><<<blocksPerGrid, threadsPerBlock, 256, *stream>>>(
      vx, xShapeInfo, xTadOffsets, vz, zShapeInfo, zTadOffsets, numOfTads, factorScalarArr->e<T>(0), dimC);
}

////////////////////////////////////////////////////////////////////////
void adjustSaturation(sd::LaunchContext* context, const NDArray* input, const NDArray* factorScalarArr, NDArray* output,
                      const int dimC) {
  auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), {dimC});
  auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), {dimC});

  const sd::LongType numOfTads = packX->numberOfTads();

  const int threadsPerBlock = SD_MAX_NUM_THREADS / 2;
  const int blocksPerGrid = (numOfTads + threadsPerBlock - 1) / threadsPerBlock;

  PointersManager manager(context, "adjustSaturation");

  NDArray::prepareSpecialUse({output}, {input, factorScalarArr});
  BUILD_SINGLE_SELECTOR(input->dataType(), adjustSaturationCudaLauncher,
                        (blocksPerGrid, threadsPerBlock, context->getCudaStream(), input->specialBuffer(),
                         input->specialShapeInfo(), packX->platformOffsets(), output->specialBuffer(),
                         output->specialShapeInfo(), packZ->platformOffsets(), numOfTads, factorScalarArr, dimC),
                        SD_FLOAT_TYPES);
  NDArray::registerSpecialUse({output}, {input, factorScalarArr});

  manager.synchronize();
}



}  // namespace helpers
}  // namespace ops
}  // namespace sd
