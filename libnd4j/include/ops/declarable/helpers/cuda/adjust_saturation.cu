/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//

#include <ops/declarable/helpers/adjust_saturation.h>


namespace nd4j {
namespace ops {
namespace helpers {

    template <typename T>
    static void _adjust_saturation_single(NDArray *array, NDArray *output, float delta, bool isNHWC) {

    }

    void _adjust_saturation(NDArray *array, NDArray *output, NDArray* delta, bool isNHWC) {
        auto xType = array->dataType();

        float d = delta->e<float>(0);
        if (array->rankOf() == 4) {
            auto tadsIn = array->allTensorsAlongDimension({0});
            auto tadsOut = output->allTensorsAlongDimension({0});

            // FIXME: template selector should be moved out of loop
#pragma omp parallel for
            for (int e = 0; e < tadsIn->size(); e++) {
                BUILD_SINGLE_SELECTOR(xType, _adjust_saturation_single, (tadsIn->at(e), tadsOut->at(e), d, isNHWC);, FLOAT_TYPES);
            }
            

            delete tadsIn;
            delete tadsOut;
        } else {
            BUILD_SINGLE_SELECTOR(xType, _adjust_saturation_single, (array, output, d, isNHWC);, FLOAT_TYPES);
        }
    }

    BUILD_SINGLE_TEMPLATE(template void _adjust_saturation_single, (NDArray *array, NDArray *output, float delta, bool isNHWC), FLOAT_TYPES);

}
}
}