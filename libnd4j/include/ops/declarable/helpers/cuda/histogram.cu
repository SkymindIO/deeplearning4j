#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//

#include <ops/declarable/helpers/histogram.h>
#include <NDArrayFactory.h>

namespace nd4j {
    namespace ops {
        namespace helpers {
            template <typename X, typename Z>
            void _CUDA_G histogramKernel(void *xBuffer, Nd4jLong *xShapeInfo, void *zBuffer, Nd4jLong *zShapeInfo, void *allocationPointer, void *reductionPointer, Nd4jLong numBins, double min_val, double max_val) {
                int tid = blockIdx.x * blockDim.x + threadIdx.x;
                auto dx = reinterpret_cast<X*>(xBuffer);
                auto result = reinterpret_cast<Z*>(zBuffer);

                __shared__ Z *bins;
                __shared__ int length;
                __shared__ Z *reductor;
                if (threadIdx.x == 0) {
                    extern __shared__ unsigned char shmem[];
                    bins = (Z *) shmem;
                    reductor = ((Z *) allocationPointer) + (numBins * blockIdx.x);

                    length = shape::length(xShapeInfo);
                }
                __syncthreads();

                Z binSize = (max_val - min_val) / (numBins);

                for (int e = threadIdx.x; e < numBins; e += blockDim.x) {
                    bins[e] = (Z) 0.0f;
                }
                __syncthreads();

                for (int e = tid; e < length; e+= blockDim.x * gridDim.x) {
                    int idx = (int) ((dx[e] - min_val) / binSize);
                    if (idx < 0) idx = 0;
                    else if (idx >= numBins) idx = numBins - 1;

                    nd4j::math::atomics::nd4j_atomicAdd(&bins[idx], (Z) 1.0f);
                }
                __syncthreads();

                // transfer shared memory to reduction memory


                if (gridDim.x > 1) {
                    unsigned int *tc = (unsigned int *)reductionPointer;
                    __shared__ bool amLast;

                    for (int e = threadIdx.x; e < numBins; e += blockDim.x) {
                        reductor[e] = bins[e];
                    }
                    __threadfence();
                    __syncthreads();

                    if (threadIdx.x == 0) {
                        unsigned int ticket = atomicInc(&tc[16384], gridDim.x);
                        amLast = (ticket == gridDim.x - 1);
                    }
                    __syncthreads();

                    if (amLast) {
                        tc[16384] = 0;

                        // nullify shared memory for future accumulation
                        for (int e = threadIdx.x; e < numBins; e += blockDim.x) {
                            bins[e] = (Z) 0.0f;
                        }

                        // accumulate reduced bins
                        for (int r = 0; r < gridDim.x; r++) {
                            Z *ptrBuf = ((Z *)allocationPointer) + (r * numBins);

                            for (int e = threadIdx.x; e < numBins; e += blockDim.x) {
                                bins[e] += ptrBuf[e];
                            }
                        }
                        __syncthreads();

                        // write them out to Z
                        for (int e = threadIdx.x; e < numBins; e += blockDim.x) {
                            result[e] = bins[e];
                        }
                    }
                } else {
                    // if there's only 1 block - just write away data
                    for (int e = threadIdx.x; e < numBins; e += blockDim.x) {
                        result[e] = bins[e];
                    }
                }
            }

            template <typename X, typename Z>
            static void histogram_(nd4j::LaunchContext *context, void *xBuffer, Nd4jLong *xShapeInfo, void *zBuffer, Nd4jLong *zShapeInfo, Nd4jLong numBins, double min_val, double max_val) {
                int numThreads = 256;
                int numBlocks = nd4j::math::nd4j_max<int>(256, nd4j::math::nd4j_min<int>(1, shape::length(xShapeInfo) / numThreads));
                int workspaceSize = numBlocks * numBins;
                auto tmp = NDArrayFactory::create<Z>('c',{workspaceSize});

                histogramKernel<X, Z><<<numBlocks, numThreads, 32768, *context->getCudaStream()>>>(xBuffer, xShapeInfo, zBuffer, zShapeInfo, tmp.getSpecialBuffer(), context->getReductionPointer(), numBins, min_val, max_val);

                hipStreamSynchronize(*context->getCudaStream());
            }

            void histogramHelper(nd4j::LaunchContext *context, NDArray &input, NDArray &output) {
                Nd4jLong numBins = output.lengthOf();
                double min_val = input.reduceNumber(reduce::SameOps::Min).e<double>(0);
                double max_val = input.reduceNumber(reduce::SameOps::Max).e<double>(0);

                BUILD_DOUBLE_SELECTOR(input.dataType(), output.dataType(), histogram_, (context, input.specialBuffer(), input.specialShapeInfo(), output.getSpecialBuffer(), output.getSpecialShapeInfo(), numBins, min_val, max_val), LIBND4J_TYPES, INTEGER_TYPES);

                NDArray::registerSpecialUse({&output}, {&input});
            }
        }
    }
}