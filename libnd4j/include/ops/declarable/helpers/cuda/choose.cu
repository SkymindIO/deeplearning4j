/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author sgazeos@gmail.com
//

#include <ops/declarable/helpers/choose.h>
#include <NDArrayFactory.h>

namespace nd4j {
namespace ops {
namespace helpers {


    template <typename T>
    nd4j::NDArray* processCondition_(int mode,nd4j::NDArray *arg, nd4j::NDArray *comp, nd4j::NDArray *output, nd4j::NDArray *numResult, nd4j::NDArray& compScalar) {
        return output;
    }

    nd4j::NDArray* processCondition(int mode,nd4j::NDArray *arg, nd4j::NDArray *comp, nd4j::NDArray *output, nd4j::NDArray *numResult, nd4j::NDArray& compScalar) {
        BUILD_SINGLE_SELECTOR(arg->dataType(), return processCondition_, (mode, arg, comp, output, numResult, compScalar), FLOAT_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template NDArray* processCondition_, (int mode,nd4j::NDArray *arg, nd4j::NDArray *comp, nd4j::NDArray *output, nd4j::NDArray *numResult, nd4j::NDArray& compScalar), FLOAT_TYPES);

    template <typename T>
    T processElementCondition(int mode,T d1,T d2) {
        T modePointer = (T ) mode;
        T input[3] = {d2, (T) EPS, (T) mode};
        T res = simdOps::MatchCondition<T,T>::op(d1, input);
        return res;
    }

    void chooseFunctorArray(NDArray* arg, NDArray* comp, int mode, NDArray* result, NDArray* numResults) {
        if(arg->isScalar() || comp->isScalar()) {
            if(arg->isScalar()) {
                processCondition(mode,comp,nullptr,result,numResults, *arg);
            }
            else {
                processCondition(mode,arg,nullptr,result,numResults, *comp);
            }
        }
        else {
            auto zero = NDArrayFactory::create<float>(0);
            processCondition(mode,arg,comp,result,numResults, zero);
        }
    }

    void chooseFunctorScalar(NDArray* arg, double scalar, int mode, NDArray* result, NDArray* numResults) {
        NDArray scalarA = NDArrayFactory::create(scalar);
        processCondition(mode, arg, nullptr,result, numResults, scalarA);
    }

}
}
}
