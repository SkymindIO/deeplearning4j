#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author sgazeos@gmail.com
//

#include <ops/declarable/helpers/random.h>
//#include <NativeOps.h>
#include <vector>
#include <memory>
#include <graph/Context.h>
#include <helpers/RandomLauncher.h>
#include <ShapeUtils.h>
#include <NDArrayFactory.h>

namespace nd4j {
namespace ops {
namespace helpers {

    /*
     * fillGammaKernel - fill up output with gamma distributed values
     *
     *  uList - uniformly distributed values set
     *  uLength - length of uList
     *  alpha - alpha param
     *  beta - beta param
     *  output - distributed output.
     * */
    template <typename T>
    static __global__ void fillGammaKernel(T* uList, Nd4jLong uLength, T* alpha, Nd4jLong* alphaShape,
            T* beta, Nd4jLong* betaShape, T* output, Nd4jLong* outputShape) {
        // fill up
        __shared__ Nd4jLong aLength;
        if (threadIdx.x == 0) {
            aLength = shape::length(alphaShape);
        }
        __syncthreads();

        for (auto k = blockIdx.x; k < (int)uLength; k += gridDim.x) {
            auto pos = k * aLength;
            auto u = uList[k];
            for (auto e = threadIdx.x; e < (int)aLength; e += blockDim.x) {
                auto betaV = T(beta != nullptr ? beta[e] * u : u);
                output[pos + e] = math::nd4j_igamma<float, float, float>(alpha[e], betaV);
            }
        }
    }

    template <typename T>
    static void fillRandomGamma_(LaunchContext* context, graph::RandomGenerator& rng, NDArray* alpha, NDArray* beta, NDArray* output) {
        // To fill up output need to broadcast alpha and beta to the same shape and in
        Nd4jLong* broadcasted = nullptr;
        if (beta != nullptr)
            ShapeUtils::evalBroadcastShapeInfo(*alpha, *beta, true, broadcasted, context->getWorkspace());
        else
            broadcasted = alpha->shapeInfo();
        auto step = shape::length(broadcasted);
        auto shift = output->lengthOf() / step;

        auto copyAlpha = alpha;
        auto copyBeta = beta;
        if (beta != nullptr) {
            NDArray alphaBroadcasted(broadcasted, alpha->dataType(), true, context);
            NDArray betaBroadcasted(broadcasted, beta->dataType(), true, context);

            copyAlpha = (alphaBroadcasted.applyTrueBroadcast(BroadcastOpsTuple::Assign(), alpha));
            copyBeta = (betaBroadcasted.applyTrueBroadcast(BroadcastOpsTuple::Assign(), beta));
            copyAlpha->tickWriteDevice(); copyBeta->tickWriteDevice();
        }

        auto stream = context->getCudaStream();
        NDArray uniform = NDArrayFactory::create<T>('c', {shift}, context);
        uniform.syncToDevice();
        // fill up uniform with given length
        RandomLauncher::fillUniform(context, rng, &uniform, 0., 1.);

        fillGammaKernel<T><<<128, 128, 256, *stream>>>(uniform.dataBuffer()->specialAsT<T>(), shift,
                copyAlpha->dataBuffer()->specialAsT<T>(), copyAlpha->specialShapeInfo(),
                beta?copyBeta->dataBuffer()->specialAsT<T>():(T*)nullptr,
                beta?copyBeta->specialShapeInfo():(Nd4jLong*)nullptr,
                output->dataBuffer()->specialAsT<T>(), output->specialShapeInfo());

        if (beta != nullptr) {
            delete copyAlpha;
            delete copyBeta;
            //delete broadcasted;
        }

    }

    void fillRandomGamma(LaunchContext* context, graph::RandomGenerator& rng, NDArray* alpha, NDArray* beta, NDArray* output) {
        if (beta)
            NDArray::prepareSpecialUse({output}, {alpha, beta});
        else
            NDArray::prepareSpecialUse({output}, {alpha});
        BUILD_SINGLE_SELECTOR(output->dataType(), fillRandomGamma_, (context, rng, alpha, beta, output), FLOAT_NATIVE);
        if (beta)
            NDArray::registerSpecialUse({output}, {alpha, beta});
        else
            NDArray::prepareSpecialUse({output}, {alpha});
    }
    BUILD_SINGLE_TEMPLATE(template void fillRandomGamma_, (LaunchContext* context, graph::RandomGenerator& rng, NDArray* alpha, NDArray* beta, NDArray* output), FLOAT_NATIVE);


    /*
     * algorithm Poisson generator based upon the inversion by sequential search
     *
    init:
         Let x ← 0, p ← e−λ, s ← p.
         using uniformly random sequence U (u in U) distributed at [0, 1].
    while u > s do:
         x ← x + 1.
         p ← p * λ / x.
         s ← s + p.
    return x.
     * */
    template <typename T>
    static __global__ void fillPoissonKernel(T* uList, Nd4jLong uLength, T* lambda, Nd4jLong* lambdaShape, T* output,
            Nd4jLong* outputShape) {

        __shared__ Nd4jLong step;

        if (threadIdx.x == 0) {
            step = shape::length(lambdaShape);
        }
        __syncthreads();

        for (auto k = blockIdx.x; k < (int)uLength; k += gridDim.x) {
            auto pos = k * step;
            auto u = uList[k];
            for (auto e = threadIdx.x; e < step; e += blockDim.x) {
                auto p = math::nd4j_exp<T,T>(-lambda[e]);
                auto s = p;
                auto x = T(0.f);
                while (u > s) {
                    x += T(1.);
                    p *= lambda[e] / x;
                    s += p;
                }
                output[pos + e] = x;
            }
        }
    }

    template <typename T>
    static void fillRandomPoisson_(LaunchContext* context, graph::RandomGenerator& rng, NDArray* lambda, NDArray* output) {
        auto shift = output->lengthOf() / lambda->lengthOf();
        NDArray uniform('c', {shift}, output->dataType());
        auto stream = context->getCudaStream();
        // fill up uniform with given length
        RandomLauncher::fillUniform(context, rng, &uniform, 0., 1.);
        fillPoissonKernel<T><<<128, 256, 128, *stream>>>(uniform.dataBuffer()->specialAsT<T>(), uniform.lengthOf(),
                lambda->dataBuffer()->specialAsT<T>(), lambda->specialShapeInfo(),
                output->dataBuffer()->specialAsT<T>(), output->specialShapeInfo());
    }

    void fillRandomPoisson(LaunchContext* context, graph::RandomGenerator& rng, NDArray* lambda, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {lambda});
        BUILD_SINGLE_SELECTOR(output->dataType(), fillRandomPoisson_, (context, rng, lambda, output), FLOAT_NATIVE);
        NDArray::registerSpecialUse({output}, {lambda});
    }

    BUILD_SINGLE_TEMPLATE(template void fillRandomPoisson_, (LaunchContext* context, graph::RandomGenerator& rng, NDArray* lambda, NDArray* output), FLOAT_NATIVE);
}
}
}