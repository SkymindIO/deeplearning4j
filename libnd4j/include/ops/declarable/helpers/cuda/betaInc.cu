#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// Created by Yurii Shyrma on 11.12.2017
//

#include<cmath> 
#include <DataTypeUtils.h>
#include<ops/declarable/helpers/betaInc.h>
#include <NDArrayFactory.h>

namespace nd4j {
namespace ops {
namespace helpers {

const int maxIter = 10000;				// max number of loop iterations in function for continued fractions 
const int maxValue = 3000;				// if a and b are both > maxValue, then apply Gauss-Legendre quadrature.


// 18 values of abscissas and weights for 36-point Gauss-Legendre integration,
// take a note - weights and abscissas are symmetric around the midpoint of the range of integration: 36/2 = 18
const double abscissas[18] = {0.0021695375159141994,
0.011413521097787704,0.027972308950302116,0.051727015600492421,
0.082502225484340941, 0.12007019910960293,0.16415283300752470,
0.21442376986779355, 0.27051082840644336, 0.33199876341447887,
0.39843234186401943, 0.46931971407375483, 0.54413605556657973,
0.62232745288031077, 0.70331500465597174, 0.78649910768313447,
0.87126389619061517, 0.95698180152629142};
const double weights[18] = {0.0055657196642445571,
0.012915947284065419,0.020181515297735382,0.027298621498568734,
0.034213810770299537,0.040875750923643261,0.047235083490265582,
0.053244713977759692,0.058860144245324798,0.064039797355015485,
0.068745323835736408,0.072941885005653087,0.076598410645870640,
0.079687828912071670,0.082187266704339706,0.084078218979661945,
0.085346685739338721,0.085983275670394821};




///////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////
// modified Lentz’s algorithm for continued fractions, 
// reference: Lentz, W.J. 1976, “Generating Bessel Functions in Mie Scattering Calculations Using Continued Fractions,” 
template <typename T> 
static T continFract(const T a, const T b, const T x) {	
    return (T) 0;
}

///////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////
// evaluates incomplete beta integral using Gauss-Legendre quadrature method
template <typename T>
static T gausLegQuad(const T a, const T b, const T x) {
	return (T) 0;
}


///////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////
// evaluates incomplete beta function for positive a and b, and x between 0 and 1.
template <typename T> 
static T betaIncTA(T a, T b, T x) {
	return (T) 0.0f;
}

template<typename T>
NDArray betaIncT(const NDArray& a, const NDArray& b, const NDArray& x) {
	auto result = NDArray(&x, false, x.getContext());

	return result;
}

///////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////
// overload betaInc for arrays, shapes of a, b and x must be the same !!!
NDArray betaInc(const NDArray& a, const NDArray& b, const NDArray& x) {
	auto xType = a.dataType();
	BUILD_SINGLE_SELECTOR(xType, return betaIncT, (a, b, x), FLOAT_TYPES);
	return a;
}


template float   continFract<float>  (const float   a, const float   b, const float   x);
template float16 continFract<float16>(const float16 a, const float16 b, const float16 x);
template bfloat16 continFract<bfloat16>(const bfloat16 a, const bfloat16 b, const bfloat16 x);
template double  continFract<double> (const double  a, const double  b, const double  x);

template float   gausLegQuad<float>  (const float   a, const float   b, const float   x);
template float16 gausLegQuad<float16>(const float16 a, const float16 b, const float16 x);
template bfloat16 gausLegQuad<bfloat16>(const bfloat16 a, const bfloat16 b, const bfloat16 x);
template double  gausLegQuad<double> (const double  a, const double  b, const double  x);

template float   betaIncTA<float>  (const float   a, const float   b, const float   x);
template float16 betaIncTA<float16>(const float16 a, const float16 b, const float16 x);
template bfloat16 betaIncTA<bfloat16>(const bfloat16 a, const bfloat16 b, const bfloat16 x);
template double  betaIncTA<double> (const double  a, const double  b, const double  x);

template NDArray betaIncT<float>  (const NDArray&   a, const NDArray&   b, const NDArray&  x);
template NDArray betaIncT<float16>(const NDArray& a, const NDArray& b, const NDArray& x);
template NDArray betaIncT<bfloat16>(const NDArray& a, const NDArray& b, const NDArray& x);
template NDArray betaIncT<double> (const NDArray&  a, const NDArray&  b, const NDArray& x);


}
}
}

