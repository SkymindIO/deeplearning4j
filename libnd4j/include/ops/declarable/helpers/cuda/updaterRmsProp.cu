#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Oleh Semeniv (oleg.semeniv@gmail.com)
//
#include <helpers/PointersManager.h>
#include <math/platformmath.h>
#include <math/templatemath.h>
#include <ops/declarable/helpers/updatersHelpers.h>
#include <system/op_boilerplate.h>

#include "execution/cuda/LaunchDims.h"
#include "helpers/DebugHelper.h"


namespace sd {
namespace ops {
namespace helpers {

///////////////////////////////////////////////////////////////////
template <typename T>
SD_KERNEL void rmsPropUpdaterCuda(const void *vx, const LongType *xShapeInfo, const void *vin,
                                  const LongType *inShapeInfo, void *vz, const LongType *zShapeInfo, void *vst,
                                  const LongType *stShapeInfo, const T lr, const T rmsDecay, const T epsilon) {
  const auto x = reinterpret_cast<const T *>(vx);
  const auto init = reinterpret_cast<const T *>(vin);

  auto up = reinterpret_cast<T *>(vz);
  auto st = reinterpret_cast<T *>(vst);

  __shared__ LongType xLen;
  __shared__ bool bOrdering, bXZsame, bXInSame, bXStSame;

  if (threadIdx.x == 0) {
    xLen = shape::length(xShapeInfo);

    bOrdering = shape::order(zShapeInfo) == shape::order(xShapeInfo) &&
                shape::order(xShapeInfo) == shape::order(stShapeInfo) &&
                shape::order(xShapeInfo) == shape::order(inShapeInfo);
    bXZsame = shape::haveSameShapeAndStrides(xShapeInfo, zShapeInfo);
    bXInSame = shape::haveSameShapeAndStrides(xShapeInfo, inShapeInfo);
    bXStSame = shape::haveSameShapeAndStrides(xShapeInfo, stShapeInfo);
  }
  __syncthreads();

  LongType coords[SD_MAX_RANK];

  for (LongType i = blockIdx.x * blockDim.x + threadIdx.x; i < xLen; i += gridDim.x * blockDim.x) {
    LongType xOffset, zOffset, initOffset, stOffset;

    INDEX2COORDS(i, shape::rank(xShapeInfo), shape::shapeOf(xShapeInfo), coords);
    COORDS2INDEX(shape::rank(xShapeInfo), shape::stride(xShapeInfo), coords, xOffset);
    if (bXZsame) {
      zOffset = xOffset;
    } else {
      COORDS2INDEX(shape::rank(zShapeInfo), shape::stride(zShapeInfo), coords, zOffset);
    }

    if (bXInSame) {
      initOffset = xOffset;
    } else {
      COORDS2INDEX(shape::rank(inShapeInfo), shape::stride(inShapeInfo), coords, initOffset);
    }

    if (bXStSame) {
      stOffset = xOffset;
    } else {
      COORDS2INDEX(shape::rank(stShapeInfo), shape::stride(stShapeInfo), coords, stOffset);
    }
    st[stOffset] = init[initOffset] * rmsDecay + x[xOffset] * x[xOffset] * (1 - rmsDecay);
    up[zOffset] = (lr * x[xOffset]) / (math::sd_sqrt<T, T>(st[stOffset]) + epsilon);
  }
}

///////////////////////////////////////////////////////////////////
template <typename T>
void rmsPropUpdaterCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMemory,
                                const hipStream_t *stream, const void *vx, const LongType *xShapeInfo,
                                const void *vin, const LongType *inShapeInfo, void *vz,
                                const LongType *zShapeInfo, void *vst, const LongType *stShapeInfo,
                                const double dLr, const double dRmsDecay, const double dEpsilon) {
  const T lr = static_cast<T>(dLr);
  const T rmsDecay = static_cast<T>(dRmsDecay);
  T epsilon = static_cast<T>(dEpsilon);
  //fp16 to prevent underflow
  if(epsilon == 0.0) {
    epsilon = static_cast<T>(1e-7);
  }
  rmsPropUpdaterCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMemory, *stream>>>(
      vx, xShapeInfo, vin, inShapeInfo, vz, zShapeInfo, vst, stShapeInfo, lr, rmsDecay, epsilon);
  sd::DebugHelper::checkErrorCode(const_cast<hipStream_t *>(stream), "rmsPropUpdaterCudaLauncher failed");

}

///////////////////////////////////////////////////////////////////
void updaterRmsProp(LaunchContext *context, NDArray&gradient, NDArray&initState, NDArray &update,
                    NDArray &stateG, const double dLr, const double dRmsDecay, const double dEpsilon) {
  PointersManager manager(context, "rmsPropUpdater");

  dim3 launchDims = updaterDims(gradient.lengthOf());
  NDArray::prepareSpecialUse({&update, &stateG}, {&gradient, &initState});

  BUILD_SINGLE_SELECTOR(
      gradient.dataType(), rmsPropUpdaterCudaLauncher,
      (launchDims.y, launchDims.x,launchDims.z, context->getCudaStream(), gradient.specialBuffer(), gradient.specialShapeInfo(),
       initState.specialBuffer(), initState.specialShapeInfo(), update.specialBuffer(), update.specialShapeInfo(),
       stateG.specialBuffer(), stateG.specialShapeInfo(), dLr, dRmsDecay, dEpsilon),
      SD_FLOAT_TYPES);

  NDArray::registerSpecialUse({&update, &stateG}, {&gradient, &initState});

  manager.synchronize();
}

}  // namespace helpers
}  // namespace ops
}  // namespace sd
