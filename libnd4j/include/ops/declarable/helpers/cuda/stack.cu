/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// Created by Yurii Shyrma on 02.01.2018
//

#include <ops/declarable/helpers/stack.h>
#include <helpers/ShapeUtils.h>
#include <array/ResultSet.h>


namespace nd4j {
namespace ops {
namespace helpers {


	///////////////////////////////////////////////////////////////////
	template <typename T>
	static void stack_(const std::vector<NDArray*>& inArrs, NDArray& outArr, const int dim) {

	}

	void stack(const std::vector<NDArray*>& inArrs, NDArray& outArr, const int dim) {
		BUILD_SINGLE_SELECTOR(outArr.dataType(), stack_, (inArrs, outArr, dim), LIBND4J_TYPES);
	}

	BUILD_SINGLE_TEMPLATE(template void stack_ , (const std::vector<NDArray*>& inArrs, NDArray& outArr, const int dim), LIBND4J_TYPES);

}
}
}

