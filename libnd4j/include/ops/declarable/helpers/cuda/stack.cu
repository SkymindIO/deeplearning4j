#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// Created by Yurii Shyrma on 02.01.2018
//
#include <array/ResultSet.h>
#include <exceptions/cuda_exception.h>
#include <helpers/ConstantTadHelper.h>
#include <helpers/PointersManager.h>
#include <helpers/ShapeUtils.h>
#include <helpers/TAD.h>
#include <ops/declarable/helpers/stack.h>

namespace sd {
namespace ops {
namespace helpers {

///////////////////////////////////////////////////////////////////
template <typename T>
static SD_KERNEL void stackScalarsCuda(void* pVx, void* vz, const sd::LongType* zShapeInfo) {
  T* z = reinterpret_cast<T*>(vz);

  __shared__ sd::LongType zLen, totalThreads;

  if (threadIdx.x == 0) {
    zLen = shape::length(zShapeInfo);
    totalThreads = gridDim.x * blockDim.x;
  }
  __syncthreads();

  const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

  for (sd::LongType i = tid; i < zLen; i += totalThreads) {
    const T* x = reinterpret_cast<const T*>(reinterpret_cast<void**>(pVx)[i]);
    z[shape::getIndexOffset(i, zShapeInfo)] = *x;
  }
}

///////////////////////////////////////////////////////////////////
template <typename T>
SD_HOST static void stackScalarsCudaLauncher(const int blocksPerGrid, const int threadsPerBlock,
                                             const hipStream_t* stream, void* pVx, void* vz,
                                             const sd::LongType* zShapeInfo) {
  stackScalarsCuda<T><<<blocksPerGrid, threadsPerBlock, 256, *stream>>>(pVx, vz, zShapeInfo);
}

///////////////////////////////////////////////////////////////////
template <typename T>
static void stack_(sd::LaunchContext* context, const std::vector<const NDArray*>& inArrs, NDArray& output,
                   const int dim) {
  const int numOfSubArrs = inArrs.size();

  NDArray::prepareSpecialUse({&output}, inArrs);

  if (inArrs[0]->rankOf() == 0) {
    std::vector<void const*> hInBuffers(numOfSubArrs);

    for (int i = 0; i < numOfSubArrs; ++i) hInBuffers[i] = inArrs[i]->specialBuffer();

    PointersManager manager(context, "helpers::stack cuda");

    void* dInBuffers = manager.replicatePointer(hInBuffers.data(), hInBuffers.size() * sizeof(void*));

    const int threadsPerBlock = SD_MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (output.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;

    stackScalarsCudaLauncher<T>(blocksPerGrid, threadsPerBlock, context->getCudaStream(), dInBuffers,
                                output.specialBuffer(), output.specialShapeInfo());

    manager.synchronize();
  } else {
    auto zTadPack = ConstantTadHelper::getInstance().tadForDimensions(
        output.shapeInfo(), ShapeUtils::evalDimsToExclude(output.rankOf(), {dim}));
    auto zTadShapeInfo = zTadPack->primaryShapeInfo();

    for (sd::LongType i = 0; i < numOfSubArrs; ++i) {
      void* zBuff = output.specialBufferWithOffset(zTadPack->primaryOffsets()[i]);

      NativeOpExecutioner::execTransformAny(context, transform::Assign, nullptr, inArrs[i]->shapeInfo(),
                                            inArrs[i]->specialBuffer(), inArrs[i]->specialShapeInfo(), nullptr,
                                            zTadShapeInfo, zBuff, zTadPack->specialShapeInfo(), nullptr, nullptr,
                                            nullptr, false /*allowParallelism*/);
    }
  }

  NDArray::registerSpecialUse({&output}, inArrs);
}

////////////////////////////////////////////////////////////////////////
void stack(sd::LaunchContext* context, const std::vector<const NDArray*>& inArrs, NDArray& output, const int dim) {
  BUILD_SINGLE_SELECTOR(output.dataType(), stack_, (context, inArrs, output, dim), SD_COMMON_TYPES);
}
BUILD_SINGLE_TEMPLATE(template void stack_,
                      (sd::LaunchContext * context, const std::vector<const NDArray*>& inArrs, NDArray& output,
                       const int dim),
                      SD_COMMON_TYPES);

///////////////////////////////////////////////////////////////////
template <typename T>
static SD_KERNEL void unstackScalarsCuda(const void* vx, const sd::LongType* xShapeInfo, void* pVz) {
  const T* x = reinterpret_cast<const T*>(vx);

  __shared__ sd::LongType xLen, totalThreads;

  if (threadIdx.x == 0) {
    xLen = shape::length(xShapeInfo);
    totalThreads = gridDim.x * blockDim.x;
  }
  __syncthreads();

  const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

  for (sd::LongType i = tid; i < xLen; i += totalThreads) {
    T* z = reinterpret_cast<T*>(reinterpret_cast<void**>(pVz)[i]);
    *z = x[shape::getIndexOffset(i, xShapeInfo)];
  }
}

///////////////////////////////////////////////////////////////////
template <typename T>
SD_HOST static void unstackScalarsCudaLauncher(const int blocksPerGrid, const int threadsPerBlock,
                                               const hipStream_t* stream, const void* vx,
                                               const sd::LongType* xShapeInfo, void* pVz) {
  unstackScalarsCuda<T><<<blocksPerGrid, threadsPerBlock, 256, *stream>>>(vx, xShapeInfo, pVz);
}

///////////////////////////////////////////////////////////////////
template <typename T>
static void unstack_(sd::LaunchContext* context, const NDArray& input, const std::vector<NDArray*>& outArrs,
                     const int dim) {
  const int numOfSubArrs = outArrs.size();

  // NDArray::prepareSpecialUse(outArrs, {&input});
  input.syncToDevice();
  for (const auto a : outArrs) a->getDataBuffer()->allocateSpecial();

  if (outArrs[0]->rankOf() == 0) {
    std::vector<void*> hOutBuffers(numOfSubArrs);

    for (int i = 0; i < numOfSubArrs; ++i) hOutBuffers[i] = outArrs[i]->specialBuffer();

    PointersManager manager(context, "helpers::unstack cuda");

    void* dOutBuffers = manager.replicatePointer(hOutBuffers.data(), hOutBuffers.size() * sizeof(void*));

    const int threadsPerBlock = SD_MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (input.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;

    unstackScalarsCudaLauncher<T>(blocksPerGrid, threadsPerBlock, context->getCudaStream(), input.specialBuffer(),
                                  input.specialShapeInfo(), dOutBuffers);

    manager.synchronize();
  } else {
    auto xTadPack = ConstantTadHelper::getInstance().tadForDimensions(
        input.shapeInfo(), ShapeUtils::evalDimsToExclude(input.rankOf(), {dim}));
    auto xTadShapeInfo = xTadPack->primaryShapeInfo();

    for (sd::LongType i = 0; i < numOfSubArrs; ++i) {
      auto xBuff = input.specialBufferWithOffset(xTadPack->primaryOffsets()[i]);

      NativeOpExecutioner::execTransformAny(input.getContext(), transform::Assign, nullptr, xTadShapeInfo, xBuff,
                                            xTadPack->specialShapeInfo(), nullptr, outArrs[i]->shapeInfo(),
                                            outArrs[i]->specialBuffer(), outArrs[i]->specialShapeInfo(), nullptr,
                                            nullptr, nullptr, false /*allowParallelism*/);
    }
  }

  // NDArray::registerSpecialUse(outArrs, {&input});
  input.tickReadDevice();
  for (const auto p : outArrs) p->tickWriteDevice();
}

////////////////////////////////////////////////////////////////////////
void unstack(sd::LaunchContext* context, const NDArray& input, const std::vector<NDArray*>& outArrs, const int dim) {
  BUILD_SINGLE_SELECTOR(input.dataType(), unstack_, (context, input, outArrs, dim), SD_COMMON_TYPES);
}
BUILD_SINGLE_TEMPLATE(template void unstack_,
                      (sd::LaunchContext * context, const NDArray& input, const std::vector<NDArray*>& outArrs,
                       const int dim),
                      SD_COMMON_TYPES);



}  // namespace helpers
}  // namespace ops
}  // namespace sd
