/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//

#include <ops/declarable/helpers/dilation2d.h>
#include <array/DataTypeUtils.h>

namespace nd4j {
namespace ops {
namespace helpers {
    template <typename X, typename Y>
    static void dilation2d_(NDArray *input, NDArray *weights, NDArray *output, int stride_rows, int stride_cols, int rate_rows, int rate_cols, int pad_top, int pad_left) {

    };

    void dilation2d(NDArray *input, NDArray *weights, NDArray *output, int stride_rows, int stride_cols, int rate_rows, int rate_cols, int pad_top, int pad_left) {        
        BUILD_DOUBLE_SELECTOR(input->dataType(), output->dataType(), dilation2d_, (input, weights, output, stride_rows, stride_cols, rate_rows, rate_cols, pad_top, pad_left), LIBND4J_TYPES, FLOAT_TYPES);
    }

    BUILD_DOUBLE_TEMPLATE(template void dilation2d_, (NDArray *input, NDArray *weights, NDArray *output, int stride_rows, int stride_cols, int rate_rows, int rate_cols, int pad_top, int pad_left), LIBND4J_TYPES, FLOAT_TYPES);

}
}
}