/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author sgazeos@gmail.com
//

#include <ops/declarable/helpers/unique.h>
#include <Status.h>

namespace nd4j {
namespace ops {
namespace helpers {

    template <typename T>
    static Nd4jLong uniqueCount_(NDArray* input) {
        Nd4jLong count = 0;
        return count;
    }

    Nd4jLong uniqueCount(NDArray* input) {
        BUILD_SINGLE_SELECTOR(input->dataType(), return uniqueCount_, (input), LIBND4J_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template Nd4jLong uniqueCount_, (NDArray* input), LIBND4J_TYPES);


    template <typename T>
    static Nd4jStatus uniqueFunctor_(NDArray* input, NDArray* values, NDArray* indices, NDArray* counts) {
        return Status::OK();
    }

    Nd4jStatus uniqueFunctor(NDArray* input, NDArray* values, NDArray* indices, NDArray* counts) {
        BUILD_SINGLE_SELECTOR(input->dataType(), return uniqueFunctor_,(input, values, indices, counts), LIBND4J_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template Nd4jStatus uniqueFunctor_, (NDArray* input, NDArray* values, NDArray* indices, NDArray* counts), LIBND4J_TYPES);

}
}
}