/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//

#include <op_boilerplate.h>
#include <types/float16.h>
#include <ops/declarable/helpers/batched_gemm.h>
#include <helpers/BlasHelper.h>


namespace nd4j {
    namespace ops {
        namespace helpers {
        

            template <typename T>
            void __bgemm(std::vector<NDArray*>& vA, std::vector<NDArray*>& vB, std::vector<NDArray*>& vC, NDArray* alphas, NDArray* betas, int transA, int transB, int M, int N, int K, int ldA, int ldB, int ldC) {

            };


            void _bgemm(std::vector<NDArray*>& vA, std::vector<NDArray*>& vB, std::vector<NDArray*>& vC, NDArray* alphas, NDArray* betas, int transA, int transB, int M, int N, int K, int ldA, int ldB, int ldC) {
                auto xType = vA.at(0)->dataType();

                BUILD_SINGLE_SELECTOR(xType, __bgemm, (vA, vB, vC, alphas, betas, transA, transB, M, N, K, ldA, ldB, ldC), FLOAT_TYPES);
            }

            BUILD_SINGLE_TEMPLATE(template void __bgemm, (std::vector<NDArray*>& vA, std::vector<NDArray*>& vB, std::vector<NDArray*>& vC, NDArray* alphas, NDArray* betas, int transA, int transB, int M, int N, int K, int ldA, int ldB, int ldC), FLOAT_TYPES);
        }
    }
}