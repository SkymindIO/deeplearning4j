/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author sgazeos@gmail.com
//
#ifndef __MIN_I_MAX_H_HELPERS__
#define __MIN_I_MAX_H_HELPERS__
#include <op_boilerplate.h>
#include <NDArray.h>
#include <helpers/ShapeUtils.h>

namespace nd4j {
namespace ops {
namespace helpers {

    template <typename T> 
    static void minimumBPFunctor_(NDArray* x, NDArray* y, NDArray* epsNext, NDArray* gradX, NDArray* gradY) {

    }

    template <typename T>
    void maximumBPFunctor_(NDArray* x, NDArray* y, NDArray* epsNext, NDArray* gradX, NDArray* gradY) {

    }

    void minimumBPFunctor(NDArray* x, NDArray* y, NDArray* epsNext, NDArray* gradX, NDArray* gradY) {
        BUILD_SINGLE_SELECTOR(x->dataType(), minimumBPFunctor_, (x, y, epsNext, gradX, gradY), NUMERIC_TYPES);
    }

    void maximumBPFunctor(NDArray* x, NDArray* y, NDArray* epsNext, NDArray* gradX, NDArray* gradY) {
        BUILD_SINGLE_SELECTOR(x->dataType(), maximumBPFunctor_, (x, y, epsNext, gradX, gradY), NUMERIC_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template void minimumBPFunctor_, (NDArray* x, NDArray* y, NDArray* epsNext, NDArray* gradX, NDArray* gradY), NUMERIC_TYPES);
    BUILD_SINGLE_TEMPLATE(template void maximumBPFunctor_, (NDArray* x, NDArray* y, NDArray* epsNext, NDArray* gradX, NDArray* gradY), NUMERIC_TYPES);

}
}
}
#endif
