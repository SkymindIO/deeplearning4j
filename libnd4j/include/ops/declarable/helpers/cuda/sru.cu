/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// implementation of operations for Simple Recurrent Unit: arXiv:1709.02755v2 [cs.CL] 12 Sep 2017
//
//  @author Yurii Shyrma, created on 05.12.2017
//

#include<ops/declarable/helpers/sru.h>
#include <NDArrayFactory.h>

namespace nd4j    {
namespace ops     {
namespace helpers {

    //////////////////////////////////////////////////////////////////////////
    static FORCEINLINE NDArray activation(const NDArray& arr) {
        // return (const_cast<NDArray<T>&>(arr)).template transform<simdOps::Tanh<T>>();
        auto result = NDArray(&arr, false, arr.getContext());
        (const_cast<NDArray&>(arr)).applyTransform(transform::Tanh, &result);
        return result;
    }


    //////////////////////////////////////////////////////////////////////////
    static FORCEINLINE NDArray sigmoid(const NDArray& arr) {
        return (const_cast<NDArray&>(arr)).transform(transform::Sigmoid);
    }


    //////////////////////////////////////////////////////////////////////////
    void sruCell(const NDArray* x, const NDArray* c0, const NDArray* w, const NDArray* b, NDArray* h, NDArray* c) {

    }

    //////////////////////////////////////////////////////////////////////////
    void sruTimeLoop(const NDArray* x, const NDArray* c0, const NDArray* w, const NDArray* b, NDArray* h, NDArray* c) {

    }

    //////////////////////////////////////////////////////////////////////////
    template <typename T>
    static void sruBI_(NDArray* x, const NDArray* w, const NDArray* b, const NDArray* c0, const NDArray* mask, NDArray* ht, NDArray* ct) {

    }

    //////////////////////////////////////////////////////////////////////////
    template <typename T>
    static void sruBIBP_(NDArray* x, const NDArray* w, const NDArray* b, const NDArray* c0, const NDArray* ct, const NDArray* inGradC0, const NDArray* inGradHt, const NDArray* mask,
                     NDArray* gradI, NDArray* gradW, NDArray* gradB, NDArray* gradC0) {
    }


    void sruBI(NDArray* x, const NDArray* w, const NDArray* b, const NDArray* c0, const NDArray* mask, NDArray* ht, NDArray* ct) {
        BUILD_SINGLE_SELECTOR(x->dataType(), sruBI_, (x, w, b, c0, mask, ht, ct), FLOAT_TYPES);
    }

    void sruBIBP(NDArray* x, const NDArray* w, const NDArray* b, const NDArray* c0, const NDArray* ct, const NDArray* inGradC0, const NDArray* inGradH, const NDArray* mask, NDArray* gradI, NDArray* gradW, NDArray* gradB, NDArray* gradC0) {
        BUILD_SINGLE_SELECTOR(x->dataType(), sruBIBP_, (x, w, b, c0, ct, inGradC0, inGradH, mask, gradI, gradW, gradB, gradC0), FLOAT_TYPES);
    }


    BUILD_SINGLE_TEMPLATE(template void sruBI_,   (NDArray* x, const NDArray* w, const NDArray* b, const NDArray* c0, const NDArray* mask, NDArray* ht, NDArray* ct), FLOAT_TYPES);
    BUILD_SINGLE_TEMPLATE(template void sruBIBP_, (NDArray* x, const NDArray* w, const NDArray* b, const NDArray* c0, const NDArray* ct, const NDArray* inGradC0, const NDArray* inGradH, const NDArray* mask, NDArray* gradI, NDArray* gradW, NDArray* gradB, NDArray* gradC0), FLOAT_TYPES);

}
}
}