#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

#include <hip/hip_runtime.h>
#include <exceptions/cuda_exception.h>
#include <exceptions/datatype_exception.h>
#include <execution/AffinityManager.h>

#include <helpers/BlasHelper.h>
#include <helpers/CudaLaunchHelper.h>
#include <helpers/DebugHelper.h>
#include <helpers/PointersManager.h>
#include <helpers/threshold.h>
#include <legacy/NativeOpExecutioner.h>
#include <legacy/NativeOps.h>
#include <loops/reduce_bool.h>
#include <loops/reduce_long.h>
#include <loops/scalar.h>
#include <loops/transform_any.h>
#include <ops/declarable/CustomOperations.h>
#include <ops/specials_cuda.h>
#include <system/buffer.h>


#include <hiprand.h>
#include <helpers/DebugHelper.h>

using namespace sd;
#include <execution/cuda/LaunchDims.h>
#include <loops/special_kernels.h>

#include "../../array/ShapeList.h"
#include "../../ops/declarable/DeclarableOp.h"
#include "../../system/common.h"
#include "../NativeOps.h"
hipDeviceProp_t *deviceProperties;
hipFuncAttributes *funcAttributes = new hipFuncAttributes[64];
int blockLimit = 128;
int maxThreads = 512;
bool allowedP2P = false;
bool supportedP2P = false;



//note we only include this if we're running gcc linux
//and should not be enabled in default builds.
#if defined(SD_GCC_FUNCTRACE)
#include <cxxabi.h>  // needed  __cxa_demangle
#include <dlfcn.h>   // needed for dladdr

#include "exceptions/backward.hpp"
#include "execution/cuda/LaunchDims.h"


//note this is outside extern C. This is fine.


#endif





int minThreads = 32;

__constant__ char deviceConstantMemory[49152];



// this method just does type conversion in fancy way
int getDeviceId(Pointer ptrToDeviceId) { return (int)(LongType)ptrToDeviceId; }
// Function to execute a custom operation with context
sd::Status execCustomOp2(sd::Pointer *extraPointers, sd::LongType  hash, Context *opContext) {
  try {
    // Retrieve the operation based on the hash
    auto op = sd::ops::OpRegistrator::getInstance().getOperation(hash);
    if (op == nullptr) {
      throw std::invalid_argument("Operation not found for the given hash.");
    }

    // Execute the custom operation with the provided context
    auto result = op->execute(opContext);

    // Synchronize the CUDA stream to ensure operation completion
    auto res = hipStreamSynchronize(*opContext->launchContext()->getCudaStream());
    if (res != hipSuccess) {
      std::string errorMessage;
      errorMessage += "CUDA stream synchronization failed with error code: ";
      errorMessage += std::to_string(res);
      THROW_EXCEPTION(errorMessage.c_str());
    }

    // Synchronize fastpath inputs
    for (auto v : opContext->fastpath_in()) {
      if (!v->isEmpty()) v->syncToDevice();
    }

    // Synchronize fastpath outputs
    for (auto v : opContext->fastpath_out()) {
      if (!v->isEmpty()) v->syncToDevice();
    }

    return result;
  }
  catch (std::exception &e) {
    // Handle exceptions by setting error codes and messages
    sd::LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    sd::LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    return sd::Status::KERNEL_FAILURE;
  }
}
/*
 * Basic CUDA constants here: number of blocks per MP
 */
int getDeviceBlockThreshold(int deviceId) {
  int ccMinor = deviceProperties[deviceId].minor;
  int ccMajor = deviceProperties[deviceId].major;

  int blockThreshold = 8;

  if (ccMajor >= 5)
    blockThreshold = 32;
  else if (ccMajor == 3)
    blockThreshold = 16;
  else if (ccMajor < 3)
    blockThreshold = 8;

  return blockThreshold;
}

/*
 * This message returns shared memory threshold value. default overflow ratio is 0.3
 */
int getDeviceSharedThreshold(int deviceId) {
  int ccMinor = deviceProperties[deviceId].minor;
  int ccMajor = deviceProperties[deviceId].major;

  // please note threshold isn't multiple of 32, and that's NOT a mistake

  int shmemThreshold;
  if (ccMajor == 6 && ccMinor == 0)
    shmemThreshold = 65536;
  else if (ccMajor == 6 && ccMinor == 1)
    shmemThreshold = 49152;
  else if (ccMajor == 5 && ccMinor == 2)
    shmemThreshold = 98304;
  else if (ccMajor == 5)
    shmemThreshold = 65536;
  else if (ccMajor == 3 && ccMinor == 7)
    shmemThreshold = 114688;
  else
    shmemThreshold = 49152;

  return shmemThreshold / 0.3;
}

buffer::Buffer<LongType> *createScalarBuffer(hipStream_t stream) {
  auto scalarShapeInfo = shape::createScalarShapeInfo();
  auto buff = buffer::createBuffer(scalarShapeInfo, shape::shapeInfoLength(2), stream);
  copyDataToGpu(&buff, stream);
  return buff;
}

class ScalarShapeInformation {
 private:
  buffer::Buffer<LongType> *scalarDimension;
  buffer::Buffer<LongType> *scalarShapeInfo;

 public:
  ScalarShapeInformation(hipStream_t stream) {
    auto scalarDimensionBuff = reinterpret_cast<LongType *>(malloc(sizeof(LongType)));

    CHECK_ALLOC(scalarDimensionBuff, "Failed to allocate ShapeInfoBuffer", sizeof(sd::LongType));

    scalarDimensionBuff[0] = SD_MAX_DIMENSION;
    scalarDimension = buffer::createBuffer(scalarDimensionBuff, 1, stream);
    scalarShapeInfo = createScalarBuffer(stream);
  }
  ~ScalarShapeInformation() {
    freeBuffer(&scalarShapeInfo);
    freeBuffer(&scalarDimension);
  }

  LongType *getShapeInfoHostPointer() { return scalarShapeInfo->data; }

  LongType *getShapeInfoGpuPointer() { return scalarShapeInfo->gData; }

  LongType *getDimensionHostPointer() { return scalarDimension->data; }

  LongType *getDimensionGpuPointer() { return scalarDimension->gData; }
};

template <typename T>
SD_KERNEL  void _printBuffers(void* buffer, LongType bufferLength) {
  T * inputBuffer = reinterpret_cast<T *>(buffer);
  const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
  if(tid == 0) {
    printf("DEVICE buffer: ");
  }
  const auto step = gridDim.x * blockDim.x;
  for (int t = tid; t < bufferLength; t += step) {
    if(t == 0) {
      printf("DEVICE buffer: ");
    }
    printf(" %f ",(double) inputBuffer[t]);
    if(t == bufferLength - 1) {
      printf("\n");
    }
  }



}


template <typename T>
void _printHostBuffer(OpaqueDataBuffer *buffer, sd::LongType offset) {
  auto xType = buffer->dataBuffer()->getDataType();
  LongType len = buffer->dataBuffer()->getNumElements();
  auto buff = buffer->dataBuffer()->template primaryAsT<T>();
  sd_printf("Data type %s: ", DataTypeUtils::asString(xType).c_str());
  sd_printf("Host buffer: ",0);
  for(int i = offset; i < len; i++) {
    sd_printf("%f ",(double) buff[i]);
  }

  sd_printf("\n",0);
}

void printDeviceBuffer(OpaqueDataBuffer *buffer, sd::LongType offset) {
  if(buffer->special() != nullptr) {
    sd_printf("Device pointer address: %d\n", buffer->special());
  } else {
    sd_printf("Device pointer address: none\n",0);
  }

  if(buffer->primary() != nullptr) {
    sd_printf("Host pointer address: %d\n", buffer->primary());
  } else  {
    sd_printf("Host pointer address: none\n",0);
  }

  auto xType = buffer->dataBuffer()->getDataType();
  BUILD_SINGLE_SELECTOR(xType, _printHostBuffer,(buffer,offset),SD_COMMON_TYPES_ALL);


}

template <typename T>
void _printDeviceBuffer(InteropDataBuffer *buffer) {
  auto xType = buffer->dataBuffer()->getDataType();
  LongType len = buffer->dataBuffer()->getNumElements();
  _printBuffers<T><<<256, 512, 1024>>>(buffer->special(),len);
  hipDeviceSynchronize();
  DebugHelper::checkGlobalErrorCode("print device buffer(...) failed");


}

void printDeviceBuffer(InteropDataBuffer *buffer) {
  auto xType = buffer->dataBuffer()->getDataType();
  sd_printf("Data type %s: ", DataTypeUtils::asString(xType).c_str());

  if(buffer->special() != nullptr) {
    sd_printf("Device pointer address: %d\n", reinterpret_cast<sd::LongType>(buffer->special()));
  } else {
    sd_printf("Device pointer address: none\n",0);
  }
  BUILD_SINGLE_SELECTOR(xType, _printDeviceBuffer,(buffer),SD_COMMON_TYPES_ALL);


  if(buffer->primary() != nullptr) {
    sd_printf("Host pointer address: %d\n",  reinterpret_cast<sd::LongType>(buffer->primary()));
  } else  {
    sd_printf("Host pointer address: none\n",0);
  }


}



void execPairwiseTransform(Pointer *extraPointers, int opNum, OpaqueNDArray x, OpaqueNDArray y, OpaqueNDArray z, void *extraParams) {
  try {
    x->prepareSpecialUse({z}, {x, y});

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execPairwiseTransform(
        &lc, opNum,
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
        x->shapeInfo(),
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
        x->specialShapeInfo(),
        shape::isEmptyConst(y->shapeInfo()) ? nullptr : y->buffer(),
        y->shapeInfo(),
        shape::isEmptyConst(y->shapeInfo()) ? nullptr : y->specialBuffer(),
        y->specialShapeInfo(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
        z->shapeInfo(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
        z->specialShapeInfo(),
        extraParams);

    x->registerSpecialUse({z}, {x, y});
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}

////////////////////////////////////////////////////////////////////////
void execPairwiseTransformBool(Pointer *extraPointers, int opNum, OpaqueNDArray x, OpaqueNDArray y, void *extraParams, OpaqueNDArray z) {
  try {
    x->prepareSpecialUse({z}, {x, y});

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execPairwiseBoolTransform(
        &lc, opNum,
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
        x->shapeInfo(),
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
        x->specialShapeInfo(),
        shape::isEmptyConst(y->shapeInfo()) ? nullptr : y->buffer(),
        y->shapeInfo(),
        shape::isEmptyConst(y->shapeInfo()) ? nullptr : y->specialBuffer(),
        y->specialShapeInfo(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
        z->shapeInfo(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
        z->specialShapeInfo(),
        extraParams);

    x->registerSpecialUse({z}, {x, y});
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}


////////////////////////////////////////////////////////////////////////
void execSummaryStatsScalar(Pointer *extraPointers, int opNum, OpaqueNDArray x, void *extraParams, OpaqueNDArray z, bool biasCorrected) {
  try {
    x->prepareSpecialUse({z}, {x});

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execSummaryStatsScalar(
        &lc, opNum,
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
        x->shapeInfo(),
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
        x->specialShapeInfo(),
        extraParams,
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
        z->shapeInfo(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
        z->specialShapeInfo(),
        biasCorrected);

    x->registerSpecialUse({z}, {x});
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}
////////////////////////////////////////////////////////////////////////
void execBroadcastBool(Pointer *extraPointers, int opNum, OpaqueNDArray x, OpaqueNDArray y, OpaqueNDArray z, void *extraParams, OpaqueNDArray dimension) {
  try {
    x->prepareSpecialUse({z}, {x, y, dimension});

    auto dimensionBuffer = dimension != nullptr ? reinterpret_cast<LongType *>(dimension->buffer()) : nullptr;
    LongType dimensionLength = static_cast<LongType>(shape::length(dimension->shapeInfo()));

    auto hTADShapeInfo = reinterpret_cast<LongType *>(extraPointers[9]);
    auto tadOnlyShapeInfo = reinterpret_cast<LongType *>(extraPointers[10]);
    auto tadOffsets = reinterpret_cast<LongType *>(extraPointers[11]);
    auto tadOnlyShapeInfoZ = reinterpret_cast<LongType *>(extraPointers[12]);
    auto tadOffsetsZ = reinterpret_cast<LongType *>(extraPointers[13]);

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execBroadcastBool(
        &lc, opNum,
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
        x->shapeInfo(),
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
        x->specialShapeInfo(),
        shape::isEmptyConst(y->shapeInfo()) ? nullptr : y->buffer(),
        y->shapeInfo(),
        shape::isEmptyConst(y->shapeInfo()) ? nullptr : y->specialBuffer(),
        y->specialShapeInfo(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
        z->shapeInfo(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
        z->specialShapeInfo(),
        extraParams,
        dimensionBuffer,
        dimensionLength,
        tadOnlyShapeInfo,
        tadOffsets,
        tadOnlyShapeInfoZ,
        tadOffsetsZ);

    x->registerSpecialUse({z}, {x, y, dimension});
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}

/**
 *
 * @param opNum
 * @param dX
 * @param dXShapeInfo
 * @param dY
 * @param dYShapeInfo
 * @param dZ
 * @param dZShapeInfo
 * @param dimension
 * @param dimensionLength
 */
void execBroadcast(Pointer *extraPointers, int opNum, OpaqueNDArray x, OpaqueNDArray y, OpaqueNDArray z, void *extraParams, OpaqueNDArray dimension) {
  try {
    x->prepareSpecialUse({z}, {x, y, dimension});

    auto dimensionBuffer = dimension != nullptr ? reinterpret_cast<LongType *>(dimension->buffer()) : nullptr;
    LongType dimensionLength = static_cast<LongType>(shape::length(dimension->shapeInfo()));

    auto hTADShapeInfo = reinterpret_cast<LongType *>(extraPointers[9]);
    auto tadOnlyShapeInfo = reinterpret_cast<LongType *>(extraPointers[10]);
    auto tadOffsets = reinterpret_cast<LongType *>(extraPointers[11]);
    auto tadOnlyShapeInfoZ = reinterpret_cast<LongType *>(extraPointers[12]);
    auto tadOffsetsZ = reinterpret_cast<LongType *>(extraPointers[13]);

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execBroadcast(
        &lc, opNum,
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
        x->shapeInfo(),
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
        x->specialShapeInfo(),
        shape::isEmptyConst(y->shapeInfo()) ? nullptr : y->buffer(),
        y->shapeInfo(),
        shape::isEmptyConst(y->shapeInfo()) ? nullptr : y->specialBuffer(),
        y->specialShapeInfo(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
        z->shapeInfo(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
        z->specialShapeInfo(),
        dimensionBuffer,
        dimensionLength,
        tadOnlyShapeInfo,
        tadOffsets,
        tadOnlyShapeInfoZ,
        tadOffsetsZ);

    x->registerSpecialUse({z}, {x, y, dimension});
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}

/**
 *
 * @param opNum
 * @param dX
 * @param dXShapeInfo
 * @param extraParams
 * @param dZ
 * @param dZShapeInfo
 */
////////////////////////////////////////////////////////////////////////
void execReduceFloat(Pointer *extraPointers, int opNum, OpaqueNDArray x, void *extraParams, OpaqueNDArray z) {
  try {
    x->prepareSpecialUse({z}, {x});

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execReduceFloatScalar(
        &lc, opNum,
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
        x->shapeInfo(),
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
        x->specialShapeInfo(),
        extraParams,
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
        z->shapeInfo(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
        z->specialShapeInfo());

    x->registerSpecialUse({z}, {x});
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}

////////////////////////////////////////////////////////////////////////
void execReduceSame(Pointer *extraPointers,
                    int opNum,
                    OpaqueNDArray x,
                    void *extraParams,
                    OpaqueNDArray z) {
  try {


    x->prepareSpecialUse({z}, {x});
    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execReduceSameScalar(
        &lc, opNum,
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
        x->shapeInfo(),
        shape::isEmptyConst(x->shapeInfo()) ? nullptr: x->specialBuffer(),
        x->specialShapeInfo(), extraParams,
        z->buffer(),
        z->shapeInfo(),
        shape::isEmptyConst(z->shapeInfo())  ? nullptr : z->specialBuffer(),
        z->specialShapeInfo());

   x->registerSpecialUse({z}, {x});
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}

////////////////////////////////////////////////////////////////////////
void execReduceSame2(Pointer *extraPointers, int opNum, OpaqueNDArray x, void *extraParams, OpaqueNDArray z, OpaqueNDArray dimension) {
  try {
    x->prepareSpecialUse({z}, {x});
    dimension->preparePrimaryUse({}, {dimension});

    auto dimensionData = dimension != nullptr ? reinterpret_cast<LongType *>(dimension->buffer()) : nullptr;
    LongType dimensionLength = static_cast<LongType>(shape::length(dimension->shapeInfo()));

    const auto zLen = shape::length(z->shapeInfo());

    std::vector<LongType> dimensions(dimensionData, dimensionData + dimensionLength);

    const LongType *zShapeInfoH = z->shapeInfo();

    if (shape::rank(x->shapeInfo()) - dimensionLength != shape::rank(z->shapeInfo()) && zLen != 1) {
      auto zPack = ConstantShapeHelper::getInstance().createShapeInfoWithNoUnitiesForReduce(z->shapeInfo(), &dimensions);
      zShapeInfoH = reinterpret_cast<LongType const *>(zPack->primary());
    }

    std::vector<LongType> *dims =
        (zLen != 1) ? ShapeUtils::evalDimsForReduceOp(shape::rank(x->shapeInfo()), &dimensions) : new std::vector<LongType>();
    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execReduceSame(&lc,
                                        opNum,
                                        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
                                        x->shapeInfo(),
                                        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
                                        x->specialShapeInfo(), extraParams,
                                        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
                                        zShapeInfoH,
                                        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
                                        z->specialShapeInfo(),
                                        dims->data(), dims->size());

    x->registerSpecialUse({z}, {x});

    delete dims;
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}

////////////////////////////////////////////////////////////////////////
void execReduceLong2(Pointer *extraPointers, int opNum, OpaqueNDArray x, void *extraParams, OpaqueNDArray z, OpaqueNDArray dimension) {
  try {
    x->prepareSpecialUse({z}, {x});
    dimension->preparePrimaryUse({}, {dimension});

    auto dimensionData = dimension != nullptr ? reinterpret_cast<LongType *>(dimension->buffer()) : nullptr;
    LongType dimensionLength = static_cast<LongType>(shape::length(dimension->shapeInfo()));

    const auto zLen = shape::length(z->shapeInfo());

    std::vector<LongType> dimensions(dimensionData, dimensionData + dimensionLength);

    const LongType *zShapeInfoH = z->shapeInfo();

    if (shape::rank(x->shapeInfo()) - dimensionLength != shape::rank(z->shapeInfo()) && zLen != 1) {
      auto zPack = ConstantShapeHelper::getInstance().createShapeInfoWithNoUnitiesForReduce(z->shapeInfo(), &dimensions);
      zShapeInfoH = reinterpret_cast<LongType const *>(zPack->primary());
    }

    std::vector<LongType> *dims =
        (zLen != 1) ? ShapeUtils::evalDimsForReduceOp(shape::rank(x->shapeInfo()), &dimensions) : new std::vector<LongType>();

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execReduceLong(&lc, opNum,
                                        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
                                        x->shapeInfo(),
                                        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
                                        x->specialShapeInfo(), extraParams,
                                        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
                                        zShapeInfoH,
                                        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
                                        z->specialShapeInfo(),
                                        dims->data(), dims->size());

    x->registerSpecialUse({z}, {x});

    delete dims;
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}

////////////////////////////////////////////////////////////////////////
void execReduceLong(Pointer *extraPointers, int opNum, OpaqueNDArray x, void *extraParams, OpaqueNDArray z, OpaqueNDArray dimension) {
  try {
    x->prepareSpecialUse({z}, {x});
    dimension->preparePrimaryUse({}, {dimension});

    auto dimensionData = dimension != nullptr ? reinterpret_cast<LongType *>(dimension->buffer()) : nullptr;
    LongType dimensionLength = static_cast<LongType>(shape::length(dimension->shapeInfo()));

    const auto zLen = shape::length(z->shapeInfo());

    std::vector<LongType> dimensions(dimensionData, dimensionData + dimensionLength);

    const LongType *zShapeInfoH = z->shapeInfo();

    if (shape::rank(x->shapeInfo()) - dimensionLength != shape::rank(z->shapeInfo()) && zLen != 1) {
      auto zPack = ConstantShapeHelper::getInstance().createShapeInfoWithNoUnitiesForReduce(z->shapeInfo(), &dimensions);
      zShapeInfoH = reinterpret_cast<LongType const *>(zPack->primary());
    }

    std::vector<LongType> *dims =
        (zLen != 1) ? ShapeUtils::evalDimsForReduceOp(shape::rank(x->shapeInfo()), &dimensions) : new std::vector<LongType>();

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execReduceLong(&lc, opNum,
                                        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
                                        x->shapeInfo(),
                                        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
                                        x->specialShapeInfo(), extraParams,
                                        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
                                        zShapeInfoH,
                                        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
                                        z->specialShapeInfo(),
                                        dims->data(), dims->size());

    x->registerSpecialUse({z}, {x});

    delete dims;
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}

////////////////////////////////////////////////////////////////////////
void execReduceBool2(Pointer *extraPointers, int opNum, OpaqueNDArray x, void *extraParams, OpaqueNDArray z, OpaqueNDArray dimension) {
  try {
    x->prepareSpecialUse({z}, {x});
    dimension->preparePrimaryUse({}, {dimension});

    auto dimensionData = dimension != nullptr ? reinterpret_cast<LongType *>(dimension->buffer()) : nullptr;
    LongType dimensionLength = static_cast<LongType>(shape::length(dimension->shapeInfo()));

    const auto zLen = shape::length(z->shapeInfo());

    std::vector<LongType> dimensions(dimensionData, dimensionData + dimensionLength);

    const LongType *zShapeInfoH = z->shapeInfo();

    if (shape::rank(x->shapeInfo()) - dimensionLength != shape::rank(z->shapeInfo()) && zLen != 1) {
      auto zPack = ConstantShapeHelper::getInstance().createShapeInfoWithNoUnitiesForReduce(z->shapeInfo(), &dimensions);
      zShapeInfoH = reinterpret_cast<LongType const *>(zPack->primary());
    }

    std::vector<LongType> *dims =
        (zLen != 1) ? ShapeUtils::evalDimsForReduceOp(shape::rank(x->shapeInfo()), &dimensions) : new std::vector<LongType>();

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execReduceBool(&lc,
                                        opNum,
                                        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
                                        x->shapeInfo(),
                                        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
                                        x->specialShapeInfo(), extraParams,
                                        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
                                        zShapeInfoH,
                                        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
                                        z->specialShapeInfo(),
                                        dims->data(), dims->size());

    x->registerSpecialUse({z}, {x});

    delete dims;
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}

////////////////////////////////////////////////////////////////////////
void execReduceBool(Pointer *extraPointers, int opNum, OpaqueNDArray x, void *extraParams, OpaqueNDArray z, OpaqueNDArray dimension) {
  try {
    x->prepareSpecialUse({z}, {x});
    dimension->preparePrimaryUse({}, {dimension});

    auto dimensionData = dimension != nullptr ? reinterpret_cast<LongType *>(dimension->buffer()) : nullptr;
    LongType dimensionLength = static_cast<LongType>(shape::length(dimension->shapeInfo()));

    const auto zLen = shape::length(z->shapeInfo());

    std::vector<LongType> dimensions(dimensionData, dimensionData + dimensionLength);

    const LongType *zShapeInfoH = z->shapeInfo();

    if (shape::rank(x->shapeInfo()) - dimensionLength != shape::rank(z->shapeInfo()) && zLen != 1) {
      auto zPack = ConstantShapeHelper::getInstance().createShapeInfoWithNoUnitiesForReduce(z->shapeInfo(), &dimensions);
      zShapeInfoH = reinterpret_cast<LongType const *>(zPack->primary());
    }

    std::vector<LongType> *dims =
        (zLen != 1) ? ShapeUtils::evalDimsForReduceOp(shape::rank(x->shapeInfo()), &dimensions) : new std::vector<LongType>();

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execReduceBool(&lc,
                                        opNum,
                                        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
                                        x->shapeInfo(),
                                        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
                                        x->specialShapeInfo(), extraParams,
                                        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
                                        zShapeInfoH,
                                        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
                                        z->specialShapeInfo(),
                                        dims->data(), dims->size());

    x->registerSpecialUse({z}, {x});

    delete dims;
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}

/**
 *
 * @param opNum
 * @param dX
 * @param dXShapeInfo
 * @param extraParams
 * @param dZ
 * @param dZShapeInfo
 * @param dimension
 * @param dimensionLength
 */
////////////////////////////////////////////////////////////////////////
void execIndexReduce(Pointer *extraPointers, int opNum, OpaqueNDArray x, void *extraParams, OpaqueNDArray z, OpaqueNDArray dimension) {
  try {
    x->prepareSpecialUse({z}, {x});
    dimension->preparePrimaryUse({}, {dimension});

    auto dimensionData = dimension != nullptr ? reinterpret_cast<LongType *>(dimension->buffer()) : nullptr;
    LongType dimensionLength = static_cast<LongType>(shape::length(dimension->shapeInfo()));

    auto tadPack = ConstantTadHelper::getInstance().tadForDimensions(x->shapeInfo(), dimensionData, dimensionLength);

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execIndexReduce(
        &lc, opNum,
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
        x->shapeInfo(),
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
        x->specialShapeInfo(), extraParams,
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
        z->shapeInfo(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
        z->specialShapeInfo(),
        dimensionData, dimensionLength, tadPack->specialShapeInfo(), tadPack->specialOffsets());

    x->registerSpecialUse({z}, {x});
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}

/**
 *
 * @param opNum
 * @param dX
 * @param dXShapeInfo
 * @param extraParams
 * @param dZ
 * @param dZShapeInfo
 */
////////////////////////////////////////////////////////////////////////
void execReduceFloat2(Pointer *extraPointers, int opNum, OpaqueNDArray x, void *extraParams, OpaqueNDArray z, OpaqueNDArray dimension) {
  try {
    x->prepareSpecialUse({z}, {x});
    dimension->preparePrimaryUse({}, {dimension});

    auto dimensionData = dimension != nullptr ? reinterpret_cast<LongType *>(dimension->buffer()) : nullptr;
    LongType dimensionLength = static_cast<LongType>(shape::length(dimension->shapeInfo()));

    const auto zLen = shape::length(z->shapeInfo());

    std::vector<LongType> dimensions(dimensionData, dimensionData + dimensionLength);

    const LongType *zShapeInfoH = z->shapeInfo();

    if (shape::rank(x->shapeInfo()) - dimensionLength != shape::rank(z->shapeInfo()) && zLen != 1) {
      auto zPack = ConstantShapeHelper::getInstance().createShapeInfoWithNoUnitiesForReduce(z->shapeInfo(), &dimensions);
      zShapeInfoH = reinterpret_cast<LongType const *>(zPack->primary());
    }

    std::vector<LongType> *dims =
        (zLen != 1) ? ShapeUtils::evalDimsForReduceOp(shape::rank(x->shapeInfo()), &dimensions) : new std::vector<LongType>();

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execReduceFloat(&lc,
                                         opNum,
                                         shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
                                         x->shapeInfo(),
                                         shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
                                         x->specialShapeInfo(), extraParams,
                                         shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
                                         zShapeInfoH,
                                         shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
                                         z->specialShapeInfo(),
                                         dims->data(), dims->size());

    x->registerSpecialUse({z}, {x});
    delete dims;
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}
/**
 *
 * @param opNum
 * @param dX
 * @param dXShapeInfo
 * @param extraParams
 */
////////////////////////////////////////////////////////////////////////
void execIndexReduceScalar(Pointer *extraPointers, int opNum, OpaqueNDArray x, void *extraParams, OpaqueNDArray z) {
  try {
    x->prepareSpecialUse({z}, {x});

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execIndexReduceScalar(
        &lc, opNum,
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
        x->shapeInfo(),
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
        x->specialShapeInfo(), extraParams,
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
        z->shapeInfo(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
        z->specialShapeInfo());

    x->registerSpecialUse({z}, {x});
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}
////////////////////////////////////////////////////////////////////////
void execTransformSame(Pointer *extraPointers, int opNum, OpaqueNDArray x, void *extraParams, OpaqueNDArray z) {
  try {
    x->prepareSpecialUse({z}, {x});

    auto tadShapeInfo = reinterpret_cast<LongType *>(extraPointers != nullptr ? extraPointers[0] : nullptr);
    auto tadOffsets = reinterpret_cast<LongType *>(extraPointers != nullptr ? extraPointers[1] : nullptr);

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execTransformSame(&lc, opNum,
                                           shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
                                           x->shapeInfo(),
                                           shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
                                           x->specialShapeInfo(),
                                           shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
                                           z->shapeInfo(),
                                           shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
                                           z->specialShapeInfo(),
                                           extraParams, tadShapeInfo, tadOffsets);

    x->registerSpecialUse({z}, {x});
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}
////////////////////////////////////////////////////////////////////////
void execTransformBool(Pointer *extraPointers, int opNum, OpaqueNDArray x, void *extraParams, OpaqueNDArray z) {
  try {
    x->prepareSpecialUse({z}, {x});

    auto tadShapeInfo = reinterpret_cast<LongType *>(extraPointers != nullptr ? extraPointers[0] : nullptr);
    auto tadOffsets = reinterpret_cast<LongType *>(extraPointers != nullptr ? extraPointers[1] : nullptr);

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execTransformBool(&lc,
                                           opNum,
                                           shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
                                           x->shapeInfo(),
                                           shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
                                           x->specialShapeInfo(),
                                           shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
                                           z->shapeInfo(),
                                           shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
                                           z->specialShapeInfo(),
                                           extraParams,
                                           tadShapeInfo,
                                           tadOffsets);

    x->registerSpecialUse({z}, {x});
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}
////////////////////////////////////////////////////////////////////////
void execTransformAny(Pointer *extraPointers, int opNum, OpaqueNDArray x, void *extraParams, OpaqueNDArray z) {
  try {
    x->prepareSpecialUse({z}, {x});
    auto stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
    auto streamSpecial = reinterpret_cast<hipStream_t &>(extraPointers[4]);
    LaunchContext lc(stream, streamSpecial, extraPointers[5], extraPointers[3], reinterpret_cast<int *>(extraPointers[6]));

    NativeOpExecutioner::execTransformAny(&lc,
                                          opNum,
                                          shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
                                          x->shapeInfo(),
                                          shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
                                          x->specialShapeInfo(),
                                          shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
                                          z->shapeInfo(),
                                          shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
                                          z->specialShapeInfo(),
                                          extraParams, nullptr, nullptr);

    x->registerSpecialUse({z}, {x});
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}

////////////////////////////////////////////////////////////////////////
void execTransformStrict(Pointer *extraPointers, int opNum, OpaqueNDArray x, void *extraParams, OpaqueNDArray z) {
  try {
    x->prepareSpecialUse({z}, {x});

    auto tadShapeInfo = reinterpret_cast<LongType *>(extraPointers != nullptr ? extraPointers[10] : nullptr);
    auto tadOffsets = reinterpret_cast<LongType *>(extraPointers != nullptr ? extraPointers[11] : nullptr);

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execTransformStrict(
        &lc, opNum,
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
        x->shapeInfo(),
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
        x->specialShapeInfo(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
        z->shapeInfo(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
        z->specialShapeInfo(), extraParams,
        tadShapeInfo, tadOffsets);

    x->registerSpecialUse({z}, {x});
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}

////////////////////////////////////////////////////////////////////////
void execTransformFloat(Pointer *extraPointers, int opNum, OpaqueNDArray x, void *extraParams, OpaqueNDArray z) {
  try {
    x->prepareSpecialUse({z}, {x});

    auto tadShapeInfo = reinterpret_cast<LongType *>(extraPointers != nullptr ? extraPointers[10] : nullptr);
    auto tadOffsets = reinterpret_cast<LongType *>(extraPointers != nullptr ? extraPointers[11] : nullptr);

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execTransformFloat(
        &lc,
        opNum,
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
        x->shapeInfo(),
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
        x->specialShapeInfo(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
        z->shapeInfo(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
        z->specialShapeInfo(), extraParams,
        tadShapeInfo,
        tadOffsets);

    x->registerSpecialUse({z}, {x});
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}

void checkP2P() {
  int curDevice = 0;

  hipGetDevice(&curDevice);

  int devCnt = 0;
  hipGetDeviceCount(&devCnt);

  if (curDevice < 0 && curDevice > devCnt) curDevice = 0;

  bool tempSupport = true;

  if (devCnt > 1) {
    for (int dX = 0; dX < devCnt; dX++) {
      for (int dY = 0; dY < devCnt; dY++) {
        if (dX == dY) continue;

        int canAccess = 0;
        hipSetDevice(dX);

        hipDeviceCanAccessPeer(&canAccess, dX, dY);

        if (!canAccess) {
          tempSupport = false;
          break;
        }
      }
    }

    supportedP2P = tempSupport;

    hipSetDevice(curDevice);
  } else {
    // if we have only 1 device - we say that we support P2P, since all data will be on 1 device
    supportedP2P = true;
  }
}

void enableP2P(bool enable) {
  if (enable == allowedP2P) return;

  int curDevice = 0;

  hipGetDevice(&curDevice);

  int devCnt = 0;
  hipGetDeviceCount(&devCnt);

  if (curDevice < 0 && curDevice > devCnt) curDevice = 0;

  if (devCnt > 1) {
    for (int dX = 0; dX < devCnt; dX++) {
      for (int dY = 0; dY < devCnt; dY++) {
        if (dX == dY) continue;

        int canAccess = 0;
        hipSetDevice(dX);

        hipDeviceCanAccessPeer(&canAccess, dX, dY);

        if (canAccess) {
          if (enable) {
            hipDeviceEnablePeerAccess(dY, 0);
          } else {
            hipDeviceDisablePeerAccess(dY);
          }
        } else {
          if (Environment::getInstance().isVerbose()) printf("Peer access [%i] -> [%i] isn't possible\n", dX, dY);
        }
      }
    }

    hipSetDevice(curDevice);
  }

  allowedP2P = enable;

  hipSetDevice(curDevice);
}

bool isP2PAvailable() { return supportedP2P; }

void initializeDevicesAndFunctions() {
  try {
    int devCnt = 0;
    hipGetDeviceCount(&devCnt);
    deviceProperties = new hipDeviceProp_t[devCnt];
    for (int i = 0; i < devCnt; i++) {
      hipSetDevice(i);
      hipGetDeviceProperties(&deviceProperties[i], i);

      hipDeviceSetLimit(hipLimitStackSize, 4096);
    }

    hipSetDevice(0);

    checkP2P();

    // enabling p2p gpu access if it's supported
    if (supportedP2P && devCnt > 1) enableP2P(allowedP2P);
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}

void initializeFunctions(Pointer *functions) { BlasHelper::getInstance().initializeDeviceFunctions(functions);
}


/**
 * This method acquires memory chunk of requested size on host side
 *
 * @param pointer pointer that'll be used for allocation
 * @param memorySize memory size, in bytes
 * @param flags optional parameter
 */
Pointer mallocHost(LongType memorySize, int flags) {
  Pointer pointer;
  // hipHostMallocMapped |hipHostMallocPortable
  auto res = hipHostAlloc(reinterpret_cast<void **>(&pointer), memorySize + 8, hipHostMallocDefault);
  if (res != 0) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(res);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipHostAlloc failed");
  }

  return reinterpret_cast<int8_t *>(pointer);
}

/**
 * This method acquires memory chunk of requested size on specified device
 *
 * @param pointer pointer that'll be used for allocation
 * @param memorySize memory size, in bytes
 * @param ptrToDeviceId pointer to deviceId. For cuda that's just and int, for OpenCL that's pointer to device_id, etc
 * @param flags optional parameter
 */
Pointer mallocDevice(LongType memorySize, int deviceId, int flags) {
  Pointer pointer;
  auto res = hipMalloc(reinterpret_cast<void **>(&pointer), memorySize + 8);
  if (res != 0) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(res);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipMalloc failed");
  }

  return reinterpret_cast<int8_t *>(pointer);
}

/**
 * This method releases previously allocated host memory space
 *
 * @param pointer pointer that'll be freed
 */
int freeHost(Pointer pointer) {
  auto res = hipHostFree(reinterpret_cast<void *>(pointer));
  if (res != 0) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(res);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipHostFree failed");
  }

  return 1L;
}

/**
 * This method releases previously allocated memory space on device
 *
 * @param pointer pointer that'll be freed
 * @param ptrToDeviceId pointer to deviceId.
 */
int freeDevice(Pointer pointer, int deviceId) {
  auto res = hipFree(reinterpret_cast<void *>(pointer));

  // we're intentionally skipping
  if (res != 0 && res != 1) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(res);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipFree failed");
  }

  return res == 0 ? 1L : 0L;
}

Pointer createContext() { return 0L; }

Pointer createStream() {
  auto stream = new hipStream_t();
  auto dZ = hipStreamCreate(stream);
  if (dZ != 0) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(dZ);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipStreamCreate failed");
  }

  return stream;
}

Pointer createEvent() {
  Pointer nativeEvent = (Pointer)malloc(sizeof(hipEvent_t));

  CHECK_ALLOC(nativeEvent, "Failed to allocate new CUDA event buffer", sizeof(hipEvent_t));

  auto dZ = hipEventCreateWithFlags(reinterpret_cast<hipEvent_t *>(&nativeEvent), hipEventDisableTiming);
  if (dZ != 0) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(dZ);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipEventCreateWithFlags failed");
  }

  return nativeEvent;
}

int registerEvent(Pointer event, Pointer stream) {
  auto pEvent = reinterpret_cast<hipEvent_t *>(&event);
  auto pStream = reinterpret_cast<hipStream_t *>(stream);

  auto dZ = hipEventRecord(*pEvent, *pStream);
  if (dZ != 0) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(dZ);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipEventRecord failed");
  }

  return 1;
}

int setDevice(int deviceId) {
  AffinityManager::setCurrentDevice(deviceId);
  return 1;
}

LongType getDeviceFreeMemoryDefault() {
  size_t memFree = 0;
  size_t memTotal = 0;

  hipMemGetInfo(&memFree, &memTotal);

  return (LongType)memFree;
}

LongType getDeviceFreeMemory(int device) {
  int orig = -1;

  hipGetDevice(&orig);

  if (device >= 0 && device != orig) {
    hipSetDevice(device);
  }

  size_t memFree = 0;
  size_t memTotal = 0;

  hipMemGetInfo(&memFree, &memTotal);

  if (device >= 0 && device != orig) {
    hipSetDevice(orig);
  }

  return (LongType)memFree;
}

LongType getDeviceTotalMemory(int device) {
  int orig = -1;

  hipGetDevice(&orig);

  if (device >= 0 && device != orig) {
    hipSetDevice(device);
  }
  size_t memFree = 0;
  size_t memTotal = 0;

  hipMemGetInfo(&memFree, &memTotal);

  if (device >= 0 && device != orig) {
    hipSetDevice(orig);
  }

  return (LongType)memTotal;
}

int memcpySync(Pointer dst, Pointer src, LongType size, int flags, Pointer reserved) {
  hipMemcpyKind kind;

  switch (flags) {
    case 0: {
      kind = hipMemcpyHostToHost;
    } break;
    case 1: {
      kind = hipMemcpyHostToDevice;
    } break;
    case 2: {
      kind = hipMemcpyDeviceToHost;
    } break;
    case 3: {
      kind = hipMemcpyDeviceToDevice;
    } break;
    default: {
      LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
      LaunchContext::defaultContext()->errorReference()->setErrorMessage("UNDEFNED MEMCPY");
      return 0;
    }
  }

  auto dZ = hipMemcpy(reinterpret_cast<void *>(dst), const_cast<const void *>(reinterpret_cast<void *>(src)),
                       static_cast<size_t>(size), kind);
  if (dZ != 0) {
    printf("Failed on [%p] -> [%p], size: [%i], direction: [%i], dZ: [%i]\n", src, dst, size, flags,
           static_cast<int>(dZ));
    fflush(stdout);
    fflush(stderr);
    LaunchContext::defaultContext()->errorReference()->setErrorCode(dZ);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipMemcpy failed");
    return 0;
  }

  return 1;
}

int memcpyAsync(Pointer dst, Pointer src, LongType size, int flags, Pointer reserved) {
  auto pStream = reinterpret_cast<hipStream_t *>(reserved);

  hipMemcpyKind kind;


  switch (flags) {
    case 0: {
      kind = hipMemcpyHostToHost;
    } break;
    case 1: {
      kind = hipMemcpyHostToDevice;
    } break;
    case 2: {
      kind = hipMemcpyDeviceToHost;
    } break;
    case 3: {
      kind = hipMemcpyDeviceToDevice;
    } break;
    default: {
      LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
      LaunchContext::defaultContext()->errorReference()->setErrorMessage("UNDEFINED MEMCPY");
      return 0;
    }
  }

  auto dZ = hipMemcpyAsync(reinterpret_cast<void *>(dst), const_cast<const void *>(reinterpret_cast<void *>(src)),
                            static_cast<size_t>(size), kind, *pStream);

  if (dZ != 0) {
    printf("Failed on [%p] -> [%p], size: [%i], direction: [%i], dZ: [%i]\n", src, dst, size, flags,
           static_cast<int>(dZ));

    fflush(stdout);
    fflush(stderr);
    LaunchContext::defaultContext()->errorReference()->setErrorCode(dZ);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipMemcpyAsync failed");
    return 0;
  }


  return 1;
}

int memsetSync(Pointer dst, int value, LongType size, int flags, Pointer reserved) {
  auto dZ = hipMemset(reinterpret_cast<void *>(dst), value, static_cast<size_t>(size));
  if (dZ != 0) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(dZ);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipMemset failed");
  }

  return 1;
}

int memsetAsync(Pointer dst, int value, LongType size, int flags, Pointer reserved) {
  auto pStream = reinterpret_cast<hipStream_t *>(reserved);

  auto dZ = hipMemsetAsync(reinterpret_cast<void *>(dst), value, static_cast<size_t>(size), *pStream);
  if (dZ != 0) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(dZ);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipMemsetAsync failed");
  }

  return 1;
}

int destroyEvent(Pointer event) {
  auto pEvent = reinterpret_cast<hipEvent_t *>(&event);
  auto dZ = hipEventDestroy(*pEvent);
  if (dZ != 0) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(dZ);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipEventDestroy failed");
  }

  return 1;
}

int streamSynchronize(Pointer stream) {
  auto pStream = reinterpret_cast<hipStream_t *>(stream);

  auto dZ = hipStreamSynchronize(*pStream);
  if (dZ != 0) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(dZ);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipStreamSynchronize failed");
  }

  return 1L;
}

int eventSynchronize(Pointer event) {
  auto pEvent = reinterpret_cast<hipEvent_t *>(&event);

  auto dZ = hipEventSynchronize(*pEvent);
  if (dZ != 0) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(dZ);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipEventSynchronize failed");
  }

  return 1L;
}

int getAvailableDevices() {
  int devCnt = 0;
  hipGetDeviceCount(&devCnt);
  return devCnt;
}

void enableDebugMode(bool reallyEnable) { Environment::getInstance().setDebug(reallyEnable); }

void setGridLimit(int gridSize) {
  if (gridSize > 8192) gridSize = 8192;
  if (gridSize < 1) gridSize = 1;
  blockLimit = gridSize;
}

int ompGetMaxThreads() { return maxThreads; }

int ompGetNumThreads() { return maxThreads; }

void setOmpNumThreads(int threads) {
  if (threads > 1024) threads = 1024;
  if (threads < 32) threads = 32;
  maxThreads = threads;
}

void enableVerboseMode(bool reallyEnable) { Environment::getInstance().setVerbose(reallyEnable); }

int getDeviceMajor(int device) { return deviceProperties[device].major; }

int getDeviceMinor(int device) { return deviceProperties[device].minor; }

const char *getDeviceName(int device) { return deviceProperties[device].name; }



void saveNpy(std::string fname, const InteropDataBuffer *data, const unsigned int *shape, const unsigned int ndims,
             std::string mode) {
  auto dtype = data->getDataBuffer()->getDataType();
  BUILD_SINGLE_SELECTOR(dtype,cnpy::npy_save,(fname,data->getDataBuffer()->primary(),shape,ndims,mode),SD_COMMON_TYPES);
}


/**
 * This method saves
 */
TadPack *tadOnlyShapeInfo(const LongType *hXShapeInfo, LongType *dimension, LongType dimensionLength) {
  try {
    auto pack = ConstantTadHelper::getInstance().tadForDimensions(hXShapeInfo, dimension, dimensionLength);
    return pack;
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
    return nullptr;
  }
}


int memcpyConstantAsync(LongType dst, Pointer src, LongType size, int flags, Pointer reserved) {
  hipStream_t *pStream = reinterpret_cast<hipStream_t *>(reserved);

  hipMemcpyKind kind;

  DEBUG_KERNEL(pStream, -1);

  switch (flags) {
    case 0: {
      kind = hipMemcpyHostToHost;
    } break;
    case 1: {
      kind = hipMemcpyHostToDevice;
    } break;
    case 2: {
      kind = hipMemcpyDeviceToHost;
    }
    case 3: {
      kind = hipMemcpyDeviceToDevice;
    } break;
  }
  auto dZ = hipMemcpyToSymbolAsync(HIP_SYMBOL(deviceConstantMemory), const_cast<const void *>(src), size, dst, kind, *pStream);
  if (dZ != 0) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(dZ);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipMemcpyToSymbolAsync failed");
  }

  return 1;
}

Pointer getConstantSpace() {
  Pointer dConstAddr;
  hipError_t dZ = hipGetSymbolAddress(reinterpret_cast<void **>(&dConstAddr), deviceConstantMemory);

  if (dZ != 0) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(dZ);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage("hipGetSymbolAddress failed");
  }

  return dConstAddr;
}

void pullRows(Pointer *extraPointers, OpaqueNDArray x, OpaqueNDArray z, LongType n, OpaqueNDArray indexes, sd::LongType dimension) {
  try {
    x->prepareSpecialUse({z}, {x});


    hipStream_t *stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
    dim3 launchDims = getLaunchDims("pullRows");
    auto xType = x->dataType();

    std::vector<void*> xBuffers(n);
    std::vector<const LongType*> tadShapeInfoBuffers(n);
    std::vector<const LongType*> tadOffsetsBuffers(n);

    for (int i = 0; i < n; ++i) {
      // Calculate TADs for each x
      auto tadPackX = sd::ConstantTadHelper::getInstance().tadForDimensions(x->shapeInfo(), &dimension, 1);
      tadShapeInfoBuffers[i] = const_cast<LongType*>(tadPackX->specialShapeInfo());
      tadOffsetsBuffers[i] = const_cast<LongType*>(tadPackX->specialOffsets());
    }

    void* zBuffer = z->specialBuffer();
    LongType* zShapeInfo = const_cast<LongType*>(z->specialShapeInfo());

    // Calculate TADs for z
    auto tadPackZ = sd::ConstantTadHelper::getInstance().tadForDimensions(z->shapeInfo(), &dimension, 1);
    LongType* zTadShapeInfoBuffer = const_cast<LongType*>(tadPackZ->specialShapeInfo());
    LongType* zTadOffsetsBuffer = const_cast<LongType*>(tadPackZ->specialOffsets());

    // Use the special buffer for indexes
    LongType* indexesBuffer = reinterpret_cast<LongType*>(indexes->specialBuffer());

    BUILD_SINGLE_SELECTOR(xType, pullRowsKernelGeneric,
                          (launchDims, stream, x->specialBuffer(), zBuffer, n, indexesBuffer,
                           *tadShapeInfoBuffers.data(), *tadOffsetsBuffers.data(), zTadShapeInfoBuffer, zTadOffsetsBuffer),
                          SD_COMMON_TYPES);

    DEBUG_KERNEL(stream, -1);

    for (int i = 0; i < n; ++i) {
      x->registerSpecialUse({z}, {x});
    }
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}

void average(Pointer *extras,
             OpaqueNDArrayArr x,
             OpaqueNDArray z,int n, LongType length, bool propagate) {
  try {
    hipStream_t *stream = reinterpret_cast<hipStream_t *>(extras[1]);
    int mode = getDeviceId(extras[3]);

    if (Environment::getInstance().isDebugAndVerbose()) printf("averageFloat called\n");

    auto xType = x[0]->dataType();

    // launching on gpu
    if (mode == 0) {
      dim3 launchDims = getLaunchDims("average");
      std::vector<void*> xBuffers(n);
      for (int i = 0; i < n; ++i) {
        xBuffers[i] = x[i]->specialBuffer();
      }

      BUILD_SINGLE_SELECTOR(xType, averagingKernelGeneric, (launchDims, stream, xBuffers.data(), z->specialBuffer(), n, length, propagate), SD_COMMON_TYPES);
      DebugHelper::checkErrorCode(stream, "AverageFloat(...) failed");
    } else {
      // launching on host memory
      BUILD_SINGLE_SELECTOR(xType, sd::SpecialMethods, ::averageGeneric(x, z, n, length, propagate), SD_COMMON_TYPES);
    }
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}

void accumulate(Pointer *extras, OpaqueNDArrayArr x,  OpaqueNDArray z, int n, LongType length) {
  try {
    auto stream = reinterpret_cast<hipStream_t *>(extras[1]);
    int mode = getDeviceId(extras[3]);

    if (Environment::getInstance().isDebugAndVerbose()) printf("accumulateFloat called\n");
    auto xType = x[0]->dataType();


    // launching on gpu
    if (mode == 0) {
      // Extract buffers from each NDArray in the array
      std::vector<void*> xBuffers(n);
      for (int i = 0; i < n; ++i) {
        xBuffers[i] = x[i]->specialBuffer();
      }

      void* zBuffer = z->specialBuffer();

      dim3 launchDims = getAccumDims(n);
      BUILD_SINGLE_SELECTOR(xType, accumulateKernelGeneric, (launchDims, stream, xBuffers.data(), zBuffer, n, length), SD_COMMON_TYPES);
      DebugHelper::checkErrorCode(stream, "AccumulateFloat(...) failed");
    } else {
      std::vector<NDArray*> xBuffers(n);
      for (int i = 0; i < n; ++i) {
        xBuffers[i] = x[i];
      }


      // launching on host memory
      BUILD_SINGLE_SELECTOR(xType, sd::SpecialMethods, ::accumulateGeneric(xBuffers.data(), z, n, length), SD_COMMON_TYPES);
    }
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}


bool isExperimentalEnabled() { return Environment::getInstance().isExperimentalBuild(); }
void shuffle(Pointer *extras,
             OpaqueNDArrayArr x,
             OpaqueNDArrayArr z,
             int N,
             OpaqueNDArray dimension,
             OpaqueNDArray shuffleMap) {
  try {
    hipStream_t *stream = reinterpret_cast<hipStream_t *>(extras[1]);

    auto xType = x[0]->dataType();
    dim3 launchDims = getLaunchDims("shuffle");

    // Extract buffers from each NDArray in the array
    std::vector<void*> xBuffers(N);
    std::vector<LongType*> xShapeInfos(N);
    std::vector<LongType*> tadShapeInfoBuffers(N);
    std::vector<LongType*> tadOffsetsBuffers(N);
    std::vector<void*> zBuffers(N);
    std::vector<LongType*> zShapeInfos(N);
    std::vector<LongType*> zTadShapeInfoBuffers(N);
    std::vector<LongType*> zTadOffsetsBuffers(N);

    for (int i = 0; i < N; ++i) {
      xBuffers[i] = x[i]->specialBuffer();
      xShapeInfos[i] = const_cast<LongType*>(x[i]->specialShapeInfo());

      zBuffers[i] = z[i]->specialBuffer();
      zShapeInfos[i] = const_cast<LongType*>(z[i]->specialShapeInfo());

      // Extract dimensions for each x[i] and z[i] from the array of arrays
      LongType* dimensions = reinterpret_cast<LongType*>(dimension->buffer());
      LongType dimLength = shape::length(dimension->shapeInfo());

      // Calculate TADs for each x
      auto tadPackX = sd::ConstantTadHelper::getInstance().tadForDimensions(x[i]->shapeInfo(), dimensions, dimLength);
      tadShapeInfoBuffers[i] = const_cast<LongType*>(tadPackX->specialShapeInfo());
      tadOffsetsBuffers[i] = const_cast<LongType*>(tadPackX->specialOffsets());

      // Calculate TADs for each z
      auto tadPackZ = sd::ConstantTadHelper::getInstance().tadForDimensions(z[i]->shapeInfo(), dimensions, dimLength);
      zTadShapeInfoBuffers[i] = const_cast<LongType*>(tadPackZ->specialShapeInfo());
      zTadOffsetsBuffers[i] = const_cast<LongType*>(tadPackZ->specialOffsets());
    }

    BUILD_SINGLE_SELECTOR(xType, shuffleKernelGeneric,
                          (launchDims, stream, xBuffers.data(), xShapeInfos.data(), zBuffers.data(), N, reinterpret_cast<int*>(shuffleMap->buffer()), tadShapeInfoBuffers.data(), tadOffsetsBuffers.data(), zTadShapeInfoBuffers.data(), zTadOffsetsBuffers.data()),
                          SD_COMMON_TYPES);

    DebugHelper::checkErrorCode(stream, "shuffle(...) failed");
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}

void setOmpMinThreads(int threads) {
  minThreads = sd::math::sd_max<int>(32, threads);
  minThreads = sd::math::sd_min<int>(maxThreads, minThreads);
}

int getDevice() { return AffinityManager::currentDeviceId(); }

////////////////////////////////////////////////////////////////////////
void execSummaryStats(Pointer *extraPointers, int opNum, OpaqueNDArray x, void *extraParams, OpaqueNDArray z, bool biasCorrected) {
  try {
    x->prepareSpecialUse({z}, {x});

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execSummaryStats(&lc,
                                          opNum,
                                          shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
                                          x->shapeInfo(),
                                          shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
                                          ConstantShapeHelper::getInstance().bufferForShapeInfo(x->shapeInfo())->special(),
                                          extraParams,
                                          shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
                                          z->shapeInfo(),
                                          shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
                                          ConstantShapeHelper::getInstance().bufferForShapeInfo(z->shapeInfo())->special(),
                                          biasCorrected);

    x->registerSpecialUse({z}, {x});
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}

////////////////////////////////////////////////////////////////////////
void execSummaryStatsTad(Pointer *extraPointers, int opNum, OpaqueNDArray x, void *extraParams, OpaqueNDArray z,
                         OpaqueNDArray dimension, bool biasCorrected) {
  try {
    x->prepareSpecialUse({z}, {x});
    dimension->preparePrimaryUse({}, {dimension});

    auto dimensionData = dimension != nullptr ? reinterpret_cast<LongType *>(dimension->buffer()) : nullptr;
    int dimensionLength = static_cast<int>(shape::length(dimension->shapeInfo()));

    auto tadPack = ConstantTadHelper::getInstance().tadForDimensions(x->shapeInfo(), dimensionData, dimensionLength);
    auto tadShapeInfo = tadPack->primaryShapeInfo();
    auto tadOffsets = tadPack->primaryOffsets();

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execSummaryStats(
        &lc, opNum,
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
        x->shapeInfo(),
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
        ConstantShapeHelper::getInstance().bufferForShapeInfo(x->shapeInfo())->special(),
        extraParams,
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
        z->shapeInfo(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
        ConstantShapeHelper::getInstance().bufferForShapeInfo(z->shapeInfo())->special(),
        dimensionData, dimensionLength, tadShapeInfo, tadOffsets, biasCorrected);

    x->registerSpecialUse({z}, {x});
    dimension->registerSpecialUse({}, {dimension});
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}
////////////////////////////////////////////////////////////////////////
void execReduce3(Pointer *extraPointers, int opNum, OpaqueNDArray x, void *extraParams, OpaqueNDArray y, OpaqueNDArray z) {
  try {
    x->prepareSpecialUse({z}, {x, y});

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execReduce3(
        &lc,
        opNum,
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
        x->shapeInfo(),
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
        x->specialShapeInfo(),
        extraParams,
        shape::isEmptyConst(y->shapeInfo()) ? nullptr : y->buffer(),
        y->shapeInfo(),
        shape::isEmptyConst(y->shapeInfo()) ? nullptr : y->specialBuffer(),
        y->specialShapeInfo(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
        z->shapeInfo(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
        z->specialShapeInfo());

    x->registerSpecialUse({z}, {x, y});
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}
////////////////////////////////////////////////////////////////////////
void execReduce3Tad(Pointer *extraPointers, int opNum, OpaqueNDArray x, void *extraParams, OpaqueNDArray y, OpaqueNDArray z, OpaqueNDArray dimension) {
  try {
    x->prepareSpecialUse({z}, {x, y});
    dimension->preparePrimaryUse({}, {dimension});

    auto dim = dimension != nullptr ? reinterpret_cast<LongType *>(dimension->buffer()) : nullptr;
    LongType dimensionLength = static_cast<LongType>(shape::length(dimension->shapeInfo()));

    auto xTadPack = ConstantTadHelper::getInstance().tadForDimensions(x->shapeInfo(), dim, dimensionLength);
    auto xTadShapeInfo = xTadPack->primaryShapeInfo();
    auto xOffsets = xTadPack->primaryOffsets();

    auto yTadPack = ConstantTadHelper::getInstance().tadForDimensions(y->shapeInfo(), dim, dimensionLength);
    auto yTadShapeInfo = yTadPack->primaryShapeInfo();
    auto yOffsets = yTadPack->primaryOffsets();

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);

    NativeOpExecutioner::execReduce3TAD(
        &lc, opNum,
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
        x->shapeInfo(),
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
        x->specialShapeInfo(), extraParams,
        shape::isEmptyConst(y->shapeInfo()) ? nullptr : y->buffer(),
        y->shapeInfo(),
        shape::isEmptyConst(y->shapeInfo()) ? nullptr : y->specialBuffer(),
        y->specialShapeInfo(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
        z->shapeInfo(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
        z->specialShapeInfo(),
        dim, dimensionLength,
        xTadShapeInfo, xOffsets, yTadShapeInfo, yOffsets);

    x->registerSpecialUse({z}, {x, y});
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}
////////////////////////////////////////////////////////////////////////
void execReduce3Scalar(Pointer *extraPointers, int opNum, OpaqueNDArray x, void *extraParams, OpaqueNDArray y, OpaqueNDArray z) {
  try {
    x->prepareSpecialUse({z}, {x, y});

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execReduce3Scalar(
        &lc, opNum,
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
        x->shapeInfo(),
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
        ConstantShapeHelper::getInstance().bufferForShapeInfo(x->shapeInfo())->special(), extraParams,
        shape::isEmptyConst(y->shapeInfo()) ? nullptr : y->buffer(),
        y->shapeInfo(),
        shape::isEmptyConst(y->shapeInfo()) ? nullptr : y->specialBuffer(),
        ConstantShapeHelper::getInstance().bufferForShapeInfo(y->shapeInfo())->special(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
        z->shapeInfo(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
        ConstantShapeHelper::getInstance().bufferForShapeInfo(z->shapeInfo())->special());

    x->registerSpecialUse({z}, {x, y});
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}

////////////////////////////////////////////////////////////////////////
void execScalarBool(Pointer *extraPointers, int opNum, OpaqueNDArray x, OpaqueNDArray z, OpaqueNDArray scalar, void *extraParams) {
  try {
    x->prepareSpecialUse({z}, {x, scalar});

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execScalarBool(
        &lc, opNum,
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
        x->shapeInfo(),
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
        ConstantShapeHelper::getInstance().bufferForShapeInfo(x->shapeInfo())->special(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
        z->shapeInfo(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
        ConstantShapeHelper::getInstance().bufferForShapeInfo(z->shapeInfo())->special(),
        shape::isEmptyConst(scalar->shapeInfo()) ? nullptr : scalar->buffer(),
        scalar->shapeInfo(),
        shape::isEmptyConst(scalar->shapeInfo()) ? nullptr : scalar->specialBuffer(),
        ConstantShapeHelper::getInstance().bufferForShapeInfo(scalar->shapeInfo())->special(), extraParams);

    x->registerSpecialUse({z}, {x, scalar});
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}

////////////////////////////////////////////////////////////////////////
void execScalarBoolTad(Pointer *extraPointers, int opNum, OpaqueNDArray x, OpaqueNDArray z, OpaqueNDArray scalar, void *extraParams, OpaqueNDArray dimension) {
  try {
    x->prepareSpecialUse({z}, {x, scalar});
    dimension->preparePrimaryUse({}, {dimension});

    auto dim = dimension != nullptr ? reinterpret_cast<LongType *>(dimension->buffer()) : nullptr;
    LongType dimensionLength = static_cast<LongType>(shape::length(dimension->shapeInfo()));

    auto xTadPack = ConstantTadHelper::getInstance().tadForDimensions(x->shapeInfo(), dim, dimensionLength);
    auto xTadShapeInfo = xTadPack->primaryShapeInfo();
    auto xOffsets = xTadPack->primaryOffsets();

    auto zTadPack = ConstantTadHelper::getInstance().tadForDimensions(z->shapeInfo(), dim, dimensionLength);
    auto zTadShapeInfo = zTadPack->primaryShapeInfo();
    auto zOffsets = zTadPack->primaryOffsets();

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execScalarBool(
        &lc, opNum,
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
        x->shapeInfo(),
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
        ConstantShapeHelper::getInstance().bufferForShapeInfo(x->shapeInfo())->special(),
        extraParams,
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
        z->shapeInfo(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
        ConstantShapeHelper::getInstance().bufferForShapeInfo(z->shapeInfo())->special(),
        shape::isEmptyConst(scalar->shapeInfo()) ? nullptr : scalar->buffer(),
        scalar->shapeInfo(),
        shape::isEmptyConst(scalar->shapeInfo()) ? nullptr : scalar->specialBuffer(),
        ConstantShapeHelper::getInstance().bufferForShapeInfo(scalar->shapeInfo())->special(),
        dim, dimensionLength,
        xTadShapeInfo, xOffsets, zTadShapeInfo, zOffsets);

    x->registerSpecialUse({z}, {x, scalar});
    dimension->registerSpecialUse({}, {dimension});
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
  }
}

////////////////////////////////////////////////////////////////////////
void execScalar(Pointer *extraPointers, int opNum, OpaqueNDArray x, OpaqueNDArray z, OpaqueNDArray scalar, void *extraParams) {
  try {
    x->prepareSpecialUse({z}, {x, scalar});

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execScalar(
        &lc, opNum,
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
        x->shapeInfo(),
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
        ConstantShapeHelper::getInstance().bufferForShapeInfo(x->shapeInfo())->special(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
        z->shapeInfo(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
        ConstantShapeHelper::getInstance().bufferForShapeInfo(z->shapeInfo())->special(),
        shape::isEmptyConst(scalar->shapeInfo()) ? nullptr : scalar->buffer(),
        scalar->shapeInfo(),
        shape::isEmptyConst(scalar->shapeInfo()) ? nullptr : scalar->specialBuffer(),
        ConstantShapeHelper::getInstance().bufferForShapeInfo(scalar->shapeInfo())->special(), extraParams);

    x->registerSpecialUse({z}, {x, scalar});
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}

////////////////////////////////////////////////////////////////////////
void execScalarTad(Pointer *extraPointers, int opNum, OpaqueNDArray x, OpaqueNDArray z, OpaqueNDArray scalar, void *extraParams, OpaqueNDArray dimension) {
  try {
    x->prepareSpecialUse({z}, {x, scalar});
    dimension->preparePrimaryUse({}, {dimension});

    auto dimensionPtr = dimension != nullptr ? reinterpret_cast<LongType *>(dimension->buffer()) : nullptr;
    LongType dimensionLength = static_cast<LongType>(shape::length(dimension->shapeInfo()));

    auto xTadPack = ConstantTadHelper::getInstance().tadForDimensions(x->shapeInfo(), dimensionPtr, dimensionLength);
    auto xTadShapeInfo = xTadPack->primaryShapeInfo();
    auto xOffsets = xTadPack->primaryOffsets();

    auto zTadPack = ConstantTadHelper::getInstance().tadForDimensions(z->shapeInfo(), dimensionPtr, dimensionLength);
    auto zTadShapeInfo = zTadPack->primaryShapeInfo();
    auto zOffsets = zTadPack->primaryOffsets();

    hipStream_t *stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);

    auto xType = ArrayOptions::dataType(x->shapeInfo());
    auto yType = ArrayOptions::dataType(scalar->shapeInfo());
    auto zType = ArrayOptions::dataType(z->shapeInfo());

    if (yType != xType && yType != BOOL && !isExperimentalEnabled())
      throw datatype_exception::build("execScalar both operands must have same data type", xType, yType);

    dim3 launchDims = getLaunchDims("scalarTad");

    BUILD_SINGLE_SELECTOR_THRICE(
        xType, functions::scalar::ScalarTransform,
        ::executeCudaAlongDimension(
            launchDims, stream, opNum,
            shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
            xTadShapeInfo,
            shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
            zTadShapeInfo,
            shape::isEmptyConst(scalar->shapeInfo()) ? nullptr : scalar->specialBuffer(),
            extraParams, dimensionPtr, dimensionLength, xTadShapeInfo, xOffsets, zTadShapeInfo, zOffsets),
        SD_COMMON_TYPES);

    DEBUG_KERNEL(stream, opNum);

    x->registerSpecialUse({z}, {x, scalar});
    dimension->registerSpecialUse({}, {dimension});
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}

////////////////////////////////////////////////////////////////////////
void execRandom(Pointer *extraPointers, int opNum, Pointer stateHost, OpaqueNDArray z, void *extraArguments) {
  try {
    z->prepareSpecialUse({}, {z});

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);

    NativeOpExecutioner::execRandom(
        &lc, opNum, stateHost,
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
        z->shapeInfo(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
        ConstantShapeHelper::getInstance().bufferForShapeInfo(z->shapeInfo())->special(),
        extraArguments);

    z->registerSpecialUse({}, {z});
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}

////////////////////////////////////////////////////////////////////////
void execRandom2(Pointer *extraPointers, int opNum, Pointer stateHost, OpaqueNDArray x, OpaqueNDArray z, void *extraArguments) {
  try {
    x->prepareSpecialUse({z}, {x});

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);

    NativeOpExecutioner::execRandom(
        &lc, opNum, stateHost,
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
        x->shapeInfo(),
        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
        ConstantShapeHelper::getInstance().bufferForShapeInfo(x->shapeInfo())->special(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
        z->shapeInfo(),
        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
        ConstantShapeHelper::getInstance().bufferForShapeInfo(z->shapeInfo())->special(), extraArguments);

    x->registerSpecialUse({z}, {x});
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}
////////////////////////////////////////////////////////////////////////
void execRandom3(Pointer *extraPointers, int opNum, Pointer stateHost, OpaqueNDArray x,
                 OpaqueNDArray y, OpaqueNDArray z, void *extraArguments) {
  try {
    x->prepareSpecialUse({z}, {x, y});

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execRandom(&lc, opNum, stateHost,
                                    shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
                                    x->shapeInfo(),
                                    shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
                                    x->specialShapeInfo(),
                                    shape::isEmptyConst(y->shapeInfo()) ? nullptr : y->buffer(),
                                    y->shapeInfo(),
                                    shape::isEmptyConst(y->shapeInfo()) ? nullptr : y->specialBuffer(),
                                    y->specialShapeInfo(),
                                    shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
                                    z->shapeInfo(),
                                    shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
                                    z->specialShapeInfo(),
                                    extraArguments);

    x->registerSpecialUse({z}, {x, y});
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}

Pointer initRandom(Pointer *extraPointers, long seed, long bufferSize, Pointer ptrToBuffer) {
  unsigned long long *ptrHost = reinterpret_cast<unsigned long long *>(extraPointers[0]);
  hipStream_t *stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);

  // we don't synchronize at random initialization, it's safe to go async here

  auto ptrDev = reinterpret_cast<unsigned long long *>(ptrToBuffer);
  auto buffer = new random::RandomBuffer(seed, bufferSize, reinterpret_cast<uint64_t *>(ptrHost),
                                         reinterpret_cast<uint64_t *>(ptrDev));
  buffer->propagateToDevice(buffer, *stream);

  DebugHelper::checkErrorCode(stream, "initRandom(...) failed A");

  // we generate sequence in the host memory
  random::Xoroshiro128 generator(buffer);
  generator.refreshBuffer();

  // and copy it to gpu
  hipMemcpyAsync(ptrDev, ptrHost, bufferSize * 8, hipMemcpyHostToDevice, *stream);
  DebugHelper::checkErrorCode(stream, "initRandom(...) failed B");

  return buffer;
}

void destroyRandom(Pointer ptrBuffer) {
  random::RandomBuffer *buffer = reinterpret_cast<random::RandomBuffer *>(ptrBuffer);

  // FIXME: it's bad thing, but we can't know in advance, which stream(s) where using this generator in practice
  hipDeviceSynchronize();

  delete buffer;
}

void refreshBuffer(Pointer *extraPointers, long seed, Pointer ptrRandom) {
  random::RandomBuffer *buffer = reinterpret_cast<random::RandomBuffer *>(ptrRandom);

  unsigned long long *ptrHost = reinterpret_cast<unsigned long long *>(extraPointers[0]);
  hipStream_t *stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
  hipStreamSynchronize(*stream);

  uint64_t *ptrDev = buffer->getDeviceBuffer();

  // update rng state
  buffer->setSeed(seed);
  buffer->setOffset(0);
  buffer->propagateToDevice(buffer, *stream);

  // refresh buffer on host size
  random::Xoroshiro128 generator(buffer);
  generator.refreshBuffer();

  // copy back to gpu
  hipMemcpyAsync(ptrDev, ptrHost, buffer->getSize() * 8, hipMemcpyHostToDevice, *stream);
}

void reSeedBuffer(Pointer *extraPointers, long seed, Pointer ptrRandom) {
  random::RandomBuffer *buffer = reinterpret_cast<random::RandomBuffer *>(ptrRandom);

  hipStream_t *stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
  hipStreamSynchronize(*stream);

  // update rng state
  buffer->reSeed(seed);
  buffer->setOffset(0);
  buffer->propagateToDevice(buffer, *stream);
}

/**
 * Return the length of a shape buffer
 * based on the pointer
 * @param buffer  the buffer pointer to check
 * @return
 */
int lengthForShapeBufferPointer(Pointer buffer) {
  auto shapeBuffer = reinterpret_cast<LongType *>(buffer);
  return shape::shapeInfoLength(shape::rank(shapeBuffer));
}

/**
 * The pointer to get the address for
 *
 * @param address the address to get the pointer
 * @return the pointer for the given address
 */

Pointer pointerForAddress(LongType address) { return reinterpret_cast<Pointer>(address); }



void prescanArrayRecursive(Pointer *extras, int *dZ, int *dX, int numElements, int level) {
  auto stream = reinterpret_cast<hipStream_t *>(extras[1]);
  auto g_scanBlockSums = reinterpret_cast<int **>(extras[2]);

  int blockSize = 512;  // max size of the thread blocks
  int numBlocks = sd::math::sd_max<int>(1, static_cast<int>(ceil(static_cast<float>(numElements) / (2.f * blockSize))));
  int numThreads;

  if (numBlocks > 1)
    numThreads = blockSize;
  else if (isPowerOfTwo(numElements))
    numThreads = numElements / 2;
  else
    numThreads = floorPow2(numElements);

  int numEltsPerBlock = numThreads * 2;

  // if this is a non-power-of-2 array, the last block will be non-full
  // compute the smallest power of 2 able to compute its scan.
  int numEltsLastBlock = numElements - (numBlocks - 1) * numEltsPerBlock;
  int numThreadsLastBlock = sd::math::sd_max<int>(1, numEltsLastBlock / 2);
  int np2LastBlock = 0;
  int sharedMemLastBlock = 0;

  if (numEltsLastBlock != numEltsPerBlock) {
    np2LastBlock = 1;

    if (!isPowerOfTwo(numEltsLastBlock)) numThreadsLastBlock = floorPow2(numEltsLastBlock);

    unsigned int extraSpace = (2 * numThreadsLastBlock) / NUM_BANKS;
    sharedMemLastBlock = sizeof(int) * (2 * numThreadsLastBlock + extraSpace);
  }

  // padding space is used to avoid shared memory bank conflicts
  int extraSpace = numEltsPerBlock / NUM_BANKS;
  int sharedMemSize = sizeof(int) * (numEltsPerBlock + extraSpace);

  // setup execution parameters
  // if NP2, we process the last block separately
  dim3 grid(sd::math::sd_max<int>(1, numBlocks - np2LastBlock), 1, 1);
  dim3 threads(numThreads, 1, 1);
  dim3 gridOnes(1, 1, 1);
  dim3 threadsOnes(numThreadsLastBlock, 1, 1);

  if (sharedMemSize < 2048) sharedMemSize = 2048;

  if (sharedMemLastBlock < 2048) sharedMemLastBlock = 2048;

  // execute the scan
  if (numBlocks > 1) {
    sd::prescanLauncher<true, false>(grid, threads, sharedMemSize, stream, dZ, dX, g_scanBlockSums[level],
                                     numThreads * 2, 0, 0);
    if (np2LastBlock) {
      sd::prescanLauncher<true, true>(gridOnes, threadsOnes, sharedMemLastBlock, stream, dZ, dX, g_scanBlockSums[level],
                                      numEltsLastBlock, numBlocks - 1, numElements - numEltsLastBlock);
    }

    // After scanning all the sub-blocks, we are mostly done.  But now we
    // need to take all of the last values of the sub-blocks and scan those.
    // This will give us a new value that must be sdded to each block to
    // get the final results.
    // recursive (CPU) call
    prescanArrayRecursive(extras, g_scanBlockSums[level], g_scanBlockSums[level], numBlocks, level + 1);

    uniformAdd<<<grid, threads, 1024, *stream>>>(dZ, g_scanBlockSums[level], numElements - numEltsLastBlock, 0, 0);
    DebugHelper::checkGlobalErrorCode("uniform addfailed(...) failed");

    if (np2LastBlock) {
      uniformAdd<<<1, numThreadsLastBlock, 1024, *stream>>>(dZ, g_scanBlockSums[level], numEltsLastBlock, numBlocks - 1,
                                                            numElements - numEltsLastBlock);
      DebugHelper::checkGlobalErrorCode("concat general case failed(...) failed");

    }
  } else if (isPowerOfTwo(numElements)) {
    sd::prescanLauncher<false, false>(grid, threads, sharedMemSize, stream, dZ, dX, 0, numThreads * 2, 0, 0);

  } else {
    sd::prescanLauncher<false, true>(grid, threads, sharedMemSize, stream, dZ, dX, 0, numElements, 0, 0);
  }

  DebugHelper::checkErrorCode(stream, "prescanArray(...) failed");
}



////////////////////////////////////////////////////////////////////////
void execReduce3All(Pointer *extraPointers, int opNum, OpaqueNDArray x, OpaqueNDArray y, OpaqueNDArray z, void *extraParamsVals, OpaqueNDArray dimension) {
  try {
    x->prepareSpecialUse({z}, {x, y, dimension});
    x->preparePrimaryUse({}, {dimension});

    auto dimensionPtr = dimension != nullptr ? reinterpret_cast<LongType *>(dimension->buffer()) : nullptr;
    LongType dimensionLength = static_cast<LongType>(shape::length(dimension->shapeInfo()));

    auto xTadPack = ConstantTadHelper::getInstance().tadForDimensions(x->shapeInfo(), dimensionPtr, dimensionLength);
    auto xTadShapeInfo = xTadPack->primaryShapeInfo();
    auto xOffsets = xTadPack->primaryOffsets();

    auto yTadPack = ConstantTadHelper::getInstance().tadForDimensions(y->shapeInfo(), dimensionPtr, dimensionLength);
    auto yTadShapeInfo = yTadPack->primaryShapeInfo();
    auto yOffsets = yTadPack->primaryOffsets();

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execReduce3All(&lc, opNum,
                                        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->buffer(),
                                        x->shapeInfo(),
                                        shape::isEmptyConst(x->shapeInfo()) ? nullptr : x->specialBuffer(),
                                        x->specialShapeInfo(),
                                        extraParamsVals,
                                        shape::isEmptyConst(y->shapeInfo()) ? nullptr : y->buffer(),
                                        y->shapeInfo(),
                                        shape::isEmptyConst(y->shapeInfo()) ? nullptr : y->specialBuffer(),
                                        y->specialShapeInfo(),
                                        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->buffer(),
                                        z->shapeInfo(),
                                        shape::isEmptyConst(z->shapeInfo()) ? nullptr : z->specialBuffer(),
                                        z->specialShapeInfo(),
                                        dimensionPtr,
                                        dimensionLength, xTadShapeInfo,
                                        xOffsets, yTadShapeInfo, yOffsets);

    x->registerSpecialUse({z}, {x, y});
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}

void sort(Pointer *extraPointers, OpaqueNDArray x, bool descending) {
  try {
    // Retrieve the CUDA stream from extraPointers
    hipStream_t *stream = nullptr;
    if (extraPointers != nullptr && extraPointers[1] != nullptr) {
      stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
    } else {
      // If no stream is provided, use the default stream
      stream = LaunchContext::defaultContext()->getCudaStream();
    }

    // Extract shape information from NDArray*
    const LongType *xShapeInfo = x->shapeInfo();
    const LongType *dXShapeInfo = x->specialShapeInfo();

    // Calculate the length of the array
    auto xLength = shape::length(xShapeInfo);

    // Get element-wise stride (not used in original logic but retrieved for consistency)
    auto xEWS = shape::elementWiseStride(xShapeInfo);

    // Determine the data type of the array
    auto xType = ArrayOptions::dataType(xShapeInfo);

    // Check if xLength is a power of 2 and within the specified limit
    if ((xLength != 0) && ((xLength & (xLength - 1)) == 0) && (xLength <= 1024 * 1024 * 10)) {
      // Get the launch dimensions for full sort
      dim3 launchDims = getSortFullDims(xLength);

      // Perform bitonic sort steps
      for (int k = 2; k <= xLength; k *= 2) {
        for (int j = k >> 1; j > 0; j >>= 1) {
          BUILD_SINGLE_SELECTOR(xType, bitonicSortStepGeneric,
                                (launchDims, stream, x->specialBuffer(), dXShapeInfo, j, k, xLength, descending),
                                SD_COMMON_TYPES);
        }
      }
    } else {
      // Get the launch dimensions for arbitrary sort
      dim3 launchDims = getSortFullDims(xLength);

      // Determine the maximum window size
      int max = 2, dg = 0;
      while (max < xLength) {
        max <<= 1;
        dg++;
      }
      max <<= 1;

      // Perform bitonic sort steps for arbitrary window sizes
      for (int window = 2; window < max; window <<= 1) {
        int n = window;
        int rev = 0;
        do {
          int half = n >> 1;
          BUILD_SINGLE_SELECTOR(xType, bitonicArbitraryStepGeneric,
                                (launchDims, stream, x->specialBuffer(), dXShapeInfo, n, xLength, rev, descending),
                                SD_COMMON_TYPES);
          n >>= 1;
          rev = 1;
        } while (n > 1);
      }
    }

    // Check for CUDA errors after sort execution
    DebugHelper::checkErrorCode(stream, "sort(...) failed");
  } catch (std::exception &e) {
    // Handle exceptions by setting error codes and messages in the LaunchContext
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}


void sortByKey(Pointer *extraPointers, NDArray *x,
               NDArray *y, bool descending) {
  try {
    // Retrieve the CUDA stream from extraPointers[1]
    hipStream_t *stream = nullptr;
    if (extraPointers != nullptr && extraPointers[1] != nullptr) {
      stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
    } else {
      // If no stream is provided, use the default stream from LaunchContext
      stream = LaunchContext::defaultContext()->getCudaStream();
    }

    // Extract shape information from NDArray* objects
    const LongType *xShapeInfo = x->shapeInfo();
    const LongType *dXShapeInfo = x->specialShapeInfo();
    const LongType *yShapeInfo = y->shapeInfo();
    const LongType *dyShapeInfo = y->specialShapeInfo();

    // Calculate the lengths of the arrays
    auto xLength = shape::length(xShapeInfo);
    auto yLength = shape::length(yShapeInfo);

    // Get element-wise strides (optional, based on original logic)
    auto xEWS = shape::elementWiseStride(xShapeInfo);

    // Determine the data types of the arrays
    auto xType = ArrayOptions::dataType(xShapeInfo);
    auto yType = ArrayOptions::dataType(yShapeInfo);

    // Check if either array is empty
    if (shape::isEmptyConst(xShapeInfo) || shape::isEmptyConst(yShapeInfo)) return;

    // Ensure that keys and values have the same length
    if (xLength != yLength) THROW_EXCEPTION("sortByKey: keys and values must have the same size");

    // Check if xLength is a power of 2 and within the specified limit
    if ((xLength != 0) && ((xLength & (xLength - 1)) == 0) && (xLength <= 1024 * 1024 * 10)) {
      // Get the launch dimensions for full sort
      dim3 launchDims = getSortFullDims(xLength);

      // Perform bitonic sort steps
      for (int k = 2; k <= xLength; k *= 2) {
        for (int j = k >> 1; j > 0; j >>= 1) {
          BUILD_DOUBLE_SELECTOR(xType, yType, bitonicSortStepGenericKey,
                                (launchDims, stream, x->specialBuffer(),
                                    dXShapeInfo, y->specialBuffer(), dyShapeInfo, j, k, xLength, descending),
                                SD_COMMON_TYPES, SD_COMMON_TYPES);
        }
      }
    } else {
      // Determine the number of threads and blocks
      int numThreads = sd::math::sd_min<int>(512, xLength);
      int numBlocks = xLength / numThreads;
      if (xLength % numThreads > 0 || numBlocks == 0) numBlocks++;
      numBlocks = sd::math::sd_min<int>(512, numBlocks);
      dim3 launchDims(numBlocks, numThreads, 32768);

      // Determine the maximum window size
      int max = 2;
      while (max < xLength) {
        max <<= 1;
      }
      max <<= 1;

      // Perform bitonic sort steps for arbitrary window sizes
      for (int window = 2; window < max; window <<= 1) {
        int n = window;
        int rev = 0;
        do {
          BUILD_DOUBLE_SELECTOR(xType, yType, bitonicArbitraryStepGenericKey,
                                (launchDims, stream, x->specialBuffer(),
                                    dXShapeInfo, y->specialBuffer(), dyShapeInfo, n, xLength, rev, descending),
                                SD_COMMON_TYPES, SD_COMMON_TYPES);
          n >>= 1;
          rev = 1;
        } while (n > 1);
      }
    }

    // Check for CUDA errors after sort execution
    DebugHelper::checkErrorCode(stream, "sortByKey(...) failed");
  }
  catch (std::exception &e) {
    // Handle exceptions by setting error codes and messages in the LaunchContext
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}


void sortByValue(Pointer *extraPointers,OpaqueNDArray x,
                 OpaqueNDArray y, bool descending) {
  try {
    // Retrieve the CUDA stream from extraPointers[1]
    hipStream_t *stream = nullptr;
    if (extraPointers != nullptr && extraPointers[1] != nullptr) {
      stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
    } else {
      // If no stream is provided, use the default stream from LaunchContext
      stream = LaunchContext::defaultContext()->getCudaStream();
    }

    // Extract shape information from NDArray* objects
    const LongType *xShapeInfo = x->shapeInfo();
    const LongType *dXShapeInfo = x->specialShapeInfo();
    const LongType *yShapeInfo = y->shapeInfo();
    const LongType *dyShapeInfo = y->specialShapeInfo();

    // Calculate the lengths of the arrays
    auto xLength = shape::length(xShapeInfo);
    auto yLength = shape::length(yShapeInfo);

    // Get element-wise strides (optional, based on original logic)
    auto xEWS = shape::elementWiseStride(xShapeInfo);

    // Determine the data types of the arrays
    auto xType = ArrayOptions::dataType(yShapeInfo); // Note the swapped types in original code
    auto yType = ArrayOptions::dataType(xShapeInfo);

    // Check if either array is empty
    if (shape::isEmptyConst(xShapeInfo) || shape::isEmptyConst(yShapeInfo)) return;

    // Ensure that keys and values have the same length
    if (xLength != yLength) THROW_EXCEPTION("sortByValue: keys and values must have the same size");

    // Check if xLength is a power of 2 and within the specified limit
    if ((xLength != 0) && ((xLength & (xLength - 1)) == 0) && (xLength <= 1024 * 1024 * 10)) {
      // Get the launch dimensions for full sort
      dim3 launchDims = getSortFullDims(xLength);

      // Perform bitonic sort steps
      for (int k = 2; k <= xLength; k *= 2) {
        for (int j = k >> 1; j > 0; j >>= 1) {
          BUILD_DOUBLE_SELECTOR(xType, yType, bitonicSortStepGenericKey,
                                (launchDims, stream, y->specialBuffer(),
                                    dyShapeInfo, x->specialBuffer(),
                                    dXShapeInfo, j, k, xLength, descending),
                                SD_COMMON_TYPES, SD_COMMON_TYPES);
        }
      }
    } else {
      // Determine the number of threads and blocks
      dim3 launchDims = getSortFullDims(xLength);

      // Determine the maximum window size
      int max = 2;
      while (max < xLength) {
        max <<= 1;
      }
      max <<= 1;

      // Perform bitonic sort steps for arbitrary window sizes
      for (int window = 2; window < max; window <<= 1) {
        int n = window;
        int rev = 0;
        do {
          BUILD_DOUBLE_SELECTOR(xType, yType, bitonicArbitraryStepGenericKey,
                                (launchDims, stream, y->specialBuffer(),
                                    dyShapeInfo, x->specialBuffer(), dXShapeInfo, n, xLength, rev, descending),
                                SD_COMMON_TYPES, SD_COMMON_TYPES);
          n >>= 1;
          rev = 1;
        } while (n > 1);
      }
    }

    // Check for CUDA errors after sort execution
    DebugHelper::checkErrorCode(stream, "sortByValue(...) failed");
  }
  catch (std::exception &e) {
    // Handle exceptions by setting error codes and messages in the LaunchContext
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}


void sortTadByKey(Pointer *extraPointers,
                  OpaqueNDArray x,
                  OpaqueNDArray y,
                  OpaqueNDArray dimension,
                  bool descending) {
  try {
    // Retrieve the CUDA stream from extraPointers[1]
    hipStream_t *stream = nullptr;
    if (extraPointers != nullptr && extraPointers[1] != nullptr) {
      stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
    } else {
      // If no stream is provided, use the default stream from LaunchContext
      stream = LaunchContext::defaultContext()->getCudaStream();
    }

    // Extract shape information from NDArray* objects
    const LongType *xShapeInfo = x->shapeInfo();
    const LongType *dXShapeInfo = x->specialShapeInfo();
    const LongType *yShapeInfo = y->shapeInfo();
    const LongType *dyShapeInfo = y->specialShapeInfo();

    // Determine the data types of the arrays
    auto xType = ArrayOptions::dataType(xShapeInfo);
    auto yType = ArrayOptions::dataType(yShapeInfo);

    // Get the dimension buffer and length
    auto dimensionPtr = reinterpret_cast<LongType *>(dimension->buffer());
    LongType dimensionLength = static_cast<LongType>(shape::length(dimension->shapeInfo()));

    // Get the TAD pack for the given dimensions
    auto tadPack = ConstantTadHelper::getInstance().tadForDimensions(xShapeInfo, dimensionPtr, dimensionLength);

    // Get the number of TADs
    auto numTads = tadPack->numberOfTads();

    // Get the launch dimensions for sorting TADs
    dim3 launchDims = getSortTadDims(numTads);

    // Execute the sortTadByKey operation based on data types
    BUILD_DOUBLE_SELECTOR(xType, yType, oesTadGenericKey,
                          (launchDims, stream, x->specialBuffer(),
                           dXShapeInfo, y->specialBuffer(), dyShapeInfo,
                           dimensionPtr, dimensionLength, tadPack->platformShapeInfo(), tadPack->platformOffsets(), descending),
                          SD_COMMON_TYPES, SD_COMMON_TYPES);

    // Check for CUDA errors after sort execution
    DebugHelper::checkErrorCode(stream, "sortTadByKey(...) failed");
  }
  catch (std::exception &e) {
    // Handle exceptions by setting error codes and messages in the LaunchContext
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}


void sortTadByValue(Pointer *extraPointers,
                    OpaqueNDArray x,
                    OpaqueNDArray y,
                    OpaqueNDArray dimension,
                    bool descending) {
  try {
    // Retrieve the CUDA stream from extraPointers[1]
    hipStream_t *stream = nullptr;
    if (extraPointers != nullptr && extraPointers[1] != nullptr) {
      stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
    } else {
      // If no stream is provided, use the default stream from LaunchContext
      stream = LaunchContext::defaultContext()->getCudaStream();
    }

    // Extract shape information from NDArray* objects
    const LongType *xShapeInfo = x->shapeInfo();
    const LongType *dXShapeInfo = x->specialShapeInfo();
    const LongType *yShapeInfo = y->shapeInfo();
    const LongType *dyShapeInfo = y->specialShapeInfo();

    // Determine the data types of the arrays
    auto xType = ArrayOptions::dataType(yShapeInfo); // Note the swapped types in original code
    auto yType = ArrayOptions::dataType(xShapeInfo);

    // Get the dimension buffer and length
    auto dimensionPtr = reinterpret_cast<LongType *>(dimension->buffer());
    LongType dimensionLength = static_cast<LongType>(shape::length(dimension->shapeInfo()));

    // Get the TAD pack for the given dimensions
    auto tadPack = ConstantTadHelper::getInstance().tadForDimensions(xShapeInfo, dimensionPtr, dimensionLength);

    // Get the number of TADs
    auto numTads = tadPack->numberOfTads();

    // Get the launch dimensions for sorting TADs
    dim3 launchDims = getSortTadDims(numTads);

    // Execute the sortTadByValue operation based on data types
    BUILD_DOUBLE_SELECTOR(xType, yType, oesTadGenericKey,
                          (launchDims, stream, y->specialBuffer(), dyShapeInfo, x->specialBuffer(), dXShapeInfo,
                           dimensionPtr, dimensionLength, tadPack->platformShapeInfo(), tadPack->platformOffsets(), descending),
                          SD_COMMON_TYPES, SD_COMMON_TYPES);

    // Check for CUDA errors after sort execution
    DebugHelper::checkErrorCode(stream, "sortTadByValue(...) failed");
  }
  catch (std::exception &e) {
    // Handle exceptions by setting error codes and messages in the LaunchContext
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}


void sortTad(Pointer *extraPointers, OpaqueNDArray  x,
             LongType *dimension, LongType dimensionLength,
              LongType *tadShapeInfo,  LongType *tadOffsets, bool descending) {
  try {
    // Retrieve the CUDA stream from extraPointers[1]
    hipStream_t *stream = nullptr;
    if (extraPointers != nullptr && extraPointers[1] != nullptr) {
      stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
    } else {
      // If no stream is provided, use the default stream from LaunchContext
      stream = LaunchContext::defaultContext()->getCudaStream();
    }

    // Extract shape information from NDArray* objects
    const LongType *xShapeInfo = x->shapeInfo();
    const LongType *dXShapeInfo = x->specialShapeInfo();

    // Determine the data type of the array
    auto xType = ArrayOptions::dataType(xShapeInfo);

    // Get the TAD pack for the given dimensions
    auto tadPack = ConstantTadHelper::getInstance().tadForDimensions(xShapeInfo, dimension, dimensionLength);

    // Get the number of TADs
    auto numTads = tadPack->numberOfTads();

    // Get the launch dimensions for sorting TADs
    dim3 launchDims = getSortTadLarge(numTads);

    // Execute the sortTad operation based on data type
    BUILD_SINGLE_SELECTOR(
        xType, oesTadGeneric,
        (launchDims, stream, x->specialBuffer(), dXShapeInfo, dimension, dimensionLength, tadShapeInfo, tadOffsets, descending),
        SD_COMMON_TYPES
    );

    // Check for CUDA errors after sort execution
    DebugHelper::checkErrorCode(stream, "sortTad(...) failed");
  }
  catch (std::exception &e) {
    // Handle exceptions by setting error codes and messages in the LaunchContext
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}


void sortCooIndices(sd::Pointer *extraPointers, OpaqueNDArray indices, OpaqueNDArray values) {
  THROW_EXCEPTION("sortCooIndices:: Not implemented yet");
}

void ravelMultiIndex(Pointer *extraPointers, NDArray *indices, NDArray *flatIndices,
                     NDArray *shapeInfo, int mode) {
  try {
    NativeOpExecutioner::execRavelMultiIndex(indices->bufferAsT<LongType>(),
                                             flatIndices->bufferAsT<LongType>(),
                                             flatIndices->lengthOf(),
                                             shapeInfo->bufferAsT<LongType>(), mode);
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}

void unravelIndex(Pointer *extraPointers, NDArray *indices, NDArray *flatIndices,
                  NDArray *shapeInfo) {
  try {
    NativeOpExecutioner::execUnravelIndex(indices->bufferAsT<LongType>(),
                                          flatIndices->bufferAsT<LongType>(),
                                          flatIndices->lengthOf(),
                                          shapeInfo->bufferAsT<LongType>());
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}


/*
 * TypeDef:
 *     void convertTypes(sd::Pointer *extras, int srcType, sd::Pointer dX, long N, int dstType, sd::Pointer dZ);
 */
void convertTypes(Pointer *extras, int srcType, Pointer dX, LongType N, int dstType, Pointer dZ) {
  try {
    auto dx = reinterpret_cast<void *>(dX);
    auto dz = reinterpret_cast<void *>(dZ);

    if (srcType == ND4J_FLOAT8) {
      if (dstType == ND4J_FLOAT8) {
        // convertKernel<double, sd::float8>(extras, dx, N, dz);
      } else if (dstType == ND4J_INT8) {
        // sd::TypeCast::convertGenericCuda<sd::float8, sd::int8>(extras, dx, N, dz);
      } else if (dstType == ND4J_UINT8) {
        // sd::TypeCast::convertGenericCuda<sd::float8, sd::uint8>(extras, dx, N, dz);
      } else if (dstType == ND4J_FLOAT16) {
        // sd::TypeCast::convertGenericCuda<sd::float8, float16>(extras, dx, N, dz);
      } else if (dstType == ND4J_INT16) {
        // sd::TypeCast::convertGenericCuda<sd::float8, sd::int16>(extras, dx, N, dz);
      } else if (dstType == ND4J_UINT16) {
        // sd::TypeCast::convertGenericCuda<sd::float8, sd::uint16>(extras, dx, N, dz);
      } else if (dstType == ND4J_FLOAT24) {
      } else if (dstType == ND4J_FLOAT32) {
        // sd::TypeCast::convertGenericCuda<sd::float8, float>(extras, dx, N, dz);
      } else if (dstType == ND4J_DOUBLE) {
        // sd::TypeCast::convertGenericCuda<sd::float8, double>(extras, dx, N, dz);
      } else {
        sd_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
      }
    } else if (srcType == ND4J_INT8) {
      if (dstType == ND4J_FLOAT8) {
        // sd::TypeCast::convertGenericCuda<sd::int8, sd::float8>(extras, dx, N, dz);
      } else if (dstType == ND4J_INT8) {
        // convertKernel<sd::int8, sd::int8>(extras, dx, N, dz);
      } else if (dstType == ND4J_UINT8) {
        TypeCast::convertGenericCuda<int8_t, uint8_t>(extras, dx, N, dz);
      } else if (dstType == ND4J_FLOAT16) {
        TypeCast::convertGenericCuda<int8_t, float16>(extras, dx, N, dz);
      } else if (dstType == ND4J_INT16) {
        TypeCast::convertGenericCuda<int8_t, int16_t>(extras, dx, N, dz);
      } else if (dstType == ND4J_UINT16) {
        TypeCast::convertGenericCuda<int8_t, uint16_t>(extras, dx, N, dz);
      } else if (dstType == ND4J_FLOAT24) {
        // TODO: eventually we might want to add it
      } else if (dstType == ND4J_FLOAT32) {
        TypeCast::convertGenericCuda<int8_t, float>(extras, dx, N, dz);
      } else if (dstType == ND4J_DOUBLE) {
        TypeCast::convertGenericCuda<int8_t, double>(extras, dx, N, dz);
      } else {
        sd_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
      }
    } else if (srcType == ND4J_UINT8) {
      if (dstType == ND4J_FLOAT8) {
        // sd::TypeCast::convertGenericCuda<uint8_t, sd::float8>(extras, dx, N, dz);
      } else if (dstType == ND4J_INT8) {
        TypeCast::convertGenericCuda<uint8_t, int8_t>(extras, dx, N, dz);
      } else if (dstType == ND4J_UINT8) {
        TypeCast::convertGenericCuda<uint8_t, uint8_t>(extras, dx, N, dz);
      } else if (dstType == ND4J_FLOAT16) {
        TypeCast::convertGenericCuda<uint8_t, float16>(extras, dx, N, dz);
      } else if (dstType == ND4J_INT16) {
        TypeCast::convertGenericCuda<uint8_t, int16_t>(extras, dx, N, dz);
      } else if (dstType == ND4J_UINT16) {
        TypeCast::convertGenericCuda<uint8_t, uint16_t>(extras, dx, N, dz);
      } else if (dstType == ND4J_FLOAT24) {
        // TODO: still might want to add
      } else if (dstType == ND4J_FLOAT32) {
        TypeCast::convertGenericCuda<uint8_t, float>(extras, dx, N, dz);
      } else if (dstType == ND4J_DOUBLE) {
        TypeCast::convertGenericCuda<uint8_t, double>(extras, dx, N, dz);
      } else {
        sd_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
      }
    } else if (srcType == ND4J_FLOAT16) {
      if (dstType == ND4J_FLOAT8) {
        // sd::TypeCast::convertGenericCuda<float16, sd::float8>(extras, dx, N, dz);
      } else if (dstType == ND4J_INT8) {
        TypeCast::convertGenericCuda<float16, int8_t>(extras, dx, N, dz);
      } else if (dstType == ND4J_UINT8) {
        TypeCast::convertGenericCuda<float16, uint8_t>(extras, dx, N, dz);
      } else if (dstType == ND4J_FLOAT16) {
        TypeCast::convertGenericCuda<float16, float16>(extras, dx, N, dz);
      } else if (dstType == ND4J_INT16) {
        TypeCast::convertGenericCuda<float16, int16_t>(extras, dx, N, dz);
      } else if (dstType == ND4J_UINT16) {
        TypeCast::convertGenericCuda<float16, uint16_t>(extras, dx, N, dz);
      } else if (dstType == ND4J_FLOAT24) {
        // TODO: .... ^^^
      } else if (dstType == ND4J_FLOAT32) {
        TypeCast::convertGenericCuda<float16, float>(extras, dx, N, dz);
      } else if (dstType == ND4J_DOUBLE) {
        TypeCast::convertGenericCuda<float16, double>(extras, dx, N, dz);
      } else if (dstType == ND4J_THRESHOLD) {
        // sd::convertToThreshold<float16>(nullptr, dx, N, dz);
      } else {
        sd_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
      }
    } else if (srcType == ND4J_INT16) {
      if (dstType == ND4J_FLOAT8) {
        // sd::TypeCast::convertGenericCuda<int16_t, sd::float8>(extras, dx, N, dz);
      } else if (dstType == ND4J_INT8) {
        TypeCast::convertGenericCuda<int16_t, int8_t>(extras, dx, N, dz);
      } else if (dstType == ND4J_UINT8) {
        TypeCast::convertGenericCuda<int16_t, uint8_t>(extras, dx, N, dz);
      } else if (dstType == ND4J_FLOAT16) {
        TypeCast::convertGenericCuda<int16_t, float16>(extras, dx, N, dz);
      } else if (dstType == ND4J_INT16) {
        TypeCast::convertGenericCuda<int16_t, int16_t>(extras, dx, N, dz);
      } else if (dstType == ND4J_UINT16) {
        TypeCast::convertGenericCuda<int16_t, uint16_t>(extras, dx, N, dz);
      } else if (dstType == ND4J_FLOAT24) {
        // TODO...
      } else if (dstType == ND4J_FLOAT32) {
        TypeCast::convertGenericCuda<int16_t, float>(extras, dx, N, dz);
      } else if (dstType == ND4J_DOUBLE) {
        TypeCast::convertGenericCuda<int16_t, double>(extras, dx, N, dz);
      } else {
        printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
      }
    } else if (srcType == ND4J_FLOAT24) {
    } else if (srcType == ND4J_FLOAT32) {
      if (dstType == ND4J_FLOAT8) {
        // sd::TypeCast::convertGenericCuda<float, sd::float8>(extras, dx, N, dz);
      } else if (dstType == ND4J_INT8) {
        TypeCast::convertGenericCuda<float, int8_t>(extras, dx, N, dz);
      } else if (dstType == ND4J_UINT8) {
        TypeCast::convertGenericCuda<float, uint8_t>(extras, dx, N, dz);
      } else if (dstType == ND4J_FLOAT16) {
        TypeCast::convertGenericCuda<float, float16>(extras, dx, N, dz);
      } else if (dstType == ND4J_INT16) {
        TypeCast::convertGenericCuda<float, int16_t>(extras, dx, N, dz);
      } else if (dstType == ND4J_UINT16) {
        TypeCast::convertGenericCuda<float, uint16_t>(extras, dx, N, dz);
      } else if (dstType == ND4J_FLOAT24) {
      } else if (dstType == ND4J_DOUBLE) {
        TypeCast::convertGenericCuda<float, double>(extras, dx, N, dz);
      } else if (dstType == ND4J_THRESHOLD) {
        // sd::convertToThreshold<float>(nullptr, dx, N, dz);
      } else {
        sd_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
      }
    } else if (srcType == ND4J_DOUBLE) {
      if (dstType == ND4J_FLOAT8) {
        // sd::TypeCast::convertGenericCuda<double, sd::float8>(extras, dx, N, dz);
      } else if (dstType == ND4J_INT8) {
        TypeCast::convertGenericCuda<double, int8_t>(extras, dx, N, dz);
      } else if (dstType == ND4J_UINT8) {
        TypeCast::convertGenericCuda<double, uint8_t>(extras, dx, N, dz);
      } else if (dstType == ND4J_FLOAT16) {
        TypeCast::convertGenericCuda<double, float16>(extras, dx, N, dz);
      } else if (dstType == ND4J_INT16) {
        TypeCast::convertGenericCuda<double, int16_t>(extras, dx, N, dz);
      } else if (dstType == ND4J_UINT16) {
        TypeCast::convertGenericCuda<double, uint16_t>(extras, dx, N, dz);
      } else if (dstType == ND4J_FLOAT24) {
      } else if (dstType == ND4J_FLOAT32) {
        TypeCast::convertGenericCuda<double, float>(extras, dx, N, dz);
      } else if (dstType == ND4J_DOUBLE) {
        //
      } else if (dstType == ND4J_THRESHOLD) {
        // sd::convertToThreshold<double>(nullptr, dx, N, dz);
      } else {
        sd_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
      }
    } else if (srcType == ND4J_THRESHOLD) {
      if (dstType == ND4J_FLOAT16) {
        // sd::convertFromThreshold<float16>(nullptr, dx, N, dz);
      } else if (dstType == ND4J_FLOAT32) {
        // sd::convertFromThreshold<float>(nullptr, dx, N, dz);
      } else if (dstType == ND4J_DOUBLE) {
        // sd::convertFromThreshold<double>(nullptr, dx, N, dz);
      } else {
        sd_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
      }
    } else {
      sd_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
    }
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}



///////////////////////////////////////////////////////////////////
template <typename T, typename I>
SD_KERNEL static void scatterUpdateCuda(const int opCode, const int numOfSubArrs, void *vx,
                                        const LongType *xShapeInfo, const LongType *xOffsets, void *vy,
                                        const LongType *yShapeInfo, const LongType *yOffsets,
                                        const void *vindexes) {
  __shared__ T *x, *y;
  __shared__ LongType arrLenX, arrLenY;
  auto indexes = reinterpret_cast<const I *>(vindexes);

  for (int e = 0; e < numOfSubArrs; e++) {
    const auto xIndex = indexes[e];
    const bool isOwner = xIndex < gridDim.x ? blockIdx.x == xIndex : blockIdx.x == xIndex % gridDim.x;

    if (!isOwner) continue;

    if (threadIdx.x == 0) {
      x = reinterpret_cast<T *>(vx) + xOffsets[xIndex];
      y = reinterpret_cast<T *>(vy) + yOffsets[e];
      arrLenX = shape::length(xShapeInfo);
      arrLenY = shape::length(yShapeInfo);
    }
    __syncthreads();

    if (arrLenX != arrLenY) return;

    for (LongType i = threadIdx.x; i < arrLenX; i += blockDim.x) {
      const auto xOffset = shape::getIndexOffset(i, xShapeInfo);
      const auto yOffset = shape::getIndexOffset(i, yShapeInfo);

      switch (opCode) {
        case 0:
          x[xOffset] += y[yOffset];
          break;
        case 1:
          x[xOffset] -= y[yOffset];
          break;
        case 2:
          x[xOffset] *= y[yOffset];
          break;
        case 3:
          x[xOffset] /= y[yOffset];
          break;
        case 4:
          x[xOffset] = y[yOffset] - x[xOffset];
          break;
        case 5:
          x[xOffset] = y[yOffset] / x[xOffset];
          break;
        case 6:
          x[xOffset] = y[yOffset];
          break;
        default:
          continue;
      }
    }
    __syncthreads();
  }
}

template <typename T, typename I>
SD_HOST static void scatterUpdateCudaLauncher(const hipStream_t *stream,
                                              int opCode, OpaqueNDArray array, OpaqueNDArray indices, OpaqueNDArray updates, LongType *axis, LongType axisLength) {
    // Calculate TADs for x and y
    auto tadPackX = sd::ConstantTadHelper::getInstance().tadForDimensions(array->shapeInfo(),axis, axisLength);
    auto tadPackY = sd::ConstantTadHelper::getInstance().tadForDimensions(updates->shapeInfo(), axis,axisLength);

    auto xTadOffsets = tadPackX->specialOffsets();
    auto yTadOffsets = tadPackY->specialOffsets();

    scatterUpdateCuda<T, I><<<512, 256, SD_MAX_NUM_THREADS, *stream>>>(
        opCode, tadPackX->numberOfTads(), array->specialBuffer(), tadPackX->specialShapeInfo(), xTadOffsets, updates->specialBuffer(),
        tadPackY->specialShapeInfo(), yTadOffsets, indices->specialBuffer());
}
//////////////////////////////////////////////////////////////////////////
void scatterUpdate(Pointer *extraPointers, int opCode, OpaqueNDArray array, OpaqueNDArray indices, OpaqueNDArray updates, OpaqueNDArray axis) {
  try {
    auto stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);

    auto type = ArrayOptions::dataType(array->shapeInfo());
    auto iType = ArrayOptions::dataType(indices->shapeInfo());

    // Extract axis data
    LongType* axisData = reinterpret_cast<LongType*>(axis->buffer());
    LongType axisLength = shape::length(axis->shapeInfo());

    BUILD_DOUBLE_SELECTOR(
        type, iType, scatterUpdateCudaLauncher,
        (stream, opCode, array, indices, updates, axisData, axisLength),
        SD_COMMON_TYPES, SD_INDEXING_TYPES);

    DebugHelper::checkErrorCode(stream, "scatterUpdate(...) failed");
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}

void SD_KERNEL tryPointerKernel(void *p, int len) {
  auto buf = reinterpret_cast<int8_t *>(p);
  auto tid = threadIdx.x + blockIdx.x * blockDim.x;
  __shared__ int b;
  if (tid < len) atomicAdd(&b, buf[tid]);

  __syncthreads();

}

void tryPointer(Pointer extra, Pointer p, int len) {
  try {
    hipStream_t stream;
    hipStreamCreate(&stream);

    tryPointerKernel<<<256, 512, len + 64, stream>>>(p, len);
    DebugHelper::checkGlobalErrorCode("try pointer failed(...) failed");

    auto e = hipStreamSynchronize(stream);

    if (e != 0) throw cuda_exception::build("tryPointer failed", e);

    hipStreamDestroy(stream);
  } catch (std::exception &e) {
    LaunchContext::defaultContext()->errorReference()->setErrorCode(1);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage(e.what());
  }
}



bool isBlasVersionMatches(int major, int minor, int build) {
  auto result = major == Environment::getInstance()._blasMajorVersion &&
                minor == Environment::getInstance()._blasMinorVersion &&
                build == Environment::getInstance()._blasPatchVersion;

  if (!result) {
    sd_printf("CUDA/cuBLAS version mismatch. Expected: %i.%i.%i but got %i.%i.%i instead\n",
              Environment::getInstance()._blasMajorVersion, Environment::getInstance()._blasMinorVersion,
              Environment::getInstance()._blasPatchVersion, major, minor, build);
    LaunchContext::defaultContext()->errorReference()->setErrorCode(152);
    LaunchContext::defaultContext()->errorReference()->setErrorMessage("CUDA/cuBLAS version mismatch");
  }

  return result;
}


void setGraphContextCudaContext(Context *ptr, void *stream, void *reductionPointer,
                                void *allocationPointer) {
  ptr->setCudaContext(stream, reductionPointer, allocationPointer);
}




int binaryLevel() { return 0; }

int optimalLevel() { return 0; }

bool isMinimalRequirementsMet() { return true; }

bool isOptimalRequirementsMet() { return true; }








void setShapeBuffer(LongType *inputShapeData,DataType dt,LongType *bufferToSet,char order,int elementWiseStride,bool isEmpty,bool isView) {
  if(inputShapeData == nullptr)
    THROW_EXCEPTION("setShapeBuffer: inputShapeData is null");

  if(bufferToSet == nullptr)
    THROW_EXCEPTION("setShapeBuffer: bufferToSet is null");
  LongType  rank = inputShapeData[0];
  if(rank > SD_MAX_RANK || rank < 0)
    THROW_EXCEPTION("Invalid rank for shape buffer.");
  std::vector<LongType> shape;
  std::vector<LongType> strides;
  //shape, stride, data type
  for(LongType i = 1; i < rank * 2 + 1; i++) {
    if(i <= rank) {
      shape.push_back(inputShapeData[i]);
    } else if(shape.size() == rank) {
      strides.push_back(inputShapeData[i]);
    }
  }


  auto len = shape::shapeInfoLength(rank);
  for(int i = 0; i < len; i++) {
    bufferToSet[i] = inputShapeData[i];
  }

  ArrayOptions::setDataType(bufferToSet,dt);
  if(isView) {
    ArrayOptions::toggleIsView(bufferToSet);
  }
  if(!ArrayOptions::isEmpty(inputShapeData) && isEmpty) {
    ArrayOptions::toggleIsEmpty(bufferToSet);
  }


  if(rank == 0) {
    //detect when the shape buffer values are unset.
    auto len = shape::shapeInfoLength(rank);
    //min number of values in a shape info buffer
    bool allZero = true;
    for(int i = 0; i < len; i++) {
      if(bufferToSet[i] != 0) {
        allZero = false;
        break;
      }
    }

    if(allZero) {
      THROW_EXCEPTION("Found shape buffer with all zero values. Values likely unset.");
    }
  }

}
