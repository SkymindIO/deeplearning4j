#include "hip/hip_runtime.h"
/*
 *  ******************************************************************************
 *  *
 *  *
 *  * This program and the accompanying materials are made available under the
 *  * terms of the Apache License, Version 2.0 which is available at
 *  * https://www.apache.org/licenses/LICENSE-2.0.
 *  *
 *  * See the NOTICE file distributed with this work for additional
 *  * information regarding copyright ownership.
 *  * Unless required by applicable law or agreed to in writing, software
 *  * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 *  * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 *  * License for the specific language governing permissions and limitations
 *  * under the License.
 *  *
 *  * SPDX-License-Identifier: Apache-2.0
 *  *****************************************************************************
 */

//
// @author raver119@gmail.com
// @author Yurii Shyrma (iuriish@yahoo.com)
//
#include <hipblas.h>
#include <exceptions/cuda_exception.h>
#include <helpers/PointersManager.h>
#include <helpers/ShapeUtils.h>
#include <ops/specials_cuda.h>

#include <numeric>

#include "../MmulHelper.h"
#include "execution/cuda/LaunchDims.h"

namespace sd {

//////////////////////////////////////////////////////////////////////////////
// MXK x KxN = MxN              -> actual sequence of axes doesn't matter
template <typename T1, typename T2, typename T3>
static SD_KERNEL void usualCudaGemm(const void* vA, const LongType* aShapeInfo, const void* vB,
                                    const LongType* bShapeInfo, void* vC, const LongType* cShapeInfo,
                                    const int aMaxis, const int aKaxis, const int bKaxis, const int bNaxis,
                                    const int cMaxis, const int cNaxis, const double alpha, const double beta) {
  const T1* A = reinterpret_cast<const T1*>(vA);
  const T2* B = reinterpret_cast<const T2*>(vB);
  T3* C = reinterpret_cast<T3*>(vC);

  __shared__ LongType K, *coords;
  __shared__ bool betaPresent;
  __shared__ LongType cLen, totalThreads;
  __shared__ T3 alphaZ, betaZ;

  if (threadIdx.x == 0) {
    extern __shared__ unsigned char shmem[];
    coords = reinterpret_cast<LongType*>(shmem);
    cLen = shape::length(cShapeInfo);

    K = shape::shapeOf(const_cast<LongType*>(aShapeInfo))[aKaxis];

    betaPresent = beta;

    totalThreads = gridDim.x * blockDim.x;

    alphaZ = alpha;
    betaZ = beta;
  }
  __syncthreads();

  auto aCoords = coords + threadIdx.x * 6;  // 6 = (aRank + bRank + cRank)
  auto bCoords = aCoords + 2;
  auto cCoords = bCoords + 2;

  const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

  for (LongType i = tid; i < cLen; i += totalThreads) {
    // evaluate C coordinates
    shape::index2coords(i, cShapeInfo, cCoords);

    // evaluate A coordinates
    aCoords[aMaxis] = cCoords[cMaxis];
    aCoords[aKaxis] = 0;

    // evaluate B coordinates
    bCoords[bKaxis] = 0;
    bCoords[bNaxis] = cCoords[cNaxis];

    auto aOffset = shape::getOffset(aShapeInfo, aCoords);
    auto bOffset = shape::getOffset(bShapeInfo, bCoords);

    T3 val = A[aOffset] * B[bOffset];  // first iteration

    for (LongType j = 1; j < K; ++j) {  // rest iterations
      aOffset += shape::stride(aShapeInfo)[aKaxis];
      bOffset += shape::stride(bShapeInfo)[bKaxis];
      val = val + A[aOffset] * B[bOffset];
    }

    auto cOffset = shape::getOffset(cShapeInfo, cCoords);

    if (betaPresent)
      C[cOffset] = alphaZ * val + betaZ * C[cOffset];
    else
      C[cOffset] = alphaZ * val;
  }
}

////////////////////////////////////////////////////////////////////////
template <typename T1, typename T2, typename T3>
SD_HOST static void usualGemm(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem,
                              hipStream_t* stream, const void* vA, const LongType* aShapeInfo, const void* vB,
                              const LongType* bShapeInfo, void* vC, const LongType* cShapeInfo,
                              const int aMaxis, const int aKaxis, const int bKaxis, const int bNaxis, const int cMaxis,
                              const int cNaxis, const double alpha, const double beta) {
  usualCudaGemm<T1, T2, T3><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(
      vA, aShapeInfo, vB, bShapeInfo, vC, cShapeInfo, aMaxis, aKaxis, bKaxis, bNaxis, cMaxis, cNaxis, alpha, beta);
  DebugHelper::checkGlobalErrorCode("MMUL cuda gemm failed(...) failed");

}

////////////////////////////////////////////////////////////////////////
// MXN x N = M  -> actual sequence of {M,N} axes doesn't matter
template <typename T1, typename T2, typename T3>
static SD_KERNEL void usualCudaGemv(const void* vA, const LongType* aShapeInfo, const void* vX,
                                    const LongType* xShapeInfo, void* vY, const LongType* yShapeInfo,
                                    const int incx, const int incy, const int aMaxis, const double alpha,
                                    const double beta) {
  const T1* A = reinterpret_cast<const T1*>(vA);
  const T2* X = reinterpret_cast<const T2*>(vX);
  T3* Y = reinterpret_cast<T3*>(vY);

  __shared__ int M, N;
  __shared__ bool betaPresent;
  __shared__ LongType cLen, totalThreads, aNstride, aMstride;
  __shared__ T3 alphaZ, betaZ;

  if (threadIdx.x == 0) {
    N = shape::length(xShapeInfo);
    M = shape::length(yShapeInfo);

    aMstride = shape::stride(aShapeInfo)[aMaxis];
    aNstride = shape::stride(aShapeInfo)[aMaxis == 0 ? 1 : 0];

    totalThreads = gridDim.x * blockDim.x;

    betaPresent = beta;

    alphaZ = alpha;
    betaZ = beta;
  }
  __syncthreads();

  const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

  for (LongType i = tid; i < M; i += totalThreads) {
    // evaluate offsets
    auto aOffset = i * aMstride;
    auto xOffset = 0;

    T3 val = A[aOffset] * X[xOffset];  // first iteration

    for (LongType j = 1; j < N; ++j) {  // rest iterations
      aOffset += aNstride;
      xOffset += incx;
      val = val + A[aOffset] * X[xOffset];
    }

    auto yOffset = i * incy;

    if (betaPresent)
      Y[yOffset] = alphaZ * val + betaZ * Y[yOffset];
    else
      Y[yOffset] = alphaZ * val;
  }
}

////////////////////////////////////////////////////////////////////////
template <typename T1, typename T2, typename T3>
SD_HOST static void usualGemv(const int blocksPerGrid, const int threadsPerBlock, hipStream_t* stream, const void* vA,
                              const LongType* aShapeInfo, const void* vX, const LongType* xShapeInfo, void* vY,
                              const LongType* yShapeInfo, const int incx, const int incy, const int aMaxis,
                              const double alpha, const double beta) {
  usualCudaGemv<T1, T2, T3><<<blocksPerGrid, threadsPerBlock, 512, *stream>>>(
      vA, aShapeInfo, vX, xShapeInfo, vY, yShapeInfo, incx, incy, aMaxis, alpha, beta);
  DebugHelper::checkGlobalErrorCode("MMUL cuda gemv case failed(...) failed");

}

//////////////////////////////////////////////////////////////////////////////
template <typename T1, typename T2, typename T3>
static SD_KERNEL void usualCudaDot(const LongType length, const double alpha, const void* vX,
                                   const LongType incx, const void* vY, const LongType incy, const double beta,
                                   void* vZ) {
  T1* X = reinterpret_cast<T1*>(const_cast<void*>(vX));
  T2* Y = reinterpret_cast<T2*>(const_cast<void*>(vY));
  T3* Z = reinterpret_cast<T3*>(vZ);

  extern __shared__ unsigned char shmem[];
  auto pairwiseMul = reinterpret_cast<T3*>(shmem);

  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < length) pairwiseMul[tid] = X[tid * incx] * Y[tid * incy];

  __syncthreads();

  if (tid == 0) {
    T3 sum = 0;
    for (LongType i = 0; i < length; ++i) sum = sum + pairwiseMul[i];

    if (beta)
      *Z = (T3)alpha * sum + (T3)beta * *Z;
    else
      *Z = (T3)alpha * sum;
  }
}

////////////////////////////////////////////////////////////////////////
template <typename T1, typename T2, typename T3>
SD_HOST static void usualDot(const dim3& launchDims, hipStream_t* stream,
                             const LongType length, const double alpha, const void* vX, const LongType incx,
                             const void* vY, const LongType incy, const double beta, void* vZ) {
  usualCudaDot<T1, T2, T3><<<launchDims.x, launchDims.y,launchDims.z, *stream>>>(
      length, alpha, vX, incx, vY, incy, beta, vZ);
  DebugHelper::checkGlobalErrorCode("concat dot failed(...) failed");

}

//////////////////////////////////////////////////////////////////////////////
// MXK x KxN = MxN
NDArray* MmulHelper::mmulMxM(const NDArray* A, const NDArray* B, NDArray* C, double alpha, double beta,
                             const char outOrder) {
  if (A->rankOf() != 2) THROW_EXCEPTION("MmulHelper::mmulMxM cuda: rank of A array is not equal 2 !");
  if (B->rankOf() != 2) THROW_EXCEPTION("MmulHelper::mmulMxM cuda: rank of B array is not equal 2 !");

  const auto M = A->sizeAt(0);
  const auto K = A->sizeAt(1);
  const auto N = B->sizeAt(1);

  if (C != nullptr && C->rankOf() != 2)
    THROW_EXCEPTION("MmulHelper::mmulMxM cuda: rank of C array is not equal 2 !");
  if (B->sizeAt(0) != K) THROW_EXCEPTION("MmulHelper::mmulMxM cuda: B array has wrong number of rows !");
  if (C != nullptr && C->sizeAt(0) != M)
    THROW_EXCEPTION("MmulHelper::mmulMxM cuda: C array has wrong number of rows !");
  if (C != nullptr && C->sizeAt(1) != N)
    THROW_EXCEPTION("MmulHelper::mmulMxM cuda: C array has wrong number of columns !");

  std::vector<LongType> cShape = {M, N};
  if (C == nullptr)
    C = new NDArray(outOrder, cShape, DataTypeUtils::pickPairwiseResultType(A->dataType(), B->dataType()),
                    A->getContext());

  if (C->isEmpty()) return C;

  const int major = Environment::getInstance().capabilities()[AffinityManager::currentDeviceId()].first();

  const auto aType = A->dataType();
  const auto bType = B->dataType();
  const auto cType = C->dataType();

  const bool AB(aType == bType), AC(aType == cType), ABC(AB && AC);

  const bool typeDouble = ABC && aType == DOUBLE;
  const bool typeFloat = ABC && aType == FLOAT32;
  const bool typeHalf = ABC && aType == HALF && major >= 6;
  const bool typeIntFloat = AB && aType == INT8 && cType == FLOAT32 && major >= 6;
  const bool typeHalfFloat = AB && aType == HALF && cType == FLOAT32 && major >= 6;

  std::lock_guard<std::mutex> lock(*LaunchContext::deviceMutex());

  auto handle = reinterpret_cast<hipblasHandle_t*>(A->getContext()->getCublasHandle());
  auto stream = A->getContext()->getCudaStream();

  auto status = hipblasSetStream(*handle, *stream);
  if (status != HIPBLAS_STATUS_SUCCESS) throw cuda_exception::build("MmulHelper::mmulMxM cuda failed !", status);

  if (!typeDouble && !typeFloat && !typeHalf && !typeIntFloat && !typeHalfFloat) {
    dim3 dims = getMMulDims(C->lengthOf(),DataTypeUtils::sizeOf(cType));
    NDArray::prepareSpecialUse({C}, {A, B});
    BUILD_SINGLE_SELECTOR_THRICE(aType, usualGemm,
                                 (dims.y, dims.x, dims.z, stream, A->specialBuffer(),
                                     A->specialShapeInfo(), B->specialBuffer(), B->specialShapeInfo(), C->specialBuffer(),
                                     C->specialShapeInfo(), 0, 1, 0, 1, 0, 1, alpha, beta),
                                 SD_NUMERIC_TYPES)
    NDArray::registerSpecialUse({C}, {A, B});

    auto cudaResult = hipStreamSynchronize(*stream);
    if (cudaResult != 0) throw cuda_exception::build("MmulHelper::mmulMxM cuda failed !", cudaResult);
  } else {
    std::vector<NDArray*> toDelete;

    NDArray *pA(const_cast<NDArray*>(A)), *pB(const_cast<NDArray*>(B)), *pC(const_cast<NDArray*>(C));

    bool aMcont = M == 1 || A->strideAt(0) == 1;
    bool aKcont = K == 1 || A->strideAt(1) == 1;
    bool bKcont = K == 1 || B->strideAt(0) == 1;
    bool bNcont = N == 1 || B->strideAt(1) == 1;
    bool cMcont = M == 1 || C->strideAt(0) == 1;
    bool cNcont = N == 1 || C->strideAt(1) == 1;

    if (!aMcont && !aKcont) {
      pA = new NDArray(A->dup('f'));
      toDelete.push_back(pA);
      aMcont = true;
    }
    if (!bKcont && !bNcont) {
      pB = new NDArray(B->dup('f'));
      toDelete.push_back(pB);
      bKcont = true;
    }
    if (!cMcont) {
      pC = new NDArray(C->dup('f'));
      toDelete.push_back(pC);
      cMcont = true;
    }

    const bool transA = !aMcont;
    const bool transB = !bKcont;

    const int lda = (aMcont && aKcont) ? M : transA ? pA->strideAt(0) : pA->strideAt(1);
    const int ldb = (bKcont && bNcont) ? K : transB ? pB->strideAt(0) : pB->strideAt(1);
    const int ldc = (cMcont && cNcont) ? M : pC->strideAt(1);

    const hipblasOperation_t transAblas = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    const hipblasOperation_t transBblas = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    NDArray::prepareSpecialUse({pC}, {pA, pB});

    // choose appropriate cuda gemm api depending on data types
    if (typeDouble) {
      status = hipblasDgemm(*handle, transAblas, transBblas, M, N, K, &alpha, (double*)pA->specialBuffer(), lda,
                           (double*)pB->specialBuffer(), ldb, &beta, (double*)pC->specialBuffer(), ldc);
    } else if (typeFloat) {
      float alphaF(alpha), betaF(beta);
      status = hipblasSgemm(*handle, transAblas, transBblas, M, N, K, &alphaF, (float*)pA->specialBuffer(), lda,
                           (float*)pB->specialBuffer(), ldb, &betaF, (float*)pC->specialBuffer(), ldc);
    } else if (typeHalf) {
      float16 alphaH(alpha), betaH(beta);
      status = hipblasHgemm(*handle, transAblas, transBblas, M, N, K, &alphaH.data, (__half*)pA->specialBuffer(), lda,
                           (__half*)pB->specialBuffer(), ldb, &betaH.data, (__half*)pC->specialBuffer(), ldc);
    } else if (typeIntFloat) {
      float alphaF(alpha), betaF(beta);
      status = cublasSgemmEx(*handle, transAblas, transBblas, M, N, K, &alphaF, pA->specialBuffer(), HIP_R_8I, lda,
                             pB->specialBuffer(), HIP_R_8I, ldb, &betaF, pC->specialBuffer(), HIP_R_32F, ldc);
    } else if (typeHalfFloat) {
      float alphaF(alpha), betaF(beta);
      status = cublasSgemmEx(*handle, transAblas, transBblas, M, N, K, &alphaF, pA->specialBuffer(), HIP_R_16F, lda,
                             pB->specialBuffer(), HIP_R_16F, ldb, &betaF, pC->specialBuffer(), HIP_R_32F, ldc);
    }

    if (status != HIPBLAS_STATUS_SUCCESS) throw cuda_exception::build("MmulHelper::mmulMxM cuda failed !", status);

    NDArray::registerSpecialUse({pC}, {pA, pB});

    auto cudaResult = hipStreamSynchronize(*stream);
    if (cudaResult != 0) throw cuda_exception::build("MmulHelper::mmulMxM cuda failed !", cudaResult);

    if (C != pC) C->assign(pC);

    for (int i = toDelete.size() - 1; i >= 0; --i) delete toDelete[i];
  }

  return C;
}

////////////////////////////////////////////////////////////////////////////
// MXN x N = M
NDArray* MmulHelper::mmulMxV(const NDArray* A, const NDArray* X, NDArray* Y, const double alpha, const double beta,
                             const char outOrder) {
  LongType xLenDim, yLenDim(0);

  if (A->rankOf() != 2) THROW_EXCEPTION("MmulHelper::mmulMxV cuda: rank of A array is not equal 2 !");
  if (!shape::isCommonVector(X->shapeInfo(), xLenDim))
    THROW_EXCEPTION("MmulHelper::mmulMxV cuda: X array must be vector !");

  const auto M = A->sizeAt(0);
  const auto N = A->sizeAt(1);

  if (Y != nullptr && !shape::isCommonVector(Y->shapeInfo(), yLenDim))
    THROW_EXCEPTION("MmulHelper::mmulMxV cuda: Y array must be vector !");
  if (X->lengthOf() != N) THROW_EXCEPTION("MmulHelper::mmulMxV cuda: X vector has wrong length !");
  if (Y != nullptr && Y->lengthOf() != M)
    THROW_EXCEPTION("MmulHelper::mmulMxV cuda: Y array has wrong length !");

  std::vector<LongType> yShape = {M};
  if (Y == nullptr)
    Y = new NDArray(outOrder, yShape, DataTypeUtils::pickPairwiseResultType(A->dataType(), X->dataType()),
                    A->getContext());

  if (Y->isEmpty()) return Y;

  const int incx = X->strideAt(xLenDim);
  const int incy = Y->strideAt(yLenDim);

  const auto aType = A->dataType();
  const auto xType = X->dataType();
  const auto yType = Y->dataType();

  const bool AX(aType == xType), AY(aType == yType), AXY(AX && AY);

  const bool typeDouble = AXY && aType == DOUBLE;
  const bool typeFloat = AXY && aType == FLOAT32;

  std::lock_guard<std::mutex> lock(*LaunchContext::deviceMutex());

  auto handle = reinterpret_cast<hipblasHandle_t*>(A->getContext()->getCublasHandle());
  auto stream = A->getContext()->getCudaStream();

  auto status = hipblasSetStream(*handle, *stream);
  if (status != HIPBLAS_STATUS_SUCCESS) throw cuda_exception::build("MmulHelper::mmulMxV cuda failed !", status);

  if (!typeDouble && !typeFloat) {
    dim3 dims = getGemVDims(M);
    NDArray::prepareSpecialUse({Y}, {A, X});

    const int blocksPerGrid = dims.x;
    const int threadsPerBlock = dims.y;
    BUILD_SINGLE_SELECTOR_THRICE(
        xType, usualGemv,
        (blocksPerGrid,threadsPerBlock,stream, A->specialBuffer(), A->specialShapeInfo(), X->specialBuffer(),
            X->specialShapeInfo(), Y->specialBuffer(), Y->specialShapeInfo(), incx, incy, 0, alpha, beta),
        SD_NUMERIC_TYPES)
    NDArray::registerSpecialUse({Y}, {A, X});

    auto cudaResult = hipStreamSynchronize(*stream);
    if (cudaResult != 0) throw cuda_exception::build("MmulHelper::mmulMxV cuda failed !", cudaResult);

  } else {
    NDArray* pA(const_cast<NDArray*>(A));

    bool aMcont = M == 1 || A->strideAt(0) == 1;
    bool aNcont = N == 1 || A->strideAt(1) == 1;

    if (!aMcont && !aNcont) {
      pA = new NDArray(A->dup('f'));
      aMcont = true;
    }

    const bool transA = !aMcont;

    const int lda = (aMcont && aNcont) ? M : transA ? pA->strideAt(0) : pA->strideAt(1);

    const hipblasOperation_t transAblas = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    NDArray::prepareSpecialUse({Y}, {pA, X});

    // choose appropriate cuda gemm api depending on data types
    if (typeDouble) {
      status = hipblasDgemv(*handle, transAblas, transA ? N : M, transA ? M : N, &alpha, (double*)pA->specialBuffer(),
                           lda, (double*)X->specialBuffer(), incx, &beta, (double*)Y->specialBuffer(), incy);
    } else if (typeFloat) {
      float alphaF(alpha), betaF(beta);
      status = hipblasSgemv(*handle, transAblas, transA ? N : M, transA ? M : N, &alphaF, (float*)pA->specialBuffer(),
                           lda, (float*)X->specialBuffer(), incx, &betaF, (float*)Y->specialBuffer(), incy);
    }

    if (status != HIPBLAS_STATUS_SUCCESS) throw cuda_exception::build("MmulHelper::mmulMxV cuda failed !", status);

    auto cudaResult = hipStreamSynchronize(*stream);
    if (cudaResult != 0) throw cuda_exception::build("MmulHelper::mmulMxV cuda failed !", cudaResult);

    NDArray::registerSpecialUse({Y}, {pA, X});

    if (pA != A) delete pA;
  }

  return Y;
}

////////////////////////////////////////////////////////////////////////////
// (X * Y) = Z[0]
NDArray* MmulHelper::dot(const NDArray* X, const NDArray* Y, NDArray* Z, const double alpha, const double beta) {
  LongType xLenDim(0), yLenDim(0);

  if (!shape::isCommonVector(X->shapeInfo(), xLenDim))
    THROW_EXCEPTION("MmulHelper::dot cuda: X array must be vector !");
  if (!shape::isCommonVector(Y->shapeInfo(), yLenDim))
    THROW_EXCEPTION("MmulHelper::dot cuda: Y array must be vector !");
  if (Z != nullptr && Z->lengthOf() > 1) {
    THROW_EXCEPTION("MmulHelper::dot: Z array must be scalar !");
  }

  const auto length = X->lengthOf();

  if (Y->lengthOf() != length)
    THROW_EXCEPTION("MmulHelper::dot cuda: lengths of input vectors are different !");

  if (Z == nullptr)
    Z = new NDArray(DataTypeUtils::pickPairwiseResultType(X->dataType(), Y->dataType()), X->getContext());

  const LongType incx = X->strideAt(xLenDim);
  const LongType incy = Y->strideAt(yLenDim);

  const auto xType = X->dataType();
  const auto yType = Y->dataType();
  const auto zType = Z->dataType();

  if (!X->isActualOnDeviceSide()) X->syncToDevice();
  if (!Y->isActualOnDeviceSide()) Y->syncToDevice();
  if (!Z->isActualOnDeviceSide()) Z->syncToDevice();

  hipStream_t* stream = X->getContext()->getCudaStream();

  dim3 dims = getMMulDims(length,DataTypeUtils::sizeOf(zType));

  NDArray::prepareSpecialUse({Z}, {X, Y});


  BUILD_SINGLE_SELECTOR_THRICE(xType, usualDot,
                               (dims, stream, length, alpha, X->specialBuffer(), incx,
                                   Y->specialBuffer(), incy, beta, Z->specialBuffer()),
                               SD_NUMERIC_TYPES)

  auto cudaResult = hipStreamSynchronize(*stream);
  if (cudaResult != 0) throw cuda_exception::build("MmulHelper::dot cuda failed !", cudaResult);

  NDArray::registerSpecialUse({Z}, {X, Y});

  return Z;
}

//////////////////////////////////////////////////////////////////////////////
// [bS,M,K] x [bS,K,N] = [bS,M,N]
// [bS,M,K] x    [K,N] = [bS,M,N]
//    [M,K] x [bS,K,N] = [bS,M,N]
// bS could stand for several axes
template <typename T1, typename T2, typename T3>
static SD_KERNEL void batchedCudaGemm(const void* vA, const LongType* aShapeInfo, const void* vB,
                                      const LongType* bShapeInfo, void* vC, const LongType* cShapeInfo,
                                      const LongType* aBatchDims, const LongType* bBatchDims,
                                      const LongType* cBatchDims, const LongType aMaxis, const LongType aKaxis,
                                      const LongType bKaxis, const LongType bNaxis, const LongType cMaxis,
                                      const LongType cNaxis, const double alpha, const double beta) {
  const T1* A = reinterpret_cast<const T1*>(vA);
  const T2* B = reinterpret_cast<const T2*>(vB);
  T3* C = reinterpret_cast<T3*>(vC);

  __shared__ bool betaPresent;
  __shared__ LongType aRank, bRank, cRank, K, *coords;
  __shared__ LongType cLen, totalThreads;
  __shared__ T3 alphaZ, betaZ;

  if (threadIdx.x == 0) {
    extern __shared__ unsigned char shmem[];
    coords = reinterpret_cast<LongType*>(shmem);
    cLen = shape::length(cShapeInfo);

    K = shape::shapeOf(const_cast<LongType*>(aShapeInfo))[aKaxis];

    totalThreads = gridDim.x * blockDim.x;
    aRank = shape::rank(aShapeInfo);
    bRank = shape::rank(bShapeInfo);
    cRank = shape::rank(cShapeInfo);

    betaPresent = beta;

    alphaZ = alpha;
    betaZ = beta;
  }
  __syncthreads();

  auto aCoords = coords + threadIdx.x * (aRank + bRank + cRank);
  auto bCoords = aCoords + aRank;
  auto cCoords = bCoords + bRank;

  const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

  for (LongType i = tid; i < cLen; i += totalThreads) {
    // evaluate C coordinates
    shape::index2coords(i, cShapeInfo, cCoords);

    // calculate index of current batch
    LongType batchInd;
    if (cBatchDims != nullptr) batchInd = shape::coords2index(cShapeInfo, cBatchDims, cRank - 2, cCoords);

    // evaluate A coordinates
    if (aBatchDims != nullptr) shape::index2coords(batchInd, aShapeInfo, aBatchDims, aRank - 2, aCoords);
    aCoords[aMaxis] = cCoords[cMaxis];
    aCoords[aKaxis] = 0;

    // evaluate B coordinates
    if (bBatchDims != nullptr) shape::index2coords(batchInd, bShapeInfo, bBatchDims, bRank - 2, bCoords);
    bCoords[bKaxis] = 0;
    bCoords[bNaxis] = cCoords[cNaxis];

    auto aOffset = shape::getOffset(aShapeInfo, aCoords);
    auto bOffset = shape::getOffset(bShapeInfo, bCoords);

    T3 val = A[aOffset] * B[bOffset];  // first iteration

    for (LongType j = 1; j < K; ++j) {  // rest iterations
      aOffset += shape::stride(aShapeInfo)[aKaxis];
      bOffset += shape::stride(bShapeInfo)[bKaxis];
      val = val + A[aOffset] * B[bOffset];
    }

    auto cOffset = shape::getOffset(cShapeInfo, cCoords);

    if (betaPresent)
      C[cOffset] = alphaZ * val + betaZ * C[cOffset];
    else
      C[cOffset] = alphaZ * val;
  }
}

////////////////////////////////////////////////////////////////////////
template <typename T1, typename T2, typename T3>
SD_HOST static void batchedGemm(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem,
                                hipStream_t* stream, const void* vA, const LongType* aShapeInfo, const void* vB,
                                const LongType* bShapeInfo, void* vC, const LongType* cShapeInfo,
                                const LongType* aBatchDims, const LongType* bBatchDims, const LongType* cBatchDims,
                                const LongType aMaxis, const LongType aKaxis, const LongType bKaxis,
                                const LongType bNaxis, const LongType cMaxis, const LongType cNaxis, const double alpha, const double beta) {
  batchedCudaGemm<T1, T2, T3><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(
      vA, aShapeInfo, vB, bShapeInfo, vC, cShapeInfo, aBatchDims, bBatchDims, cBatchDims, aMaxis, aKaxis, bKaxis,
      bNaxis, cMaxis, cNaxis, alpha, beta);
  DebugHelper::checkGlobalErrorCode("batch gemm failed(...) failed");

}

///////////////////////////////////////////////////////////////////
NDArray* MmulHelper::mmulNxN(NDArray* A, NDArray* B, NDArray* C, const double alpha, const double beta,
                             const char outOrder) {
  const LongType aRank = A->rankOf();
  const LongType bRank = B->rankOf();

  // input ranks validation
  if (aRank > bRank && bRank != 2) {
    THROW_EXCEPTION("MmulHelper::mmulNxN: rank of B array should be equal 2 !");
  }
  else if (bRank > aRank && aRank != 2) {
    THROW_EXCEPTION("MmulHelper::mmulNxN: rank of A array should be equal 2 !");
  }
  else if (aRank == bRank) {
    for (int i = 0; i < aRank - 2; ++i)
      if (A->sizeAt(i) != B->sizeAt(i))
        THROW_EXCEPTION(
            "MmulHelper::mmulNxN: shapes of A and B arrays are not suitable for matrix multiplication !");
  }

  if (A->sizeAt(-1) != B->sizeAt(-2)) {
    THROW_EXCEPTION("MmulHelper::mmulNxN: shapes of A and B arrays are not suitable for matrix multiplication !");
  }
  // validation of C array
  std::vector<LongType> cExpectedShape = aRank > bRank ? A->getShapeAsVector() : B->getShapeAsVector();
  cExpectedShape[cExpectedShape.size() - 2] = A->sizeAt(-2);
  cExpectedShape[cExpectedShape.size() - 1] = B->sizeAt(-1);

  if (C != nullptr) {
    if (!C->isSameShape(cExpectedShape))
      THROW_EXCEPTION("MmulHelper::mmulNxN: shape of C array is not suitable for AxB matrix multiplication !");
  } else
    C = new NDArray(outOrder, cExpectedShape, DataTypeUtils::pickPairwiseResultType(A->dataType(), B->dataType()),
                    A->getContext());

  if (C->isEmpty()) return C;

  const LongType cRank = C->rankOf();

  const LongType aMaxis(aRank - 2), aKaxis(aRank - 1), bKaxis(bRank - 2), bNaxis(bRank - 1), cMaxis(cRank - 2),
      cNaxis(cRank - 1);

  const int threadsPerBlock = SD_MAX_NUM_THREADS / 8;
  const int blocksPerGrid = (C->lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
  const int sharedMem = threadsPerBlock * sizeof(LongType) * (aRank + bRank + cRank) + 128;

  PointersManager manager(A->getContext(), "MmulHelper::mmulNxN");

  const LongType *aBatchDims(nullptr), *bBatchDims(nullptr), *cBatchDims(nullptr);

  std::vector<LongType> aDimsVec = {aMaxis,aKaxis};
  std::vector<LongType> *aDims = ShapeUtils::evalDimsToExclude(aRank, 2,aDimsVec.data());

  std::vector<LongType> bDimsVec = {bKaxis, bNaxis};
  std::vector<LongType> *bDims =  ShapeUtils::evalDimsToExclude(bRank,2, bDimsVec.data());


  std::vector<LongType> cDimsVec = {cMaxis,2, cNaxis};
  std::vector<LongType> *cDims = ShapeUtils::evalDimsToExclude(cRank, cDimsVec.size(),cDimsVec.data());
  if (aRank > 2)
    aBatchDims = reinterpret_cast<LongType*>(manager.replicatePointer(
        aDims->data(), (aRank - 2) * sizeof(LongType)));
  if (bRank > 2)
    bBatchDims = reinterpret_cast<LongType*>(manager.replicatePointer(
        bDims->data(), (bRank - 2) * sizeof(LongType)));
  if (cRank > 2)
    cBatchDims = reinterpret_cast<LongType*>(manager.replicatePointer(
        cDims->data(), (cRank - 2) * sizeof(LongType)));

  NDArray::prepareSpecialUse({C}, {A, B});
  BUILD_SINGLE_SELECTOR_THRICE(
      A->dataType(), batchedGemm,
      (blocksPerGrid, threadsPerBlock, sharedMem, A->getContext()->getCudaStream(), A->specialBuffer(),
          A->specialShapeInfo(), B->specialBuffer(), B->specialShapeInfo(), C->specialBuffer(), C->specialShapeInfo(),
          aBatchDims, bBatchDims, cBatchDims, aMaxis, aKaxis, bKaxis, bNaxis, cMaxis, cNaxis, alpha, beta),
      SD_NUMERIC_TYPES)
  NDArray::registerSpecialUse({C}, {A, B});

  manager.synchronize();

  delete aDims;
  delete bDims;
  delete cDims;

  return C;
}


}  // namespace sd
