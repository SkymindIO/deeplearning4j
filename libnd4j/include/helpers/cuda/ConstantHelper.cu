/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//

#include <exceptions/cuda_exception.h>
#include <ConstantHelper.h>
#include <execution/LaunchContext.h>
#include <logger.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CONSTANT_LIMIT 49152

__constant__ char deviceConstantMemory[CONSTANT_LIMIT];

namespace nd4j {
    static void* getConstantSpace() {
        Nd4jPointer dConstAddr;
        auto dZ = hipGetSymbolAddress(reinterpret_cast<void **>(&dConstAddr), deviceConstantMemory);

        if (dZ != 0)
            throw cuda_exception::build("hipGetSymbolAddress(...) failed", dZ);

        return dConstAddr;
    }

    int ConstantHelper::getCurrentDevice() {
        int dev = 0;
        auto res = hipGetDevice(&dev);

        if (res != 0)
            throw cuda_exception::build("hipGetDevice failed", res);

        return dev;
    }

    int ConstantHelper::getNumberOfDevices() {
        int dev = 0;
        auto res = hipGetDeviceCount(&dev);

        if (res != 0)
            throw cuda_exception::build("hipGetDeviceCount failed", res);

        return dev;
    }


    ConstantHelper::ConstantHelper() {
        auto initialDevice = getCurrentDevice();

        auto numDevices = getNumberOfDevices();
        _devicePointers.resize(numDevices);
        _deviceOffsets.resize(numDevices);
        // filling all pointers
        for (int e = 0; e < numDevices; e++) {
            auto res = hipSetDevice(e);
            if (res != 0)
                throw cuda_exception::build("hipSetDevice failed", res);
             auto constant = getConstantSpace();

            _devicePointers[e] = constant;
            _deviceOffsets[e] = 0;
        }

        //
        auto res = hipSetDevice(initialDevice);
        if (res != 0)
            throw cuda_exception::build("Final hipSetDevice failed", res);
    }

    ConstantHelper* ConstantHelper::getInstance() {
        if (!_INSTANCE)
            _INSTANCE = new nd4j::ConstantHelper();

        return _INSTANCE;
    }

    void* ConstantHelper::replicatePointer(void *src, size_t numBytes, memory::Workspace *workspace) {
        _mutex.lock();

        auto deviceId = getCurrentDevice();
        Nd4jPointer constantPtr = nullptr;
        Nd4jLong constantOffset = 0L;
        if (_devicePointers[deviceId] == 0) {
            auto constant = getConstantSpace();

            // filling default ptr, which will be 0 probably
            _devicePointers[deviceId] = constant;
            _deviceOffsets[deviceId] = 0;
            constantPtr = constant;
        } else {
            constantPtr = _devicePointers[deviceId];
            constantOffset = _deviceOffsets[deviceId];
        }
        if (constantOffset + numBytes >= CONSTANT_LIMIT) {
            int8_t *ptr = nullptr;
            ALLOCATE_SPECIAL(ptr, workspace, numBytes, int8_t);
            auto res = hipMemcpyAsync(ptr, src, numBytes, hipMemcpyHostToDevice, *nd4j::LaunchContext ::defaultContext()->getCudaSpecialStream());
            if (res != 0)
                throw cuda_exception::build("hipMemcpyToSymbolAsync failed", res);

            res = hipStreamSynchronize(*nd4j::LaunchContext ::defaultContext()->getCudaSpecialStream());
            if (res != 0)
                throw cuda_exception::build("hipStreamSynchronize failed", res);

            _mutex.unlock();
            return ptr;
        } else {
            auto originalBytes = numBytes;
            auto rem = numBytes % 8;
            if (rem != 0)
                numBytes += 8 - rem;

            _deviceOffsets[deviceId] += numBytes;

            auto res = hipMemcpyToSymbolAsync(HIP_SYMBOL(deviceConstantMemory), const_cast<const void *>(src), originalBytes, constantOffset, hipMemcpyHostToDevice, *nd4j::LaunchContext ::defaultContext()->getCudaSpecialStream());
            if (res != 0)
                throw cuda_exception::build("hipMemcpyToSymbolAsync failed", res);

            res = hipStreamSynchronize(*nd4j::LaunchContext ::defaultContext()->getCudaSpecialStream());
            if (res != 0)
                throw cuda_exception::build("hipStreamSynchronize failed", res);

            _mutex.unlock();
            return reinterpret_cast<int8_t *>(constantPtr) + constantOffset;
        }
    }

    ConstantDataBuffer* ConstantHelper::constantBuffer(ConstantDescriptor &descriptor, nd4j::DataType dataType) {
        const auto deviceId = getCurrentDevice();

//        if (_cache[deviceId].count(descriptor) == 0)
//            _cache[deviceId].[descriptor]

//        auto constant = _cache[deviceId][descriptor];



        return nullptr;
    }

    nd4j::ConstantHelper* nd4j::ConstantHelper::_INSTANCE = 0;
}