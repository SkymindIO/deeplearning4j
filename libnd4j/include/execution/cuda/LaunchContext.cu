/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//
#include <exceptions/cuda_exception.h>
#include <execution/AffinityManager.h>
#include <execution/LaunchContext.h>
#include <helpers/cublasHelper.h>
#include <helpers/logger.h>

#include <thread>

thread_local sd::ContextBuffers contextBuffers = sd::ContextBuffers();

namespace sd {

std::vector<std::shared_ptr<LaunchContext>> LaunchContext::_contexts = std::vector<std::shared_ptr<LaunchContext>>();
std::mutex LaunchContext::_mutex;
SD_MAP_IMPL<int, std::mutex*> LaunchContext::_deviceMutexes;

////////////////////////////////////////////////////////////////////////
LaunchContext::LaunchContext(hipStream_t* cudaStream, hipStream_t& specialCudaStream, void* reductionPointer,
                             void* scalarPointer, int* allocationPointer) {

  _workspace = nullptr;
  _isAllocated = false;
}

std::mutex* LaunchContext::deviceMutex() {
  auto deviceId = AffinityManager::currentDeviceId();
  return _deviceMutexes[deviceId];
}

LaunchContext::~LaunchContext() {
  if (_isAllocated) {
  }
}

////////////////////////////////////////////////////////////////////////
LaunchContext::LaunchContext() {
  // default constructor, just to make clang/ranlib happy
  _workspace = nullptr;
  _deviceID = 0;

  _isAllocated = true;
}

LaunchContext::LaunchContext(Pointer cudaStream, Pointer reductionPointer, Pointer scalarPointer,
                             Pointer allocationPointer) {
  _isAllocated = false;

}

LaunchContext* LaunchContext::defaultContext() {
  /**
   * This method returns LaunchContext, that has multiple entities within:
   * 1) temporary buffers. they must be per-thread
   * 2) CUDA stream. it must be either per-thread or per-device
   * 3) cuBLAS handle. it must be per-device
   */
  auto deviceId = AffinityManager::currentDeviceId();

  {
    // we need this block synchronous, to avoid double initialization etc
    std::lock_guard<std::mutex> lock(_mutex);
    if (_contexts.empty()) {
      // create one context per device
      auto numDevices = AffinityManager::numberOfDevices();

      _contexts.resize(numDevices);
      for (int e = 0; e < numDevices; e++) {
        _deviceMutexes[e] = new std::mutex();

        AffinityManager::setCurrentNativeDevice(e);

        _contexts[e] = std::make_shared<LaunchContext>();
      }

      // don't forget to restore device back again
      AffinityManager::setCurrentNativeDevice(deviceId);
    }
  }

  // return context for current device
  return _contexts[deviceId].get();
}

void* LaunchContext::getReductionPointer() const { return contextBuffers.reductionBuffer(); };

void* LaunchContext::getScalarPointer() const { return contextBuffers.scalarBuffer(); };

LongType* LaunchContext::getAllocationPointer() const { return reinterpret_cast<LongType*>(contextBuffers.allocationBuffer()); };

void* LaunchContext::getCublasHandle() const { return CublasHelper::getInstance().handle(); };

void* LaunchContext::getCusolverHandle() const { return CublasHelper::getInstance().solver(); };

hipStream_t* LaunchContext::getCudaStream() const {
  return reinterpret_cast<hipStream_t*>(contextBuffers.execStream());
};

hipStream_t* LaunchContext::getCudaSpecialStream() const {
  return reinterpret_cast<hipStream_t*>(contextBuffers.specialStream());
  ;
};

void LaunchContext::setReductionPointer(void* reductionPointer) {
  contextBuffers.setReductionBuffer(reductionPointer);
};

void LaunchContext::setScalarPointer(void* scalarPointer) { contextBuffers.setScalarBuffer(scalarPointer); };

void LaunchContext::setAllocationPointer(int* allocationPointer) {
  contextBuffers.setAllocationBuffer(allocationPointer);
};

void LaunchContext::setCudaStream(hipStream_t* cudaStream){
};

void LaunchContext::setCudaSpecialStream(hipStream_t* cudaStream){
};

void LaunchContext::setCublasHandle(void* handle) { _cublasHandle = handle; };

void LaunchContext::swapContextBuffers(ContextBuffers& buffers) { contextBuffers = buffers; };

void LaunchContext::releaseBuffers() {
  contextBuffers.release();
}

bool LaunchContext::isInitialized() { return contextBuffers.isInitialized(); }

void* LaunchContext::getCuDnnHandle() const { return CublasHelper::getInstance().cudnn(); }

ErrorReference* LaunchContext::errorReference() { return contextBuffers.errorReference(); }

void* LaunchContext::engine() { return _engine; }
}  // namespace sd
