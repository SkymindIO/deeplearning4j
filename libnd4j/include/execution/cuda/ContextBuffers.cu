/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//

#include <execution/ContextBuffers.h>
#include <exceptions/cuda_exception.h>
#include <logger.h>
#include <AffinityManager.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>

namespace nd4j {
    ContextBuffers::ContextBuffers() {
        nd4j_printf("Creating ContextBuffers for device [%i]\n", AffinityManager::currentDeviceId());
        _deviceId = AffinityManager::currentDeviceId();
    }

    ContextBuffers::~ContextBuffers() {
        if (_allocated) {
            nd4j_printf("Releasing ContextBuffers\n","");

            if (_allocationPointer != nullptr)
                hipFree(_allocationPointer);

            if (_scalarPointer != nullptr)
                hipFree(_scalarPointer);

            if (_allocationPointer != nullptr)
                hipFree(_reductionPointer);

            auto _cudaStream = reinterpret_cast<hipStream_t*>(_execStream);
            auto _cudaSpecialStream = reinterpret_cast<hipStream_t*>(_specialStream);

            hipStreamSynchronize(*_cudaStream);
            hipStreamSynchronize(*_cudaSpecialStream);

            hipStreamDestroy(*_cudaStream);
            hipStreamDestroy(*_cudaSpecialStream);

            delete _cudaStream;
            delete _cudaSpecialStream;
        }
    }

    ContextBuffers::ContextBuffers(void* rPointer, void* sPointer, void* aPointer, bool isOwner) {
        _reductionPointer = rPointer;
        _scalarPointer = sPointer;
        _allocationPointer = aPointer;
        _allocated = isOwner;
    }

    void ContextBuffers::initialize() {
        nd4j_printf("Initializing buffers on deviceId [%i]\n", AffinityManager::currentNativeDeviceId());

        auto res = hipMalloc(reinterpret_cast<void**>(&_reductionPointer), 1024 * 1024 * 8);
        if (res != 0)
            throw std::runtime_error("_reductionPointer allocation failed");

        res = hipMalloc(reinterpret_cast<void**>(&_scalarPointer), 16);
        if (res != 0)
            throw std::runtime_error("_scalarPointer allocation failed");

        res = hipMalloc(reinterpret_cast<void**>(&_allocationPointer), 1024 * 1024 * 8);
        if (res != 0)
            throw std::runtime_error("_allocationPointer allocation failed");

        _execStream  = new hipStream_t();
        _specialStream = new hipStream_t();
        if (nullptr == _execStream || nullptr == _specialStream)
            throw std::runtime_error("Failed to allocate memory for new CUDA stream");

        res = hipStreamCreate(reinterpret_cast<hipStream_t*>(_execStream));
        if (res != 0)
            throw cuda_exception::build("Failed to create default CUDA stream with launch context", res);

        res = hipStreamCreate(reinterpret_cast<hipStream_t*>(_specialStream));
        if (res != 0)
            throw cuda_exception::build("Failed to create special CUDA stream with launch context", res);

        _allocated = true;
    }

    void* ContextBuffers::reductionBuffer() {
        if (_reductionPointer == nullptr)
            initialize();

        return _reductionPointer;
    }

    void* ContextBuffers::scalarBuffer() {
        if (_scalarPointer == nullptr)
            initialize();

        return _scalarPointer;
    }

    void* ContextBuffers::allocationBuffer() {
        if (_allocationPointer == nullptr)
            initialize();

        return _allocationPointer;
    }

    void ContextBuffers::setReductionBuffer(void* pointer) {
        _reductionPointer = pointer;
    }

    void ContextBuffers::setScalarBuffer(void* pointer) {
        _scalarPointer = pointer;
    }

    void ContextBuffers::setAllocationBuffer(void* pointer) {
        _allocationPointer = pointer;
    }

    void ContextBuffers::triggerOwnership(bool isOwner) {
        _allocated = isOwner;
    }

    int ContextBuffers::deviceId() {
        return _deviceId;
    }

    void* ContextBuffers::execStream() {
        if (_execStream == nullptr)
            initialize();

        return _execStream;
    }

    void* ContextBuffers::specialStream() {
        if (_specialStream == nullptr)
            initialize();

        return _specialStream;
    }
}
