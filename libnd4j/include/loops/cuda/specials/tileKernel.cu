/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author GS <sgazeos@gmail.com>, created on 16.01.2019
//
#include <loops/special_kernels.h>

#include <execution/cuda/LaunchDims.h>


namespace sd {


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//  tileKernel:
//  input: (inputBuffer and inputShape) - NDArray buffer and shape to tile
//  output: (outputBuffer and outputShape) - NDArray to tile input
//  resultLength - length for output array
template <typename T>
static SD_KERNEL void tileKernel(void const* inputBuffer, LongType const* inputShape, void* outputBuffer,
                                 LongType const* outputShape, LongType resultLength) {
  auto tid = blockIdx.x * blockDim.x + threadIdx.x;  // copy linear sequence of elements, so one-level threading
  int totalThreads = gridDim.x * blockDim.x;

  if (shape::order(outputShape) == 'c') {
    for (int i = tid; i < resultLength; i += totalThreads) {
      sd::LongType yCoords[SD_MAX_RANK];
      sd::LongType yOffset;

      INDEX2COORDS(i, shape::rank(outputShape), outputShape, yCoords);
      COORDS2INDEX(shape::rank(outputShape), shape::shapeOf(outputShape), yCoords, yOffset);

      *(reinterpret_cast<T*>(outputBuffer) + i) = *(reinterpret_cast<T const*>(inputBuffer) + yOffset);
    }
  } else {
    for (int i = tid; i < resultLength; i += totalThreads) {
      sd::LongType xCoords[SD_MAX_RANK];
      sd::LongType yCoords[SD_MAX_RANK];
      sd::LongType xOffset;
      sd::LongType yOffset;

      INDEX2COORDS(i, shape::rank(outputShape), outputShape, xCoords);
      COORDS2INDEX(shape::rank(outputShape), shape::shapeOf(outputShape), xCoords, xOffset);
      INDEX2COORDS(i, shape::rank(inputShape), inputShape, yCoords);
      COORDS2INDEX(shape::rank(inputShape), shape::shapeOf(inputShape), yCoords, yOffset);

      *(reinterpret_cast<T*>(outputBuffer) + xOffset) = *(reinterpret_cast<T const*>(inputBuffer) + yOffset);
    }
  }
}

BUILD_SINGLE_TEMPLATE(template SD_KERNEL void tileKernel,
                      (void const* inputBuffer, sd::LongType const* inputShape, void* outputBuffer,
                       sd::LongType const* outputShape, sd::LongType resultLength),
                      SD_COMMON_TYPES);

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
template <typename T>
void tileKernelH(void const* inputBuffer, LongType const* inputShape, void* outputBuffer, LongType const* outputShape,
                 LongType resultLength, hipStream_t* stream) {
  dim3 launchDims = getLaunchDims("tile");
  tileKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(inputBuffer, inputShape, outputBuffer,
                                                                       outputShape, resultLength);
  sd::DebugHelper::checkErrorCode(stream, "tileKernel  failed");


}

BUILD_SINGLE_TEMPLATE(template void tileKernelH,
                      (void const* inputBuffer, sd::LongType const* inputShape, void* outputBuffer,
                       sd::LongType const* outputShape, sd::LongType resultLength, hipStream_t* stream),
                      SD_COMMON_TYPES);

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// enhancement for tileKernel to different input and output data types: X - output type, Y - input type
template <typename X, typename Y>
static SD_KERNEL void tileKernelDouble(void const* inputBuffer, LongType const* inputShape, void* outputBuffer,
                                       LongType const* outputShape, LongType resultLength) {
  char ordering = shape::order(outputShape);
  auto tid = blockIdx.x * blockDim.x + threadIdx.x;
  int totalThreads = gridDim.x * blockDim.x;

  if (ordering == 'c') {
    for (int i = tid; i < resultLength; i += totalThreads) {
      sd::LongType yCoords[SD_MAX_RANK];
      sd::LongType yOffset;

      INDEX2COORDS(i, shape::rank(outputShape), outputShape, yCoords);
      COORDS2INDEX(shape::rank(outputShape), shape::shapeOf(outputShape), yCoords, yOffset);

      *(reinterpret_cast<X*>(outputBuffer) + i) = static_cast<X>(*(reinterpret_cast<Y const*>(inputBuffer) + yOffset));
    }
  } else {
    for (int i = tid; i < resultLength; i += totalThreads) {
      sd::LongType xCoords[SD_MAX_RANK];
      sd::LongType yCoords[SD_MAX_RANK];
      sd::LongType xOffset;
      sd::LongType yOffset;

      INDEX2COORDS(i, shape::rank(outputShape), outputShape, xCoords);
      COORDS2INDEX(shape::rank(outputShape), shape::shapeOf(outputShape), xCoords, xOffset);
      INDEX2COORDS(i, shape::rank(inputShape), inputShape, yCoords);
      COORDS2INDEX(shape::rank(inputShape), shape::shapeOf(inputShape), yCoords, yOffset);

      *(reinterpret_cast<X*>(outputBuffer) + xOffset) = static_cast<X>(*(reinterpret_cast<Y const*>(inputBuffer) + yOffset));
    }
  }
}
BUILD_SINGLE_TEMPLATE_TWICE(template SD_KERNEL void tileKernelDouble,
                            (void const* inputBuffer, sd::LongType const* inputShape, void* outputBuffer,
                             sd::LongType const* outputShape, sd::LongType resultLength, sd::LongType ews),
                            SD_COMMON_TYPES);

template <typename X, typename Y>
void tileKernelHH(void const* inputBuffer, LongType const* inputShape, void* outputBuffer, LongType const* outputShape,
                  LongType resultLength, LongType ews, hipStream_t* stream) {
  dim3 launchDims = getLaunchDims("tile");
  tileKernelDouble<X, Y><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(inputBuffer, inputShape, outputBuffer,
                                                                                outputShape, resultLength, ews);

  DebugHelper::checkErrorCode(stream,"templatedSwapUnsafe(...) failed");

}

BUILD_SINGLE_TEMPLATE_TWICE(template void tileKernelHH,
                            (void const* inputBuffer, sd::LongType const* inputShape, void* outputBuffer,
                             sd::LongType const* outputShape, sd::LongType resultLength, sd::LongType ews,
                             hipStream_t* stream),
                            SD_COMMON_TYPES);
}  // namespace sd
