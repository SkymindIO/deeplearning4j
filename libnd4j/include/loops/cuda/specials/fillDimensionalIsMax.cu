#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 15.11.2018
//
#include <loops/special_kernels.h>


namespace sd {

////////////////////////////////////////////////////////////////////////
template <typename T>
SD_DEVICE void fillDimensionalIsMax(const void *vdX, void *vdZ, const LongType *zShapeInfo,
                                    const LongType *tadOnlyShapeInfo, LongType *dimension, LongType dimensionLength,
                                    const LongType *tadOffsets) {
  auto dX = reinterpret_cast<const LongType *>(vdX);
  auto dZ = reinterpret_cast<T *>(vdZ);

  __shared__ int tadLength;
  __shared__ int numTads;

  if (threadIdx.x == 0) {
    tadLength = shape::length(tadOnlyShapeInfo);
    numTads = shape::length(zShapeInfo) / tadLength;
  }
  __syncthreads();

  for (int r = blockIdx.x; r < numTads; r += gridDim.x) {
    auto tadOffsetForBlock = tadOffsets[r];
    auto highestElement = dX[r];

    if (dimensionLength > 1) {
      for (LongType e = threadIdx.x; e < tadLength; e += blockDim.x) {
        sd::LongType xCoords[SD_MAX_RANK];
        sd::LongType xOffset;
        INDEX2COORDS(e, shape::rank(tadOnlyShapeInfo), shape::shapeOf(tadOnlyShapeInfo), xCoords);
        COORDS2INDEX(shape::rank(tadOnlyShapeInfo), shape::stride(tadOnlyShapeInfo), xCoords, xOffset);
        auto finalOffset = tadOffsetForBlock + xOffset;
        dZ[finalOffset] = (e == highestElement ? (T)1 : (T)0);
      }
    } else {
      for (LongType e = threadIdx.x; e < tadLength; e += blockDim.x) {
        sd::LongType xCoords[SD_MAX_RANK];
        sd::LongType xOffset;
        INDEX2COORDS(e, shape::rank(tadOnlyShapeInfo), shape::shapeOf(tadOnlyShapeInfo), xCoords);
        COORDS2INDEX(shape::rank(tadOnlyShapeInfo), shape::stride(tadOnlyShapeInfo), xCoords, xOffset);
        auto finalOffset = tadOffsetForBlock + xOffset;
        dZ[finalOffset] = (e == highestElement ? (T)1 : (T)0);
      }
    }
  }
}
////////////////////////////////////////////////////////////////////////
template <typename T>
SD_KERNEL void execfillDimensionalIsMax(const void *dX, void *dZ, const LongType *zShapeInfo,
                                        const LongType *tadOnlyShapeInfo, LongType *dimension, LongType dimensionLength,
                                        const LongType *tadOffsets) {
  fillDimensionalIsMax<T>(dX, dZ, zShapeInfo, tadOnlyShapeInfo, dimension, dimensionLength, tadOffsets);
}

////////////////////////////////////////////////////////////////////////
template <typename T>
SD_HOST void fillDimensionalIsMaxGeneric(dim3 &launchDims, hipStream_t *stream, const void *dX, void *dZ,
                                         const LongType *zShapeInfo, const LongType *tadOnlyShapeInfo,
                                         LongType *dimension, LongType dimensionLength, const LongType *tadOffsets) {
  execfillDimensionalIsMax<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(
      dX, dZ, zShapeInfo, tadOnlyShapeInfo, dimension, dimensionLength, tadOffsets);
  DebugHelper::checkErrorCode(stream, "fillDimensionalIsMax(...) failed");
}
BUILD_SINGLE_TEMPLATE(template void fillDimensionalIsMaxGeneric,
                      (dim3 & launchDims, hipStream_t *stream, const void *dX, void *dZ,
                       const sd::LongType *zShapeInfo, const sd::LongType *tadOnlyShapeInfo, sd::LongType *dimension,
                       sd::LongType dimensionLength, const sd::LongType *tadOffsets),
                      SD_COMMON_TYPES);
}  // namespace sd
